
/**
 Cで学ぶアルゴリズムとデータ構造
 ステップバイステップでＮ−クイーン問題を最適化
 一般社団法人  共同通信社  情報技術局  鈴木  維一郎(suzuki.iichiro@kyodonews.jp)

 実行
 $ gcc -Wall -W -O3 -g -ftrapv -std=c99 GCC10.c && ./a.out [-c|-r]


 １０．クイーンの位置による分岐BOUND1,2

  前章のコードは全ての解を求めた後に、ユニーク解以外の対称解を除去していた
  ある意味、「生成検査法（generate ＆ test）」と同じである
  問題の性質を分析し、バックトラッキング/前方検査法と同じように、無駄な探索を省略することを考える
  ユニーク解に対する左右対称解を予め削除するには、1行目のループのところで、
  右半分だけにクイーンを配置するようにすればよい
  Nが奇数の場合、クイーンを1行目中央に配置する解は無い。
  他の3辺のクィーンが中央に無い場合、その辺が上辺に来るよう回転し、場合により左右反転することで、
  最小値解とすることが可能だから、中央に配置したものしかユニーク解には成り得ない
  しかし、上辺とその他の辺の中央にクィーンは互いの効きになるので、配置することが出来ない


  1. １行目角にクイーンがある場合、とそうでない場合で処理を分ける
    １行目かどうかの条件判断はループ外に出してもよい
    処理時間的に有意な差はないので、分かりやすいコードを示した
  2.１行目角にクイーンがある場合、回転対称形チェックを省略することが出来る
    １行目角にクイーンがある場合、他の角にクイーンを配置することは不可
    鏡像についても、主対角線鏡像のみを判定すればよい
    ２行目、２列目を数値とみなし、２行目＜２列目という条件を課せばよい

  １行目角にクイーンが無い場合、クイーン位置より右位置の８対称位置にクイーンを置くことはできない
  置いた場合、回転・鏡像変換により得られる状態のユニーク判定値が明らかに大きくなる
    ☓☓・・・Ｑ☓☓
    ☓・・・／｜＼☓
    ｃ・・／・｜・rt
    ・・／・・｜・・
    ・／・・・｜・・
    lt・・・・｜・ａ
    ☓・・・・｜・☓
    ☓☓ｂ・・dn☓☓
    
  １行目位置が確定した時点で、配置可能位置を計算しておく（☓の位置）
  lt, dn, lt 位置は効きチェックで配置不可能となる
  回転対称チェックが必要となるのは、クイーンがａ, ｂ, ｃにある場合だけなので、
  90度、180度、270度回転した状態のユニーク判定値との比較を行うだけで済む


bash-3.2$ gcc -Wall -W -O3 -g -ftrapv -std=c99 -pthread GCC10.c && ./a.out -r
１０．CPUR 再帰 クイーンの位置による分岐BOUND1,2
 N:        Total       Unique        hh:mm:ss.ms
 4:            2               1            0.00
 5:           10               2            0.00
 6:            4               1            0.00
 7:           40               6            0.00
 8:           92              12            0.00
 9:          352              46            0.00
10:          724              92            0.00
11:         2680             341            0.00
12:        14200            1787            0.01
13:        73712            9233            0.05
14:       365596           45752            0.31
15:      2279184          285053            1.94
16:     14772512         1846955           14.54
17:     95815104        11977939         1:38.93


bash-3.2$ gcc -Wall -W -O3 -g -ftrapv -std=c99 -pthread GCC10.c && ./a.out -c
１０．CPU 非再帰 クイーンの位置による分岐BOUND1,2
 N:        Total       Unique        hh:mm:ss.ms
 4:            2               1            0.00
 5:           10               2            0.00
 6:            4               1            0.00
 7:           40               6            0.00
 8:           92              12            0.00
 9:          352              46            0.00
10:          724              92            0.00
11:         2680             341            0.00
12:        14200            1787            0.01
13:        73712            9233            0.05
14:       365596           45752            0.28
15:      2279184          285053            1.80
16:     14772512         1846955           13.55
17:     95815104        11977939         1:33.01

bash-3.2$ nvcc CUDA10_N-Queen.cu && ./a.out -g
１０．GPU 非再帰 クイーンの位置による分岐BOUND1,2
 N:        Total      Unique      dd:hh:mm:ss.ms
 4:            2               1  00:00:00:00.03
 5:           10               2  00:00:00:00.00
 6:            4               1  00:00:00:00.00
 7:           40               6  00:00:00:00.02
 8:           92              12  00:00:00:00.02
 9:          352              46  00:00:00:00.03
10:          724              92  00:00:00:00.05
11:         2680             341  00:00:00:00.09
12:        14200            1787  00:00:00:00.25
13:        73712            9233  00:00:00:00.97
14:       365596           45752  00:00:00:01.23
15:      2279184          285053  00:00:00:04.58
16:     14772512         1846955  00:00:00:28.91
17:     95815104        11977939  00:00:03:44.38
*/

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define THREAD_NUM		96
#define MAX 27
//変数宣言
int down[2*MAX-1];  //CPUで使用 down:flagA 縦 配置フラグ　
int left[2*MAX-1];  //CPUで使用 left:flagB 斜め配置フラグ　
int right[2*MAX-1]; //CPUで使用 right:flagC 斜め配置フラグ　
/***07 aBoard*************************************/
unsigned int aBoard[MAX];//CPU,GPUで使用
/****************************************/
int aT[MAX];//CPUで使用
int aS[MAX];//CPUで使用
long TOTAL=0;//GPU,CPUで使用
/***07 uniq*************************************/
long UNIQUE=0;//GPU,CPUで使用
/****************************************/
int COUNT2,COUNT4,COUNT8;//CPUで使用
int BOUND1,BOUND2,TOPBIT,ENDBIT,SIDEMASK,LASTMASK;//CPUで使用 11からGPUで使用
//関数宣言 GPU
/***07 d_uniq,t_aBoard,h_row追加に伴いコメント*************************************/
//void cuda_kernel(
//    int size,int mark,
//    unsigned int* t_down,unsigned int* t_left,unsigned int* t_right,
//    unsigned int* d_results,int totalCond,unsigned);
/****************************************/
/***07 d_uniq,t_aBoard,h_row追加*************************************/
__global__
void cuda_kernel(
    register int size,register int mark,
    unsigned int* t_down,unsigned int* t_left,unsigned int* t_right,
    unsigned int* d_results,unsigned int* d_uniq,int totalCond,unsigned int* t_aBoard,int h_row,int* aT,int* aS);
/****************************************/
/***10 関数名をsolve_nqueen_cudaからbacktrackに変更するためコメント*************************************/
//long solve_nqueen_cuda(int size,int mask,int row,int n_left,int n_down,int n_right,int steps);
/****************************************/
/***10 関数名をsolve_nqueen_cudaからbacktrackに変更する*************************************/
long backTrack1G(int size,int mask,int row,int n_left,int n_down,int n_right,int steps);
long backTrack2G(int size,int mask,int row,int n_left,int n_down,int n_right,int steps);
/****************************************/
void NQueenG(int size,int mask,int row,int steps);
__device__ int symmetryOps_bitmap_gpu(int si,unsigned int *d_aBoard,int *d_aT,int *d_aS);
//関数宣言 GPU
bool InitCUDA();
//関数宣言 SGPU
__global__ void sgpu_cuda_kernel(int size,int mark,unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,unsigned int* results,int totalCond);
long long sgpu_solve_nqueen_cuda(int size,int steps);
//関数宣言 CPU/GPU
__device__ __host__ void rotate_bitmap(int bf[],int af[],int si);
__device__ __host__ void vMirror_bitmap(int bf[],int af[],int si);
__device__ __host__ int intncmp(int lt[],int rt[],int n);
__device__ __host__ int rh(int a,int size);
//関数宣言
void TimeFormat(clock_t utime,char *form);
long getUnique();
long getTotal();
void symmetryOps_bitmap(int si);
//関数宣言 CPU
void solve_nqueen(int size,int mask, int row,int* left,int* down,int* right,int* bitmap);
void NQueen(int size,int mask);
//関数宣言 CPUR
void solve_nqueenr(int size,int mask, int row,int left,int down,int right);
void NQueenR(int size,int mask);
//関数宣言 通常版
void NQueenD(int size,int mask);
void NQueenDR(int size,int mask,int row,int left,int down,int right,int ex1,int ex2);
//
//
__global__ void sgpu_cuda_kernel(int size,int mark,unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,unsigned int* results,int totalCond){
  const int tid=threadIdx.x;
  const int bid=blockIdx.x;
  const int idx=bid*blockDim.x+tid;
  __shared__ unsigned int down[THREAD_NUM][10];
  __shared__ unsigned int left[THREAD_NUM][10];
  __shared__ unsigned int right[THREAD_NUM][10];
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  __shared__ unsigned int sum[THREAD_NUM];
  const unsigned int mask=(1<<size)-1;
  int total=0;
  int row=0;
  unsigned int bit;
  if(idx<totalCond){
    down[tid][row]=totalDown[idx];
    left[tid][row]=totalLeft[idx];
    right[tid][row]=totalRight[idx];
    bitmap[tid][row]=down[tid][row]|left[tid][row]|right[tid][row];
    while(row>=0){
      if((bitmap[tid][row]&mask)==mask){row--;}
      else{
        bit=(bitmap[tid][row]+1)&~bitmap[tid][row];
        bitmap[tid][row]|=bit;
        if((bit&mask)!=0){
          if(row+1==mark){total++;row--;}
          else{
            down[tid][row+1]=down[tid][row]|bit;
            left[tid][row+1]=(left[tid][row]|bit)<<1;
            right[tid][row+1]=(right[tid][row]|bit)>>1;
            bitmap[tid][row+1]=(down[tid][row+1]|left[tid][row+1]|right[tid][row+1]);
            row++;
          }
        }else{row--;}
      }
    }
    sum[tid]=total;
  }else{sum[tid]=0;} 
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){sum[tid]+=sum[tid+64];} 
  __syncthreads();if(tid<32){sum[tid]+=sum[tid+32];} 
  __syncthreads();if(tid<16){sum[tid]+=sum[tid+16];} 
  __syncthreads();if(tid<8){sum[tid]+=sum[tid+8];} 
  __syncthreads();if(tid<4){sum[tid]+=sum[tid+4];} 
  __syncthreads();if(tid<2){sum[tid]+=sum[tid+2];} 
  __syncthreads();if(tid<1){sum[tid]+=sum[tid+1];} 
  __syncthreads();if(tid==0){results[bid]=sum[0];}
}
//
long long sgpu_solve_nqueen_cuda(int size,int steps) {
  unsigned int down[32];
  unsigned int left[32];
  unsigned int right[32];
  unsigned int bitmap[32];
  unsigned int bit;
  if(size<=0||size>32){return 0;}
  unsigned int* totalDown=new unsigned int[steps];
  unsigned int* totalLeft=new unsigned int[steps];
  unsigned int* totalRight=new unsigned int[steps];
  unsigned int* results=new unsigned int[steps];
  unsigned int* downCuda;
  unsigned int* leftCuda;
  unsigned int* rightCuda;
  unsigned int* resultsCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);
  const unsigned int mask=(1<<size)-1;
  const unsigned int mark=size>11?size-10:2;
  long long total=0;
  int totalCond=0;
  int row=0;
  down[0]=0;
  left[0]=0;
  right[0]=0;
  bitmap[0]=0;
  bool matched=false;
  for(int col=0;col<size/2;col++){
    bit=(1<<col);
    bitmap[0]|=bit;
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=(down[1]|left[1]|right[1]);
    row=1;
    while(row>0){
      if((bitmap[row]&mask)==mask){row--;}
      else{
        bit=(bitmap[row]+1)&~bitmap[row];
        bitmap[row]|=bit;
        if((bit&mask)!=0){
          down[row+1]=down[row]|bit;
          left[row+1]=(left[row]|bit)<<1;
          right[row+1]=(right[row]|bit)>>1;
          bitmap[row+1]=(down[row+1]|left[row+1]|right[row+1]);
          row++;
          if(row==mark){
            totalDown[totalCond]=down[row];
            totalLeft[totalCond]=left[row];
            totalRight[totalCond]=right[row];
            totalCond++;
            if(totalCond==steps){
              if(matched){
                hipMemcpy(results,resultsCuda,
                    sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
                matched=false;
              }
              hipMemcpy(downCuda,totalDown,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              matched=true;
              totalCond=0;
            }
            row--;
          }
        }else{row--;}
      }
    }
  }
  if(matched){
    hipMemcpy(results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
    matched=false;
  }
  hipMemcpy(downCuda,totalDown,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  /** backTrack+bitmap*/
  sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
    >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
  hipMemcpy(results,resultsCuda,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}	
  total*=2;


  if(size%2==1){
    matched=false;
    totalCond=0;
    bit=(1<<(size-1)/2);
    bitmap[0]|=bit;
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=(down[1]|left[1]|right[1]);
    row=1;
    while(row>0){
      if((bitmap[row]&mask)==mask){row--;}
      else{
        bit=(bitmap[row]+1)&~bitmap[row];
        bitmap[row]|=bit;
        if((bit&mask)!=0){
          down[row+1]=down[row]|bit;
          left[row+1]=(left[row]|bit)<<1;
          right[row+1]=(right[row]|bit)>>1;
          bitmap[row+1]=(down[row+1]|left[row+1]|right[row+1]);
          row++;
          if(row==mark){
            totalDown[totalCond]=down[row];
            totalLeft[totalCond]=left[row];
            totalRight[totalCond]=right[row];
            totalCond++;
            if(totalCond==steps){
              if(matched){
                hipMemcpy(results,resultsCuda,
                    sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
                matched=false;
              }
              hipMemcpy(downCuda,totalDown,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              matched=true;
              totalCond=0;
            }
            row--;
          }
        }else{row--;}
      }
    }
    if(matched){
      hipMemcpy(results,resultsCuda,
          sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
      for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
      matched=false;
    }
    hipMemcpy(downCuda,totalDown,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(leftCuda,totalLeft,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(rightCuda,totalRight,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    /** backTrack+bitmap*/
    sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
      >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
    hipMemcpy(results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
  }
  hipFree(downCuda);
  hipFree(leftCuda);
  hipFree(rightCuda);
  hipFree(resultsCuda);
  delete[] totalDown;
  delete[] totalLeft;
  delete[] totalRight;
  delete[] results;
  return total;
}
//
/** CUDA 初期化 **/
bool InitCUDA(){
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
//hh:mm:ss.ms形式に処理時間を出力
void TimeFormat(clock_t utime,char *form){
  int dd,hh,mm;
  float ftime,ss;
  ftime=(float)utime/CLOCKS_PER_SEC;
  mm=(int)ftime/60;
  ss=ftime-(int)(mm*60);
  dd=mm/(24*60);
  mm=mm%(24*60);
  hh=mm/60;
  mm=mm%60;
  if(dd)
    sprintf(form,"%4d %02d:%02d:%05.2f",dd,hh,mm,ss);
  else if(hh)
    sprintf(form,"     %2d:%02d:%05.2f",hh,mm,ss);
  else if(mm)
    sprintf(form,"        %2d:%05.2f",mm,ss);
  else
    sprintf(form,"           %5.2f",ss);
}
//
__device__ __host__
int rh(int a,int sz){
  int tmp=0;
  for(int i=0;i<=sz;i++){
    if(a&(1<<i)){ return tmp|=(1<<(sz-i)); }
  }
  return tmp;
}
//
__device__ __host__
void vMirror_bitmap(int bf[],int af[],int si){
  int score ;
  for(int i=0;i<si;i++) {
    score=bf[i];
    af[i]=rh(score,si-1);
  }
}
//
__device__ __host__
void rotate_bitmap(int bf[],int af[],int si){
  for(int i=0;i<si;i++){
    int t=0;
    for(int j=0;j<si;j++){
      t|=((bf[j]>>i)&1)<<(si-j-1); // x[j] の i ビット目を
    }
    af[i]=t;                        // y[i] の j ビット目にする
  }
}
//
__device__ __host__
int intncmp(unsigned int lt[],int rt[],int n){
  int rtn=0;
  for(int k=0;k<n;k++){
    rtn=lt[k]-rt[k];
    if(rtn!=0){
      break;
    }
  }
  return rtn;
}
//
long getUnique(){
  return COUNT2+COUNT4+COUNT8;
}
//
long getTotal(){
  return COUNT2*2+COUNT4*4+COUNT8*8;
}
//
void symmetryOps_bitmap(int si){
  int nEquiv;
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<si;i++){ aT[i]=aBoard[i];}
  rotate_bitmap(aT,aS,si);    //時計回りに90度回転
  int k=intncmp(aBoard,aS,si);
  if(k>0)return;
  if(k==0){ nEquiv=2;}else{
    rotate_bitmap(aS,aT,si);  //時計回りに180度回転
    k=intncmp(aBoard,aT,si);
    if(k>0)return;
    if(k==0){ nEquiv=4;}else{
      rotate_bitmap(aT,aS,si);//時計回りに270度回転
      k=intncmp(aBoard,aS,si);
      if(k>0){ return;}
      nEquiv=8;
    }
  }
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<si;i++){ aS[i]=aBoard[i];}
  vMirror_bitmap(aS,aT,si);   //垂直反転
  k=intncmp(aBoard,aT,si);
  if(k>0){ return; }
  if(nEquiv>2){             //-90度回転 対角鏡と同等
    rotate_bitmap(aT,aS,si);
    k=intncmp(aBoard,aS,si);
    if(k>0){return;}
    if(nEquiv>4){           //-180度回転 水平鏡像と同等
      rotate_bitmap(aS,aT,si);
      k=intncmp(aBoard,aT,si);
      if(k>0){ return;}       //-270度回転 反対角鏡と同等
      rotate_bitmap(aT,aS,si);
      k=intncmp(aBoard,aS,si);
      if(k>0){ return;}
    }
  }
  if(nEquiv==2){COUNT2++;}
  if(nEquiv==4){COUNT4++;}
  if(nEquiv==8){COUNT8++;}
}
//

//
__device__
int symmetryOps_bitmap_gpu(int si,unsigned int *d_aBoard,int *d_aT,int *d_aS){
  int nEquiv;
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<si;i++){ d_aT[i]=d_aBoard[i];}
  rotate_bitmap(d_aT,d_aS,si);    //時計回りに90度回転
  int k=intncmp(d_aBoard,d_aS,si);
  //printf("1_k:%d\n",k);
  if(k>0)return 0;
  if(k==0){ nEquiv=2;}else{
    rotate_bitmap(d_aS,d_aT,si);  //時計回りに180度回転
    k=intncmp(d_aBoard,d_aT,si);
    //printf("2_k:%d\n",k);
    if(k>0)return 0;
    if(k==0){ nEquiv=4;}else{
      rotate_bitmap(d_aT,d_aS,si);//時計回りに270度回転
      k=intncmp(d_aBoard,d_aS,si);
      //printf("3_k:%d\n",k);
      if(k>0){ return 0;}
      nEquiv=8;
    }
  }
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<si;i++){ d_aS[i]=d_aBoard[i];}
  vMirror_bitmap(d_aS,d_aT,si);   //垂直反転
  k=intncmp(d_aBoard,d_aT,si);
  //printf("4_k:%d\n",k);
  if(k>0){ return 0; }
  if(nEquiv>2){             //-90度回転 対角鏡と同等
    rotate_bitmap(d_aT,d_aS,si);
    k=intncmp(d_aBoard,d_aS,si);
    //printf("5_k:%d\n",k);
    if(k>0){return 0;}
    if(nEquiv>4){           //-180度回転 水平鏡像と同等
      rotate_bitmap(d_aS,d_aT,si);
      k=intncmp(d_aBoard,d_aT,si);
      //printf("6_k:%d\n",k);

      if(k>0){ return 0;}       //-270度回転 反対角鏡と同等
      rotate_bitmap(d_aT,d_aS,si);
      k=intncmp(d_aBoard,d_aS,si);
      //printf("7_k:%d\n",k);

      if(k>0){ return 0;}
    }
  }
  //printf("eq:%d\n",nEquiv);
  return nEquiv;
  
}

//GPU
/***07 引数 追加に伴いコメント*********************/
//__global__ 
//void cuda_kernel(int size,int mark,unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,unsigned int* d_results,int totalCond)
/************************/
/***07 引数 d_uniq,t_aBoard,h_row追加 uniq,aBoardのため*********************/
__global__
void cuda_kernel(
    int size,
    int mark,
    unsigned int* totalDown,
    unsigned int* totalLeft,
    unsigned int* totalRight,
    unsigned int* d_results,
    unsigned int* d_uniq,
    register int totalCond,
    unsigned int* t_aBoard,
    int h_row,
    int* aT,
    int* aS)
{
  /************************/
  register const unsigned int mask=(1<<size)-1;
  register unsigned int total=0;
  /***07 uniq,aBoard追加*********************/
  register unsigned int unique=0;
  //int aT[MAX];
  //int aS[MAX];
  /************************/
  //row=0となってるが1行目からやっているわけではなく
  //mask行目以降からスタート 
  //n=8 なら mask==2 なので そこからスタート
  register int row=0;
  register unsigned int bit;
  //
  //スレッド
  //
  //ブロック内のスレッドID
  register unsigned const int tid=threadIdx.x;
  //グリッド内のブロックID
  register unsigned const int bid=blockIdx.x;
  //全体通してのID
  register unsigned const int idx=bid*blockDim.x+tid;
  //
  //シェアードメモリ
  //
  //sharedメモリを使う ブロック内スレッドで共有
  //10固定なのは現在のmask設定で
  //GPUで実行するのは最大10だから
  //THREAD_NUMはブロックあたりのスレッド数
  __shared__ unsigned int down[THREAD_NUM][10];
  down[tid][row]=totalDown[idx];
  __shared__ unsigned int left[THREAD_NUM][10];
  left[tid][row]=totalLeft[idx];
  __shared__ unsigned int right[THREAD_NUM][10];
  right[tid][row]=totalRight[idx];
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  //down,left,rightからbitmapを出す
  bitmap[tid][row]
    =mask&~(
         down[tid][row]
        |left[tid][row]
        |right[tid][row]);
  __shared__ unsigned int sum[THREAD_NUM];
  /***07 aBoard,uniq追加*********************/
  /***07 shared に変更 **********************/
  __shared__ unsigned int usum[THREAD_NUM];
  /***07 registerに変更 *********************/
  register int c_aT[MAX];
  register int c_aS[MAX];
  register unsigned int c_aBoard[MAX];
  /************************/
  //
  //余分なスレッドは動かさない 
  //GPUはsteps数起動するがtotalCond以上は空回しする
  if(idx<totalCond){
    //totalDown,totalLeft,totalRightの情報を
    //down,left,rightに詰め直す 
    //CPU で詰め込んだ t_はsteps個あるが
    //ブロック内ではブロックあたりのスレッド数に限定
    //されるので idxでよい
    //
    /***07 aBoard追加*********************/
    for(int i=0;i<h_row;i++){
      //c_aBoard[tid][i]=t_aBoard[idx][i];   
      c_aBoard[i]=t_aBoard[idx*h_row+i]; //２次元配列だが1次元的に利用  
    }
    /************************/
    /**07 スカラー変数に置き換えた**********/
    register unsigned int bitmap_tid_row;
    register unsigned int down_tid_row;
    register unsigned int left_tid_row;
    register unsigned int right_tid_row;
    while(row>=0){
      //bitmap[tid][row]をスカラー変数に置き換え
      bitmap_tid_row=bitmap[tid][row];
      down_tid_row=down[tid][row];
      left_tid_row=left[tid][row];
      right_tid_row=right[tid][row];
    /***************************************/
      //
      //bitmap[tid][row]=00000000 クイーンを
      //どこにも置けないので1行上に戻る
      /**07 スカラー変数に置き換えた**********/
      //if(bitmap[tid][row]==0){
      if(bitmap_tid_row==0){
      /***************************************/
        row--;
      }else{
        //クイーンを置く
        //bitmap[tid][row]
        //  ^=bit
        //  =(-bitmap[tid][row]&bitmap[tid][row]);
        //置く場所があるかどうか
        /***07 aBoard追加*********************/
        bitmap[tid][row]
          ^=c_aBoard[row+h_row]
          =bit
          /**07 スカラー変数に置き換えた**********/
          //=(-bitmap[tid][row]&bitmap[tid][row]);       
          =(-bitmap_tid_row&bitmap_tid_row);       
          /***************************************/
        /************************/
        if((bit&mask)!=0){
          //最終行?最終行から１個前の行まで
          //無事到達したら 加算する
          if(row+1==mark){
           /***07 symmetryOpsの処理を追加*********************/
           int s=symmetryOps_bitmap_gpu(size,c_aBoard,c_aT,c_aS); 
           //int s=0;//=symmetryOps_bitmap_gpu(size,c_aBoard[tid],aT,aS); 
           if(s!=0){
           //print(size); //print()でTOTALを++しない
           //ホストに戻す配列にTOTALを入れる
           //スレッドが１つの場合は配列は１個
              unique++; 
              total+=s;   //対称解除で得られた解数を加算
           }
           /************************/
           /***07 symmetryOpsの処理追加に伴いコメント*********************/
           //total++;
           /************************/
            row--;
          }else{
            int rowP=row+1;
            /**07スカラー変数に置き換えてregister対応 ****/
            //down[tid][rowP]=down[tid][row]|bit;
            down[tid][rowP]=down_tid_row|bit;
            //left[tid][rowP]=(left[tid][row]|bit)<<1;
            left[tid][rowP]=(left_tid_row|bit)<<1;
            //right[tid][rowP]=(right[tid][row]|bit)>>1;
            right[tid][rowP]=(right_tid_row|bit)>>1;
            bitmap[tid][rowP]
              =mask&~(
                  down[tid][rowP]
                  |left[tid][rowP]
                  |right[tid][rowP]);
            row++;
          }
        }else{
          //置く場所がなければ１個上に
          row--;
        }
      }
    }
    //最後sum[tid]に加算する
    sum[tid]=total;
    /***07 uniq追加*********************/
    usum[tid]=unique;
    /************************/
  }else{
    //totalCond未満は空回しするのでtotalは加算しない
    sum[tid]=0;
    /***07 uniq追加*********************/
    usum[tid]=0;
    /************************/
  } 
  //__syncthreads()でブロック内のスレッド間の同期
  //全てのスレッドが__syncthreads()に辿り着くのを待つ
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){
    sum[tid]+=sum[tid+64];
    /***07 uniq追加*********************/
    usum[tid]+=usum[tid+64];
    /************************/
  }
  __syncthreads();if(tid<32){
    sum[tid]+=sum[tid+32];
    /***07 uniq追加*********************/
    usum[tid]+=usum[tid+32];
    /************************/
  } 
  __syncthreads();if(tid<16){
    sum[tid]+=sum[tid+16];
    /***07 uniq追加*********************/
    usum[tid]+=usum[tid+16];
    /************************/  
  } 
  __syncthreads();if(tid<8){
    sum[tid]+=sum[tid+8];
    /***07 uniq追加*********************/
    usum[tid]+=usum[tid+8];
    /************************/
  } 
  __syncthreads();if(tid<4){
    sum[tid]+=sum[tid+4];
    /***07 uniq追加*********************/
    usum[tid]+=usum[tid+4];
    /************************/  
  } 
  __syncthreads();if(tid<2){
    sum[tid]+=sum[tid+2];
    /***07 uniq追加*********************/
    usum[tid]+=usum[tid+2];
    /************************/  
  } 
  __syncthreads();if(tid<1){
    sum[tid]+=sum[tid+1];
    /***07 uniq追加*********************/
    usum[tid]+=usum[tid+1];
    /************************/  
  } 
  __syncthreads();if(tid==0){
    d_results[bid]=sum[0];
    /****07 uniq追加********************/
    d_uniq[bid]=usum[0];
    /************************/
  }
}
//
// GPU
/***10 関数名をsolve_nqueen_cudaからbackTrack1,2に変更するためコメント*********************/
//long solve_nqueen_cuda(int size,int mask,int row,int n_left,int n_down,int n_right,int steps)
/************************/
/***10 関数名をsolve_nqueen_cudaからbackTrack1,2に変更する*********************/
long backTrack1G(int size,int mask,int row,int n_left,int n_down,int n_right,int steps) //NQueenに相当
/************************/
{
  //何行目からGPUで行くか。ここの設定は変更可能、設定値を多くするほどGPUで並行して動く
  /***08 クイーンを２行目まで固定で置くためmarkが3以上必要のためコメント*********************/
  //const unsigned int mark=size>11?size-10:2;
  /************************/
  /***08 クイーンを２行目まで固定で置くためmarkが3以上必要*********************/
  const unsigned int mark=size>12?size-10:3;
  /************************/  
  const unsigned int h_mark=row;
  long total=0;
  int totalCond=0;
  bool matched=false;
  //host
  unsigned int down[32];  down[row]=n_down;
  unsigned int right[32]; right[row]=n_right;
  unsigned int left[32];  left[row]=n_left;
  //bitmapを配列で持つことにより
  //stackを使わないで1行前に戻れる
  unsigned int bitmap[32];
  //bitmap[row]=(left[row]|down[row]|right[row]);
  /***07 aBoard追加に伴いbit処理をGPU*********************/
  bitmap[row]=mask&~(left[row]|down[row]|right[row]);
  /************************/
  unsigned int bit;

  //unsigned int* totalDown=new unsigned int[steps];
  unsigned int* totalDown;
  hipHostMalloc((void**) &totalDown,sizeof(int)*steps, hipHostMallocDefault);

  //unsigned int* totalLeft=new unsigned int[steps];
  unsigned int* totalLeft;
  hipHostMalloc((void**) &totalLeft,sizeof(int)*steps, hipHostMallocDefault);

  //unsigned int* totalRight=new unsigned int[steps];
  unsigned int* totalRight;
  hipHostMalloc((void**) &totalRight,sizeof(int)*steps, hipHostMallocDefault);

  //unsigned int* h_results=new unsigned int[steps];
  unsigned int* h_results;
  hipHostMalloc((void**) &h_results,sizeof(int)*steps, hipHostMallocDefault);

  /***07 uniq,aBoard追加*********************/
  //unsigned int* h_uniq=new unsigned int[steps];
  unsigned int* h_uniq;
  hipHostMalloc((void**) &h_uniq,sizeof(int)*steps, hipHostMallocDefault);

  //unsigned int* t_aBoard=new unsigned int[steps*mark];
  unsigned int* t_aBoard;
  hipHostMalloc((void**) &t_aBoard,sizeof(int)*steps*mark, hipHostMallocDefault);
  /************************/
  //device
  unsigned int* downCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  unsigned int* leftCuda;
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  unsigned int* rightCuda;
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  unsigned int* resultsCuda;
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);
  /***07 uniq,aBoard追加*********************/
  unsigned int* d_aT;
  hipMalloc((void**) &d_aT,sizeof(int)*steps*MAX);
  unsigned int* d_aS;
  hipMalloc((void**) &d_aS,sizeof(int)*steps*MAX);

  unsigned int* d_uniq;
  hipMalloc((void**) &d_uniq,sizeof(int)*steps/THREAD_NUM);
  unsigned int* d_aBoard;
  hipMalloc((void**) &d_aBoard,sizeof(int)*steps*mark);
  /************************/
  //12行目までは3行目までCPU->row==mark以下で 3行目までの
  //down,left,right情報を totalDown,totalLeft,totalRight
  //に格納
  //する->3行目以降をGPUマルチスレッドで実行し結果を取得
  //13行目以降はCPUで実行する行数が１個ずつ増えて行く
  //例えばn15だとrow=5までCPUで実行し、
  //それ以降はGPU(現在の設定だとGPUでは最大10行実行する
  //ようになっている)
  //while(row>=0) {
  register int rowP=0;
  while(row>=h_mark) {
    //bitmap[row]=00000000 クイーンを
    //どこにも置けないので1行上に戻る
    /***07 aBoard追加に伴いbit操作変更*********************/
    //06GPU こっちのほうが優秀
    if(bitmap[row]==0){ row--; }
    /************************/
    /***07 aBoard追加に伴いbit操作変更でコメント*********************/
    //06SGPU
    //if((bitmap[row]&mask)==mask){row--;}
    /************************/
    else{//おける場所があれば進む
      //06SGPU
      /***07 aBoard追加に伴いbit操作変更でコメント*********************/
      //bit=(bitmap[row]+1)&~bitmap[row];
      //bitmap[row]|=bit;
      /************************/
      //06GPU こっちのほうが優秀
      //bitmap[row]^=bit=(-bitmap[row]&bitmap[row]); //クイーンを置く
      /***07 aBoard追加*********************/
      bitmap[row]^=aBoard[row]=bit=(-bitmap[row]&bitmap[row]);
      /************************/ 
      if((bit&mask)!=0){//置く場所があれば先に進む
        rowP=row+1;
        down[rowP]=down[row]|bit;
        left[rowP]=(left[row]|bit)<<1;
        right[rowP]=(right[row]|bit)>>1;
        /***07 aBoard追加に伴いbit操作変更でコメント*********************/
        //bitmap[rowP]=(down[rowP]|left[rowP]|right[rowP]);
        /************************/
        /***07 aBoard追加に伴いbit操作変更*********************/
        bitmap[rowP]=mask&~(down[rowP]|left[rowP]|right[rowP]);
        /************************/
        row++;
        if(row==mark){
          //3行目(mark)にクイーンを１個ずつ置いていって、
          //down,left,right情報を格納、
          //その次の行へは進まない。その行で可能な場所にクイー
          //ン置き終わったらGPU並列実行
          //totalCond がthreadIdになる 各スレッドに down,left,right情報を渡す
          //row=2(13行目以降は増えていく。例えばn15だとrow=5)の情報を
          //totalDown,totalLeft,totalRightに格納する
          totalDown[totalCond]=down[row];
          totalLeft[totalCond]=left[row];
          totalRight[totalCond]=right[row];
          /***07 aBoard追加*********************/
          for(int i=0;i<mark;i++){
            //t_aBoard[totalCond][i]=aBoard[i];
            t_aBoard[totalCond*mark+i]=aBoard[i];
          }
          /************************/
          //スレッド数をインクリメントする
          totalCond++;
          //最大GPU数に達してしまったら一旦ここでGPUを実行する。stepsはGPUの同
          //時並行稼働数を制御
          //nの数が少ないうちはtotalCondがstepsを超えることはないがnの数が増え
          //て行くと超えるようになる。
          //ここではtotalCond==stepsの場合だけこの中へ         
          if(totalCond==steps){
            //matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUか
            //ら出たらmatched=trueになってる
            if(matched){
              hipMemcpy(h_results,resultsCuda,
                  sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
              /***07 uniq追加*********************/
              hipMemcpy(h_uniq,d_uniq,
                  sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
              /************************/
              for(int col=0;col<steps/THREAD_NUM;col++){
                total+=h_results[col];
                /****07 uniq追加********************/
                UNIQUE+=h_uniq[col];
                /************************/                                        
              }
              matched=false;
            }
            hipMemcpy(downCuda,totalDown,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(leftCuda,totalLeft,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(rightCuda,totalRight,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            /***07 aBoard追加*********************/
            hipMemcpy(d_aBoard,t_aBoard,
                sizeof(int)*totalCond*mark,hipMemcpyHostToDevice);
            /************************/
            /** backTrack+bitmap*/
            //size-mark は何行GPUを実行するか totalCondはスレッド数
            /***07 d_uniq,d_aBoard,row追加に伴いコメント*********************/
            //cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
            //  >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
            /************************/
            /***07 d_uniq,d_aBoard,row追加*********************/
            cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
              >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,d_aBoard,row,aT,aS);
            /************************/          
            //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われ
            //るのはtotalCondの数だけでそれ以外は空回しになる
            //GPU内でカウントしているので、GPUから出たらmatched=trueになってる
            matched=true;
            //totalCond==stepsルートでGPUを実行したらスレッドをまた0から開始す
            //る(これによりなんどもsteps数分だけGPUを起動できる)
            totalCond=0;           
          }
          //totalDown,totalLeft,totalRightに情報を格納したら1行上に上がる
          //これを繰り返すことにより row=2で可能な場所全てにクイーンを置いて
          //totalDown,totalLeft,totalRightに情報を格納する
          row--;
        }
      }else{
        //置く場所がなければ上に上がる。row==mark行に達するまではCPU側で普通に
        //nqueenをやる
        row--;
      }
    }
  }
  //matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUから出たら
  //matched=trueになってる
  if(matched){
    hipMemcpy(h_results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    /***07 uniq追加*********************/
    hipMemcpy(h_uniq,d_uniq,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    /************************/
   
    for(int col=0;col<steps/THREAD_NUM;col++){
      total+=h_results[col];
      /***07 uniq追加*********************/
      UNIQUE+=h_uniq[col];
      /************************/    
    }
    matched=false;
  }
  hipMemcpy(downCuda,totalDown,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  /***07 aBoard追加*********************/
  hipMemcpy(d_aBoard,t_aBoard,
      sizeof(int)*totalCond*mark,hipMemcpyHostToDevice);
  /************************/ 
  /** backTrack+bitmap*/
  //size-mark は何行GPUを実行するか totalCondはスレッド数
  //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われるのは
  //totalCondの数だけでそれ以外は空回しになる
  /***07 d_uniq,d_aBoard,mark追加に伴いコメント*********************/   
  //cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
  //  >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
  /***07 d_uniq,d_aBoard,mark追加*********************/  
  cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
    >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,d_aBoard,mark,aT,aS);
  /************************/
  hipMemcpy(h_results,resultsCuda,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  /***07 uniq追加*********************/
  hipMemcpy(h_uniq,d_uniq,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  /************************/   
  for(int col=0;col<steps/THREAD_NUM;col++){
    total+=h_results[col];
    /***07 uniq追加*********************/
    UNIQUE+=h_uniq[col];
    /************************/    
  }
  //
  hipFree(downCuda);
  hipFree(leftCuda);
  hipFree(rightCuda);
  hipFree(resultsCuda);
  /***07 uniq,aBoard追加 cudaFreeHostへ変更**/
  hipFree(d_uniq);
  hipFree(d_aBoard);
  //delete[] totalDown;
  hipHostFree(totalDown);
  //delete[] totalLeft;
  hipHostFree(totalLeft);
  //delete[] totalRight;
  hipHostFree(totalRight);
  //delete[] h_results;
  hipHostFree(h_results);
  //delete[] h_uniq;
  hipHostFree(h_uniq);
  //delete[] t_aBoard;
  hipHostFree(t_aBoard);
  /************************/
  return total;
}
/***10 関数名をsolve_nqueen_cudaからbackTrack1,2に変更するためコメント*********************/
//long solve_nqueen_cuda(int size,int mask,int row,int n_left,int n_down,int n_right,int steps)
/************************/
/***10 関数名をsolve_nqueen_cudaからbackTrack1,2に変更する*********************/
long backTrack2G(int size,int mask,int row,int n_left,int n_down,int n_right,int steps) //NQueenに相当
/************************/
{
  //何行目からGPUで行くか。ここの設定は変更可能、設定値を多くするほどGPUで並行して動く
  /***08 クイーンを２行目まで固定で置くためmarkが3以上必要のためコメント*********************/
  //const unsigned int mark=size>11?size-10:2;
  /************************/
  /***08 クイーンを２行目まで固定で置くためmarkが3以上必要*********************/
  const unsigned int mark=size>12?size-10:3;
  /************************/  
  const unsigned int h_mark=row;
  long total=0;
  int totalCond=0;
  bool matched=false;
  //host
  unsigned int down[32];  down[row]=n_down;
  unsigned int right[32]; right[row]=n_right;
  unsigned int left[32];  left[row]=n_left;
  //bitmapを配列で持つことにより
  //stackを使わないで1行前に戻れる
  unsigned int bitmap[32];
  //bitmap[row]=(left[row]|down[row]|right[row]);
  /***07 aBoard追加に伴いbit処理をGPU*********************/
  bitmap[row]=mask&~(left[row]|down[row]|right[row]);
  /************************/
  unsigned int bit;

  //unsigned int* totalDown=new unsigned int[steps];
  unsigned int* totalDown;
  hipHostMalloc((void**) &totalDown,sizeof(int)*steps, hipHostMallocDefault);

  //unsigned int* totalLeft=new unsigned int[steps];
  unsigned int* totalLeft;
  hipHostMalloc((void**) &totalLeft,sizeof(int)*steps, hipHostMallocDefault);

  //unsigned int* totalRight=new unsigned int[steps];
  unsigned int* totalRight;
  hipHostMalloc((void**) &totalRight,sizeof(int)*steps, hipHostMallocDefault);

  //unsigned int* h_results=new unsigned int[steps];
  unsigned int* h_results;
  hipHostMalloc((void**) &h_results,sizeof(int)*steps, hipHostMallocDefault);

  /***07 uniq,aBoard追加*********************/
  //unsigned int* h_uniq=new unsigned int[steps];
  unsigned int* h_uniq;
  hipHostMalloc((void**) &h_uniq,sizeof(int)*steps, hipHostMallocDefault);

  //unsigned int* t_aBoard=new unsigned int[steps*mark];
  unsigned int* t_aBoard;
  hipHostMalloc((void**) &t_aBoard,sizeof(int)*steps*mark, hipHostMallocDefault);
  /************************/
  //device
  unsigned int* downCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  unsigned int* leftCuda;
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  unsigned int* rightCuda;
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  unsigned int* resultsCuda;
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);
  /***07 uniq,aBoard追加*********************/
  unsigned int* d_aT;
  hipMalloc((void**) &d_aT,sizeof(int)*steps*MAX);
  unsigned int* d_aS;
  hipMalloc((void**) &d_aS,sizeof(int)*steps*MAX);

  unsigned int* d_uniq;
  hipMalloc((void**) &d_uniq,sizeof(int)*steps/THREAD_NUM);
  unsigned int* d_aBoard;
  hipMalloc((void**) &d_aBoard,sizeof(int)*steps*mark);
  /************************/
  //12行目までは3行目までCPU->row==mark以下で 3行目までの
  //down,left,right情報を totalDown,totalLeft,totalRight
  //に格納
  //する->3行目以降をGPUマルチスレッドで実行し結果を取得
  //13行目以降はCPUで実行する行数が１個ずつ増えて行く
  //例えばn15だとrow=5までCPUで実行し、
  //それ以降はGPU(現在の設定だとGPUでは最大10行実行する
  //ようになっている)
  //while(row>=0) {
  register int rowP=0;
  while(row>=h_mark) {
    //bitmap[row]=00000000 クイーンを
    //どこにも置けないので1行上に戻る
    /***07 aBoard追加に伴いbit操作変更*********************/
    //06GPU こっちのほうが優秀
    if(bitmap[row]==0){ row--; }
    /************************/
    /***07 aBoard追加に伴いbit操作変更でコメント*********************/
    //06SGPU
    //if((bitmap[row]&mask)==mask){row--;}
    /************************/
    else{//おける場所があれば進む
      //06SGPU
      /***07 aBoard追加に伴いbit操作変更でコメント*********************/
      //bit=(bitmap[row]+1)&~bitmap[row];
      //bitmap[row]|=bit;
      /************************/
      //06GPU こっちのほうが優秀
      //bitmap[row]^=bit=(-bitmap[row]&bitmap[row]); //クイーンを置く
      /***07 aBoard追加*********************/
      bitmap[row]^=aBoard[row]=bit=(-bitmap[row]&bitmap[row]);
      /************************/ 
      if((bit&mask)!=0){//置く場所があれば先に進む
        rowP=row+1;
        down[rowP]=down[row]|bit;
        left[rowP]=(left[row]|bit)<<1;
        right[rowP]=(right[row]|bit)>>1;
        /***07 aBoard追加に伴いbit操作変更でコメント*********************/
        //bitmap[rowP]=(down[rowP]|left[rowP]|right[rowP]);
        /************************/
        /***07 aBoard追加に伴いbit操作変更*********************/
        bitmap[rowP]=mask&~(down[rowP]|left[rowP]|right[rowP]);
        /************************/
        row++;
        if(row==mark){
          //3行目(mark)にクイーンを１個ずつ置いていって、
          //down,left,right情報を格納、
          //その次の行へは進まない。その行で可能な場所にクイー
          //ン置き終わったらGPU並列実行
          //totalCond がthreadIdになる 各スレッドに down,left,right情報を渡す
          //row=2(13行目以降は増えていく。例えばn15だとrow=5)の情報を
          //totalDown,totalLeft,totalRightに格納する
          totalDown[totalCond]=down[row];
          totalLeft[totalCond]=left[row];
          totalRight[totalCond]=right[row];
          /***07 aBoard追加*********************/
          for(int i=0;i<mark;i++){
            //t_aBoard[totalCond][i]=aBoard[i];
            t_aBoard[totalCond*mark+i]=aBoard[i];
          }
          /************************/
          //スレッド数をインクリメントする
          totalCond++;
          //最大GPU数に達してしまったら一旦ここでGPUを実行する。stepsはGPUの同
          //時並行稼働数を制御
          //nの数が少ないうちはtotalCondがstepsを超えることはないがnの数が増え
          //て行くと超えるようになる。
          //ここではtotalCond==stepsの場合だけこの中へ         
          if(totalCond==steps){
            //matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUか
            //ら出たらmatched=trueになってる
            if(matched){
              hipMemcpy(h_results,resultsCuda,
                  sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
              /***07 uniq追加*********************/
              hipMemcpy(h_uniq,d_uniq,
                  sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
              /************************/
              for(int col=0;col<steps/THREAD_NUM;col++){
                total+=h_results[col];
                /****07 uniq追加********************/
                UNIQUE+=h_uniq[col];
                /************************/                                        
              }
              matched=false;
            }
            hipMemcpy(downCuda,totalDown,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(leftCuda,totalLeft,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(rightCuda,totalRight,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            /***07 aBoard追加*********************/
            hipMemcpy(d_aBoard,t_aBoard,
                sizeof(int)*totalCond*mark,hipMemcpyHostToDevice);
            /************************/
            /** backTrack+bitmap*/
            //size-mark は何行GPUを実行するか totalCondはスレッド数
            /***07 d_uniq,d_aBoard,row追加に伴いコメント*********************/
            //cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
            //  >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
            /************************/
            /***07 d_uniq,d_aBoard,row追加*********************/
            cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
              >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,d_aBoard,row,aT,aS);
            /************************/          
            //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われ
            //るのはtotalCondの数だけでそれ以外は空回しになる
            //GPU内でカウントしているので、GPUから出たらmatched=trueになってる
            matched=true;
            //totalCond==stepsルートでGPUを実行したらスレッドをまた0から開始す
            //る(これによりなんどもsteps数分だけGPUを起動できる)
            totalCond=0;           
          }
          //totalDown,totalLeft,totalRightに情報を格納したら1行上に上がる
          //これを繰り返すことにより row=2で可能な場所全てにクイーンを置いて
          //totalDown,totalLeft,totalRightに情報を格納する
          row--;
        }
      }else{
        //置く場所がなければ上に上がる。row==mark行に達するまではCPU側で普通に
        //nqueenをやる
        row--;
      }
    }
  }
  //matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUから出たら
  //matched=trueになってる
  if(matched){
    hipMemcpy(h_results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    /***07 uniq追加*********************/
    hipMemcpy(h_uniq,d_uniq,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    /************************/
   
    for(int col=0;col<steps/THREAD_NUM;col++){
      total+=h_results[col];
      /***07 uniq追加*********************/
      UNIQUE+=h_uniq[col];
      /************************/    
    }
    matched=false;
  }
  hipMemcpy(downCuda,totalDown,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  /***07 aBoard追加*********************/
  hipMemcpy(d_aBoard,t_aBoard,
      sizeof(int)*totalCond*mark,hipMemcpyHostToDevice);
  /************************/ 
  /** backTrack+bitmap*/
  //size-mark は何行GPUを実行するか totalCondはスレッド数
  //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われるのは
  //totalCondの数だけでそれ以外は空回しになる
  /***07 d_uniq,d_aBoard,mark追加に伴いコメント*********************/   
  //cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
  //  >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
  /***07 d_uniq,d_aBoard,mark追加*********************/  
  cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
    >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,d_aBoard,mark,aT,aS);
  /************************/
  hipMemcpy(h_results,resultsCuda,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  /***07 uniq追加*********************/
  hipMemcpy(h_uniq,d_uniq,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  /************************/   
  for(int col=0;col<steps/THREAD_NUM;col++){
    total+=h_results[col];
    /***07 uniq追加*********************/
    UNIQUE+=h_uniq[col];
    /************************/    
  }
  //
  hipFree(downCuda);
  hipFree(leftCuda);
  hipFree(rightCuda);
  hipFree(resultsCuda);
  /***07 uniq,aBoard追加 cudaFreeHostへ変更**/
  hipFree(d_uniq);
  hipFree(d_aBoard);
  //delete[] totalDown;
  hipHostFree(totalDown);
  //delete[] totalLeft;
  hipHostFree(totalLeft);
  //delete[] totalRight;
  hipHostFree(totalRight);
  //delete[] h_results;
  hipHostFree(h_results);
  //delete[] h_uniq;
  hipHostFree(h_uniq);
  //delete[] t_aBoard;
  hipHostFree(t_aBoard);
  /************************/
  return total;
}
//
void NQueenG(int size,int steps){
  int bit=0;
  int mask=(1<<size)-1;
  //10では枝借りはまだしないのでTOPBIT,SIDEMASK,LASTMASK,ENDBITは使用しない
  //backtrack1
  //1行め右端 0
  int col=0;
  aBoard[0]=bit=(1<<col);
  int left=bit<<1;
  int down=bit;
  int right=bit>>1;
  //2行目は右から3列目から左端から2列目まで
  for(int col_j=2;col_j<size-1;col_j++){
      aBoard[1]=bit=(1<<col_j);
      /***10 関数名をbacktrackにするためコメント*********************/
      //TOTAL+=solve_nqueen_cuda(size,mask,2,(left|bit)<<1,(down|bit),(right|bit)>>1,steps);
      /************************/
      /***10 関数名をbacktrackにする*********************/
      TOTAL+=backTrack1G(size,mask,2,(left|bit)<<1,(down|bit),(right|bit)>>1,steps);
      /************************/
  }
  //backtrack2
  //1行目右から2列目から
  //偶数個は1/2 n=8 なら 1,2,3 奇数個は1/2+1 n=9 なら 1,2,3,4
  for(int col=1,col2=size-2;col<col2;col++,col2--){
      aBoard[0]=bit=(1<<col);
      /***10 関数名をbacktrackにするためコメント*********************/
      //TOTAL+=solve_nqueen_cuda(size,mask,1,bit<<1,bit,bit>>1,steps);
      /************************/
      /***10 関数名をbacktrackにする*********************/
      TOTAL+=backTrack2G(size,mask,1,bit<<1,bit,bit>>1,steps);
      /************************/
  }
   /***09 backtrack1,2を意識し１行目右端とそれ以外で処理を分けるためコメント*********************/
  //偶数、奇数共通 右側半分だけクイーンを置く
	//int lim=(size%2==0)?size/2:sizeE/2;
  //for(int col=0;col<lim;col++){
  //  bit=(1<<col);
  //  TOTAL+=solve_nqueen_cuda(size,mask,1,bit<<1,bit,bit>>1,steps);
  //}
  //ミラーなのでTOTALを２倍する
  //TOTAL=TOTAL*2;
  //奇数の場合はさらに中央にクイーンを置く
  //if(size%2==1){
  //  bit=(1<<(sizeE)/2);
  //  TOTAL+=solve_nqueen_cuda(size,mask,1,bit<<1,bit,bit>>1,steps);
  //}
  /************************/

}

//
//CPU 非再帰版 backTrack2
void backTrack2_NR(int size,int mask,int row,int left,int down,int right){
  int bitmap,bit;
  int b[100], *p=b;
  int odd=size&1; //奇数:1 偶数:0
  for(int i=0;i<(1+odd);++i){
    bitmap=0;
    if(0==i){
      int half=size>>1; // size/2
      bitmap=(1<<half)-1;
    }else{
      bitmap=1<<(size>>1);
      // down[1]=bitmap;
      // right[1]=(bitmap>>1);
      // left[1]=(bitmap<<1);
      // pnStack=aStack+1;
      // *pnStack++=0;
    }
mais1:bitmap=mask&~(left|down|right);
      if(row==size){
        if(!bitmap){
          aBoard[row]=bitmap;
          symmetryOps_bitmap(size);
        }
      }else{
        if(bitmap){
outro:bitmap^=aBoard[row]=bit=-bitmap&bitmap;
      if(bitmap){
        *p++=left;
        *p++=down;
        *p++=right;
      }
      *p++=bitmap;
      row++;
      left=(left|bit)<<1;
      down=down|bit;
      right=(right|bit)>>1;
      goto mais1;
      //Backtrack2(y+1, (left | bit)<<1, down | bit, (right | bit)>>1);
volta:if(p<=b)
        return;
      row--;
      bitmap=*--p;
      if(bitmap){
        right=*--p;
        down=*--p;
        left=*--p;
        goto outro;
      }else{
        goto volta;
      }
        }
      }
      goto volta;
  }
}

//CPU 非再帰版 ロジックメソッド
void backTrack2(int size,int mask, int row,int h_left,int h_down,int h_right){
	unsigned int left[size];
    unsigned int down[size];
	unsigned int right[size];
    unsigned int bitmap[size];
	left[row]=h_left;
	down[row]=h_down;
	right[row]=h_right;
	bitmap[row]=mask&~(left[row]|down[row]|right[row]);
    unsigned int bit;
    unsigned int sizeE=size-1;
    int mark=row;
    //固定していれた行より上はいかない
    while(row>=mark){//row=1 row>=1, row=2 row>=2
      if(bitmap[row]==0){
        --row;
      }else{
        bitmap[row]^=aBoard[row]=bit=(-bitmap[row]&bitmap[row]); 
        if((bit&mask)!=0){
          if(row==sizeE){
            symmetryOps_bitmap(size);
            --row;
          }else{
            int n=row++;
            left[row]=(left[n]|bit)<<1;
            down[row]=down[n]|bit;
            right[row]=(right[n]|bit)>>1;
            bitmap[row]=mask&~(left[row]|down[row]|right[row]);
          }
        }else{
           --row;
        }
      }  
    }
}
//
void backTrack1(int size,int mask, int row,int h_left,int h_down,int h_right){
	unsigned int left[size];
    unsigned int down[size];
	unsigned int right[size];
    unsigned int bitmap[size];
	left[row]=h_left;
	down[row]=h_down;
	right[row]=h_right;
	bitmap[row]=mask&~(left[row]|down[row]|right[row]);
    unsigned int bit;
    unsigned int sizeE=size-1;
    int mark=row;
    //固定していれた行より上はいかない
    while(row>=mark){//row=1 row>=1, row=2 row>=2
      if(bitmap[row]==0){
        --row;
      }else{
        bitmap[row]^=aBoard[row]=bit=(-bitmap[row]&bitmap[row]); 
        if((bit&mask)!=0){
          if(row==sizeE){
            symmetryOps_bitmap(size);
            --row;
          }else{
            int n=row++;
            left[row]=(left[n]|bit)<<1;
            down[row]=down[n]|bit;
            right[row]=(right[n]|bit)>>1;
            bitmap[row]=mask&~(left[row]|down[row]|right[row]);
          }
        }else{
           --row;
        }
      }  
    }
}
void NQueen(int size,int mask){
  int bit=0;
  //10では枝借りはまだしないのでTOPBIT,SIDEMASK,LASTMASK,ENDBITは使用しない
  //backtrack1
  //1行め右端 0
  int col=0;
  aBoard[0]=bit=(1<<col);
  int left=bit<<1;
  int down=bit;
  int right=bit>>1;
  //2行目は右から3列目から左端から2列目まで
  for(int col_j=2;col_j<size-1;col_j++){
      aBoard[1]=bit=(1<<col_j);
      backTrack1(size,mask,2,(left|bit)<<1,(down|bit),(right|bit)>>1);
  }
  //backtrack2
  //1行目右から2列目から
  //偶数個は1/2 n=8 なら 1,2,3 奇数個は1/2+1 n=9 なら 1,2,3,4
  for(int col=1,col2=size-2;col<col2;col++,col2--){
      aBoard[0]=bit=(1<<col);
      backTrack2(size,mask,1,bit<<1,bit,bit>>1);
  }
}
//
void backTrackR1(int size,int mask, int row,int left,int down,int right){
 int bitmap=0;
 int bit=0;
 int sizeE=size-1;
 bitmap=(mask&~(left|down|right));
 if(row==sizeE){
   if(bitmap){
     aBoard[row]=(-bitmap&bitmap);
     symmetryOps_bitmap(size);
   }
  }else{
    while(bitmap){
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap);
      backTrackR1(size,mask,row+1,(left|bit)<<1, down|bit,(right|bit)>>1);
    }
  }
}
//
void backTrackR2(int size,int mask, int row,int left,int down,int right){
 int bitmap=0;
 int bit=0;
 int sizeE=size-1;
 bitmap=(mask&~(left|down|right));
 if(row==sizeE){
   if(bitmap){
     aBoard[row]=(-bitmap&bitmap);
     symmetryOps_bitmap(size);
   }
  }else{
    while(bitmap){
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap);
      backTrackR2(size,mask,row+1,(left|bit)<<1, down|bit,(right|bit)>>1);
    }
  }
}
//CPUR 再帰版 ロジックメソッド
void NQueenR(int size,int mask){
  int bit=0;
  //10では枝借りはまだしないのでTOPBIT,SIDEMASK,LASTMASK,ENDBITは使用しない
  //backtrack1
  //1行め右端 0
  int col=0;
  aBoard[0]=bit=(1<<col);
  int left=bit<<1;
  int down=bit;
  int right=bit>>1;
  //2行目は右から3列目から左端から2列目まで
  for(int col_j=2;col_j<size-1;col_j++){
      aBoard[1]=bit=(1<<col_j);
      backTrackR1(size,mask,2,(left|bit)<<1,(down|bit),(right|bit)>>1);
  }
  //backtrack2
  //1行目右から2列目から
  //偶数個は1/2 n=8 なら 1,2,3 奇数個は1/2+1 n=9 なら 1,2,3,4
  for(int col=1,col2=size-2;col<col2;col++,col2--){
      aBoard[0]=bit=(1<<col);
      backTrackR2(size,mask,1,bit<<1,bit,bit>>1);
  }
}
//CPU 非再帰版 backTrack2
void backTrack2D_NR(int size,int mask,int row,int left,int down,int right){
	int bitmap,bit;
	int b[100], *p=b;
  int odd=size&1; //奇数:1 偶数:0
  for(int i=0;i<(1+odd);++i){
    bitmap=0;
    if(0==i){
      int half=size>>1; // size/2
      bitmap=(1<<half)-1;
    }else{
      bitmap=1<<(size>>1);
      // down[1]=bitmap;
      // right[1]=(bitmap>>1);
      // left[1]=(bitmap<<1);
      // pnStack=aStack+1;
      // *pnStack++=0;
    }
    mais1:bitmap=mask&~(left|down|right);
    if(row==size){
      if(!bitmap){
        aBoard[row]=bitmap;
        symmetryOps_bitmap(size);
      }
    }else{
      if(bitmap){
        outro:bitmap^=aBoard[row]=bit=-bitmap&bitmap;
        if(bitmap){
          *p++=left;
          *p++=down;
          *p++=right;
        }
        *p++=bitmap;
        row++;
        left=(left|bit)<<1;
        down=down|bit;
        right=(right|bit)>>1;
        goto mais1;
        //Backtrack2(y+1, (left | bit)<<1, down | bit, (right | bit)>>1);
        volta:if(p<=b)
          return;
        row--;
        bitmap=*--p;
        if(bitmap){
          right=*--p;
          down=*--p;
          left=*--p;
          goto outro;
        }else{
          goto volta;
        }
      }
    }
    goto volta;
  }
}
//CPU 非再帰版 backTrack1
void backTrack1D_NR(int size,int mask,int row,int left,int down,int right){
  int bitmap,bit;
  int b[100], *p=b;
  int sizeE=size-1;
  int odd=size&1; //奇数:1 偶数:0
  for(int i=0;i<(1+odd);++i){
    bitmap=0;
    if(0==i){
      int half=size>>1; // size/2
      bitmap=(1<<half)-1;
    }else{
      bitmap=1<<(size>>1);
      // down[1]=bitmap;
      // right[1]=(bitmap>>1);
      // left[1]=(bitmap<<1);
      // pnStack=aStack+1;
      // *pnStack++=0;
    }
    b1mais1:bitmap=mask&~(left|down|right);
    if(row==sizeE){
      if(bitmap){
        aBoard[row]=bitmap;
        symmetryOps_bitmap(size);
      }
    }else{
      if(bitmap){
        b1outro:bitmap^=aBoard[row]=bit=-bitmap&bitmap;
        if(bitmap){
          *p++=left;
          *p++=down;
          *p++=right;
        }
        *p++=bitmap;
        row++;
        left=(left|bit)<<1;
        down=down|bit;
        right=(right|bit)>>1;
        goto b1mais1;
        //Backtrack1(y+1, (left | bit)<<1, down | bit, (right | bit)>>1);
        b1volta:if(p<=b)
          return;
        row--;
        bitmap=*--p;
        if(bitmap){
          right=*--p;
          down=*--p;
          left=*--p;
          goto b1outro;
        }else{
          goto b1volta;
        }
      }
    }
    goto b1volta;
  }
}
//CPU 非再帰版 ロジックメソッド
void NQueenD(int size,int mask){
  int bit;
  TOPBIT=1<<(size-1);
  aBoard[0]=1;
  for(BOUND1=2;BOUND1<size-1;BOUND1++){
    aBoard[1]=bit=(1<<BOUND1);
    //backTrack1(size,mask,2,(2|bit)<<1,(1|bit),(bit>>1));
    backTrack1D_NR(size,mask,2,(2|bit)<<1,(1|bit),(bit>>1));
  }
  SIDEMASK=LASTMASK=(TOPBIT|1);
  ENDBIT=(TOPBIT>>1);
  for(BOUND1=1,BOUND2=size-2;BOUND1<BOUND2;BOUND1++,BOUND2--){
    aBoard[0]=bit=(1<<BOUND1);
    //backTrack1(size,mask,1,bit<<1,bit,bit>>1);
    backTrack2D_NR(size,mask,1,bit<<1,bit,bit>>1);
    LASTMASK|=LASTMASK>>1|LASTMASK<<1;
    ENDBIT>>=1;
  }
}
//
//
void backTrack2D(int size,int mask,int row,int left,int down,int right){
  int bit;
  int bitmap=mask&~(left|down|right); /* 配置可能フィールド */
  if(row==size){
    aBoard[row]=bitmap; //symmetryOpsの時は代入します。
    symmetryOps_bitmap(size);
  }else{
    while(bitmap){
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap); //最も下位の１ビットを抽出
      backTrack2D(size,mask,row+1,(left|bit)<<1,down|bit,(right|bit)>>1);
    }
  }
}
//
void backTrack1D(int size,int mask,int row,int left,int down,int right){
  int bit;
  int bitmap=mask&~(left|down|right);   //BOUNDで対応済み
  if(row==size){
    aBoard[row]=bitmap; //symmetryOpsの時は代入します。
    symmetryOps_bitmap(size);
  }else{
    while(bitmap){
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap); //最も下位の１ビットを抽出
      backTrack1D(size,mask,row+1,(left|bit)<<1,down|bit,(right|bit)>>1);
    }
  }
}
//

//CPUR 再帰版 ロジックメソッド
void NQueenDR(int size,int mask){
  int bit;
  TOPBIT=1<<(size-1);
  aBoard[0]=1;
  for(BOUND1=2;BOUND1<size-1;BOUND1++){
    aBoard[1]=bit=(1<<BOUND1);
    backTrack1D(size,mask,2,(2|bit)<<1,(1|bit),(bit>>1));
  }
  SIDEMASK=LASTMASK=(TOPBIT|1);
  ENDBIT=(TOPBIT>>1);
  for(BOUND1=1,BOUND2=size-2;BOUND1<BOUND2;BOUND1++,BOUND2--){
    aBoard[0]=bit=(1<<BOUND1);
    backTrack2D(size,mask,1,bit<<1,bit,bit>>1);
    LASTMASK|=LASTMASK>>1|LASTMASK<<1;
    ENDBIT>>=1;
  }
}
//
//メインメソッド
int main(int argc,char** argv) {
  bool cpu=false,cpur=false,gpu=false,sgpu=false;
  int argstart=1,steps=24576;
  /** パラメータの処理 */
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='r'||argv[1][1]=='R'){cpur=true;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){gpu=true;}
    else if(argv[1][1]=='s'||argv[1][1]=='S'){sgpu=true;}
    else
      cpur=true;
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g|-r]\n",argv[0]);
    printf("  -c: CPU only\n");
    printf("  -r: CPUR only\n");
    printf("  -g: GPU only\n");
    printf("  -s: SGPU only\n");
    printf("Default to 8 queen\n");
  }
  /** 出力と実行 */  
  if(cpu){
    printf("\n\n１０．CPU 非再帰 クイーンの位置による分岐BOUND1,2\n");
  }else if(cpur){
    printf("\n\n１０．CPUR 再帰 クイーンの位置による分岐BOUND1,2\n");
  }else if(gpu){
    printf("\n\n１０．GPU 非再帰 クイーンの位置による分岐BOUND1,2\n");
  }else if(sgpu){
    printf("\n\n１０．SGPU 非再帰 バックトラック＋ビットマップ\n");
  }

  if(cpu||cpur){
    printf("%s\n"," N:        Total       Unique        hh:mm:ss.ms");
    clock_t st;           //速度計測用
    char t[20];           //hh:mm:ss.msを格納
    int min=4; int targetN=17;
    int mask;
    for(int i=min;i<=targetN;i++){
      //TOTAL=0; UNIQUE=0;
      COUNT2=COUNT4=COUNT8=0;
      mask=(1<<i)-1;
      st=clock();
      //初期化は不要です
      /** 非再帰は-1で初期化 */
      // for(int j=0;j<=targetN;j++){
      //   aBoard[j]=-1;
      // }
      //
      //再帰
      if(cpur){ 
        NQueenR(i,mask);
        //printf("通常版\n");
        //NQueenDR(i,mask);//通常版
      }
      //非再帰
      if(cpu){ 
        NQueen(i,mask); 
        //printf("通常版\n");
        //NQueenD(i,mask);//通常版
      }
      //
      TimeFormat(clock()-st,t); 
      printf("%2d:%13ld%16ld%s\n",i,getTotal(),getUnique(),t);
    }
  }
  if(gpu||sgpu){
    if(!InitCUDA()){return 0;}
    int min=4;int targetN=17;
    struct timeval t0;struct timeval t1;int ss;int ms;int dd;
    printf("%s\n"," N:        Total      Unique      dd:hh:mm:ss.ms");
    for(int i=min;i<=targetN;i++){
      TOTAL=0;
      UNIQUE=0;
      gettimeofday(&t0,NULL);   // 計測開始
      if(gpu){
        TOTAL=0;
        UNIQUE=0;
        NQueenG(i,steps);
      }else if(sgpu){
        TOTAL=sgpu_solve_nqueen_cuda(i,steps);
      }
      gettimeofday(&t1,NULL);   // 計測終了
      if (t1.tv_usec<t0.tv_usec) {
        dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      } else {
        dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%13ld%16ld%4.2d:%02d:%02d:%02d.%02d\n", i,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    }
  }
  return 0;
}
