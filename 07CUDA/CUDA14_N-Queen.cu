
/**
 CUDAで学ぶアルゴリズムとデータ構造
 ステップバイステップでＮ−クイーン問題を最適化
 一般社団法人  共同通信社  情報技術局  鈴木  維一郎(suzuki.iichiro@kyodonews.jp)

 コンパイルと実行
 $ nvcc -O3 CUDA**_N-Queen.cu && ./a.out (-c|-r|-g|-s)
                    -c:cpu 
                    -r cpu再帰 
                    -g GPU 
                    -s SGPU(サマーズ版と思われる)


６．バックトラック＋ビットマップ

   ビット演算を使って高速化 状態をビットマップにパックし、処理する
   単純なバックトラックよりも２０〜３０倍高速
 
 　ビットマップであれば、シフトにより高速にデータを移動できる。
  フラグ配列ではデータの移動にO(N)の時間がかかるが、ビットマップであればO(1)
  フラグ配列のように、斜め方向に 2*N-1の要素を用意するのではなく、Nビットで充
  分。

 　配置可能なビット列を flags に入れ、-flags & flags で順にビットを取り出し処理。
 　バックトラックよりも２０−３０倍高速。
 
 ===================
 考え方 1
 ===================

 　Ｎ×ＮのチェスボードをＮ個のビットフィールドで表し、ひとつの横列の状態をひと
 つのビットフィールドに対応させます。(クイーンが置いてある位置のビットをONに
 する)
 　そしてバックトラッキングは0番目のビットフィールドから「下に向かって」順にい
 ずれかのビット位置をひとつだけONにして進めていきます。

 
 -----Q--    00000100 0番目のビットフィールド
 ---Q----    00010000 1番目のビットフィールド
 ------ Q-   00000010 2番目のビットフィールド
  Q-------   10000000 3番目のビットフィールド
 -------Q    00000001 4番目のビットフィールド
 -Q------    01000000 5番目のビットフィールド
 ---- Q---   00001000 6番目のビットフィールド
 -- Q-----   00100000 7番目のビットフィールド


 ===================
 考え方 2
 ===================

 次に、効き筋をチェックするためにさらに３つのビットフィールドを用意します。

 1. 左下に効き筋が進むもの: left 
 2. 真下に効き筋が進むもの: down
 3. 右下に効き筋が進むもの: right

次に、斜めの利き筋を考えます。
 上図の場合、
 1列目の右斜め上の利き筋は 3 番目(0x08)
 2列目の右斜め上の利き筋は 2 番目(0x04) になります。
 この値は 0 列目のクイーンの位置 0x10 を 1 ビットずつ「右シフト」すれば求める
 ことができます。
 また、左斜め上の利き筋の場合、1 列目では 5 番目(0x20) で 2 列目では 6 番目(0x40)
になるので、今度は 1 ビットずつ「左シフト」すれば求めることができます。

つまり、右シフトの利き筋を right、左シフトの利き筋を left で表すことで、クイー
ンの効き筋はrightとleftを1 ビットシフトするだけで求めることができるわけです。

  *-------------
 |. . . . . .
 |. . . -3. .  0x02 -|
 |. . -2. . .  0x04  |(1 bit 右シフト right)
 |. -1. . . .  0x08 -|
 |Q . . . . .  0x10 ←(Q の位置は 4   down)
 |. +1. . . .  0x20 -| 
 |. . +2. . .  0x40  |(1 bit 左シフト left)  
 |. . . +3. .  0x80 -|
  *-------------
  図：斜めの利き筋のチェック

 n番目のビットフィールドからn+1番目のビットフィールドに探索を進めるときに、そ
 の３つのビットフィールドとn番目のビットフィールド(bit)とのOR演算をそれぞれ行
 います。leftは左にひとつシフトし、downはそのまま、rightは右にひとつシフトして
 n+1番目のビットフィールド探索に渡してやります。

 left :(left |bit)<<1
 right:(right|bit)>>1
 down :   down|bit


 ===================
 考え方 3
 ===================

   n+1番目のビットフィールドの探索では、この３つのビットフィールドをOR演算した
 ビットフィールドを作り、それがONになっている位置は効き筋に当たるので置くことが
 できない位置ということになります。次にその３つのビットフィールドをORしたビッ
 トフィールドをビット反転させます。つまり「配置可能なビットがONになったビットフィー
 ルド」に変換します。そしてこの配置可能なビットフィールドを bitmap と呼ぶとして、
 次の演算を行なってみます。
 
 bit=-bitmap & bitmap;//一番右のビットを取り出す
 
   この演算式の意味を理解するには負の値がコンピュータにおける２進法ではどのよう
 に表現されているのかを知る必要があります。負の値を２進法で具体的に表わしてみる
 と次のようになります。
 
  00000011   3
  00000010   2
  00000001   1
  00000000   0
  11111111  -1
  11111110  -2
  11111101  -3
 
   正の値nを負の値-nにするときは、nをビット反転してから+1されています。そして、
 例えばn=22としてnと-nをAND演算すると下のようになります。nを２進法で表したときの
 一番下位のONビットがひとつだけ抽出される結果が得られるのです。極めて簡単な演算
 によって1ビット抽出を実現させていることが重要です。
 
      00010110   22
  AND 11101010  -22
 ------------------
      00000010
 
   さて、そこで下のようなwhile文を書けば、このループは bitmap のONビットの数の
 回数だけループすることになります。配置可能なパターンをひとつずつ全く無駄がなく
 生成されることになります。
 
 while(bitmap) {
     bit=-bitmap & bitmap;
     bitmap ^= bit;
     //ここでは配置可能なパターンがひとつずつ生成される(bit) 
 }

 実行結果
$ nvcc -O3 CUDA06_N-Queen.cu  && ./a.out -r
６．CPUR 再帰 バックトラック＋ビットマップ
 N:        Total       Unique        hh:mm:ss.ms
 4:            2               0            0.00
 5:           10               0            0.00
 6:            4               0            0.00
 7:           40               0            0.00
 8:           92               0            0.00
 9:          352               0            0.00
10:          724               0            0.00
11:         2680               0            0.00
12:        14200               0            0.01
13:        73712               0            0.04
14:       365596               0            0.19
15:      2279184               0            1.24
16:     14772512               0            7.79
17:     95815104               0           57.57

$ nvcc -O3 CUDA06_N-Queen.cu  && ./a.out -c
６．CPU 非再帰 バックトラック＋ビットマップ
 N:        Total       Unique        hh:mm:ss.ms
 4:            2               0            0.00
 5:           10               0            0.00
 6:            4               0            0.00
 7:           40               0            0.00
 8:           92               0            0.00
 9:          352               0            0.00
10:          724               0            0.00
11:         2680               0            0.00
12:        14200               0            0.01
13:        73712               0            0.04
14:       365596               0            0.21
15:      2279184               0            1.40
16:     14772512               0            8.78
17:     95815104               0         1:05.00

$ nvcc -O3 CUDA06_N-Queen.cu  && ./a.out -s
６．SGPU 非再帰 バックトラック＋ビットマップ
 N:        Total      Unique      dd:hh:mm:ss.ms
 4:            2               0  00:00:00:00.02
 5:           10               0  00:00:00:00.00
 6:            4               0  00:00:00:00.00
 7:           40               0  00:00:00:00.00
 8:           92               0  00:00:00:00.00
 9:          352               0  00:00:00:00.00
10:          724               0  00:00:00:00.00
11:         2680               0  00:00:00:00.01
12:        14200               0  00:00:00:00.02
13:        73712               0  00:00:00:00.03
14:       365596               0  00:00:00:00.08
15:      2279184               0  00:00:00:00.48
16:     14772512               0  00:00:00:02.41
17:     95815104               0  00:00:00:18.30

$ nvcc -O3 CUDA06_N-Queen.cu  && ./a.out -g
６．GPU 非再帰 バックトラック＋ビットマップ
 N:        Total      Unique      dd:hh:mm:ss.ms
 4:            2               0  00:00:00:00.02
 5:           10               0  00:00:00:00.00
 6:            4               0  00:00:00:00.00
 7:           40               0  00:00:00:00.00
 8:           92               0  00:00:00:00.00
 9:          352               0  00:00:00:00.00
10:          724               0  00:00:00:00.00
11:         2680               0  00:00:00:00.01
12:        14200               0  00:00:00:00.05
13:        73712               0  00:00:00:00.07
14:       365596               0  00:00:00:00.07
15:      2279184               0  00:00:00:00.37
16:     14772512               0  00:00:00:02.30
17:     95815104               0  00:00:00:18.07
*/

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define THREAD_NUM		96
#define MAX 27
//変数宣言
long TOTAL=0;         //CPU,CPUR
long UNIQUE=0;        //CPU,CPUR
//関数宣言 GPU
__global__ void cuda_kernel(int size,int mark,unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,unsigned int* d_results,int totalCond);
long long solve_nqueen_cuda(int size,int steps);
void NQueenG(int size,int mask,int row,int steps);
//関数宣言 SGPU
__global__ void sgpu_cuda_kernel(int size,int mark,unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,unsigned int* results,int totalCond);
long long sgpu_solve_nqueen_cuda(int size,int steps); 
//関数宣言 CPU
void TimeFormat(clock_t utime,char *form);
//関数宣言 CPU
void solve_nqueen(int size,int mask, int row,int* left,int* down,int* right,int* bitmap);
void NQueen(int size,int mask);
//関数宣言 CPUR
void solve_nqueenr(int size,int mask, int row,int left,int down,int right);
void NQueenR(int size,int mask,int row,int left,int down,int right);
//関数宣言 通常版
//  非再帰
void NQueenD(int size,int mask,int row);
//  再帰
void NQueenDR(int size,int mask,int row,int left,int down,int right);
//
//GPU
__global__ 
void cuda_kernel(
    int size,
    int mark,
    unsigned int* totalDown,
    unsigned int* totalLeft,
    unsigned int* totalRight,
    unsigned int* d_results,
    int totalCond)
{
  register const unsigned int mask=(1<<size)-1;
  register unsigned int total=0;
  //row=0となってるが1行目からやっているわけではなく
  //mask行目以降からスタート 
  //n=8 なら mask==2 なので そこからスタート
  register int row=0;
  register unsigned int bit;
  //
  //スレッド
  //
  //ブロック内のスレッドID
  register unsigned const int tid=threadIdx.x;
  //グリッド内のブロックID
  register unsigned const int bid=blockIdx.x;
  //全体通してのID
  register unsigned const int idx=bid*blockDim.x+tid;
  //
  //シェアードメモリ
  //
  //sharedメモリを使う ブロック内スレッドで共有
  //10固定なのは現在のmask設定で
  //GPUで実行するのは最大10だから
  //THREAD_NUMはブロックあたりのスレッド数
  __shared__ unsigned int down[THREAD_NUM][10];
  down[tid][row]=totalDown[idx];
  __shared__ unsigned int left[THREAD_NUM][10];
  left[tid][row]=totalLeft[idx];
  __shared__ unsigned int right[THREAD_NUM][10];
  right[tid][row]=totalRight[idx];
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  //down,left,rightからbitmapを出す
  bitmap[tid][row]
    =mask&~(
         down[tid][row]
        |left[tid][row]
        |right[tid][row]);
  __shared__ unsigned int sum[THREAD_NUM];
  //
  //余分なスレッドは動かさない 
  //GPUはsteps数起動するがtotalCond以上は空回しする
  if(idx<totalCond){
    //totalDown,totalLeft,totalRightの情報を
    //down,left,rightに詰め直す 
    //CPU で詰め込んだ t_はsteps個あるが
    //ブロック内ではブロックあたりのスレッド数に限定
    //されるので idxでよい
    //
    /**06 スカラー変数に置き換えた**********/
    register unsigned int bitmap_tid_row;
    register unsigned int down_tid_row;
    register unsigned int left_tid_row;
    register unsigned int right_tid_row;
    while(row>=0){
      //bitmap[tid][row]をスカラー変数に置き換え
      bitmap_tid_row=bitmap[tid][row];
      down_tid_row=down[tid][row];
      left_tid_row=left[tid][row];
      right_tid_row=right[tid][row];
    /***************************************/
      //
      //bitmap[tid][row]=00000000 クイーンを
      //どこにも置けないので1行上に戻る
      /**06 スカラー変数に置き換えた**********/
      //if(bitmap[tid][row]==0){
      if(bitmap_tid_row==0){
      /***************************************/
        row--;
      }else{
        //クイーンを置く
        bitmap[tid][row]
          ^=bit
          /**06 スカラー変数に置き換えた**********/
          //=(-bitmap[tid][row]&bitmap[tid][row]);
          =(-bitmap_tid_row&bitmap_tid_row);       
          /***************************************/
        //置く場所があるかどうか
        if((bit&mask)!=0){
          //最終行?最終行から１個前の行まで
          //無事到達したら 加算する
          if(row+1==mark){
           total++;
            row--;
          }else{
            int rowP=row+1;
            /**07スカラー変数に置き換えてregister対応 ****/
            //down[tid][rowP]=down[tid][row]|bit;
            down[tid][rowP]=down_tid_row|bit;
            //left[tid][rowP]=(left[tid][row]|bit)<<1;
            left[tid][rowP]=(left_tid_row|bit)<<1;
            //right[tid][rowP]=(right[tid][row]|bit)>>1;
            right[tid][rowP]=(right_tid_row|bit)>>1;
            bitmap[tid][rowP]
              =mask&~(
                  down[tid][rowP]
                  |left[tid][rowP]
                  |right[tid][rowP]);
            row++;
          }
        }else{
          //置く場所がなければ１個上に
          row--;
        }
      }
    }
    //最後sum[tid]に加算する
    sum[tid]=total;
  }else{
    //totalCond未満は空回しするのでtotalは加算しない
    sum[tid]=0;
  } 
  //__syncthreads()でブロック内のスレッド間の同期
  //全てのスレッドが__syncthreads()に辿り着くのを待つ
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){
    sum[tid]+=sum[tid+64];
  }
  __syncthreads();if(tid<32){
    sum[tid]+=sum[tid+32];
  } 
  __syncthreads();if(tid<16){
    sum[tid]+=sum[tid+16];
  } 
  __syncthreads();if(tid<8){
    sum[tid]+=sum[tid+8];
  } 
  __syncthreads();if(tid<4){
    sum[tid]+=sum[tid+4];
  } 
  __syncthreads();if(tid<2){
    sum[tid]+=sum[tid+2];
  } 
  __syncthreads();if(tid<1){
    sum[tid]+=sum[tid+1];
  } 
  __syncthreads();if(tid==0){
    d_results[bid]=sum[0];
  }
}
//
// GPU
long solve_nqueen_cuda(int size,int mask,int row,int n_left,int n_down,int n_right,int steps)
{
  //何行目からGPUで行くか。ここの設定は変更可能、設定値を多くするほどGPUで並行して動く
  const unsigned int mark=size>11?size-10:2;
  const unsigned int h_mark=row;
  long total=0;
  int totalCond=0;
  bool matched=false;
  //host
  unsigned int down[32];  down[row]=n_down;
  unsigned int right[32]; right[row]=n_right;
  unsigned int left[32];  left[row]=n_left;
  //bitmapを配列で持つことにより
  //stackを使わないで1行前に戻れる
  unsigned int bitmap[32];
  //bitmap[row]=(left[row]|down[row]|right[row]);
  /***06 bit処理をGPU*********************/
  bitmap[row]=mask&~(left[row]|down[row]|right[row]);
  /************************/
  unsigned int bit;
  //unsigned int* totalDown=new unsigned int[steps];
  unsigned int* totalDown;
  hipHostMalloc((void**) &totalDown,sizeof(int)*steps, hipHostMallocDefault);
  //unsigned int* totalLeft=new unsigned int[steps];
  unsigned int* totalLeft;
  hipHostMalloc((void**) &totalLeft,sizeof(int)*steps, hipHostMallocDefault);
  //unsigned int* totalRight=new unsigned int[steps];
  unsigned int* totalRight;
  hipHostMalloc((void**) &totalRight,sizeof(int)*steps, hipHostMallocDefault);
  //unsigned int* h_results=new unsigned int[steps];
  unsigned int* h_results;
  hipHostMalloc((void**) &h_results,sizeof(int)*steps, hipHostMallocDefault);
  //device
  unsigned int* downCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  unsigned int* leftCuda;
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  unsigned int* rightCuda;
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  unsigned int* resultsCuda;
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);
  //12行目までは3行目までCPU->row==mark以下で 3行目までの
  //down,left,right情報を totalDown,totalLeft,totalRight
  //に格納
  //する->3行目以降をGPUマルチスレッドで実行し結果を取得
  //13行目以降はCPUで実行する行数が１個ずつ増えて行く
  //例えばn15だとrow=5までCPUで実行し、
  //それ以降はGPU(現在の設定だとGPUでは最大10行実行する
  //ようになっている)
  //while(row>=0) {
  register int rowP=0;
  while(row>=h_mark) {
    //bitmap[row]=00000000 クイーンを
    //どこにも置けないので1行上に戻る
    /***06 bit操作変更*********************/
    //06GPU こっちのほうが優秀
    if(bitmap[row]==0){ row--; }
    /************************/
    /***06 bit操作変更でコメント*********************/
    //06SGPU
    //if((bitmap[row]&mask)==mask){row--;}
    /************************/
    else{//おける場所があれば進む
      //06SGPU
      /***06 bit操作変更でコメント*********************/
      //bit=(bitmap[row]+1)&~bitmap[row];
      //bitmap[row]|=bit;
      /************************/
      //06GPU こっちのほうが優秀
      bitmap[row]^=bit=(-bitmap[row]&bitmap[row]); //クイーンを置く
      if((bit&mask)!=0){//置く場所があれば先に進む
        rowP=row+1;
        down[rowP]=down[row]|bit;
        left[rowP]=(left[row]|bit)<<1;
        right[rowP]=(right[row]|bit)>>1;
        /***06 bit操作変更でコメント*********************/
        //bitmap[rowP]=(down[rowP]|left[rowP]|right[rowP]);
        /************************/
        /***06 bit操作変更*********************/
        bitmap[rowP]=mask&~(down[rowP]|left[rowP]|right[rowP]);
        /************************/
        row++;
        if(row==mark){
          //3行目(mark)にクイーンを１個ずつ置いていって、
          //down,left,right情報を格納、
          //その次の行へは進まない。その行で可能な場所にクイー
          //ン置き終わったらGPU並列実行
          //totalCond がthreadIdになる 各スレッドに down,left,right情報を渡す
          //row=2(13行目以降は増えていく。例えばn15だとrow=5)の情報を
          //totalDown,totalLeft,totalRightに格納する
          totalDown[totalCond]=down[row];
          totalLeft[totalCond]=left[row];
          totalRight[totalCond]=right[row];
          //スレッド数をインクリメントする
          totalCond++;
          //最大GPU数に達してしまったら一旦ここでGPUを実行する。stepsはGPUの同
          //時並行稼働数を制御
          //nの数が少ないうちはtotalCondがstepsを超えることはないがnの数が増え
          //て行くと超えるようになる。
          //ここではtotalCond==stepsの場合だけこの中へ         
          if(totalCond==steps){
            //matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUか
            //ら出たらmatched=trueになってる
            if(matched){
              hipMemcpy(h_results,resultsCuda,
                  sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
              for(int col=0;col<steps/THREAD_NUM;col++){
                total+=h_results[col];
              }
              matched=false;
            }
            hipMemcpy(downCuda,totalDown,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(leftCuda,totalLeft,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(rightCuda,totalRight,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            /** backTrack+bitmap*/
            //size-mark は何行GPUを実行するか totalCondはスレッド数
            cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
              >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
            //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われ
            //るのはtotalCondの数だけでそれ以外は空回しになる
            //GPU内でカウントしているので、GPUから出たらmatched=trueになってる
            matched=true;
            //totalCond==stepsルートでGPUを実行したらスレッドをまた0から開始す
            //る(これによりなんどもsteps数分だけGPUを起動できる)
            totalCond=0;           
          }
          //totalDown,totalLeft,totalRightに情報を格納したら1行上に上がる
          //これを繰り返すことにより row=2で可能な場所全てにクイーンを置いて
          //totalDown,totalLeft,totalRightに情報を格納する
          row--;
        }
      }else{
        //置く場所がなければ上に上がる。row==mark行に達するまではCPU側で普通に
        //nqueenをやる
        row--;
      }
    }
  }
  //matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUから出たら
  //matched=trueになってる
  if(matched){
    hipMemcpy(h_results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);   
    for(int col=0;col<steps/THREAD_NUM;col++){
      total+=h_results[col];
    }
    matched=false;
  }
  hipMemcpy(downCuda,totalDown,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  /** backTrack+bitmap*/
  //size-mark は何行GPUを実行するか totalCondはスレッド数
  //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われるのは
  //totalCondの数だけでそれ以外は空回しになる
  cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
    >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
  hipMemcpy(h_results,resultsCuda,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  for(int col=0;col<steps/THREAD_NUM;col++){
    total+=h_results[col];
  }
  //
  hipFree(downCuda);
  hipFree(leftCuda);
  hipFree(rightCuda);
  hipFree(resultsCuda);
  /***06 cudaFreeHostへ変更**/
  //delete[] totalDown;
  hipHostFree(totalDown);
  //delete[] totalLeft;
  hipHostFree(totalLeft);
  //delete[] totalRight;
  hipHostFree(totalRight);
  //delete[] h_results;
  hipHostFree(h_results);
  /************************/
  return total;
}
//GPU
void NQueenG(int size,int steps)
{
  register int sizeE=size-1;
  register int bit=0;
  register int mask=((1<<size)-1);
  if(size<=0||size>32){return;}
  //偶数、奇数共通 右側半分だけクイーンを置く
	int lim=(size%2==0)?size/2:sizeE/2;
  for(int col=0;col<lim;col++){
    bit=(1<<col);
    TOTAL+=solve_nqueen_cuda(size,mask,1,bit<<1,bit,bit>>1,steps);
  }
  //ミラーなのでTOTALを２倍する
  TOTAL=TOTAL*2;
  //奇数の場合はさらに中央にクイーンを置く
  if(size%2==1){
    bit=(1<<(sizeE)/2);
    TOTAL+=solve_nqueen_cuda(size,mask,1,bit<<1,bit,bit>>1,steps);
  }
}
//
//SGPU
__global__ 
void sgpu_cuda_kernel(int size,int mark,unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,unsigned int* d_results,int totalCond)
{
  //スレッド
  const int tid=threadIdx.x;//ブロック内のスレッドID
  const int bid=blockIdx.x;//グリッド内のブロックID
  const int idx=bid*blockDim.x+tid;//全体通してのID
  //シェアードメモリ
  __shared__ unsigned int down[THREAD_NUM][10];//sharedメモリを使う ブロック内スレッドで共有
  __shared__ unsigned int left[THREAD_NUM][10];//THREAD_NUMはブロックあたりのスレッド数
  __shared__ unsigned int right[THREAD_NUM][10];//10で固定なのは現在のmaskの設定でGPUで実行するのは最大10だから
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  __shared__ unsigned int sum[THREAD_NUM];
  //
  const unsigned int mask=(1<<size)-1;
  int total=0;
  int row=0;//row=0となってるが1行目からやっているわけではなくmask行目以降からスタート n=8 なら mask==2 なので そこからスタート
  unsigned int bit;
  if(idx<totalCond){//余分なスレッドは動かさない GPUはsteps数起動するがtotalCond以上は空回しする
    down[tid][row]=totalDown[idx];//totalDown,totalLeft,totalRightの情報をdown,left,rightに詰め直す 
    left[tid][row]=totalLeft[idx];//CPU で詰め込んだ t_はsteps個あるがブロック内ではブロックあたりのスレッドすうに限定されるので idxでよい
    right[tid][row]=totalRight[idx];
    bitmap[tid][row]=down[tid][row]|left[tid][row]|right[tid][row];//down,left,rightからbitmapを出す
    while(row>=0){
      //
      //06のGPU
      //if(bitmap[tid][row]==0){//bitmap[tid][row]=00000000 クイーンをどこにも置けないので1行上に戻る
      //06のSGPU
      if((bitmap[tid][row]&mask)==mask){//bitmap[tid][row]=00000000 クイーンをどこにも置けないので1行上に戻る
      //
        row--;
      }else{
        //
        //06GPU
        //bitmap[tid][row]^=bit=(-bitmap[tid][row]&bitmap[tid][row]); //クイーンを置く
        //06SGPU
        bit=(bitmap[tid][row]+1)&~bitmap[tid][row];
        bitmap[tid][row]|=bit;
        //
        if((bit&mask)!=0){//置く場所があるかどうか
          if(row+1==mark){//最終行?最終行から１個前の行まで無事到達したら 加算する
            total++;
            row--;
          }
          else{
            down[tid][row+1]=down[tid][row]|bit;
            left[tid][row+1]=(left[tid][row]|bit)<<1;
            right[tid][row+1]=(right[tid][row]|bit)>>1;
            bitmap[tid][row+1]=(down[tid][row+1]|left[tid][row+1]|right[tid][row+1]);
            row++;
          }
        }else{//置く場所がなければ１個上に
          row--;
        }
      }
    }
    sum[tid]=total;//最後sum[tid]に加算する
  }else{//totalCond未満は空回しするので当然 totalは加算しない
    sum[tid]=0;
  } 
  //__syncthreads()で、ブロック内のスレッド間の同期をとれます。
  //同期を取るということは、全てのスレッドが__syncthreads()に辿り着くのを待つ
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){sum[tid]+=sum[tid+64];}//__syncthreads();は複数個必要1個だけ記述したら数が違った
  __syncthreads();if(tid<32){sum[tid]+=sum[tid+32];} 
  __syncthreads();if(tid<16){sum[tid]+=sum[tid+16];} 
  __syncthreads();if(tid<8){sum[tid]+=sum[tid+8];} 
  __syncthreads();if(tid<4){sum[tid]+=sum[tid+4];} 
  __syncthreads();if(tid<2){sum[tid]+=sum[tid+2];} 
  __syncthreads();if(tid<1){sum[tid]+=sum[tid+1];} 
  __syncthreads();if(tid==0){d_results[bid]=sum[0];}
}
//
//SGPU
long long sgpu_solve_nqueen_cuda(int size,int steps)
{
  unsigned int down[32];
  unsigned int left[32];
  unsigned int right[32];
  unsigned int bitmap[32];
  unsigned int bit;
  if(size<=0||size>32){return 0;}
  unsigned int* totalDown=new unsigned int[steps];
  unsigned int* totalLeft=new unsigned int[steps];
  unsigned int* totalRight=new unsigned int[steps];
  unsigned int* h_results=new unsigned int[steps];

  //device
  unsigned int* downCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  unsigned int* leftCuda;
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  unsigned int* rightCuda;
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  unsigned int* resultsCuda;
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);

  const unsigned int mask=(1<<size)-1;
  const unsigned int mark=size>11?size-10:2;
  long long total=0;
  int totalCond=0;
  int row=0;
  down[0]=0;
  left[0]=0;
  right[0]=0;
  bitmap[0]=0;
  bool matched=false;
  for(int col=0;col<size/2;col++){
    bit=(1<<col);
    bitmap[0]|=bit;
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=(down[1]|left[1]|right[1]);
    row=1;
    while(row>0){
      if((bitmap[row]&mask)==mask){row--;}
      else{
        bit=(bitmap[row]+1)&~bitmap[row];
        bitmap[row]|=bit;
        if((bit&mask)!=0){
          down[row+1]=down[row]|bit;
          left[row+1]=(left[row]|bit)<<1;
          right[row+1]=(right[row]|bit)>>1;
          bitmap[row+1]=(down[row+1]|left[row+1]|right[row+1]);
          row++;
          if(row==mark){
            totalDown[totalCond]=down[row];
            totalLeft[totalCond]=left[row];
            totalRight[totalCond]=right[row];
            totalCond++;
            if(totalCond==steps){
              if(matched){
                hipMemcpy(h_results,resultsCuda,
                    sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=h_results[col];}
                matched=false;
              }
              hipMemcpy(downCuda,totalDown,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              matched=true;
              totalCond=0;
            }
            row--;
          }
        }else{row--;}
      }
    }
  }
  if(matched){
    hipMemcpy(h_results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=h_results[col];}
    matched=false;
  }
  hipMemcpy(downCuda,totalDown,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  /** backTrack+bitmap*/
  sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
    >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
  hipMemcpy(h_results,resultsCuda,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  for(int col=0;col<steps/THREAD_NUM;col++){total+=h_results[col];}	
  total*=2;

  if(size%2==1){
    matched=false;
    totalCond=0;
    bit=(1<<(size-1)/2);
    bitmap[0]|=bit;
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=(down[1]|left[1]|right[1]);
    row=1;
    while(row>0){
      if((bitmap[row]&mask)==mask){row--;}
      else{
        bit=(bitmap[row]+1)&~bitmap[row];
        bitmap[row]|=bit;
        if((bit&mask)!=0){
          down[row+1]=down[row]|bit;
          left[row+1]=(left[row]|bit)<<1;
          right[row+1]=(right[row]|bit)>>1;
          bitmap[row+1]=(down[row+1]|left[row+1]|right[row+1]);
          row++;
          if(row==mark){
            totalDown[totalCond]=down[row];
            totalLeft[totalCond]=left[row];
            totalRight[totalCond]=right[row];
            totalCond++;
            if(totalCond==steps){
              if(matched){
                hipMemcpy(h_results,resultsCuda,
                    sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=h_results[col];}
                matched=false;
              }
              hipMemcpy(downCuda,totalDown,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              matched=true;
              totalCond=0;
            }
            row--;
          }
        }else{row--;}
      }
    }
    if(matched){
      hipMemcpy(h_results,resultsCuda,
          sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
      for(int col=0;col<steps/THREAD_NUM;col++){total+=h_results[col];}
      matched=false;
    }
    hipMemcpy(downCuda,totalDown,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(leftCuda,totalLeft,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(rightCuda,totalRight,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    /** backTrack+bitmap*/
    sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
      >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
    hipMemcpy(h_results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=h_results[col];}
  }
  hipFree(downCuda);
  hipFree(leftCuda);
  hipFree(rightCuda);
  hipFree(resultsCuda);
  delete[] totalDown;
  delete[] totalLeft;
  delete[] totalRight;
  delete[] h_results;
  return total;
}
//
//CUDA 初期化
bool InitCUDA()
{
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
//
//hh:mm:ss.ms形式に処理時間を出力
void TimeFormat(clock_t utime,char *form){
  int dd,hh,mm;
  float ftime,ss;
  ftime=(float)utime/CLOCKS_PER_SEC;
  mm=(int)ftime/60;
  ss=ftime-(int)(mm*60);
  dd=mm/(24*60);
  mm=mm%(24*60);
  hh=mm/60;
  mm=mm%60;
  if(dd)
    sprintf(form,"%4d %02d:%02d:%05.2f",dd,hh,mm,ss);
  else if(hh)
    sprintf(form,"     %2d:%02d:%05.2f",hh,mm,ss);
  else if(mm)
    sprintf(form,"        %2d:%05.2f",mm,ss);
  else
    sprintf(form,"           %5.2f",ss);
}
//
//CPU 非再帰版 ロジックメソッド
void solve_nqueen(int size,int mask, int row,int* left,int* down,int* right,int* bitmap){
  unsigned int bit;
  unsigned int sizeE=size-1;
  int mark=row;
  //固定していれた行より上はいかない
  while(row>=mark){//row=1 row>=1, row=2 row>=2
    if(bitmap[row]==0){
      --row;
    }else{
      bitmap[row]^=bit=(-bitmap[row]&bitmap[row]); 
      if((bit&mask)!=0){
        if(row==sizeE){
          TOTAL++;
          --row;
        }else{
          int n=row++;
          left[row]=(left[n]|bit)<<1;
          down[row]=down[n]|bit;
          right[row]=(right[n]|bit)>>1;
          bitmap[row]=mask&~(left[row]|down[row]|right[row]);
        }
      }else{
         --row;
      }
    }  
  }
}
//
//非再帰版
void NQueen(int size,int mask){
  register int sizeE=size-1;
  register int bitmap[size];
  register int down[size],right[size],left[size];
  register int bit;
  if(size<=0||size>32){return;}
  bit=0;
  bitmap[0]=mask;
  down[0]=left[0]=right[0]=0;
  //偶数、奇数共通
  for(int col=0;col<size/2;col++){//右側半分だけクイーンを置く
    bit=(1<<col);//
    down[1]=bit;//再帰の場合は down,left,right,bitmapは現在の行だけで良いが
    left[1]=bit<<1;//非再帰の場合は全行情報を配列に入れて行の上がり下がりをする
    right[1]=bit>>1;
    bitmap[1]=mask&~(left[1]|down[1]|right[1]);
    solve_nqueen(size,mask,1,left,down,right,bitmap);
  }
  TOTAL*=2;//ミラーなのでTOTALを２倍する
  //奇数の場合はさらに中央にクイーンを置く
  if(size%2==1){
    bit=(1<<(sizeE)/2);
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=mask&~(left[1]|down[1]|right[1]);
    solve_nqueen(size,mask,1,left,down,right,bitmap);
  }  
}
//
//CPUR 再帰版 ロジックメソッド
void solve_nqueenr(int size,int mask, int row,int left,int down,int right){
 int bitmap=0;
 int bit=0;
 int sizeE=size-1;
 bitmap=(mask&~(left|down|right));
 if(row==sizeE){
    if(bitmap){
      TOTAL++;
    }
  }else{
    while(bitmap){
      bitmap^=bit=(-bitmap&bitmap);
      solve_nqueenr(size,mask,row+1,(left|bit)<<1, down|bit,(right|bit)>>1);
    }
  }
}
//
//CPUR 再帰版 ロジックメソッド
void NQueenR(int size,int mask, int row,int left,int down,int right){
  int bit=0;
  int sizeE=size-1;
  for(int col=0;col<size/2;col++){
    bit=(1<<col);
    solve_nqueenr(size,mask,1,bit<<1,bit,bit>>1);
  }
  TOTAL*=2;
  if(size%2==1){
    bit=(1<<(sizeE)/2);
    solve_nqueenr(size,mask,1,bit<<1,bit,bit>>1);
  }
}
//
//通常版 CPU 非再帰版 ロジックメソッド
void NQueenD(int size,int mask,int row){
  int aStack[size];
  int* pnStack;
  int bit;
  int bitmap;
  int sizeE=size-1;
  int down[size],right[size],left[size];
  aStack[0]=-1; 
  pnStack=aStack+1;
  bit=0;
  bitmap=mask;
  down[0]=left[0]=right[0]=0;
  while(true){
    if(bitmap){
      bitmap^=bit=(-bitmap&bitmap); 
      if(row==sizeE){
        TOTAL++;
        bitmap=*--pnStack;
        --row;
        continue;
      }else{
        int n=row++;
        left[row]=(left[n]|bit)<<1;
        down[row]=down[n]|bit;
        right[row]=(right[n]|bit)>>1;
        *pnStack++=bitmap;
        bitmap=mask&~(left[row]|down[row]|right[row]);
        continue;
      }
    }else{ 
      bitmap=*--pnStack;
      if(pnStack==aStack){ break ; }
      --row;
      continue;
    }
  }
}
//
//通常版 CPUR 再帰版　ロジックメソッド
void NQueenDR(int size,int mask,int row,int left,int down,int right){
  int bitmap=0;
  int bit=0;
  if(row==size){
    TOTAL++;
  }else{
    bitmap=(mask&~(left|down|right));
    while(bitmap){
      bitmap^=bit=(-bitmap&bitmap);
      NQueenDR(size,mask,row+1,(left|bit)<<1, down|bit,(right|bit)>>1);
    }
  }
}
//
//メインメソッド
int main(int argc,char** argv) {
  bool cpu=false,cpur=false,gpu=false,sgpu=false;
  int argstart=1,steps=24576;
  /** パラメータの処理 */
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='r'||argv[1][1]=='R'){cpur=true;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){gpu=true;}
    else if(argv[1][1]=='s'||argv[1][1]=='S'){sgpu=true;}
    else
      cpur=true;
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g|-r|-s]\n",argv[0]);
    printf("  -c: CPU only\n");
    printf("  -r: CPUR only\n");
    printf("  -g: GPU only\n");
    printf("  -s: SGPU only\n");
    printf("Default to 8 queen\n");
  }
  /** 出力と実行 */
  if(cpu){
    printf("\n\n６．CPU 非再帰 バックトラック＋ビットマップ\n");
  }else if(cpur){
    printf("\n\n６．CPUR 再帰 バックトラック＋ビットマップ\n");
  }else if(gpu){
    printf("\n\n６．GPU 非再帰 バックトラック＋ビットマップ\n");
  }else if(sgpu){
    printf("\n\n６．SGPU 非再帰 バックトラック＋ビットマップ\n");
  }
  if(cpu||cpur){
    printf("%s\n"," N:        Total       Unique        hh:mm:ss.ms");
    clock_t st;          //速度計測用
    char t[20];          //hh:mm:ss.msを格納
    int min=4;
    int targetN=17;
    int mask;
    for(int i=min;i<=targetN;i++){
      TOTAL=0;
      UNIQUE=0;
      mask=((1<<i)-1);
      st=clock();
      //
      //CPUR
      if(cpur){ 
        NQueenR(i,mask,0,0,0,0); 
        //NQueenDR(i,mask,0,0,0,0);//通常版
      }
      //CPU
      if(cpu){ 
        NQueen(i,mask); 
        //NQueenD(i,mask,0); //通常版
      }
      //
      TimeFormat(clock()-st,t);
      printf("%2d:%13ld%16ld%s\n",i,TOTAL,UNIQUE,t);
    }
  }
  if(gpu||sgpu){
    if(!InitCUDA()){return 0;}
    int min=4;int targetN=17;
    
    struct timeval t0;struct timeval t1;
    int ss;int ms;int dd;
    printf("%s\n"," N:        Total      Unique      dd:hh:mm:ss.ms");
    for(int i=min;i<=targetN;i++){
      gettimeofday(&t0,NULL);  // 計測開始
      if(gpu){
        TOTAL=0;
        UNIQUE=0;
        NQueenG(i,steps);
      }else if(sgpu){
        TOTAL=sgpu_solve_nqueen_cuda(i,steps);
      	UNIQUE=0;
      }
      gettimeofday(&t1,NULL);  // 計測終了
      if(t1.tv_usec<t0.tv_usec) {
        dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      } else {
        dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%13ld%16ld%4.2d:%02d:%02d:%02d.%02d\n", i,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    }
  }
  return 0;
}

