
/**
 CUDAで学ぶアルゴリズムとデータ構造
 ステップバイステップでＮ−クイーン問題を最適化
 一般社団法人  共同通信社  情報技術局  鈴木  維一郎(suzuki.iichiro@kyodonews.jp)

 コンパイルと実行
 $ nvcc CUDA**_N-Queen.cu && ./a.out (-c|-r|-g|-s)
                    -c:cpu 
                    -r cpu再帰 
                    -g GPU 
                    -s SGPU(サマーズ版と思われる)


 ７．バックトラック＋ビットマップ＋対称解除法

 *     一つの解には、盤面を９０度、１８０度、２７０度回転、及びそれらの鏡像の合計
 *     ８個の対称解が存在する。対照的な解を除去し、ユニーク解から解を求める手法。
 * 
 * ■ユニーク解の判定方法
 *   全探索によって得られたある１つの解が、回転・反転などによる本質的に変わること
 * のない変換によって他の解と同型となるものが存在する場合、それを別の解とはしない
 * とする解の数え方で得られる解を「ユニーク解」といいます。つまり、ユニーク解とは、
 * 全解の中から回転・反転などによる変換によって同型になるもの同士をグループ化する
 * ことを意味しています。
 * 
 *   従って、ユニーク解はその「個数のみ」に着目され、この解はユニーク解であり、こ
 * の解はユニーク解ではないという定まった判定方法はありません。ユニーク解であるか
 * どうかの判断はユニーク解の個数を数える目的の為だけに各個人が自由に定義すること
 * になります。もちろん、どのような定義をしたとしてもユニーク解の個数それ自体は変
 * わりません。
 * 
 *   さて、Ｎクイーン問題は正方形のボードで形成されるので回転・反転による変換パター
 * ンはぜんぶで８通りあります。だからといって「全解数＝ユニーク解数×８」と単純には
 * いきません。ひとつのグループの要素数が必ず８個あるとは限らないのです。Ｎ＝５の
 * 下の例では要素数が２個のものと８個のものがあります。
 *
 *
 * Ｎ＝５の全解は１０、ユニーク解は２なのです。
 * 
 * グループ１: ユニーク解１つ目
 * - - - Q -   - Q - - -
 * Q - - - -   - - - - Q
 * - - Q - -   - - Q - -
 * - - - - Q   Q - - - -
 * - Q - - -   - - - Q -
 * 
 * グループ２: ユニーク解２つ目
 * - - - - Q   Q - - - -   - - Q - -   - - Q - -   - - - Q -   - Q - - -   Q - - - -   - - - - Q
 * - - Q - -   - - Q - -   Q - - - -   - - - - Q   - Q - - -   - - - Q -   - - - Q -   - Q - - -
 * Q - - - -   - - - - Q   - - - Q -   - Q - - -   - - - - Q   Q - - - -   - Q - - -   - - - Q -
 * - - - Q -   - Q - - -   - Q - - -   - - - Q -   - - Q - -   - - Q - -   - - - - Q   Q - - - -
 * - Q - - -   - - - Q -   - - - - Q   Q - - - -   Q - - - -   - - - - Q   - - Q - -   - - Q - -
 *
 * 
 *   それでは、ユニーク解を判定するための定義付けを行いますが、次のように定義する
 * ことにします。各行のクイーンが右から何番目にあるかを調べて、最上段の行から下
 * の行へ順番に列挙します。そしてそれをＮ桁の数値として見た場合に最小値になるもの
 * をユニーク解として数えることにします。尚、このＮ桁の数を以後は「ユニーク判定値」
 * と呼ぶことにします。
 * 
 * - - - - Q   0
 * - - Q - -   2
 * Q - - - -   4   --->  0 2 4 1 3  (ユニーク判定値)
 * - - - Q -   1
 * - Q - - -   3
 * 
 * 
 *   探索によって得られたある１つの解(オリジナル)がユニーク解であるかどうかを判定
 * するには「８通りの変換を試み、その中でオリジナルのユニーク判定値が最小であるか
 * を調べる」ことになります。しかし結論から先にいえば、ユニーク解とは成り得ないこ
 * とが明確なパターンを探索中に切り捨てるある枝刈りを組み込むことにより、３通りの
 * 変換を試みるだけでユニーク解の判定が可能になります。
 *  
 実行結果

$ nvcc CUDA07_N-Queen.cu  && ./a.out -r
７．CPUR 再帰 バックトラック＋ビットマップ＋対称解除法
 N:        Total       Unique        hh:mm:ss.ms
 4:            2               1            0.00
 5:           10               2            0.00
 6:            4               1            0.00
 7:           40               6            0.00
 8:           92              12            0.00
 9:          352              46            0.00
10:          724              92            0.00
11:         2680             341            0.00
12:        14200            1787            0.01
13:        73712            9233            0.08
14:       365596           45752            0.48
15:      2279184          285053            3.20
16:     14772512         1846955           22.49
17:     95815104        11977939         2:41.93

$ nvcc CUDA07_N-Queen.cu  && ./a.out -c
７．CPU 非再帰 バックトラック＋ビットマップ＋対称解除法
 N:        Total       Unique        hh:mm:ss.ms
 4:            2               1            0.00
 5:           10               2            0.00
 6:            4               1            0.00
 7:           40               6            0.00
 8:           92              12            0.00
 9:          352              46            0.00
10:          724              92            0.00
11:         2680             341            0.00
12:        14200            1787            0.01
13:        73712            9233            0.09
14:       365596           45752            0.49
15:      2279184          285053            3.25
16:     14772512         1846955           22.96
17:     95815104        11977939         2:43.94

bash-3.2$ nvcc CUDA06_N-Queen.cu && ./a.out -s
６．SGPU 非再帰 バックトラック＋ビットマップ
 N:        Total      Unique      dd:hh:mm:ss.ms
 4:            2               0  00:00:00:00.02
 5:           10               0  00:00:00:00.00
 6:            4               0  00:00:00:00.00
 7:           40               0  00:00:00:00.00
 8:           92               0  00:00:00:00.00
 9:          352               0  00:00:00:00.00
10:          724               0  00:00:00:00.00
11:         2680               0  00:00:00:00.00
12:        14200               0  00:00:00:00.02
13:        73712               0  00:00:00:00.03
14:       365596               0  00:00:00:00.07
15:      2279184               0  00:00:00:00.48
16:     14772512               0  00:00:00:02.40
17:     95815104               0  00:00:00:18.30

$ nvcc CUDA07_N-Queen.cu  && ./a.out -g
．GPU 非再帰 バックトラック＋ビットマップ＋対称解除法
 N:        Total      Unique      dd:hh:mm:ss.ms
 4:            2               1  00:00:00:00.02
 5:           10               2  00:00:00:00.00
 6:            4               1  00:00:00:00.00
 7:           40               6  00:00:00:00.00
 8:           92              12  00:00:00:00.01
 9:          352              46  00:00:00:00.01
10:          724              92  00:00:00:00.04
11:         2680             341  00:00:00:00.13
12:        14200            1787  00:00:00:00.54
13:        73712            9233  00:00:00:01.05
14:       365596           45752  00:00:00:01.11
15:      2279184          285053  00:00:00:06.65
16:     14772512         1846955  00:00:00:41.28
17:     95815104        11977939  00:00:05:35.54
*/
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define THREAD_NUM		96
//#define THREAD_NUM		1
#define MAX 27
//変数宣言
long TOTAL=0; //GPU,CPUで使用
/***07 uniq*************************************/
long UNIQUE=0;//GPU,CPUで使用
/****************************************/
int down[2*MAX-1]; //down:flagA 縦 配置フラグ　//CPUで使用
int left[2*MAX-1];  //left:flagB 斜め配置フラグ　//CPUで使用
int right[2*MAX-1];  //right:flagC 斜め配置フラグ　//CPUで使用
/***07 aBoard*************************************/
unsigned int aBoard[MAX];//CPU,GPUで使用
/****************************************/
int aT[MAX];//CPUで使用
int aS[MAX];//CPUで使用
int COUNT2,COUNT4,COUNT8;//CPUで使用
//関数宣言 GPU
//関数宣言 GPU/CPU
__device__ __host__ int rh(int a,int sz);
__device__ __host__ void vMirror_bitmap(int bf[],int af[],int si);
__device__ __host__ void rotate_bitmap(int bf[],int af[],int si);
__device__ __host__ int intncmp(int lt[],int rt[],int n);
__device__ int symmetryOps_bitmap_gpu(int si,int *d_aBoard,int *d_aT,int *d_aS);
__global__
/***07 d_uniq,t_aBoard,h_row追加に伴いコメント*************************************/
//void cuda_kernel(
//    int size,int mark,
//    unsigned int* t_down,unsigned int* t_left,unsigned int* t_right,
//    unsigned int* d_results,int totalCond,unsigned);
/****************************************/
/***07 d_uniq,t_aBoard,h_row追加*************************************/
void cuda_kernel(
    int size,int mark,
    unsigned int* t_down,unsigned int* t_left,unsigned int* t_right,
    unsigned int* d_results,unsigned int* d_uniq,int totalCond,unsigned int* t_aBoard,int h_row,int* aT,int* aS);
/****************************************/
long long solve_nqueen_cuda(int size,int steps);
void NQueenG(int size,int mask,int row,int steps);
//関数宣言 SGPU
__global__ 
void sgpu_cuda_kernel(int size,int mark,unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,unsigned int* results,int totalCond);
long long sgpu_solve_nqueen_cuda(int size,int steps);
bool InitCUDA();
//関数宣言 CPU
void TimeFormat(clock_t utime,char *form);
long getUnique();
long getTotal();
void symmetryOps_bitmap(int si);
//関数宣言 非再帰版
void solve_nqueen(int size,int mask, int row,int* left,int* down,int* right,int* bitmap);
void NQueen(int size,int mask);
//関数宣言 GPUへの移行再帰版
void solve_nqueenr(int size,int mask, int row,int left,int down,int right);
void NQueenR(int size,int mask);
//関数宣言 通常版
//  再帰
void NQueenDR(int size,int mask,int row,int left,int down,int right);
//  非再帰
void NQueenD(int size,int mask,int row);
//
//GPU マルチスレッド
//
/***07 symmetryOps*************************************/
__device__ __host__
int rh(int a,int sz)
{
  int tmp=0;
  for(int i=0;i<=sz;i++){
    if(a&(1<<i)){ return tmp|=(1<<(sz-i)); }
  }
  return tmp;
}
/****************************************/
//
/***07 symmetryOps*************************************/
__device__ __host__
void vMirror_bitmap(int bf[],int af[],int si)
{
  int score ;
  for(int i=0;i<si;i++) {
    score=bf[i];
    af[i]=rh(score,si-1);
  }
}
/****************************************/
//
/***07 symmetryOps*************************************/
__device__ __host__
void rotate_bitmap(int bf[],int af[],int si)
{
  for(int i=0;i<si;i++){
    int t=0;
    for(int j=0;j<si;j++){
      t|=((bf[j]>>i)&1)<<(si-j-1); // x[j] の i ビット目を
    }
    af[i]=t;                        // y[i] の j ビット目にする
  }
}
/****************************************/
//
/***07 symmetryOps*************************************/
__device__ __host__
int intncmp(unsigned int lt[],int rt[],int n)
{
  int rtn=0;
  for(int k=0;k<n;k++){
    rtn=lt[k]-rt[k];
    if(rtn!=0){
      break;
    }
  }
  return rtn;
}
/****************************************/
/***07 symmetryOps*************************************/
__device__
int symmetryOps_bitmap_gpu(int si,unsigned int *d_aBoard,int *d_aT,int *d_aS)
{
  int nEquiv;
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<si;i++){ d_aT[i]=d_aBoard[i];}
  rotate_bitmap(d_aT,d_aS,si);    //時計回りに90度回転
  int k=intncmp(d_aBoard,d_aS,si);
  //printf("1_k:%d\n",k);
  if(k>0)return 0;
  if(k==0){ nEquiv=2;}else{
    rotate_bitmap(d_aS,d_aT,si);  //時計回りに180度回転
    k=intncmp(d_aBoard,d_aT,si);
    //printf("2_k:%d\n",k);
    if(k>0)return 0;
    if(k==0){ nEquiv=4;}else{
      rotate_bitmap(d_aT,d_aS,si);//時計回りに270度回転
      k=intncmp(d_aBoard,d_aS,si);
      //printf("3_k:%d\n",k);
      if(k>0){ return 0;}
      nEquiv=8;
    }
  }
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<si;i++){ d_aS[i]=d_aBoard[i];}
  vMirror_bitmap(d_aS,d_aT,si);   //垂直反転
  k=intncmp(d_aBoard,d_aT,si);
  //printf("4_k:%d\n",k);
  if(k>0){ return 0; }
  if(nEquiv>2){             //-90度回転 対角鏡と同等
    rotate_bitmap(d_aT,d_aS,si);
    k=intncmp(d_aBoard,d_aS,si);
    //printf("5_k:%d\n",k);
    if(k>0){return 0;}
    if(nEquiv>4){           //-180度回転 水平鏡像と同等
      rotate_bitmap(d_aS,d_aT,si);
      k=intncmp(d_aBoard,d_aT,si);
      //printf("6_k:%d\n",k);
      if(k>0){ return 0;}       //-270度回転 反対角鏡と同等
      rotate_bitmap(d_aT,d_aS,si);
      k=intncmp(d_aBoard,d_aS,si);
      //printf("7_k:%d\n",k);
      if(k>0){ return 0;}
    }
  }
  //printf("eq:%d\n",nEquiv);
  return nEquiv;  
}
/****************************************/
//
//GPU
/***07 引数 追加に伴いコメント*********************/
//__global__ 
//void cuda_kernel(int size,int mark,unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,unsigned int* d_results,int totalCond)
/************************/
/***07 引数 d_uniq,t_aBoard,h_row追加 uniq,aBoardのため*********************/
__global__
void cuda_kernel(int size,int mark,unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,unsigned int* d_results,unsigned int* d_uniq,int totalCond,unsigned int* t_aBoard,int h_row,int* aT,int* aS)
{
  /************************/
  const unsigned int mask=(1<<size)-1;
  int total=0;
  /***07 uniq,aBoard追加*********************/
  int unique=0;
  //int aT[MAX];
  //int aS[MAX];
  /************************/
  //row=0となってるが1行目からやっているわけではなく
  //mask行目以降からスタート 
  //n=8 なら mask==2 なので そこからスタート
  int row=0;
  unsigned int bit;
  //
  //スレッド
  //
  //ブロック内のスレッドID
  const int tid=threadIdx.x;
  //グリッド内のブロックID
  const int bid=blockIdx.x;
  //全体通してのID
  const int idx=bid*blockDim.x+tid;
  //
  //シェアードメモリ
  //
  //sharedメモリを使う ブロック内スレッドで共有
  //10固定なのは現在のmask設定で
  //GPUで実行するのは最大10だから
  //THREAD_NUMはブロックあたりのスレッド数
  __shared__ unsigned int down[THREAD_NUM][10];
  down[tid][row]=totalDown[idx];
  __shared__ unsigned int left[THREAD_NUM][10];
  left[tid][row]=totalLeft[idx];
  __shared__ unsigned int right[THREAD_NUM][10];
  right[tid][row]=totalRight[idx];
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  //down,left,rightからbitmapを出す
  bitmap[tid][row]
    =mask&~(
         down[tid][row]
        |left[tid][row]
        |right[tid][row]);
  __shared__ unsigned int sum[THREAD_NUM];
  /***07 aBoard,uniq追加*********************/
  unsigned int c_aBoard[MAX];
  int c_aT[MAX];
  int c_aS[MAX];
  __shared__ unsigned int usum[THREAD_NUM];
  /************************/
  //
  //余分なスレッドは動かさない 
  //GPUはsteps数起動するがtotalCond以上は空回しする
  if(idx<totalCond){
    //totalDown,totalLeft,totalRightの情報を
    //down,left,rightに詰め直す 
    //CPU で詰め込んだ t_はsteps個あるが
    //ブロック内ではブロックあたりのスレッド数に限定
    //されるので idxでよい
    //
    /***07 aBoard追加*********************/
    for(int i=0;i<size;i++){
      //c_aBoard[tid][i]=t_aBoard[idx][i];   
      c_aBoard[i]=t_aBoard[idx*MAX+i]; //２次元配列だが1次元的に利用  
    }
    /************************/
    while(row>=0){
      //
      //bitmap[tid][row]=00000000 クイーンを
      //どこにも置けないので1行上に戻る
      if(bitmap[tid][row]==0){
        row--;
      }else{
        //クイーンを置く
        //bitmap[tid][row]
        //  ^=bit
        //  =(-bitmap[tid][row]&bitmap[tid][row]);
        //置く場所があるかどうか
        /***07 aBoard追加*********************/
        bitmap[tid][row]
          ^=c_aBoard[row+h_row]
          =bit
          =(-bitmap[tid][row]&bitmap[tid][row]);       
        /************************/
        if((bit&mask)!=0){
          //最終行?最終行から１個前の行まで
          //無事到達したら 加算する
          if(row+1==mark){
           /***07 symmetryOpsの処理を追加*********************/
           int s=symmetryOps_bitmap_gpu(size,c_aBoard,c_aT,c_aS); 
           //int s=0;//=symmetryOps_bitmap_gpu(size,c_aBoard[tid],aT,aS); 
           if(s!=0){
           //print(size); //print()でTOTALを++しない
           //ホストに戻す配列にTOTALを入れる
           //スレッドが１つの場合は配列は１個
              unique++; 
              total+=s;   //対称解除で得られた解数を加算
           }
           /************************/
           /***07 symmetryOpsの処理追加に伴いコメント*********************/
           //total++;
           /************************/
            row--;
          }else{
            int rowP=row+1;
            down[tid][rowP]=down[tid][row]|bit;
            left[tid][rowP]=(left[tid][row]|bit)<<1;
            right[tid][rowP]=(right[tid][row]|bit)>>1;
            bitmap[tid][rowP]
              =mask&~(
                  down[tid][rowP]
                  |left[tid][rowP]
                  |right[tid][rowP]);
            row++;
          }
        }else{
          //置く場所がなければ１個上に
          row--;
        }
      }
    }
    //最後sum[tid]に加算する
    sum[tid]=total;
    /***07 uniq追加*********************/
    usum[tid]=unique;
    /************************/
  }else{
    //totalCond未満は空回しするのでtotalは加算しない
    sum[tid]=0;
    /***07 uniq追加*********************/
    usum[tid]=0;
    /************************/
  } 
  //__syncthreads()でブロック内のスレッド間の同期
  //全てのスレッドが__syncthreads()に辿り着くのを待つ
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){
    sum[tid]+=sum[tid+64];
    /***07 uniq追加*********************/
    usum[tid]+=usum[tid+64];
    /************************/
  }
  __syncthreads();if(tid<32){
    sum[tid]+=sum[tid+32];
    /***07 uniq追加*********************/
    usum[tid]+=usum[tid+32];
    /************************/
  } 
  __syncthreads();if(tid<16){
    sum[tid]+=sum[tid+16];
    /***07 uniq追加*********************/
    usum[tid]+=usum[tid+16];
    /************************/  
  } 
  __syncthreads();if(tid<8){
    sum[tid]+=sum[tid+8];
    /***07 uniq追加*********************/
    usum[tid]+=usum[tid+8];
    /************************/
  } 
  __syncthreads();if(tid<4){
    sum[tid]+=sum[tid+4];
    /***07 uniq追加*********************/
    usum[tid]+=usum[tid+4];
    /************************/  
  } 
  __syncthreads();if(tid<2){
    sum[tid]+=sum[tid+2];
    /***07 uniq追加*********************/
    usum[tid]+=usum[tid+2];
    /************************/  
  } 
  __syncthreads();if(tid<1){
    sum[tid]+=sum[tid+1];
    /***07 uniq追加*********************/
    usum[tid]+=usum[tid+1];
    /************************/  
  } 
  __syncthreads();if(tid==0){
    d_results[bid]=sum[0];
    /****07 uniq追加********************/
    d_uniq[bid]=usum[0];
    /************************/
  }
}
//
// GPU
long solve_nqueen_cuda(int size,int mask,int row,int n_left,int n_down,int n_right,int steps)
{
  //何行目からGPUで行くか。ここの設定は変更可能、設定値を多くするほどGPUで並行して動く
  const unsigned int mark=size>11?size-10:2;
  const unsigned int h_mark=row;
  long total=0;
  int totalCond=0;
  bool matched=false;
  //host
  unsigned int down[32];
  down[row]=n_down;
  unsigned int right[32];
  right[row]=n_right;
  unsigned int left[32];
  left[row]=n_left;
  //bitmapを配列で持つことにより
  //stackを使わないで1行前に戻れる
  unsigned int bitmap[32];
  //bitmap[row]=(left[row]|down[row]|right[row]);
  /***07 aBoard追加に伴いbit処理をGPU*********************/
  bitmap[row]=mask&~(left[row]|down[row]|right[row]);
  /************************/
  unsigned int bit;
  unsigned int* totalDown=new unsigned int[steps];
  unsigned int* totalLeft=new unsigned int[steps];
  unsigned int* totalRight=new unsigned int[steps];
  unsigned int* h_results=new unsigned int[steps];
  /***07 uniq,aBoard追加*********************/
  unsigned int* h_uniq=new unsigned int[steps];
  unsigned int* t_aBoard=new unsigned int[steps*MAX];
  /************************/
  //device
  unsigned int* downCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  unsigned int* leftCuda;
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  unsigned int* rightCuda;
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  unsigned int* resultsCuda;
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);
  /***07 uniq,aBoard追加*********************/
  unsigned int* d_aT;
  hipMalloc((void**) &d_aT,sizeof(int)*steps*MAX);
  unsigned int* d_aS;
  hipMalloc((void**) &d_aS,sizeof(int)*steps*MAX);

  unsigned int* d_uniq;
  hipMalloc((void**) &d_uniq,sizeof(int)*steps/THREAD_NUM);
  unsigned int* d_aBoard;
  hipMalloc((void**) &d_aBoard,sizeof(int)*steps*MAX);
  /************************/
  //12行目までは3行目までCPU->row==mark以下で 3行目までの
  //down,left,right情報を totalDown,totalLeft,totalRight
  //に格納
  //する->3行目以降をGPUマルチスレッドで実行し結果を取得
  //13行目以降はCPUで実行する行数が１個ずつ増えて行く
  //例えばn15だとrow=5までCPUで実行し、
  //それ以降はGPU(現在の設定だとGPUでは最大10行実行する
  //ようになっている)
  //while(row>=0) {
  int rowP=0;
  while(row>=h_mark) {
    //bitmap[row]=00000000 クイーンを
    //どこにも置けないので1行上に戻る
    /***07 aBoard追加に伴いbit操作変更*********************/
    //06GPU こっちのほうが優秀
    if(bitmap[row]==0){ row--; }
    /************************/
    /***07 aBoard追加に伴いbit操作変更でコメント*********************/
    //06SGPU
    //if((bitmap[row]&mask)==mask){row--;}
    /************************/
    else{//おける場所があれば進む
      //06SGPU
      /***07 aBoard追加に伴いbit操作変更でコメント*********************/
      //bit=(bitmap[row]+1)&~bitmap[row];
      //bitmap[row]|=bit;
      /************************/
      //06GPU こっちのほうが優秀
      //bitmap[row]^=bit=(-bitmap[row]&bitmap[row]); //クイーンを置く
      /***07 aBoard追加*********************/
      bitmap[row]^=aBoard[row]=bit=(-bitmap[row]&bitmap[row]);
      /************************/ 
      if((bit&mask)!=0){//置く場所があれば先に進む
        rowP=row+1;
        down[rowP]=down[row]|bit;
        left[rowP]=(left[row]|bit)<<1;
        right[rowP]=(right[row]|bit)>>1;
        /***07 aBoard追加に伴いbit操作変更でコメント*********************/
        //bitmap[rowP]=(down[rowP]|left[rowP]|right[rowP]);
        /************************/
        /***07 aBoard追加に伴いbit操作変更*********************/
        bitmap[rowP]=mask&~(down[rowP]|left[rowP]|right[rowP]);
        /************************/
        row++;
        if(row==mark){
          //3行目(mark)にクイーンを１個ずつ置いていって、
          //down,left,right情報を格納、
          //その次の行へは進まない。その行で可能な場所にクイー
          //ン置き終わったらGPU並列実行
          //totalCond がthreadIdになる 各スレッドに down,left,right情報を渡す
          //row=2(13行目以降は増えていく。例えばn15だとrow=5)の情報を
          //totalDown,totalLeft,totalRightに格納する
          totalDown[totalCond]=down[row];
          totalLeft[totalCond]=left[row];
          totalRight[totalCond]=right[row];
          /***07 aBoard追加*********************/
          for(int i=0;i<size;i++){
            //t_aBoard[totalCond][i]=aBoard[i];
            t_aBoard[totalCond*MAX+i]=aBoard[i];
          }
          /************************/
          //スレッド数をインクリメントする
          totalCond++;
          //最大GPU数に達してしまったら一旦ここでGPUを実行する。stepsはGPUの同
          //時並行稼働数を制御
          //nの数が少ないうちはtotalCondがstepsを超えることはないがnの数が増え
          //て行くと超えるようになる。
          //ここではtotalCond==stepsの場合だけこの中へ         
          if(totalCond==steps){
            //matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUか
            //ら出たらmatched=trueになってる
            if(matched){
              hipMemcpy(h_results,resultsCuda,
                  sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
              /***07 uniq追加*********************/
              hipMemcpy(h_uniq,d_uniq,
                  sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
              /************************/
              for(int col=0;col<steps/THREAD_NUM;col++){
                total+=h_results[col];
                /****07 uniq追加********************/
                UNIQUE+=h_uniq[col];
                /************************/                                        
              }
              matched=false;
            }
            hipMemcpy(downCuda,totalDown,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(leftCuda,totalLeft,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(rightCuda,totalRight,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            /***07 aBoard追加*********************/
            hipMemcpy(d_aBoard,t_aBoard,
                sizeof(int)*totalCond*MAX,hipMemcpyHostToDevice);
            /************************/
            /** backTrack+bitmap*/
            //size-mark は何行GPUを実行するか totalCondはスレッド数
            /***07 d_uniq,d_aBoard,row追加に伴いコメント*********************/
            //cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
            //  >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
            /************************/
            /***07 d_uniq,d_aBoard,row追加*********************/
            cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
              >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,d_aBoard,row,aT,aS);
            /************************/          
            //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われ
            //るのはtotalCondの数だけでそれ以外は空回しになる
            //GPU内でカウントしているので、GPUから出たらmatched=trueになってる
            matched=true;
            //totalCond==stepsルートでGPUを実行したらスレッドをまた0から開始す
            //る(これによりなんどもsteps数分だけGPUを起動できる)
            totalCond=0;           
          }
          //totalDown,totalLeft,totalRightに情報を格納したら1行上に上がる
          //これを繰り返すことにより row=2で可能な場所全てにクイーンを置いて
          //totalDown,totalLeft,totalRightに情報を格納する
          row--;
        }
      }else{
        //置く場所がなければ上に上がる。row==mark行に達するまではCPU側で普通に
        //nqueenをやる
        row--;
      }
    }
  }
  //matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUから出たら
  //matched=trueになってる
  if(matched){
    hipMemcpy(h_results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    /***07 uniq追加*********************/
    hipMemcpy(h_uniq,d_uniq,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    /************************/
   
    for(int col=0;col<steps/THREAD_NUM;col++){
      total+=h_results[col];
      /***07 uniq追加*********************/
      UNIQUE+=h_uniq[col];
      /************************/    
    }
    matched=false;
  }
  hipMemcpy(downCuda,totalDown,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  /***07 aBoard追加*********************/
  hipMemcpy(d_aBoard,t_aBoard,
      sizeof(int)*totalCond*MAX,hipMemcpyHostToDevice);
  /************************/ 
  /** backTrack+bitmap*/
  //size-mark は何行GPUを実行するか totalCondはスレッド数
  //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われるのは
  //totalCondの数だけでそれ以外は空回しになる
  /***07 d_uniq,d_aBoard,mark追加に伴いコメント*********************/   
  //cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
  //  >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
  /***07 d_uniq,d_aBoard,mark追加*********************/  
  cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
    >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,d_aBoard,mark,aT,aS);
  /************************/
  hipMemcpy(h_results,resultsCuda,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  /***07 uniq追加*********************/
  hipMemcpy(h_uniq,d_uniq,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  /************************/   
  for(int col=0;col<steps/THREAD_NUM;col++){
    total+=h_results[col];
    /***07 uniq追加*********************/
    UNIQUE+=h_uniq[col];
    /************************/    
  }
  //
  hipFree(downCuda);
  hipFree(leftCuda);
  hipFree(rightCuda);
  hipFree(resultsCuda);
  /***07 uniq,aBoard追加*********************/
  hipFree(d_uniq);
  hipFree(d_aBoard);
  /************************/
  delete[] totalDown;
  delete[] totalLeft;
  delete[] totalRight;
  delete[] h_results;
  /***07 uniq aBoard追加*********************/
  delete[] h_uniq;
  delete[] t_aBoard;
  /************************/
  return total;
}
//
//GPU
void NQueenG(int size,int steps)
{
  //register int sizeE=size-1;
  register int bit=0;
  register int mask=((1<<size)-1);
  if(size<=0||size>32){return;}
  /***07 ミラーリングしない*********************/
  //偶数、奇数共通
  for(int col=0;col<size;col++){
    aBoard[0]=bit=(1<<col);
    TOTAL+=solve_nqueen_cuda(size,mask,1,bit<<1,bit,bit>>1,steps);
  }
  /************************/
  /***07 ミラーリングしないためコメント*********************/
  //偶数、奇数共通 右側半分だけクイーンを置く
	//int lim=(size%2==0)?size/2:sizeE/2;
  //for(int col=0;col<lim;col++){
  //  bit=(1<<col);
  //  TOTAL+=solve_nqueen_cuda(size,mask,1,bit<<1,bit,bit>>1,steps);
  //}
  //ミラーなのでTOTALを２倍する
  //TOTAL=TOTAL*2;
  //奇数の場合はさらに中央にクイーンを置く
  //if(size%2==1){
  //  bit=(1<<(sizeE)/2);
  //  TOTAL+=solve_nqueen_cuda(size,mask,1,bit<<1,bit,bit>>1,steps);
  //}
  /************************/
}
//SGPU
__global__ 
void sgpu_cuda_kernel(int size,int mark,unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,unsigned int* results,int totalCond)
{
  const int tid=threadIdx.x;
  const int bid=blockIdx.x;
  const int idx=bid*blockDim.x+tid;
  __shared__ unsigned int down[THREAD_NUM][10];
  __shared__ unsigned int left[THREAD_NUM][10];
  __shared__ unsigned int right[THREAD_NUM][10];
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  __shared__ unsigned int sum[THREAD_NUM];
  const unsigned int mask=(1<<size)-1;
  int total=0;
  int row=0;
  unsigned int bit;
  if(idx<totalCond){
    down[tid][row]=totalDown[idx];
    left[tid][row]=totalLeft[idx];
    right[tid][row]=totalRight[idx];
    bitmap[tid][row]=down[tid][row]|left[tid][row]|right[tid][row];
    while(row>=0){
      if((bitmap[tid][row]&mask)==mask){row--;}
      else{
        bit=(bitmap[tid][row]+1)&~bitmap[tid][row];
        bitmap[tid][row]|=bit;
        if((bit&mask)!=0){
          if(row+1==mark){total++;row--;}
          else{
            down[tid][row+1]=down[tid][row]|bit;
            left[tid][row+1]=(left[tid][row]|bit)<<1;
            right[tid][row+1]=(right[tid][row]|bit)>>1;
            bitmap[tid][row+1]=(down[tid][row+1]|left[tid][row+1]|right[tid][row+1]);
            row++;
          }
        }else{row--;}
      }
    }
    sum[tid]=total;
  }else{sum[tid]=0;} 
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){sum[tid]+=sum[tid+64];} 
  __syncthreads();if(tid<32){sum[tid]+=sum[tid+32];} 
  __syncthreads();if(tid<16){sum[tid]+=sum[tid+16];} 
  __syncthreads();if(tid<8){sum[tid]+=sum[tid+8];} 
  __syncthreads();if(tid<4){sum[tid]+=sum[tid+4];} 
  __syncthreads();if(tid<2){sum[tid]+=sum[tid+2];} 
  __syncthreads();if(tid<1){sum[tid]+=sum[tid+1];} 
  __syncthreads();if(tid==0){results[bid]=sum[0];}
}
//SGPU
long long sgpu_solve_nqueen_cuda(int size,int steps)
{
  unsigned int down[32];
  unsigned int left[32];
  unsigned int right[32];
  unsigned int bitmap[32];
  unsigned int bit;
  if(size<=0||size>32){return 0;}
  unsigned int* totalDown=new unsigned int[steps];
  unsigned int* totalLeft=new unsigned int[steps];
  unsigned int* totalRight=new unsigned int[steps];
  unsigned int* results=new unsigned int[steps];
  unsigned int* downCuda;
  unsigned int* leftCuda;
  unsigned int* rightCuda;
  unsigned int* resultsCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);
  const unsigned int mask=(1<<size)-1;
  const unsigned int mark=size>11?size-10:2;
  long long total=0;
  int totalCond=0;
  int row=0;
  down[0]=0;
  left[0]=0;
  right[0]=0;
  bitmap[0]=0;
  bool matched=false;
  for(int col=0;col<size/2;col++){
    bit=(1<<col);
    bitmap[0]|=bit;
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=(down[1]|left[1]|right[1]);
    row=1;
    while(row>0){
      if((bitmap[row]&mask)==mask){row--;}
      else{
        bit=(bitmap[row]+1)&~bitmap[row];
        bitmap[row]|=bit;
        if((bit&mask)!=0){
          down[row+1]=down[row]|bit;
          left[row+1]=(left[row]|bit)<<1;
          right[row+1]=(right[row]|bit)>>1;
          bitmap[row+1]=(down[row+1]|left[row+1]|right[row+1]);
          row++;
          if(row==mark){
            totalDown[totalCond]=down[row];
            totalLeft[totalCond]=left[row];
            totalRight[totalCond]=right[row];
            totalCond++;
            if(totalCond==steps){
              if(matched){
                hipMemcpy(results,resultsCuda,
                    sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
                matched=false;
              }
              hipMemcpy(downCuda,totalDown,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              matched=true;
              totalCond=0;
            }
            row--;
          }
        }else{row--;}
      }
    }
  }
  if(matched){
    hipMemcpy(results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
    matched=false;
  }
  hipMemcpy(downCuda,totalDown,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  /** backTrack+bitmap*/
  sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
    >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
  hipMemcpy(results,resultsCuda,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}	
  total*=2;
  if(size%2==1){
    matched=false;
    totalCond=0;
    bit=(1<<(size-1)/2);
    bitmap[0]|=bit;
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=(down[1]|left[1]|right[1]);
    row=1;
    while(row>0){
      if((bitmap[row]&mask)==mask){row--;}
      else{
        bit=(bitmap[row]+1)&~bitmap[row];
        bitmap[row]|=bit;
        if((bit&mask)!=0){
          down[row+1]=down[row]|bit;
          left[row+1]=(left[row]|bit)<<1;
          right[row+1]=(right[row]|bit)>>1;
          bitmap[row+1]=(down[row+1]|left[row+1]|right[row+1]);
          row++;
          if(row==mark){
            totalDown[totalCond]=down[row];
            totalLeft[totalCond]=left[row];
            totalRight[totalCond]=right[row];
            totalCond++;
            if(totalCond==steps){
              if(matched){
                hipMemcpy(results,resultsCuda,
                    sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
                matched=false;
              }
              hipMemcpy(downCuda,totalDown,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              matched=true;
              totalCond=0;
            }
            row--;
          }
        }else{row--;}
      }
    }
    if(matched){
      hipMemcpy(results,resultsCuda,
          sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
      for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
      matched=false;
    }
    hipMemcpy(downCuda,totalDown,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(leftCuda,totalLeft,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(rightCuda,totalRight,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    /** backTrack+bitmap*/
    sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
      >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
    hipMemcpy(results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
  }
  hipFree(downCuda);
  hipFree(leftCuda);
  hipFree(rightCuda);
  hipFree(resultsCuda);
  delete[] totalDown;
  delete[] totalLeft;
  delete[] totalRight;
  delete[] results;
  return total;
}
/** GPU/SGPU CUDA 初期化 **/
bool InitCUDA()
{
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
//CPU/GPU
//hh:mm:ss.ms形式に処理時間を出力
void TimeFormat(clock_t utime,char *form)
{
  int dd,hh,mm;
  float ftime,ss;
  ftime=(float)utime/CLOCKS_PER_SEC;
  mm=(int)ftime/60;
  ss=ftime-(int)(mm*60);
  dd=mm/(24*60);
  mm=mm%(24*60);
  hh=mm/60;
  mm=mm%60;
  if(dd)
    sprintf(form,"%4d %02d:%02d:%05.2f",dd,hh,mm,ss);
  else if(hh)
    sprintf(form,"     %2d:%02d:%05.2f",hh,mm,ss);
  else if(mm)
    sprintf(form,"        %2d:%05.2f",mm,ss);
  else
    sprintf(form,"           %5.2f",ss);
}
//
long getUnique()
{
  return COUNT2+COUNT4+COUNT8;
}
//
long getTotal()
{
  return COUNT2*2+COUNT4*4+COUNT8*8;
}
//CPU
void symmetryOps_bitmap(int si)
{
  int nEquiv;
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<si;i++){ aT[i]=aBoard[i];}
  rotate_bitmap(aT,aS,si);    //時計回りに90度回転
  int k=intncmp(aBoard,aS,si);
  if(k>0)return;
  if(k==0){ nEquiv=2;}else{
    rotate_bitmap(aS,aT,si);  //時計回りに180度回転
    k=intncmp(aBoard,aT,si);
    if(k>0)return;
    if(k==0){ nEquiv=4;}else{
      rotate_bitmap(aT,aS,si);//時計回りに270度回転
      k=intncmp(aBoard,aS,si);
      if(k>0){ return;}
      nEquiv=8;
    }
  }
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<si;i++){ aS[i]=aBoard[i];}
  vMirror_bitmap(aS,aT,si);   //垂直反転
  k=intncmp(aBoard,aT,si);
  if(k>0){ return; }
  if(nEquiv>2){             //-90度回転 対角鏡と同等
    rotate_bitmap(aT,aS,si);
    k=intncmp(aBoard,aS,si);
    if(k>0){return;}
    if(nEquiv>4){           //-180度回転 水平鏡像と同等
      rotate_bitmap(aS,aT,si);
      k=intncmp(aBoard,aT,si);
      if(k>0){ return;}       //-270度回転 反対角鏡と同等
      rotate_bitmap(aT,aS,si);
      k=intncmp(aBoard,aS,si);
      if(k>0){ return;}
    }
  }
  if(nEquiv==2){COUNT2++;}
  if(nEquiv==4){COUNT4++;}
  if(nEquiv==8){COUNT8++;}
}
//
//CPU 非再帰版 ロジックメソッド
void solve_nqueen(int size,int mask, int row,int* left,int* down,int* right,int* bitmap)
{
    unsigned int bit;
    unsigned int sizeE=size-1;
    int mark=row;
    //固定していれた行より上はいかない
    while(row>=mark){//row=1 row>=1, row=2 row>=2
      if(bitmap[row]==0){
        --row;
      }else{
        bitmap[row]^=aBoard[row]=bit=(-bitmap[row]&bitmap[row]); 
        if((bit&mask)!=0){
          if(row==sizeE){
            symmetryOps_bitmap(size);
            --row;
          }else{
            int n=row++;
            left[row]=(left[n]|bit)<<1;
            down[row]=down[n]|bit;
            right[row]=(right[n]|bit)>>1;
            bitmap[row]=mask&~(left[row]|down[row]|right[row]);
          }
        }else{
           --row;
        }
      }  
    }
}
//
//非再帰版
void NQueen(int size,int mask)
{
  register int bitmap[size];
  register int down[size],right[size],left[size];
  register int bit;
  if(size<=0||size>32){return;}
  bit=0;
  bitmap[0]=mask;
  down[0]=left[0]=right[0]=0;
  //偶数、奇数共通
  for(int col=0;col<size;col++){
    aBoard[0]=bit=(1<<col);
    down[1]=bit;//再帰の場合は down,left,right,bitmapは現在の行だけで良いが
    left[1]=bit<<1;//非再帰の場合は全行情報を配列に入れて行の上がり下がりをする
    right[1]=bit>>1;
    bitmap[1]=mask&~(left[1]|down[1]|right[1]);
    solve_nqueen(size,mask,1,left,down,right,bitmap);
  }
}
//CPUR 再帰版 ロジックメソッド
void solve_nqueenr(int size,int mask, int row,int left,int down,int right)
{
 int bitmap=0;
 int bit=0;
 int sizeE=size-1;
 bitmap=(mask&~(left|down|right));
 if(row==sizeE){
    if(bitmap){
      aBoard[row]=(-bitmap&bitmap);
      symmetryOps_bitmap(size);
    }
  }else{
    while(bitmap){
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap);
      solve_nqueenr(size,mask,row+1,(left|bit)<<1, down|bit,(right|bit)>>1);
    }
  }
}
//CPUR 再帰版 ロジックメソッド
void NQueenR(int size,int mask)
{
  int bit=0;
  //1行目全てにクイーンを置く
  for(int col=0;col<size;col++){
    aBoard[0]=bit=(1<<col);
    solve_nqueenr(size,mask,1,bit<<1,bit,bit>>1);
  }
}
//
//通常版 CPU 非再帰版 ロジックメソッド
void NQueenD(int size,int mask,int row)
{
  int aStack[size];
  int* pnStack;
  int bit;
  int bitmap;
  int sizeE=size-1;
  int down[size],right[size],left[size];
  aStack[0]=-1; 
  pnStack=aStack+1;
  bit=0;
  bitmap=mask;
  down[0]=left[0]=right[0]=0;
  while(true){
    if(bitmap){
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap); 
      if(row==sizeE){
        /* 対称解除法の追加 */
        //TOTAL++;
        symmetryOps_bitmap(size); 
        bitmap=*--pnStack;
        --row;
        continue;
      }else{
        int n=row++;
        left[row]=(left[n]|bit)<<1;
        down[row]=down[n]|bit;
        right[row]=(right[n]|bit)>>1;
        *pnStack++=bitmap;
        bitmap=mask&~(left[row]|down[row]|right[row]);
        continue;
      }
    }else{ 
      bitmap=*--pnStack;
      if(pnStack==aStack){ break ; }
      --row;
      continue;
    }
  }
}
//
//通常版 CPUR 再帰版　ロジックメソッド
void NQueenDR(int size,int mask,int row,int left,int down,int right)
{
  int bit;
  int bitmap=mask&~(left|down|right);
  if(row==size){
    /* 対称解除法の追加 */
    //TOTAL++;
    symmetryOps_bitmap(size);
  }else{
    while(bitmap){
      //bitmap^=bit=(-bitmap&bitmap);
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap);
      NQueenDR(size,mask,row+1,(left|bit)<<1,down|bit,(right|bit)>>1);
    }
  }
}
//メインメソッド
int main(int argc,char** argv)
{
  bool cpu=false,cpur=false,gpu=false,sgpu=false;
  int argstart=1,steps=24576;
  //int argstart=1,steps=1;
  
  /** パラメータの処理 */
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='r'||argv[1][1]=='R'){cpur=true;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){gpu=true;}
    else if(argv[1][1]=='s'||argv[1][1]=='S'){sgpu=true;}
    else
      cpur=true;
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g|-r|-s]\n",argv[0]);
    printf("  -c: CPU only\n");
    printf("  -r: CPUR only\n");
    printf("  -g: GPU only\n");
    printf("  -s: SGPU only\n");
    printf("Default to 8 queen\n");
  }
  /** 出力と実行 */
  if(cpu){
    printf("\n\n７．CPU 非再帰 バックトラック＋ビットマップ＋対称解除法\n");
  }else if(cpur){
    printf("\n\n７．CPUR 再帰 バックトラック＋ビットマップ＋対称解除法\n");
  }else if(gpu){
    printf("\n\n７．GPU 非再帰 バックトラック＋ビットマップ＋対称解除法\n");
  }else if(sgpu){
    printf("\n\n７．SGPU 非再帰 バックトラック＋ビットマップ\n");
  }
  if(cpu||cpur){
    printf("%s\n"," N:        Total       Unique        hh:mm:ss.ms");
    clock_t st;           //速度計測用
    char t[20];           //hh:mm:ss.msを格納
    int min=4; int targetN=17;
    int mask;
    for(int i=min;i<=targetN;i++){
      //TOTAL=0; UNIQUE=0;
      COUNT2=COUNT4=COUNT8=0;
      mask=(1<<i)-1;
      st=clock();
      //
      //【通常版】
      //if(cpur){ _NQueenR(i,mask,0,0,0,0); }
      //CPUR
      if(cpur){ 
        NQueenR(i,mask); 
        //printf("通常版\n");
        //NQueenDR(i,mask,0,0,0,0); //通常版
      }
      //CPU
      if(cpu){ 
        NQueen(i,mask); 
        //printf("通常版\n");
        //NQueenD(i,mask,0); //通常版
      }
      //
      TimeFormat(clock()-st,t); 
      printf("%2d:%13ld%16ld%s\n",i,getTotal(),getUnique(),t);
    }
  }
  if(gpu||sgpu){
    if(!InitCUDA()){return 0;}
    int min=4;int targetN=17;
    //int min=7;int targetN=7;
   
    struct timeval t0;struct timeval t1;
    int ss;int ms;int dd;
    printf("%s\n"," N:        Total      Unique      dd:hh:mm:ss.ms");
    for(int i=min;i<=targetN;i++){
      gettimeofday(&t0,NULL);   // 計測開始
      if(gpu){
        TOTAL=0;
        UNIQUE=0;
        NQueenG(i,steps);
      }else if(sgpu){
        TOTAL=sgpu_solve_nqueen_cuda(i,steps);
      }
      gettimeofday(&t1,NULL);   // 計測終了
      if (t1.tv_usec<t0.tv_usec) {
        dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      } else {
        dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%13ld%16ld%4.2d:%02d:%02d:%02d.%02d\n", i,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    }
  }
  return 0;
}