/**
 CUDAで学ぶアルゴリズムとデータ構造
 ステップバイステップでＮ−クイーン問題を最適化
 一般社団法人  共同通信社  情報技術局  鈴木  維一郎(suzuki.iichiro@kyodonews.jp)

 コンパイルと実行
 $ nvcc -O3 CUDA**_N-Queen.cu && ./a.out (-c|-r|-g)
                    -c:cpu 
                    -r cpu再帰 
                    -g GPU 

$ nvcc -O3 CUDA07_N-Queen.cu  && ./a.out -g
７．GPU 非再帰 バックトラック＋ビットマップ＋対称解除法
 N:        Total      Unique      dd:hh:mm:ss.ms
 4:            2               1  00:00:00:00.05
 5:           10               2  00:00:00:00.00
 6:            4               1  00:00:00:00.00
 7:           40               6  00:00:00:00.00
 8:           92              12  00:00:00:00.02
 9:          352              46  00:00:00:00.03
10:          724              92  00:00:00:00.07
11:         2680             341  00:00:00:00.19
12:        14200            1787  00:00:00:00.59
13:        73712            9233  00:00:00:01.13
14:       365596           45752  00:00:00:01.20
15:      2279184          285053  00:00:00:07.10
16:     14772512         1846955  00:00:00:45.59
17:     95815104        11977939  00:00:06:04.57
*/
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define THREAD_NUM		96
#define MAX 27
//変数宣言
long TOTAL=0; //GPU,CPUで使用
long UNIQUE=0;//GPU,CPUで使用
//
__device__ __host__
int* vMirror(int* bf,int* af,int si)
{
  int bf_i;
  int tmp;
  for(int i=0;i<si;i++) {
    bf_i=bf[i];
    tmp=0;
    for(int j=0;j<=si-1;j++){
      if(bf_i&(1<<j)){ 
        tmp|=(1<<(si-1-j)); 
        break;                 
      }
    }
    af[i]=tmp;
  }
  return af;
}
//
__device__ __host__
int* rotate(int* bf,int* af,int si)
{
  int t;
  for(int i=0;i<si;i++){
    t=0;
    for(int j=0;j<si;j++){
      t|=((bf[j]>>i)&1)<<(si-j-1);
    }
    af[i]=t;
  }
  return af;
}
//
__device__ __host__
int intncmp(unsigned int* lt,int* rt,int n)
{
  int rtn=0;
  for(int k=0;k<n;k++){
    rtn=lt[k]-rt[k];
    if(rtn!=0){
      break;
    }
  }
  return rtn;
}
//
__device__ __host__
int symmetryOps(int si,unsigned int *aBoard)
{
  int nEquiv=0;
  int aT[MAX];
  int aS[MAX];
  // 回転・反転・対称チェックのためにboard配列をコピー
  memcpy(aT,aBoard,sizeof(int)*si);
  //時計回りに90度回転
  rotate(aT,aS,si);
  int icmp=intncmp(aBoard,aS,si);
  if(icmp>0){ return 0; }
  else if(icmp==0){ nEquiv=2; }
  else{//時計回りに180度回転
    rotate(aS,aT,si);
    icmp=intncmp(aBoard,aT,si);
    if(icmp>0){ return 0;}
    else if(icmp==0){ nEquiv=4;}
    else{//時計回りに270度回転
      rotate(aT,aS,si);
      icmp=intncmp(aBoard,aS,si);
      if(icmp>0){ return 0;}
      nEquiv=8;
    }
  }
  // 回転・反転・対称チェックのためにboard配列をコピー
  memcpy(aS,aBoard,sizeof(int)*si);
  //垂直反転
  vMirror(aS,aT,si);   
  icmp=intncmp(aBoard,aT,si);
  if(icmp>0){ return 0; }
  //-90度回転 対角鏡と同等
  if(nEquiv>2){
    rotate(aT,aS,si);
    icmp=intncmp(aBoard,aS,si);
    if(icmp>0){return 0;}
    //-180度回転 水平鏡像と同等
    else if(nEquiv>4){
      rotate(aS,aT,si);
      icmp=intncmp(aBoard,aT,si);
      //-270度回転 反対角鏡と同等
      if(icmp>0){ return 0;}
      rotate(aT,aS,si);
      icmp=intncmp(aBoard,aS,si);
      if(icmp>0){ return 0;}
    }
  }
  return nEquiv;  
}
//
__global__
void cuda_kernel(
    register int size,
    register int mark,
    unsigned int* totalDown,
    unsigned int* totalLeft,
    unsigned int* totalRight,
    unsigned int* d_results,
    unsigned int* d_uniq,
    register int totalCond,
    unsigned int* t_aBoard,
    register int h_row)
{
  register const unsigned int mask=(1<<size)-1;
  register unsigned int total=0;
  register unsigned int unique=0;
  //row=0となってるが1行目からやっているわけではなく
  //mask行目以降からスタート 
  //n=8 なら mask==2 なので そこからスタート
  register int row=0;
  register unsigned int bit;
  //
  //スレッド
  //
  //ブロック内のスレッドID
  register unsigned const int tid=threadIdx.x;
  //グリッド内のブロックID
  register unsigned const int bid=blockIdx.x;
  //全体通してのID
  register unsigned const int idx=bid*blockDim.x+tid;
  //
  //シェアードメモリ
  //
  //sharedメモリを使う ブロック内スレッドで共有
  //10固定なのは現在のmask設定で
  //GPUで実行するのは最大10だから
  //THREAD_NUMはブロックあたりのスレッド数
  __shared__ unsigned int down[THREAD_NUM][10];
  down[tid][row]=totalDown[idx];
  __shared__ unsigned int left[THREAD_NUM][10];
  left[tid][row]=totalLeft[idx];
  __shared__ unsigned int right[THREAD_NUM][10];
  right[tid][row]=totalRight[idx];
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  //down,left,rightからbitmapを出す
  bitmap[tid][row]
    =mask&~(
         down[tid][row]
        |left[tid][row]
        |right[tid][row]);
  __shared__ unsigned int sum[THREAD_NUM];
  __shared__ unsigned int usum[THREAD_NUM];
  unsigned int c_aBoard[MAX];
  //
  //余分なスレッドは動かさない 
  //GPUはsteps数起動するがtotalCond以上は空回しする
  if(idx<totalCond){
    //totalDown,totalLeft,totalRightの情報を
    //down,left,rightに詰め直す 
    //CPU で詰め込んだ t_はsteps個あるが
    //ブロック内ではブロックあたりのスレッド数に限定
    //されるので idxでよい
    //
    for(int i=0;i<h_row;i++){
      c_aBoard[i]=t_aBoard[idx*h_row+i]; //２次元配列だが1次元的に利用  
    }
    register unsigned int bitmap_tid_row;
    register unsigned int down_tid_row;
    register unsigned int left_tid_row;
    register unsigned int right_tid_row;
    while(row>=0){
      bitmap_tid_row=bitmap[tid][row];
      down_tid_row=down[tid][row];
      left_tid_row=left[tid][row];
      right_tid_row=right[tid][row];
      //
      //bitmap[tid][row]=00000000 クイーンを
      //どこにも置けないので1行上に戻る
      if(bitmap_tid_row==0){
        row--;
      }else{
        //クイーンを置く
        //置く場所があるかどうか
        bitmap[tid][row]
          ^=c_aBoard[row+h_row]
          =bit
          =(-bitmap_tid_row&bitmap_tid_row);       
        if((bit&mask)!=0){
          //最終行?最終行から１個前の行まで
          //無事到達したら 加算する
          if(row+1==mark){
           int s=symmetryOps(size,c_aBoard); 
           if(s!=0){
           //print(size); //print()でTOTALを++しない
           //ホストに戻す配列にTOTALを入れる
           //スレッドが１つの場合は配列は１個
              unique++; 
              total+=s;   //対称解除で得られた解数を加算
           }
            row--;
          }else{
            int rowP=row+1;
            down[tid][rowP]=down_tid_row|bit;
            left[tid][rowP]=(left_tid_row|bit)<<1;
            right[tid][rowP]=(right_tid_row|bit)>>1;
            bitmap[tid][rowP]
              =mask&~(
                  down[tid][rowP]
                  |left[tid][rowP]
                  |right[tid][rowP]);
            row++;
          }
        }else{
          //置く場所がなければ１個上に
          row--;
        }
      }
    }
    //最後sum[tid]に加算する
    sum[tid]=total;
    usum[tid]=unique;
  }else{
    //totalCond未満は空回しするのでtotalは加算しない
    sum[tid]=0;
    usum[tid]=0;
  } 
  //__syncthreads()でブロック内のスレッド間の同期
  //全てのスレッドが__syncthreads()に辿り着くのを待つ
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){
    sum[tid]+=sum[tid+64];
    usum[tid]+=usum[tid+64];
  }
  __syncwarp();if(tid<32){
    sum[tid]+=sum[tid+32];
    usum[tid]+=usum[tid+32];
  } 
  __syncwarp();if(tid<16){
    sum[tid]+=sum[tid+16];
    usum[tid]+=usum[tid+16];
  } 
  __syncwarp();if(tid<8){
    sum[tid]+=sum[tid+8];
    usum[tid]+=usum[tid+8];
  } 
  __syncwarp();if(tid<4){
    sum[tid]+=sum[tid+4];
    usum[tid]+=usum[tid+4];
  } 
  __syncwarp();if(tid<2){
    sum[tid]+=sum[tid+2];
    usum[tid]+=usum[tid+2];
  } 
  __syncwarp();if(tid<1){
    sum[tid]+=sum[tid+1];
    usum[tid]+=usum[tid+1];
  } 
  __syncwarp();if(tid==0){
    d_results[bid]=sum[0];
    d_uniq[bid]=usum[0];
  }
}
//
long solve_nqueen_cuda(int size,int mask,int row,int n_left,int n_down,int n_right,int steps,unsigned int* aBoard)
{
  //何行目からGPUで行くか。ここの設定は変更可能、設定値を多くするほどGPUで並行して動く
  /***08 クイーンを２行目まで固定で置くためmarkが3以上必要*********************/
  const unsigned int mark=size>12?size-10:3;
  const unsigned int h_mark=row;
  long total=0;
  int totalCond=0;
  bool matched=false;
  //host
  unsigned int down[32];  down[row]=n_down;
  unsigned int right[32]; right[row]=n_right;
  unsigned int left[32];  left[row]=n_left;
  //bitmapを配列で持つことにより
  //stackを使わないで1行前に戻れる
  unsigned int bitmap[32];
  bitmap[row]=mask&~(left[row]|down[row]|right[row]);
  unsigned int bit;
  unsigned int* totalDown;
  hipHostMalloc((void**) &totalDown,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* totalLeft;
  hipHostMalloc((void**) &totalLeft,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* totalRight;
  hipHostMalloc((void**) &totalRight,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* h_results;
  hipHostMalloc((void**) &h_results,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* h_uniq;
  hipHostMalloc((void**) &h_uniq,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* t_aBoard;
  hipHostMalloc((void**) &t_aBoard,sizeof(int)*steps*mark, hipHostMallocDefault);
  //device
  unsigned int* downCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  unsigned int* leftCuda;
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  unsigned int* rightCuda;
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  unsigned int* resultsCuda;
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);
  unsigned int* d_uniq;
  hipMalloc((void**) &d_uniq,sizeof(int)*steps/THREAD_NUM);
  unsigned int* d_aBoard;
  hipMalloc((void**) &d_aBoard,sizeof(int)*steps*mark);
  //12行目までは3行目までCPU->row==mark以下で 3行目までの
  //down,left,right情報を totalDown,totalLeft,totalRight
  //に格納
  //する->3行目以降をGPUマルチスレッドで実行し結果を取得
  //13行目以降はCPUで実行する行数が１個ずつ増えて行く
  //例えばn15だとrow=5までCPUで実行し、
  //それ以降はGPU(現在の設定だとGPUでは最大10行実行する
  //ようになっている)
  register int rowP=0;
  while(row>=h_mark) {
    //bitmap[row]=00000000 クイーンを
    //どこにも置けないので1行上に戻る
    //06GPU こっちのほうが優秀
    if(bitmap[row]==0){ row--; }
    else{//おける場所があれば進む
      bitmap[row]^=aBoard[row]=bit=(-bitmap[row]&bitmap[row]);
      if((bit&mask)!=0){//置く場所があれば先に進む
        rowP=row+1;
        down[rowP]=down[row]|bit;
        left[rowP]=(left[row]|bit)<<1;
        right[rowP]=(right[row]|bit)>>1;
        bitmap[rowP]=mask&~(down[rowP]|left[rowP]|right[rowP]);
        row++;
        if(row==mark){
          //3行目(mark)にクイーンを１個ずつ置いていって、
          //down,left,right情報を格納、
          //その次の行へは進まない。その行で可能な場所にクイー
          //ン置き終わったらGPU並列実行
          //totalCond がthreadIdになる 各スレッドに down,left,right情報を渡す
          //row=2(13行目以降は増えていく。例えばn15だとrow=5)の情報を
          //totalDown,totalLeft,totalRightに格納する
          totalDown[totalCond]=down[row];
          totalLeft[totalCond]=left[row];
          totalRight[totalCond]=right[row];
          for(int i=0;i<mark;i++){
            t_aBoard[totalCond*mark+i]=aBoard[i];
          }
          //スレッド数をインクリメントする
          totalCond++;
          //最大GPU数に達してしまったら一旦ここでGPUを実行する。stepsはGPUの同
          //時並行稼働数を制御
          //nの数が少ないうちはtotalCondがstepsを超えることはないがnの数が増え
          //て行くと超えるようになる。
          //ここではtotalCond==stepsの場合だけこの中へ         
          if(totalCond==steps){
            //matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUか
            //ら出たらmatched=trueになってる
            if(matched){
              hipMemcpy(h_results,resultsCuda,
                  sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
              hipMemcpy(h_uniq,d_uniq,
                  sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
              for(int col=0;col<steps/THREAD_NUM;col++){
                total+=h_results[col];
                UNIQUE+=h_uniq[col];
              }
              matched=false;
            }
            hipMemcpy(downCuda,totalDown,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(leftCuda,totalLeft,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(rightCuda,totalRight,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(d_aBoard,t_aBoard,
                sizeof(int)*totalCond*mark,hipMemcpyHostToDevice);
            cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
              >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,d_aBoard,row);
            //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われ
            //るのはtotalCondの数だけでそれ以外は空回しになる
            //GPU内でカウントしているので、GPUから出たらmatched=trueになってる
            matched=true;
            //totalCond==stepsルートでGPUを実行したらスレッドをまた0から開始す
            //る(これによりなんどもsteps数分だけGPUを起動できる)
            totalCond=0;           
          }
          //totalDown,totalLeft,totalRightに情報を格納したら1行上に上がる
          //これを繰り返すことにより row=2で可能な場所全てにクイーンを置いて
          //totalDown,totalLeft,totalRightに情報を格納する
          row--;
        }
      }else{
        //置く場所がなければ上に上がる。row==mark行に達するまではCPU側で普通に
        //nqueenをやる
        row--;
      }
    }
  }
  //matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUから出たら
  //matched=trueになってる
  if(matched){
    hipMemcpy(h_results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    hipMemcpy(h_uniq,d_uniq,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){
      total+=h_results[col];
      UNIQUE+=h_uniq[col];
    }
    matched=false;
  }
  hipMemcpy(downCuda,totalDown,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(d_aBoard,t_aBoard,
      sizeof(int)*totalCond*mark,hipMemcpyHostToDevice);
  //size-mark は何行GPUを実行するか totalCondはスレッド数
  //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われるのは
  //totalCondの数だけでそれ以外は空回しになる
  cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
    >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,d_aBoard,mark);
  hipMemcpy(h_results,resultsCuda,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  hipMemcpy(h_uniq,d_uniq,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  for(int col=0;col<steps/THREAD_NUM;col++){
    total+=h_results[col];
    UNIQUE+=h_uniq[col];
  }
  //
  hipFree(downCuda);
  hipFree(leftCuda);
  hipFree(rightCuda);
  hipFree(resultsCuda);
  hipFree(d_uniq);
  hipFree(d_aBoard);
  hipHostFree(totalDown);
  hipHostFree(totalLeft);
  hipHostFree(totalRight);
  hipHostFree(h_results);
  hipHostFree(h_uniq);
  hipHostFree(t_aBoard);
  return total;
}
//
void NQueenG(int size,int steps)
{
  unsigned int aBoard[MAX];
  register int bit=0;
  register int mask=((1<<size)-1);
  if(size<=0||size>32){return;}
  //偶数、奇数共通
  for(int col=0;col<size;col++){
    aBoard[0]=bit=(1<<col);
    TOTAL+=solve_nqueen_cuda(size,mask,1,bit<<1,bit,bit>>1,steps,aBoard);
  }
}
//SGPU
__global__ 
void sgpu_cuda_kernel(int size,int mark,unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,unsigned int* results,int totalCond)
{
  const int tid=threadIdx.x;
  const int bid=blockIdx.x;
  const int idx=bid*blockDim.x+tid;
  __shared__ unsigned int down[THREAD_NUM][10];
  __shared__ unsigned int left[THREAD_NUM][10];
  __shared__ unsigned int right[THREAD_NUM][10];
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  __shared__ unsigned int sum[THREAD_NUM];
  const unsigned int mask=(1<<size)-1;
  int total=0;
  int row=0;
  unsigned int bit;
  if(idx<totalCond){
    down[tid][row]=totalDown[idx];
    left[tid][row]=totalLeft[idx];
    right[tid][row]=totalRight[idx];
    bitmap[tid][row]=down[tid][row]|left[tid][row]|right[tid][row];
    while(row>=0){
      if((bitmap[tid][row]&mask)==mask){row--;}
      else{
        bit=(bitmap[tid][row]+1)&~bitmap[tid][row];
        bitmap[tid][row]|=bit;
        if((bit&mask)!=0){
          if(row+1==mark){total++;row--;}
          else{
            down[tid][row+1]=down[tid][row]|bit;
            left[tid][row+1]=(left[tid][row]|bit)<<1;
            right[tid][row+1]=(right[tid][row]|bit)>>1;
            bitmap[tid][row+1]=(down[tid][row+1]|left[tid][row+1]|right[tid][row+1]);
            row++;
          }
        }else{row--;}
      }
    }
    sum[tid]=total;
  }else{sum[tid]=0;} 
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){sum[tid]+=sum[tid+64];} 
  __syncthreads();if(tid<32){sum[tid]+=sum[tid+32];} 
  __syncthreads();if(tid<16){sum[tid]+=sum[tid+16];} 
  __syncthreads();if(tid<8){sum[tid]+=sum[tid+8];} 
  __syncthreads();if(tid<4){sum[tid]+=sum[tid+4];} 
  __syncthreads();if(tid<2){sum[tid]+=sum[tid+2];} 
  __syncthreads();if(tid<1){sum[tid]+=sum[tid+1];} 
  __syncthreads();if(tid==0){results[bid]=sum[0];}
}
//SGPU
long long sgpu_solve_nqueen_cuda(int size,int steps)
{
  unsigned int down[32];
  unsigned int left[32];
  unsigned int right[32];
  unsigned int bitmap[32];
  unsigned int bit;
  if(size<=0||size>32){return 0;}
  unsigned int* totalDown=new unsigned int[steps];
  unsigned int* totalLeft=new unsigned int[steps];
  unsigned int* totalRight=new unsigned int[steps];
  unsigned int* results=new unsigned int[steps];
  unsigned int* downCuda;
  unsigned int* leftCuda;
  unsigned int* rightCuda;
  unsigned int* resultsCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);
  const unsigned int mask=(1<<size)-1;
  const unsigned int mark=size>11?size-10:2;
  long long total=0;
  int totalCond=0;
  int row=0;
  down[0]=0;
  left[0]=0;
  right[0]=0;
  bitmap[0]=0;
  bool matched=false;
  for(int col=0;col<size/2;col++){
    bit=(1<<col);
    bitmap[0]|=bit;
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=(down[1]|left[1]|right[1]);
    row=1;
    while(row>0){
      if((bitmap[row]&mask)==mask){row--;}
      else{
        bit=(bitmap[row]+1)&~bitmap[row];
        bitmap[row]|=bit;
        if((bit&mask)!=0){
          down[row+1]=down[row]|bit;
          left[row+1]=(left[row]|bit)<<1;
          right[row+1]=(right[row]|bit)>>1;
          bitmap[row+1]=(down[row+1]|left[row+1]|right[row+1]);
          row++;
          if(row==mark){
            totalDown[totalCond]=down[row];
            totalLeft[totalCond]=left[row];
            totalRight[totalCond]=right[row];
            totalCond++;
            if(totalCond==steps){
              if(matched){
                hipMemcpy(results,resultsCuda,
                    sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
                matched=false;
              }
              hipMemcpy(downCuda,totalDown,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              matched=true;
              totalCond=0;
            }
            row--;
          }
        }else{row--;}
      }
    }
  }
  if(matched){
    hipMemcpy(results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
    matched=false;
  }
  hipMemcpy(downCuda,totalDown,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
    >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
  hipMemcpy(results,resultsCuda,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}	
  total*=2;
  if(size%2==1){
    matched=false;
    totalCond=0;
    bit=(1<<(size-1)/2);
    bitmap[0]|=bit;
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=(down[1]|left[1]|right[1]);
    row=1;
    while(row>0){
      if((bitmap[row]&mask)==mask){row--;}
      else{
        bit=(bitmap[row]+1)&~bitmap[row];
        bitmap[row]|=bit;
        if((bit&mask)!=0){
          down[row+1]=down[row]|bit;
          left[row+1]=(left[row]|bit)<<1;
          right[row+1]=(right[row]|bit)>>1;
          bitmap[row+1]=(down[row+1]|left[row+1]|right[row+1]);
          row++;
          if(row==mark){
            totalDown[totalCond]=down[row];
            totalLeft[totalCond]=left[row];
            totalRight[totalCond]=right[row];
            totalCond++;
            if(totalCond==steps){
              if(matched){
                hipMemcpy(results,resultsCuda,
                    sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
                matched=false;
              }
              hipMemcpy(downCuda,totalDown,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
                >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              matched=true;
              totalCond=0;
            }
            row--;
          }
        }else{row--;}
      }
    }
    if(matched){
      hipMemcpy(results,resultsCuda,
          sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
      for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
      matched=false;
    }
    hipMemcpy(downCuda,totalDown,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(leftCuda,totalLeft,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(rightCuda,totalRight,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
      >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
    hipMemcpy(results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
  }
  hipFree(downCuda);
  hipFree(leftCuda);
  hipFree(rightCuda);
  hipFree(resultsCuda);
  delete[] totalDown;
  delete[] totalLeft;
  delete[] totalRight;
  delete[] results;
  return total;
}
/** GPU/SGPU CUDA 初期化 **/
bool InitCUDA()
{
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
//hh:mm:ss.ms形式に処理時間を出力
void TimeFormat(clock_t utime,char *form)
{
  int dd,hh,mm;
  float ftime,ss;
  ftime=(float)utime/CLOCKS_PER_SEC;
  mm=(int)ftime/60;
  ss=ftime-(int)(mm*60);
  dd=mm/(24*60);
  mm=mm%(24*60);
  hh=mm/60;
  mm=mm%60;
  if(dd)
    sprintf(form,"%4d %02d:%02d:%05.2f",dd,hh,mm,ss);
  else if(hh)
    sprintf(form,"     %2d:%02d:%05.2f",hh,mm,ss);
  else if(mm)
    sprintf(form,"        %2d:%05.2f",mm,ss);
  else
    sprintf(form,"           %5.2f",ss);
}
//
void solve_nqueen(int size,int mask, int row,int* left,int* down,int* right,int* bitmap,unsigned int* aBoard)
{
    unsigned int bit;
    unsigned int sizeE=size-1;
    int mark=row;
    //固定していれた行より上はいかない
    while(row>=mark){//row=1 row>=1, row=2 row>=2
      if(bitmap[row]==0){
        --row;
      }else{
        bitmap[row]^=aBoard[row]=bit=(-bitmap[row]&bitmap[row]); 
        if((bit&mask)!=0){
          if(row==sizeE){
            int s=symmetryOps(size,aBoard);
            if(s!=0){
              UNIQUE++;
              TOTAL+=s;
            }
            --row;
          }else{
            int n=row++;
            left[row]=(left[n]|bit)<<1;
            down[row]=down[n]|bit;
            right[row]=(right[n]|bit)>>1;
            bitmap[row]=mask&~(left[row]|down[row]|right[row]);
          }
        }else{
           --row;
        }
      }  
    }
}
//
//非再帰版
void NQueen(int size,int mask)
{
  register int bitmap[size];
  register int down[size],right[size],left[size];
  register int bit;
  unsigned int aBoard[MAX];
  if(size<=0||size>32){return;}
  bit=0;
  bitmap[0]=mask;
  down[0]=left[0]=right[0]=0;
  //偶数、奇数共通
  for(int col=0;col<size;col++){
    aBoard[0]=bit=(1<<col);
    down[1]=bit;//再帰の場合は down,left,right,bitmapは現在の行だけで良いが
    left[1]=bit<<1;//非再帰の場合は全行情報を配列に入れて行の上がり下がりをする
    right[1]=bit>>1;
    bitmap[1]=mask&~(left[1]|down[1]|right[1]);
    solve_nqueen(size,mask,1,left,down,right,bitmap,aBoard);
  }
}
//CPUR 再帰版 ロジックメソッド
void solve_nqueenr(int size,int mask, int row,int left,int down,int right,unsigned int* aBoard)
{
 int bitmap=0;
 int bit=0;
 int sizeE=size-1;
 bitmap=(mask&~(left|down|right));
 if(row==sizeE){
    if(bitmap){
      aBoard[row]=(-bitmap&bitmap);
      int s=symmetryOps(size,aBoard);
      if(s!=0){
        UNIQUE++;
        TOTAL+=s;
      }
    }
  }else{
    while(bitmap){
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap);
      solve_nqueenr(size,mask,row+1,(left|bit)<<1, down|bit,(right|bit)>>1,aBoard);
    }
  }
}
//CPUR 再帰版 ロジックメソッド
void NQueenR(int size,int mask)
{
  int bit=0;
  unsigned int aBoard[MAX];
  //1行目全てにクイーンを置く
  for(int col=0;col<size;col++){
    aBoard[0]=bit=(1<<col);
    solve_nqueenr(size,mask,1,bit<<1,bit,bit>>1,aBoard);
  }
}
//
//通常版 CPU 非再帰版 ロジックメソッド
void NQueenD(int size,int mask,int row)
{
  int aStack[size];
  int* pnStack;
  int bit;
  int bitmap;
  int sizeE=size-1;
  int down[size],right[size],left[size];
  unsigned int aBoard[MAX];
  aStack[0]=-1;
  pnStack=aStack+1;
  bit=0;
  bitmap=mask;
  down[0]=left[0]=right[0]=0;
  while(true){
    if(bitmap){
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap); 
      if(row==sizeE){
        /* 対称解除法の追加 */
        //TOTAL++;
        int s=symmetryOps(size,aBoard);
        if(s!=0){
          UNIQUE++;
          TOTAL+=s;
        }
        bitmap=*--pnStack;
        --row;
        continue;
      }else{
        int n=row++;
        left[row]=(left[n]|bit)<<1;
        down[row]=down[n]|bit;
        right[row]=(right[n]|bit)>>1;
        *pnStack++=bitmap;
        bitmap=mask&~(left[row]|down[row]|right[row]);
        continue;
      }
    }else{ 
      bitmap=*--pnStack;
      if(pnStack==aStack){ break ; }
      --row;
      continue;
    }
  }
}
//
//通常版 CPUR 再帰版　ロジックメソッド
void NQueenDR(
    int size,
    int mask,
    int row,
    int left,
    int down,
    int right)
{
  int bit;
  int bitmap=mask&~(left|down|right);
  unsigned int aBoard[MAX];
  if(row==size){
    /* 対称解除法の追加 */
    //TOTAL++;
    int s=symmetryOps(size,aBoard);
    if(s!=0){
      UNIQUE++;
      TOTAL+=s;
    }
  }else{
    while(bitmap){
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap);
      NQueenDR(size,mask,row+1,(left|bit)<<1,down|bit,(right|bit)>>1);
    }
  }
}
//メインメソッド
int main(int argc,char** argv)
{
  bool cpu=false,cpur=false,gpu=false,sgpu=false;
  int argstart=1,steps=24576;
  /** パラメータの処理 */
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='r'||argv[1][1]=='R'){cpur=true;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){gpu=true;}
    else if(argv[1][1]=='s'||argv[1][1]=='S'){sgpu=true;}
    else
      cpur=true;
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g|-r|-s]\n",argv[0]);
    printf("  -c: CPU only\n");
    printf("  -r: CPUR only\n");
    printf("  -g: GPU only\n");
    printf("  -s: SGPU only\n");
    printf("Default to 8 queen\n");
  }
  /** 出力と実行 */
  if(cpu){
    printf("\n\n７．CPU 非再帰 バックトラック＋ビットマップ＋対称解除法\n");
  }else if(cpur){
    printf("\n\n７．CPUR 再帰 バックトラック＋ビットマップ＋対称解除法\n");
  }else if(gpu){
    printf("\n\n７．GPU 非再帰 バックトラック＋ビットマップ＋対称解除法\n");
  }else if(sgpu){
    printf("\n\n７．SGPU 非再帰 バックトラック＋ビットマップ\n");
  }
  if(cpu||cpur){
    printf("%s\n"," N:        Total       Unique        hh:mm:ss.ms");
    clock_t st;           //速度計測用
    char t[20];           //hh:mm:ss.msを格納
    int min=4; int targetN=17;
    int mask;
    for(int i=min;i<=targetN;i++){
      TOTAL=0; UNIQUE=0;
      mask=(1<<i)-1;
      st=clock();
      //
      //【通常版】
      //if(cpur){ _NQueenR(i,mask,0,0,0,0); }
      //CPUR
      if(cpur){ 
        NQueenR(i,mask); 
        //printf("通常版\n");
        //NQueenDR(i,mask,0,0,0,0); //通常版
      }
      //CPU
      if(cpu){ 
        NQueen(i,mask); 
        //printf("通常版\n");
        //NQueenD(i,mask,0); //通常版
      }
      //
      TimeFormat(clock()-st,t); 
      printf("%2d:%13ld%16ld%s\n",i,TOTAL,UNIQUE,t);
    }
  }
  if(gpu||sgpu){
    if(!InitCUDA()){return 0;}
    int min=4;int targetN=17;
    struct timeval t0;struct timeval t1;
    int ss;int ms;int dd;
    printf("%s\n"," N:        Total      Unique      dd:hh:mm:ss.ms");
    for(int i=min;i<=targetN;i++){
      gettimeofday(&t0,NULL);   // 計測開始
      if(gpu){
        TOTAL=0;
        UNIQUE=0;
        NQueenG(i,steps);
      }else if(sgpu){
        TOTAL=sgpu_solve_nqueen_cuda(i,steps);
      }
      gettimeofday(&t1,NULL);   // 計測終了
      if (t1.tv_usec<t0.tv_usec) {
        dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      } else {
        dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%13ld%16ld%4.2d:%02d:%02d:%02d.%02d\n", i,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    }
  }
  return 0;
}
