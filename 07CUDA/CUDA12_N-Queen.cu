
/**
 CUDAで学ぶアルゴリズムとデータ構造
 ステップバイステップでＮ−クイーン問題を最適化
 一般社団法人  共同通信社  情報技術局  鈴木  維一郎(suzuki.iichiro@kyodonews.jp)

 コンパイルと実行
 $ nvcc CUDA**_N-Queen.cu && ./a.out (-c|-r|-g)
                    -c:cpu 
                    -r cpu再帰 
                    -g GPU 

１２．対称解除法の最適化

bash-3.2$ nvcc CUDA12_N-Queen.cu && ./a.out -g
１２．GPU 非再帰 枝刈り
 N:        Total      Unique      dd:hh:mm:ss.ms
 4:            2               1  00:00:00:00.04
 5:           10               2  00:00:00:00.00
 6:            4               1  00:00:00:00.00
 7:           40               6  00:00:00:00.01
 8:           92              12  00:00:00:00.02
 9:          352              46  00:00:00:00.02
10:          724              92  00:00:00:00.02
11:         2680             341  00:00:00:00.03
12:        14200            1787  00:00:00:00.07
13:        73712            9233  00:00:00:00.16
14:       365596           45752  00:00:00:00.13
15:      2279184          285053  00:00:00:00.34
16:     14772512         1846955  00:00:00:01.62
17:     95815104        11977939  00:00:00:10.87
*/

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define THREAD_NUM		96
#define MAX 27
//変数宣言
long TOTAL=0; //GPU,CPUで使用
long UNIQUE=0;//GPU,CPUで使用
//
__device__  __host__
int symmetryOps(int si,unsigned int *d_aBoard,int BOUND1,int BOUND2,int TOPBIT,int ENDBIT){
  int own,ptn,you,bit;
  //90度回転
  if(d_aBoard[BOUND2]==1){ own=1; ptn=2;
    while(own<=si-1){ bit=1; you=si-1;
      while((d_aBoard[you]!=ptn)&&(d_aBoard[own]>=bit)){ bit<<=1; you--; }
      if(d_aBoard[own]>bit){ return 0; } else if(d_aBoard[own]<bit){ break; }
      own++; ptn<<=1;
    }
    /** 90度回転して同型なら180度/270度回転も同型である */
    if(own>si-1){ return 2; }
  }
  //180度回転
  if(d_aBoard[si-1]==ENDBIT){ own=1; you=si-1-1;
    while(own<=si-1){ bit=1; ptn=TOPBIT;
      while((d_aBoard[you]!=ptn)&&(d_aBoard[own]>=bit)){ bit<<=1; ptn>>=1; }
      if(d_aBoard[own]>bit){ return 0; } else if(d_aBoard[own]<bit){ break; }
      own++; you--;
    }
    /** 90度回転が同型でなくても180度回転が同型である事もある */
    if(own>si-1){ return 4; }
  }
  //270度回転
  if(d_aBoard[BOUND1]==TOPBIT){ own=1; ptn=TOPBIT>>1;
    while(own<=si-1){ bit=1; you=0;
      while((d_aBoard[you]!=ptn)&&(d_aBoard[own]>=bit)){ bit<<=1; you++; }
      if(d_aBoard[own]>bit){ return 0; } else if(d_aBoard[own]<bit){ break; }
      own++; ptn>>=1;
    }
  }
  return 8; 
}
//
__global__
void cuda_kernel_b1(
    register int size,
    register int mark,
    unsigned int* totalDown,
    unsigned int* totalLeft,
    unsigned int* totalRight,
    unsigned int* d_results,
    unsigned int* d_uniq,
    register long totalCond,
    /**11 backTrack1ではaBoard不要のためコメント*********************/
    //unsigned int* t_aBoard,
    register int h_row,
    /**11 BOUND1追加*********************/
    int B1
    )
{
  register const unsigned int mask=(1<<size)-1;
  register unsigned long total=0;
  register unsigned int unique=0;
  register int row=0;
  register unsigned int bit;
  //
  //スレッド
  //
  //ブロック内のスレッドID
  register unsigned const int tid=threadIdx.x;
  //グリッド内のブロックID
  register unsigned const int bid=blockIdx.x;
  //全体通してのID
  register unsigned const int idx=bid*blockDim.x+tid;
  //
  //シェアードメモリ
  //
  //sharedメモリを使う ブロック内スレッドで共有
  //10固定なのは現在のmask設定で
  //GPUで実行するのは最大10だから
  //THREAD_NUMはブロックあたりのスレッド数
  __shared__ unsigned int down[THREAD_NUM][10];
  down[tid][row]=totalDown[idx];
  __shared__ unsigned int left[THREAD_NUM][10];
  left[tid][row]=totalLeft[idx];
  __shared__ unsigned int right[THREAD_NUM][10];
  right[tid][row]=totalRight[idx];
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  //down,left,rightからbitmapを出す
  bitmap[tid][row]
    =mask&~(
         down[tid][row]
        |left[tid][row]
        |right[tid][row]);
  __shared__ unsigned int sum[THREAD_NUM];
  /***11 backTrack1ではaBoard不要 *********************/
  //unsigned int c_aBoard[MAX];
  __shared__ unsigned int usum[THREAD_NUM];
  //余分なスレッドは動かさない 
  //GPUはsteps数起動するがtotalCond以上は空回しする
  if(idx<totalCond){
    //totalDown,totalLeft,totalRightの情報を
    //down,left,rightに詰め直す 
    //CPU で詰め込んだ t_はsteps個あるが
    //ブロック内ではブロックあたりのスレッド数に限定
    //されるので idxでよい
    //
    /***11 backTrack1ではaBoard不要*********************/
    //for(int i=0;i<h_row;i++){
    //  c_aBoard[i]=t_aBoard[idx*h_row+i]; //２次元配列だが1次元的に利用  
    //}
    register unsigned int bitmap_tid_row;
    register unsigned int down_tid_row;
    register unsigned int left_tid_row;
    register unsigned int right_tid_row;
    while(row>=0){
      bitmap_tid_row=bitmap[tid][row];
      down_tid_row=down[tid][row];
      left_tid_row=left[tid][row];
      right_tid_row=right[tid][row];
      if(bitmap_tid_row==0){
        row--;
      }else{
        /**11 枝刈り**********/
        if(row+h_row<B1) {
          bitmap_tid_row=bitmap[tid][row]&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
        }  
        //クイーンを置く
        //置く場所があるかどうか
        bitmap[tid][row]
          /***11 backTrack1ではaBoard不要のためコメント*********************/
          //^=c_aBoard[row+h_row]
          //=bit
          ^=bit
          =(-bitmap_tid_row&bitmap_tid_row);       
        if((bit&mask)!=0){
          //最終行?最終行から１個前の行まで
          //無事到達したら 加算する
          if(row+1==mark){
           /**11 backTradk1ではsymmetryOps不要のためコメント*********************/
           //int s=symmetryOps(size,c_aBoard); 
           //if(s!=0){
           //print(size); //print()でTOTALを++しない
           //ホストに戻す配列にTOTALを入れる
           //スレッドが１つの場合は配列は１個
            unique++; 
            total+=8;   //対称解除で得られた解数を加算
           //}
            row--;
          }else{
            int rowP=row+1;
            down[tid][rowP]=down_tid_row|bit;
            left[tid][rowP]=(left_tid_row|bit)<<1;
            right[tid][rowP]=(right_tid_row|bit)>>1;
            bitmap[tid][rowP]
              =mask&~(
                  down[tid][rowP]
                  |left[tid][rowP]
                  |right[tid][rowP]);
            row++;
          }
        }else{
          //置く場所がなければ１個上に
          row--;
        }
      }
    }
    //最後sum[tid]に加算する
    sum[tid]=total;
    usum[tid]=unique;
  }else{
    //totalCond未満は空回しするのでtotalは加算しない
    sum[tid]=0;
    usum[tid]=0;
  } 
  //__syncthreads()でブロック内のスレッド間の同期
  //全てのスレッドが__syncthreads()に辿り着くのを待つ
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){
    sum[tid]+=sum[tid+64];
    usum[tid]+=usum[tid+64];
  }
  __syncwarp();if(tid<32){
    sum[tid]+=sum[tid+32];
    usum[tid]+=usum[tid+32];
  } 
  __syncwarp();if(tid<16){
    sum[tid]+=sum[tid+16];
    usum[tid]+=usum[tid+16];
  } 
  __syncwarp();if(tid<8){
    sum[tid]+=sum[tid+8];
    usum[tid]+=usum[tid+8];
  } 
  __syncwarp();if(tid<4){
    sum[tid]+=sum[tid+4];
    usum[tid]+=usum[tid+4];
  } 
  __syncwarp();if(tid<2){
    sum[tid]+=sum[tid+2];
    usum[tid]+=usum[tid+2];
  } 
  __syncwarp();if(tid<1){
    sum[tid]+=sum[tid+1];
    usum[tid]+=usum[tid+1];
  } 
  __syncwarp();if(tid==0){
    d_results[bid]=sum[0];
    d_uniq[bid]=usum[0];
  }
}
//
//
/***11 cuda_kernel_b2新設*********************/
__global__
void cuda_kernel_b2(
    register int size,
    register int mark,
    unsigned int* totalDown,
    unsigned int* totalLeft,
    unsigned int* totalRight,
    unsigned int* d_results,
    unsigned int* d_uniq,
    register long totalCond,
    unsigned int* t_aBoard,
    register int h_row,
    register int B1,
    register int B2,
    register int SM,
    register int LM,
    /***12 symmetryOps 省力化のためTOPBIT,ENDBITを渡す*****/ 
    register int TB,
    register int EB
    )
{
  register const unsigned int mask=(1<<size)-1;
  register unsigned long total=0;
  register unsigned int unique=0;
  register int row=0;
  register unsigned int bit;
  //
  //スレッド
  //
  //ブロック内のスレッドID
  register unsigned const int tid=threadIdx.x;
  //グリッド内のブロックID
  register unsigned const int bid=blockIdx.x;
  //全体通してのID
  register unsigned const int idx=bid*blockDim.x+tid;
  //
  //シェアードメモリ
  //
  //sharedメモリを使う ブロック内スレッドで共有
  //10固定なのは現在のmask設定で
  //GPUで実行するのは最大10だから
  //THREAD_NUMはブロックあたりのスレッド数
  __shared__ unsigned int down[THREAD_NUM][10];
  down[tid][row]=totalDown[idx];
  __shared__ unsigned int left[THREAD_NUM][10];
  left[tid][row]=totalLeft[idx];
  __shared__ unsigned int right[THREAD_NUM][10];
  right[tid][row]=totalRight[idx];
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  //down,left,rightからbitmapを出す
  bitmap[tid][row]
    =mask&~(
         down[tid][row]
        |left[tid][row]
        |right[tid][row]);
  __shared__ unsigned int sum[THREAD_NUM];
  unsigned int c_aBoard[MAX];
  __shared__ unsigned int usum[THREAD_NUM];
  //余分なスレッドは動かさない 
  //GPUはsteps数起動するがtotalCond以上は空回しする
  if(idx<totalCond){
    //totalDown,totalLeft,totalRightの情報を
    //down,left,rightに詰め直す 
    //CPU で詰め込んだ t_はsteps個あるが
    //ブロック内ではブロックあたりのスレッド数に限定
    //されるので idxでよい
    //
    for(int i=0;i<h_row;i++){
      c_aBoard[i]=t_aBoard[idx*h_row+i]; //２次元配列だが1次元的に利用  
    }
    register unsigned int bitmap_tid_row;
    register unsigned int down_tid_row;
    register unsigned int left_tid_row;
    register unsigned int right_tid_row;
    while(row>=0){
      bitmap_tid_row=bitmap[tid][row];
      down_tid_row=down[tid][row];
      left_tid_row=left[tid][row];
      right_tid_row=right[tid][row];
      //
      //bitmap[tid][row]=00000000 クイーンを
      //どこにも置けないので1行上に戻る
      if(bitmap_tid_row==0){
        row--;
      }else{
        /**11 枝刈り追加**********/
        //【枝刈り】上部サイド枝刈り
	      if(row+h_row<B1){             	
        //printf("BOUND1_row:%d:h_row:%d:row+hrow:%d:bit:%d\n",row,h_row,row+h_row,bitmap[tid][row]);
          bitmap_tid_row=bitmap[tid][row]&=~SM;
        //【枝刈り】下部サイド枝刈り
        }else if(row+h_row==B2) {     	
        //printf("BOUND2_row:%d:h_row:%d:row+hrow:%d:bit:%d\n",row,h_row,row+h_row,bitmap[tid][row]);
          if((down_tid_row&SM)==0){ 
            row--; 
            continue;
            //printf("BOUND2_row\n");
          }
          if((down_tid_row&SM)!=SM){ 
             bitmap_tid_row=bitmap[tid][row]&=SM; 
            //printf("BOUND2_SIDEMASK\n");            
          }
        }
        int save_bitmap=bitmap[tid][row];
        //クイーンを置く
        //置く場所があるかどうか
        bitmap[tid][row]
          ^=c_aBoard[row+h_row]
          =bit
          =(-bitmap_tid_row&bitmap_tid_row);       
        if((bit&mask)!=0){
          //最終行?最終行から１個前の行まで
          //無事到達したら 加算する
          if(row+1==mark){
            /***11 LASTMASK枝刈り*********************/ 
            if((save_bitmap&LM)==0){ 
              /***12 symmetryOps 省力化のためBOUND1,BOUND2,TOPBIT,ENDBITを渡す*****/ 
              int s=symmetryOps(size,c_aBoard,B1,B2,TB,EB); 
              if(s!=0){
              //print(size); //print()でTOTALを++しない
              //ホストに戻す配列にTOTALを入れる
              //スレッドが１つの場合は配列は１個
                unique++; 
                total+=s;   //対称解除で得られた解数を加算
              }
              row--;
            }
          }else{
            int rowP=row+1;
            down[tid][rowP]=down_tid_row|bit;
            left[tid][rowP]=(left_tid_row|bit)<<1;
            right[tid][rowP]=(right_tid_row|bit)>>1;
            bitmap[tid][rowP]
              =mask&~(
                  down[tid][rowP]
                  |left[tid][rowP]
                  |right[tid][rowP]);
            row++;
          }
        }else{
          //置く場所がなければ１個上に
          row--;
        }
      }
    }
    //最後sum[tid]に加算する
    sum[tid]=total;
    usum[tid]=unique;
  }else{
    //totalCond未満は空回しするのでtotalは加算しない
    sum[tid]=0;
    usum[tid]=0;
  } 
  //__syncthreads()でブロック内のスレッド間の同期
  //全てのスレッドが__syncthreads()に辿り着くのを待つ
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){
    sum[tid]+=sum[tid+64];
    usum[tid]+=usum[tid+64];
  }
  __syncwarp();if(tid<32){
    sum[tid]+=sum[tid+32];
    usum[tid]+=usum[tid+32];
  } 
  __syncwarp();if(tid<16){
    sum[tid]+=sum[tid+16];
    usum[tid]+=usum[tid+16];
  } 
  __syncwarp();if(tid<8){
    sum[tid]+=sum[tid+8];
    usum[tid]+=usum[tid+8];
  } 
  __syncwarp();if(tid<4){
    sum[tid]+=sum[tid+4];
    usum[tid]+=usum[tid+4];
  } 
  __syncwarp();if(tid<2){
    sum[tid]+=sum[tid+2];
    usum[tid]+=usum[tid+2];
  } 
  __syncwarp();if(tid<1){
    sum[tid]+=sum[tid+1];
    usum[tid]+=usum[tid+1];
  } 
  __syncwarp();if(tid==0){
    d_results[bid]=sum[0];
    d_uniq[bid]=usum[0];
  }
}
//
/***12 symmetryOps 省力化のためTOPBIT,ENDBITを渡す*****/ 
//long backTrack2G(int size,int mask,int row,int n_left,int n_down,int n_right,int steps,int BOUND1,int BOUND2,int SIDEMASK,int LASTMASK,unsigned int* aBoard)
long backTrack2G(int size,int mask,int row,int n_left,int n_down,int n_right,int steps,int BOUND1,int BOUND2,int SIDEMASK,int LASTMASK,int TOPBIT,int ENDBIT,unsigned int* aBoard)
{
  //何行目からGPUで行くか。ここの設定は変更可能、設定値を多くするほどGPUで並行して動く
  /***11 size<8の時はmarkが2*********************/
  unsigned int mark=size>12?size-10:3;
  //unsigned int mark=size>11?size-9:3;
  if(size<8){ mark=2; }
  const unsigned int h_mark=row;
  long total=0;
  long totalCond=0;
  bool matched=false;
  //host
  unsigned int down[32];  down[row]=n_down;
  unsigned int right[32]; right[row]=n_right;
  unsigned int left[32];  left[row]=n_left;
  //bitmapを配列で持つことにより
  //stackを使わないで1行前に戻れる
  unsigned int bitmap[32];
  bitmap[row]=mask&~(left[row]|down[row]|right[row]);
  unsigned int bit;
  unsigned int* totalDown;
  hipHostMalloc((void**) &totalDown,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* totalLeft;
  hipHostMalloc((void**) &totalLeft,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* totalRight;
  hipHostMalloc((void**) &totalRight,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* h_results;
  hipHostMalloc((void**) &h_results,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* h_uniq;
  hipHostMalloc((void**) &h_uniq,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* t_aBoard;
  hipHostMalloc((void**) &t_aBoard,sizeof(int)*steps*mark, hipHostMallocDefault);
  //device
  unsigned int* downCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  unsigned int* leftCuda;
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  unsigned int* rightCuda;
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  unsigned int* resultsCuda;
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);
  unsigned int* d_uniq;
  hipMalloc((void**) &d_uniq,sizeof(int)*steps/THREAD_NUM);
  unsigned int* d_aBoard;
  hipMalloc((void**) &d_aBoard,sizeof(int)*steps*mark);
  //12行目までは3行目までCPU->row==mark以下で 3行目までの
  //down,left,right情報を totalDown,totalLeft,totalRight
  //に格納
  //する->3行目以降をGPUマルチスレッドで実行し結果を取得
  //13行目以降はCPUで実行する行数が１個ずつ増えて行く
  //例えばn15だとrow=5までCPUで実行し、
  //それ以降はGPU(現在の設定だとGPUでは最大10行実行する
  //ようになっている)
  register int rowP=0;
  while(row>=h_mark) {
    //bitmap[row]=00000000 クイーンを
    //どこにも置けないので1行上に戻る
    //06GPU こっちのほうが優秀
    if(bitmap[row]==0){ row--; }
    else{//おける場所があれば進む
      /***11 枝刈り追加*********************/
      //【枝刈り】上部サイド枝刈り
	    if(row<BOUND1){             	
	      bitmap[row]&=~SIDEMASK;
      //【枝刈り】下部サイド枝刈り
      }else if(row==BOUND2) {     	
        if((down[row]&SIDEMASK)==0){ row--; }
        if((down[row]&SIDEMASK)!=SIDEMASK){ bitmap[row]&=SIDEMASK; }
      }
      //06SGPU
      bitmap[row]^=aBoard[row]=bit=(-bitmap[row]&bitmap[row]);
      if((bit&mask)!=0){//置く場所があれば先に進む
        rowP=row+1;
        down[rowP]=down[row]|bit;
        left[rowP]=(left[row]|bit)<<1;
        right[rowP]=(right[row]|bit)>>1;
        bitmap[rowP]=mask&~(down[rowP]|left[rowP]|right[rowP]);
        row++;
        if(row==mark){
          //3行目(mark)にクイーンを１個ずつ置いていって、
          //down,left,right情報を格納、
          //その次の行へは進まない。その行で可能な場所にクイー
          //ン置き終わったらGPU並列実行
          //totalCond がthreadIdになる 各スレッドに down,left,right情報を渡す
          //row=2(13行目以降は増えていく。例えばn15だとrow=5)の情報を
          //totalDown,totalLeft,totalRightに格納する
          totalDown[totalCond]=down[row];
          totalLeft[totalCond]=left[row];
          totalRight[totalCond]=right[row];
          for(int i=0;i<mark;i++){
            t_aBoard[totalCond*mark+i]=aBoard[i];
          }
          //スレッド数をインクリメントする
          totalCond++;
          //最大GPU数に達してしまったら一旦ここでGPUを実行する。stepsはGPUの同
          //時並行稼働数を制御
          //nの数が少ないうちはtotalCondがstepsを超えることはないがnの数が増え
          //て行くと超えるようになる。
          //ここではtotalCond==stepsの場合だけこの中へ         
          if(totalCond==steps){
            //matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUか
            //ら出たらmatched=trueになってる
            if(matched){
              hipMemcpy(h_results,resultsCuda,
                  sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
              hipMemcpy(h_uniq,d_uniq,
                  sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
              for(int col=0;col<steps/THREAD_NUM;col++){
                total+=h_results[col];
                UNIQUE+=h_uniq[col];
              }
              matched=false;
            }
            hipMemcpy(downCuda,totalDown,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(leftCuda,totalLeft,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(rightCuda,totalRight,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(d_aBoard,t_aBoard,
                sizeof(int)*totalCond*mark,hipMemcpyHostToDevice);
            /***12 TOPBIT,ENDBIT追加*********************/
            //cuda_kernel_b2<<<steps/THREAD_NUM,THREAD_NUM
            //  >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,d_aBoard,row,BOUND1,BOUND2,SIDEMASK,LASTMASK);
            cuda_kernel_b2<<<steps/THREAD_NUM,THREAD_NUM
              >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,d_aBoard,row,BOUND1,BOUND2,SIDEMASK,LASTMASK,TOPBIT,ENDBIT);
            //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われ
            //るのはtotalCondの数だけでそれ以外は空回しになる
            //GPU内でカウントしているので、GPUから出たらmatched=trueになってる
            matched=true;
            //totalCond==stepsルートでGPUを実行したらスレッドをまた0から開始す
            //る(これによりなんどもsteps数分だけGPUを起動できる)
            totalCond=0;           
          }
          //totalDown,totalLeft,totalRightに情報を格納したら1行上に上がる
          //これを繰り返すことにより row=2で可能な場所全てにクイーンを置いて
          //totalDown,totalLeft,totalRightに情報を格納する
          row--;
        }
      }else{
        //置く場所がなければ上に上がる。row==mark行に達するまではCPU側で普通に
        //nqueenをやる
        row--;
      }
    }
  }
  //matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUから出たら
  //matched=trueになってる
  if(matched){
    hipMemcpy(h_results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    hipMemcpy(h_uniq,d_uniq,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){
      total+=h_results[col];
      UNIQUE+=h_uniq[col];
    }
    matched=false;
  }
  hipMemcpy(downCuda,totalDown,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(d_aBoard,t_aBoard,
      sizeof(int)*totalCond*mark,hipMemcpyHostToDevice);
  //size-mark は何行GPUを実行するか totalCondはスレッド数
  //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われるのは
  //totalCondの数だけでそれ以外は空回しになる
  /***12 TOPBIT,ENDBIT追加*********************/
  //cuda_kernel_b2<<<steps/THREAD_NUM,THREAD_NUM
  //  >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,d_aBoard,mark,BOUND1,BOUND2,SIDEMASK,LASTMASK);
  cuda_kernel_b2<<<steps/THREAD_NUM,THREAD_NUM
    >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,d_aBoard,mark,BOUND1,BOUND2,SIDEMASK,LASTMASK,TOPBIT,ENDBIT);
  hipMemcpy(h_results,resultsCuda,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  hipMemcpy(h_uniq,d_uniq,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  for(int col=0;col<steps/THREAD_NUM;col++){
    total+=h_results[col];
    UNIQUE+=h_uniq[col];
  }
  //
  hipFree(downCuda);
  hipFree(leftCuda);
  hipFree(rightCuda);
  hipFree(resultsCuda);
  hipFree(d_uniq);
  hipFree(d_aBoard);
  hipHostFree(totalDown);
  hipHostFree(totalLeft);
  hipHostFree(totalRight);
  hipHostFree(h_results);
  hipHostFree(h_uniq);
  hipHostFree(t_aBoard);
  return total;
}
//
/***11 枝刈りをするので引数を追加,aBoardは不要*********************/
//long backTrack1G(int size,int mask,int row,int n_left,int n_down,int n_right,int steps,unsigned int* aBoard)
long backTrack1G(int size,int mask,int row,int n_left,int n_down,int n_right,int steps,int BOUND1)
{
  //何行目からGPUで行くか。ここの設定は変更可能、設定値を多くするほどGPUで並行して動く
  /***08 クイーンを２行目まで固定で置くためmarkが3以上必要*********************/
  const unsigned int mark=size>12?size-10:3;
  const unsigned int h_mark=row;
  long total=0;
  long totalCond=0;
  bool matched=false;
  //host
  unsigned int down[32];  down[row]=n_down;
  unsigned int right[32]; right[row]=n_right;
  unsigned int left[32];  left[row]=n_left;
  //bitmapを配列で持つことにより
  //stackを使わないで1行前に戻れる
  unsigned int bitmap[32];
  bitmap[row]=mask&~(left[row]|down[row]|right[row]);
  unsigned int bit;
  unsigned int* totalDown;
  hipHostMalloc((void**) &totalDown,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* totalLeft;
  hipHostMalloc((void**) &totalLeft,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* totalRight;
  hipHostMalloc((void**) &totalRight,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* h_results;
  hipHostMalloc((void**) &h_results,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* h_uniq;
  hipHostMalloc((void**) &h_uniq,sizeof(int)*steps, hipHostMallocDefault);
  /***11 backTrack1ではaBoard不要のためコメント*********************/
  //unsigned int* t_aBoard;
  //cudaMallocHost((void**) &t_aBoard,sizeof(int)*steps*mark);
  //device
  unsigned int* downCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  unsigned int* leftCuda;
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  unsigned int* rightCuda;
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  unsigned int* resultsCuda;
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);
  unsigned int* d_uniq;
  hipMalloc((void**) &d_uniq,sizeof(int)*steps/THREAD_NUM);
  /***11 backTrack1ではaBoard不要のためコメント*********************/
  //unsigned int* d_aBoard;
  //cudaMalloc((void**) &d_aBoard,sizeof(int)*steps*mark);
  //12行目までは3行目までCPU->row==mark以下で 3行目までの
  //down,left,right情報を totalDown,totalLeft,totalRight
  //に格納
  //する->3行目以降をGPUマルチスレッドで実行し結果を取得
  //13行目以降はCPUで実行する行数が１個ずつ増えて行く
  //例えばn15だとrow=5までCPUで実行し、
  //それ以降はGPU(現在の設定だとGPUでは最大10行実行する
  //ようになっている)
  //while(row>=0) {
  register int rowP=0;
  while(row>=h_mark) {
    //bitmap[row]=00000000 クイーンを
    //どこにも置けないので1行上に戻る
    //06GPU こっちのほうが優秀
    if(bitmap[row]==0){ row--; }
    else{//おける場所があれば進む
      /***11 枝刈り*********************/
      if(row<BOUND1) {
          bitmap[row]&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
      }
      //06SGPU
      /***11 aBoard不要*********************/
      //bitmap[row]^=aBoard[row]=bit=(-bitmap[row]&bitmap[row]);
      bitmap[row]^=bit=(-bitmap[row]&bitmap[row]);
      if((bit&mask)!=0){//置く場所があれば先に進む
        rowP=row+1;
        down[rowP]=down[row]|bit;
        left[rowP]=(left[row]|bit)<<1;
        right[rowP]=(right[row]|bit)>>1;
        bitmap[rowP]=mask&~(down[rowP]|left[rowP]|right[rowP]);
        row++;
        if(row==mark){
          //3行目(mark)にクイーンを１個ずつ置いていって、
          //down,left,right情報を格納、
          //その次の行へは進まない。その行で可能な場所にクイー
          //ン置き終わったらGPU並列実行
          //totalCond がthreadIdになる 各スレッドに down,left,right情報を渡す
          //row=2(13行目以降は増えていく。例えばn15だとrow=5)の情報を
          //totalDown,totalLeft,totalRightに格納する         
          totalDown[totalCond]=down[row];
          totalLeft[totalCond]=left[row];
          totalRight[totalCond]=right[row];
          /***11 aBoardコメント*********************/
          //for(int i=0;i<mark;i++){
          //  t_aBoard[totalCond*mark+i]=aBoard[i];
          //}
          //スレッド数をインクリメントする
          totalCond++;
          //最大GPU数に達してしまったら一旦ここでGPUを実行する。stepsはGPUの同
          //時並行稼働数を制御
          //nの数が少ないうちはtotalCondがstepsを超えることはないがnの数が増え
          //て行くと超えるようになる。
          //ここではtotalCond==stepsの場合だけこの中へ         
          if(totalCond==steps){
            //matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUか
            //ら出たらmatched=trueになってる
            if(matched){
              hipMemcpy(h_results,resultsCuda,
                  sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
              hipMemcpy(h_uniq,d_uniq,
                  sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
              for(int col=0;col<steps/THREAD_NUM;col++){
                total+=h_results[col];
                UNIQUE+=h_uniq[col];
              }
              matched=false;
            }
            hipMemcpy(downCuda,totalDown,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(leftCuda,totalLeft,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(rightCuda,totalRight,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            /***11 aBoard不要のためコメント*********************/
            //cudaMemcpy(d_aBoard,t_aBoard,
            //    sizeof(int)*totalCond*mark,cudaMemcpyHostToDevice);
            /***11 BOUND1追加*********************/
            //cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
            //  >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,d_aBoard,row);
            cuda_kernel_b1<<<steps/THREAD_NUM,THREAD_NUM
              >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,row,BOUND1);

            //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われ
            //るのはtotalCondの数だけでそれ以外は空回しになる
            //GPU内でカウントしているので、GPUから出たらmatched=trueになってる
            matched=true;
            //totalCond==stepsルートでGPUを実行したらスレッドをまた0から開始す
            //る(これによりなんどもsteps数分だけGPUを起動できる)
            totalCond=0;           
          }
          //totalDown,totalLeft,totalRightに情報を格納したら1行上に上がる
          //これを繰り返すことにより row=2で可能な場所全てにクイーンを置いて
          //totalDown,totalLeft,totalRightに情報を格納する
          row--;
        }
      }else{
        //置く場所がなければ上に上がる。row==mark行に達するまではCPU側で普通に
        //nqueenをやる
        row--;
      }
    }
  }
  //matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUから出たら
  //matched=trueになってる
  if(matched){
    hipMemcpy(h_results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    hipMemcpy(h_uniq,d_uniq,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){
      total+=h_results[col];
      UNIQUE+=h_uniq[col];
    }
    matched=false;
  }
  hipMemcpy(downCuda,totalDown,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  /***11 aBoard不要のためコメント*********************/
  //cudaMemcpy(d_aBoard,t_aBoard,
  //    sizeof(int)*totalCond*mark,cudaMemcpyHostToDevice);
  //size-mark は何行GPUを実行するか totalCondはスレッド数
  //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われるのは
  //totalCondの数だけでそれ以外は空回しになる
   /***11 BOUND1追加*********************/
  //cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
  //  >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,d_aBoard,mark);
  cuda_kernel_b1<<<steps/THREAD_NUM,THREAD_NUM
    >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,mark,BOUND1);
  hipMemcpy(h_results,resultsCuda,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  hipMemcpy(h_uniq,d_uniq,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  for(int col=0;col<steps/THREAD_NUM;col++){
    total+=h_results[col];
    UNIQUE+=h_uniq[col];
  }
  //
  hipFree(downCuda);
  hipFree(leftCuda);
  hipFree(rightCuda);
  hipFree(resultsCuda);
  hipFree(d_uniq);
  /***11 aBoardコメント**/
  //cudaFree(d_aBoard);
  hipHostFree(totalDown);
  hipHostFree(totalLeft);
  hipHostFree(totalRight);
  hipHostFree(h_results);
  hipHostFree(h_uniq);
  /***11 aBoardコメント**/
  //cudaFreeHost(t_aBoard);
  return total;
}
//
//GPU
void NQueenG(int size,int steps)
{
  int TOPBIT;
  int ENDBIT;
  int LASTMASK;
  int SIDEMASK;
  int BOUND1;
  int BOUND2;
  unsigned int aBoard[MAX];
  register int bit=0;
  register int mask=((1<<size)-1);
  TOPBIT=1<<(size-1);
  if(size<=0||size>32){return;}
  /***09 backtrack1*********************/
  //1行め右端 0
  int col=0;
  aBoard[0]=bit=(1<<col);
  int left=bit<<1;
  int down=bit;
  int right=bit>>1;
  //2行目は右から3列目から左端から2列目まで
  for(int col_j=2;col_j<size-1;col_j++){
      aBoard[1]=bit=(1<<col_j);
      /***11 BOUND1*********************/
      BOUND1=col_j;
      /***11 枝刈りするので引数を渡す*********************/
      //TOTAL+=backTrack1G(size,mask,2,(left|bit)<<1,(down|bit),(right|bit)>>1,steps,aBoard);
      TOTAL+=backTrack1G(size,mask,2,(left|bit)<<1,(down|bit),(right|bit)>>1,steps,BOUND1);
  }
  SIDEMASK=LASTMASK=(TOPBIT|1);
  ENDBIT=(TOPBIT>>1);
  /***09 backtrack2*********************/
  //1行目右から2列目から
  //偶数個は1/2 n=8 なら 1,2,3 奇数個は1/2+1 n=9 なら 1,2,3,4
  for(int col=1,col2=size-2;col<col2;col++,col2--){
      aBoard[0]=bit=(1<<col);
      BOUND1=col;
      BOUND2=col2;
      /***12 symmetryOps 省力化のためTOPBIT,ENDBITを渡す*****/ 
      //TOTAL+=backTrack2G(size,mask,1,bit<<1,bit,bit>>1,steps,BOUND1,BOUND2,SIDEMASK,LASTMASK,aBoard);
      TOTAL+=backTrack2G(size,mask,1,bit<<1,bit,bit>>1,steps,BOUND1,BOUND2,SIDEMASK,LASTMASK,TOPBIT,ENDBIT,aBoard);
      LASTMASK|=LASTMASK>>1|LASTMASK<<1;
      ENDBIT>>=1;
  }
}
/** GPU/SGPU CUDA 初期化 **/
bool InitCUDA()
{
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
//hh:mm:ss.ms形式に処理時間を出力
void TimeFormat(clock_t utime,char *form)
{
  int dd,hh,mm;
  float ftime,ss;
  ftime=(float)utime/CLOCKS_PER_SEC;
  mm=(int)ftime/60;
  ss=ftime-(int)(mm*60);
  dd=mm/(24*60);
  mm=mm%(24*60);
  hh=mm/60;
  mm=mm%60;
  if(dd)
    sprintf(form,"%4d %02d:%02d:%05.2f",dd,hh,mm,ss);
  else if(hh)
    sprintf(form,"     %2d:%02d:%05.2f",hh,mm,ss);
  else if(mm)
    sprintf(form,"        %2d:%05.2f",mm,ss);
  else
    sprintf(form,"           %5.2f",ss);
}
//

/***12 symmetryOps 省力化のためTOPBIT,ENDBITを渡す*****/ 
//void backTrack2(int size,int mask, int row,int h_left,int h_down,int h_right,int BOUND1,int BOUND2,int SIDEMASK,int LASTMASK,unsigned int* aBoard)
void backTrack2(int size,int mask, int row,int h_left,int h_down,int h_right,int BOUND1,int BOUND2,int SIDEMASK,int LASTMASK,int TOPBIT,int ENDBIT,unsigned int* aBoard)
{
    unsigned int left[size];
    unsigned int down[size];
	  unsigned int right[size];
    unsigned int bitmap[size];
	  left[row]=h_left;
	  down[row]=h_down;
	  right[row]=h_right;
	  bitmap[row]=mask&~(left[row]|down[row]|right[row]);
    unsigned int bit;
    unsigned int sizeE=size-1;
    int mark=row;
    //固定していれた行より上はいかない
    while(row>=mark){//row=1 row>=1, row=2 row>=2
      if(bitmap[row]==0){
        --row;
      }else{
        /***11 【枝刈り】上部サイド枝刈り*********************/
        if(row<BOUND1){             	
          bitmap[row]&=~SIDEMASK;
        /***11 【枝刈り】下部サイド枝刈り*********************/
        }else if(row==BOUND2) {     	
          if((down[row]&SIDEMASK)==0){ 
              row--; 
          }
          if((down[row]&SIDEMASK)!=SIDEMASK){ 
              bitmap[row]&=SIDEMASK;
              }
        }
        int save_bitmap=bitmap[row];
        bitmap[row]^=aBoard[row]=bit=(-bitmap[row]&bitmap[row]); 
        if((bit&mask)!=0){
          if(row==sizeE){
            /***11 【枝刈り】 最下段枝刈り*********************/
            if((save_bitmap&LASTMASK)==0){ 
              /***12 symmetryOps 省力化のためBOUND1,BOUND2,TOPBIT,ENDBITを渡す*****/ 
              int s=symmetryOps(size,aBoard,BOUND1,BOUND2,TOPBIT,ENDBIT);
              if(s!=0){
                UNIQUE++;
                TOTAL+=s;
              }
              --row;
            }
          }else{
            int n=row++;
            left[row]=(left[n]|bit)<<1;
            down[row]=down[n]|bit;
            right[row]=(right[n]|bit)>>1;
            bitmap[row]=mask&~(left[row]|down[row]|right[row]);
          }
        }else{
           --row;
        }
      }  
    }
}
//
/***11 枝刈りをするので引数を追加*********************/
 //void backTrack1(int size,int mask, int row,int h_left,int h_down,int h_right,unsigned int* aBoard)
void backTrack1(int size,int mask, int row,int h_left,int h_down,int h_right,int BOUND1,unsigned int* aBoard)
{
    unsigned int left[size];
    unsigned int down[size];
	  unsigned int right[size];
    unsigned int bitmap[size];
	  left[row]=h_left;
	  down[row]=h_down;
	  right[row]=h_right;
	  bitmap[row]=mask&~(left[row]|down[row]|right[row]);
    unsigned int bit;
    unsigned int sizeE=size-1;
    int mark=row;
    //固定していれた行より上はいかない
    while(row>=mark){//row=1 row>=1, row=2 row>=2
      if(bitmap[row]==0){
        --row;
      }else{
        /***11　【枝刈り】鏡像についても主対角線鏡像のみを判定すればよい　*****/
        // ２行目、２列目を数値とみなし、２行目＜２列目という条件を課せばよい
        if(row<BOUND1) {
          bitmap[row]&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
        }
        bitmap[row]^=aBoard[row]=bit=(-bitmap[row]&bitmap[row]); 
        if((bit&mask)!=0){
          if(row==sizeE){
            /***11　【枝刈り】１行目角にクイーンがある場合回転対称チェックを省略　*****/   
            //int s=symmetryOps(size,aBoard);
            //if(s!=0){
            UNIQUE++;
            TOTAL+=8;
            //}
            --row;
          }else{
            int n=row++;
            left[row]=(left[n]|bit)<<1;
            down[row]=down[n]|bit;
            right[row]=(right[n]|bit)>>1;
            bitmap[row]=mask&~(left[row]|down[row]|right[row]);
          }
        }else{
           --row;
        }
      }  
    }
}
//
//非再帰版
void NQueen(int size,int mask)
{
  int TOPBIT;
  int ENDBIT;
  int LASTMASK;
  int SIDEMASK;
  int BOUND1;
  int BOUND2;
  register int bit;
  TOPBIT=1<<(size-1);
  unsigned int aBoard[MAX];
  bit=0;
  if(size<=0||size>32){return;}
  /***09 backtrack1***/
  //1行め右端 0
  int col=0;
  aBoard[0]=bit=(1<<col);
  int left=bit<<1;
  int down=bit;
  int right=bit>>1;
  /***09 2行目は右から3列目から左端から2列目まで***/
  for(int col_j=2;col_j<size-1;col_j++){
      aBoard[1]=bit=(1<<col_j);
      BOUND1=col_j;
      /***11 枝刈りするので引数を渡す*********************/
      //backTrack1(size,mask,2,(left|bit)<<1,(down|bit),(right|bit)>>1,aBoard);
      backTrack1(size,mask,2,(left|bit)<<1,(down|bit),(right|bit)>>1,BOUND1,aBoard);
  }
  SIDEMASK=LASTMASK=(TOPBIT|1);
  ENDBIT=(TOPBIT>>1);
  /***09 backtrack2***/
  //1行目右から2列目から
  //偶数個は1/2 n=8 なら 1,2,3 奇数個は1/2+1 n=9 なら 1,2,3,4
  for(int col=1,col2=size-2;col<col2;col++,col2--){
      aBoard[0]=bit=(1<<col);
      BOUND1=col;
      BOUND2=col2;
      /***12 symmetryOps 省力化のためTOPBIT,ENDBITを渡す*****/ 
      //backTrack2(size,mask,1,bit<<1,bit,bit>>1,BOUND1,BOUND2,SIDEMASK,LASTMASK,aBoard);
      backTrack2(size,mask,1,bit<<1,bit,bit>>1,BOUND1,BOUND2,SIDEMASK,LASTMASK,TOPBIT,ENDBIT,aBoard);
      LASTMASK|=LASTMASK>>1|LASTMASK<<1;
      ENDBIT>>=1;
  }
}
//CPUR 再帰版 ロジックメソッド
/***12 symmetryOps 省力化のためTOPBIT,ENDBITを渡す*****/ 
//void backTrackR2(int size,int mask, int row,int left,int down,int right,int BOUND1,int BOUND2,int SIDEMASK,int LASTMASK,unsigned int* aBoard)
void backTrackR2(int size,int mask, int row,int left,int down,int right,int BOUND1,int BOUND2,int SIDEMASK,int LASTMASK,int TOPBIT,int ENDBIT,unsigned int* aBoard)
{
 int bitmap=0;
 int bit=0;
 int sizeE=size-1;
 bitmap=(mask&~(left|down|right));
 if(row==sizeE){
    if(bitmap){
      /***11 【枝刈り】 最下段枝刈り*********************/
      if((bitmap&LASTMASK)==0){ 
        aBoard[row]=(-bitmap&bitmap);
        /***12 symmetryOps 省力化のためBOUND1,BOUND2,TOPBIT,ENDBITを渡す*****/ 
        int s=symmetryOps(size,aBoard,BOUND1,BOUND2,TOPBIT,ENDBIT);
        if(s!=0){
          UNIQUE++;
          TOTAL+=s;
        }
      }
    }
  }else{
    /***11 【枝刈り】上部サイド枝刈*********************/
    if(row<BOUND1){             	
      bitmap&=~SIDEMASK;
    /***11 【枝刈り】下部サイド枝刈り*********************/
    }else if(row==BOUND2) {     	
      if((down&SIDEMASK)==0){ return; }
      if((down&SIDEMASK)!=SIDEMASK){ bitmap&=SIDEMASK; }
    }
    while(bitmap){
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap);
      backTrackR2(size,mask,row+1,(left|bit)<<1, down|bit,(right|bit)>>1,BOUND1,BOUND2,SIDEMASK,LASTMASK,TOPBIT,ENDBIT,aBoard);
    }
  }
}
//
/***11 枝刈りをするので引数を追加する*********************/
//void backTrackR1(int size,int mask, int row,int left,int down,int right,unsigned int* aBoard)
void backTrackR1(int size,int mask, int row,int left,int down,int right,int BOUND1,unsigned int* aBoard)
{
 int bitmap=0;
 int bit=0;
 int sizeE=size-1;
 bitmap=(mask&~(left|down|right));
 if(row==sizeE){
    if(bitmap){
      /***11　【枝刈り】１行目角にクイーンがある場合回転対称チェックを省略　*****/   
      //aBoard[row]=(-bitmap&bitmap);
      //int s=symmetryOps(size,aBoard);
      //if(s!=0){
      UNIQUE++;
      TOTAL+=8;
      //}
    }
  }else{
    /***11　【枝刈り】鏡像についても主対角線鏡像のみを判定すればよい　*****/
    // ２行目、２列目を数値とみなし、２行目＜２列目という条件を課せばよい
    if(row<BOUND1) {
      bitmap&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
    }
    while(bitmap){
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap);
      backTrackR1(size,mask,row+1,(left|bit)<<1, down|bit,(right|bit)>>1,BOUND1,aBoard);
    }
  }
}  
//CPUR 再帰版 ロジックメソッド
void NQueenR(int size,int mask)
{
  int TOPBIT;
  int ENDBIT;
  int LASTMASK;
  int SIDEMASK;
  int BOUND1;
  int BOUND2;
  int bit=0;
  TOPBIT=1<<(size-1);
  unsigned int aBoard[MAX];
  /***09 backtrack1*********************/
  //1行め右端 0
  int col=0;
  aBoard[0]=bit=(1<<col);
  int left=bit<<1;
  int down=bit;
  int right=bit>>1;
  //2行目は右から3列目から左端から2列目まで
  for(int col_j=2;col_j<size-1;col_j++){
    aBoard[1]=bit=(1<<col_j);
    BOUND1=col_j;
    //backTrackR1(size,mask,2,(left|bit)<<1,(down|bit),(right|bit)>>1,aBoard);
    backTrackR1(size,mask,2,(left|bit)<<1,(down|bit),(right|bit)>>1,BOUND1,aBoard);
  }
  SIDEMASK=LASTMASK=(TOPBIT|1);
  ENDBIT=(TOPBIT>>1);
  /***09 backtrack2*********************/
  //1行目右から2列目から
  //偶数個は1/2 n=8 なら 1,2,3 奇数個は1/2+1 n=9 なら 1,2,3,4
  for(int col=1,col2=size-2;col<col2;col++,col2--){
      aBoard[0]=bit=(1<<col);
      BOUND1=col;
      BOUND2=col2;
      /***11 枝刈りするので引数を渡す*********************/
      //backTrackR2(size,mask,1,bit<<1,bit,bit>>1,aBoard);
      /***12 symmetryOps 省力化のためTOPBIT,ENDBITを渡す*****/ 
      backTrackR2(size,mask,1,bit<<1,bit,bit>>1,BOUND1,BOUND2,SIDEMASK,LASTMASK,TOPBIT,ENDBIT,aBoard);
      LASTMASK|=LASTMASK>>1|LASTMASK<<1;
      ENDBIT>>=1;
  }
}
//
//通常版
/***12 symmetryOps 省力化のためTOPBIT,ENDBITを渡す*****/ 
void backTrack2D_NR(int size,int mask,int row,int left,int down,int right,int BOUND1,int BOUND2,int SIDEMASK,int LASTMASK,int TOPBIT,int ENDBIT,unsigned int* aBoard){
  int bitmap,bit;
  int b[100], *p=b;
  int sizeE=size-1;
  int odd=size&1; //奇数:1 偶数:0
  for(int i=0;i<(1+odd);++i){
    bitmap=0;
    if(0==i){
      int half=size>>1; // size/2
      bitmap=(1<<half)-1;
    }else{
      bitmap=1<<(size>>1);
      // down[1]=bitmap;
      // right[1]=(bitmap>>1);
      // left[1]=(bitmap<<1);
      // pnStack=aStack+1;
      // *pnStack++=0;
    }
  b1mais1:bitmap=mask&~(left|down|right);
  if(row==sizeE){
    if(bitmap){
      /***11 【枝刈り】 最下段枝刈り*****/ 
      if((bitmap&LASTMASK)==0){
        aBoard[row]=bitmap;
        /***12 symmetryOps 省力化のためTOPBIT,ENDBITを渡す*****/ 
        int s=symmetryOps(size,aBoard,BOUND1,BOUND2,TOPBIT,ENDBIT);
        if(s!=0){
          UNIQUE++;
          TOTAL+=s;
        }
      }
    }
  }else{
    /***11 【枝刈り】 上部サイド枝刈り*****/ 
    if(row<BOUND1){
      bitmap&=~SIDEMASK;
    /***11 【枝刈り】 下部サイド枝刈り*****/ 
    }else if(row==BOUND2){
      if(!(down&SIDEMASK))
        goto b1volta;
      if((down&SIDEMASK)!=SIDEMASK)
        bitmap&=SIDEMASK;
    }
    if(bitmap){
      b1outro:bitmap^=aBoard[row]=bit=-bitmap&bitmap;
      if(bitmap){
        *p++=left;
        *p++=down;
        *p++=right;
      }
      *p++=bitmap;
      row++;
      left=(left|bit)<<1;
      down=down|bit;
      right=(right|bit)>>1;
      goto b1mais1;
      //Backtrack2(y+1, (left | bit)<<1, down | bit, (right | bit)>>1);
      b1volta:if(p<=b)
        return;
      row--;
      bitmap=*--p;
      if(bitmap){
        right=*--p;
        down=*--p;
        left=*--p;
        goto b1outro;
      }else{
        goto b1volta;
      }
    }
  }
  goto b1volta;
  }
} 
/***11 枝刈りする*****/ 
void backTrack1D_NR(int size,int mask,int row,int left,int down,int right,int BOUND1,unsigned int* aBoard){
  int bitmap,bit;
  int b[100], *p=b;
  int sizeE=size-1;
  int odd=size&1; //奇数:1 偶数:0
  for(int i=0;i<(1+odd);++i){
    bitmap=0;
    if(0==i){
      int half=size>>1; // size/2
      bitmap=(1<<half)-1;
    }else{
      bitmap=1<<(size>>1);
      // down[1]=bitmap;
      // right[1]=(bitmap>>1);
      // left[1]=(bitmap<<1);
      // pnStack=aStack+1;
      // *pnStack++=0;
    }
  b1mais1:bitmap=mask&~(left|down|right);
  if(row==sizeE){
    if(bitmap){
      /***11　【枝刈り】１行目角にクイーンがある場合回転対称チェックを省略　*****/ 
      //aBoard[row]=bitmap;
      //int s=symmetryOps(size,aBoard);
      //if(s!=0){
      UNIQUE++;
      TOTAL+=8;
      //}
    }
  }else{
    /***11　【枝刈り】鏡像についても主対角線鏡像のみを判定すればよい　*****/
    // ２行目、２列目を数値とみなし、２行目＜２列目という条件を課せばよい
    if(row<BOUND1) {
      bitmap&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
    }
    if(bitmap){
      b1outro:bitmap^=aBoard[row]=bit=-bitmap&bitmap;
      if(bitmap){
        *p++=left;
        *p++=down;
        *p++=right;
      }
      *p++=bitmap;
      row++;
      left=(left|bit)<<1;
      down=down|bit;
      right=(right|bit)>>1;
      goto b1mais1;
      //Backtrack1(y+1, (left | bit)<<1, down | bit, (right | bit)>>1);
      b1volta:if(p<=b)
        return;
      row--;
      bitmap=*--p;
      if(bitmap){
        right=*--p;
        down=*--p;
        left=*--p;
        goto b1outro;
      }else{
        goto b1volta;
      }
    }
  }
  goto b1volta;
  }
}
// 
//通常版 CPU 非再帰版 ロジックメソッド
/***09 backTrack登場メソッド名だけ枝刈りはまだしない*****/  
void NQueenD(int size,int mask){
  int TOPBIT;
  int ENDBIT;
  int LASTMASK;
  int SIDEMASK;
  int BOUND1;
  int BOUND2;
  int bit;
  unsigned int aBoard[MAX];
  TOPBIT=1<<(size-1);
  aBoard[0]=1;
  for(BOUND1=2;BOUND1<size-1;BOUND1++){
    aBoard[1]=bit=(1<<BOUND1);
    //backTrack1(size,mask,2,(2|bit)<<1,(1|bit),(bit>>1));
    backTrack1D_NR(size,mask,2,(2|bit)<<1,(1|bit),(bit>>1),BOUND1,aBoard);
  }
  SIDEMASK=LASTMASK=(TOPBIT|1);
  ENDBIT=(TOPBIT>>1);
  for(BOUND1=1,BOUND2=size-2;BOUND1<BOUND2;BOUND1++,BOUND2--){
    aBoard[0]=bit=(1<<BOUND1);
    //backTrack1(size,mask,1,bit<<1,bit,bit>>1);
    /***12 symmetryOps 省力化のためTOPBIT,ENDBITを渡す*****/ 
    backTrack2D_NR(size,mask,1,bit<<1,bit,bit>>1,BOUND1,BOUND2,SIDEMASK,LASTMASK,TOPBIT,ENDBIT,aBoard);
    LASTMASK|=LASTMASK>>1|LASTMASK<<1;
    ENDBIT>>=1;
  }
}
//
// 通常版
/***12 symmetryOps 省力化のためTOPBIT,ENDBITを渡す*****/ 
void backTrack2D(int size,int mask,int row,int left,int down,int right,int BOUND1,int BOUND2,int SIDEMASK,int LASTMASK,int TOPBIT,int ENDBIT,unsigned int* aBoard){
  int bit;
  int bitmap=(mask&~(left|down|right));
  /***11 枝刈り*****/
  //if(row==size){
  if(row==size-1){
    if(bitmap){
    /***11 【枝刈り】 最下段枝刈り*****/
      if((bitmap&LASTMASK)==0){ 	
        aBoard[row]=bitmap; //symmetryOpsの時は代入します。
        /***12 symmetryOps 省力化のためTOPBIT,ENDBITを渡す*****/ 
        int s=symmetryOps(size,aBoard,BOUND1,BOUND2,TOPBIT,ENDBIT);
        if(s!=0){
          UNIQUE++;
          TOTAL+=s;
        }
      }
    }else{
      /***11 【枝刈り】上部サイド枝刈り*****/
      if(row<BOUND1){             	
        bitmap&=~SIDEMASK;
      /***11 【枝刈り】下部サイド枝刈り*****/
      }else if(row==BOUND2) {     	
        if((down&SIDEMASK)==0){ return; }
        if((down&SIDEMASK)!=SIDEMASK){ bitmap&=SIDEMASK; }
      }
      while(bitmap){
        bitmap^=aBoard[row]=bit=(-bitmap&bitmap); //ロジック用
        backTrack2D(size,mask,row+1,(left|bit)<<1,down|bit,(right|bit)>>1,BOUND1,BOUND2,SIDEMASK,LASTMASK,TOPBIT,ENDBIT,aBoard);
      }
    }
  }
}
//
void backTrack1D(int size,int mask,int row,int left,int down,int right,int BOUND1,unsigned int* aBoard){
  int bit;
  int bitmap=(mask&~(left|down|right));
  /***11 【枝刈り】１行目角にクイーンがある場合回転対称チェックを省略*****/
  //if(row==size){
  if(row==size-1){    
    //aBoard[row]=bitmap; //symmetryOpsの時は代入します。
    //int s=symmetryOps(size,aBoard);
    //if(s!=0){
    if(bitmap){
      UNIQUE++;
      TOTAL+=8;
    }
  }else{
    /***11 【枝刈り】鏡像についても主対角線鏡像のみを判定すればよい*****/
    // ２行目、２列目を数値とみなし、２行目＜２列目という条件を課せばよい
    if(row<BOUND1) {
      bitmap&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
    }
    while(bitmap){
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap); //ロジック用
      backTrack1D(size,mask,row+1,(left|bit)<<1,down|bit,(right|bit)>>1,BOUND1,aBoard);
    }
  }
}
// 
//通常版 CPUR 再帰版　ロジックメソッド
void NQueenDR(int size,int mask)
{
  int TOPBIT;
  int ENDBIT;
  int LASTMASK;
  int SIDEMASK;
  int BOUND1;
  int BOUND2;
  int bit;
  unsigned int aBoard[MAX]; 
  TOPBIT=1<<(size-1);
  aBoard[0]=1;
  for(BOUND1=2;BOUND1<size-1;BOUND1++){
    aBoard[1]=bit=(1<<BOUND1);
    backTrack1D(size,mask,2,(2|bit)<<1,(1|bit),(bit>>1),BOUND1,aBoard);
  }
  SIDEMASK=LASTMASK=(TOPBIT|1);
  ENDBIT=(TOPBIT>>1);
  for(BOUND1=1,BOUND2=size-2;BOUND1<BOUND2;BOUND1++,BOUND2--){
    aBoard[0]=bit=(1<<BOUND1);
    /***12 symmetryOps 省力化のためTOPBIT,ENDBITを渡す*****/
    backTrack2D(size,mask,1,bit<<1,bit,bit>>1,BOUND1,BOUND2,SIDEMASK,LASTMASK,TOPBIT,ENDBIT,aBoard);
    LASTMASK|=LASTMASK>>1|LASTMASK<<1;
    ENDBIT>>=1;
  }
}
//メインメソッド
int main(int argc,char** argv)
{
  bool cpu=false,cpur=false,gpu=false,sgpu=false;
  int argstart=1,steps=24576;
  /** パラメータの処理 */
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='r'||argv[1][1]=='R'){cpur=true;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){gpu=true;}
    else if(argv[1][1]=='s'||argv[1][1]=='S'){sgpu=true;}
    else
      cpur=true;
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g|-r|-s]\n",argv[0]);
    printf("  -c: CPU only\n");
    printf("  -r: CPUR only\n");
    printf("  -g: GPU only\n");
    printf("  -s: SGPU only\n");
    printf("Default to 8 queen\n");
  }
  /** 出力と実行 */
  if(cpu){
    printf("\n\n１２．CPU 非再帰 枝刈り\n");
  }else if(cpur){
    printf("\n\n１２．CPUR 再帰 枝刈り\n");
  }else if(gpu){
    printf("\n\n１２．GPU 非再帰 枝刈り\n");
  }else if(sgpu){
    printf("\n\n１２．SGPU 非再帰 バックトラック＋ビットマップ\n");
  }
  if(cpu||cpur){
    printf("%s\n"," N:        Total       Unique        hh:mm:ss.ms");
    clock_t st;           //速度計測用
    char t[20];           //hh:mm:ss.msを格納
    int min=4; int targetN=17;
    int mask;
    for(int i=min;i<=targetN;i++){
      TOTAL=0; UNIQUE=0;
      //COUNT2=COUNT4=COUNT8=0;
      mask=(1<<i)-1;
      st=clock();
      //
      //【通常版】
      //if(cpur){ _NQueenR(i,mask,0,0,0,0); }
      //CPUR
      if(cpur){ 
        NQueenR(i,mask); 
        //printf("通常版\n");
        //NQueenDR(i,mask);//通常版
      }
      //CPU
      if(cpu){ 
        NQueen(i,mask); 
        //printf("通常版\n");
        //NQueenD(i,mask,0); //通常版
      }
      //
      TimeFormat(clock()-st,t); 
      printf("%2d:%13ld%16ld%s\n",i,TOTAL,UNIQUE,t);
    }
  }
  if(gpu||sgpu){
    if(!InitCUDA()){return 0;}
    int min=4;int targetN=17;
    struct timeval t0;struct timeval t1;
    int ss;int ms;int dd;
    printf("%s\n"," N:        Total      Unique      dd:hh:mm:ss.ms");
    for(int i=min;i<=targetN;i++){
      gettimeofday(&t0,NULL);   // 計測開始
      if(gpu){
        TOTAL=0;
        UNIQUE=0;
        NQueenG(i,steps);
      }
      gettimeofday(&t1,NULL);   // 計測終了
      if (t1.tv_usec<t0.tv_usec) {
        dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      } else {
        dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%13ld%16ld%4.2d:%02d:%02d:%02d.%02d\n", i,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    }
  }
  return 0;
}
