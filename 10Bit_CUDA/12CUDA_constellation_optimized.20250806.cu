/**
  10CUDA_constellation_warp.cu複写
  21Py_constellations_optimized_codon.pyを移植

NQueens_suzuki$ nvcc -O3 -arch=sm_61 -m64 -prec-div=false 12CUDA_constellation_optimized.cu && POCL_DEBUG=all && ./a.out -g
ptxas warning : Stack size for entry function '_Z19execSolutionsKernelP13ConstellationPjii' cannot be statically determined
GPU Constellations
 N:        Total      Unique      dd:hh:mm:ss.ms
 4:                0               0     000:00:00:00.40
 5:               18               0     000:00:00:00.00
 6:                4               0     000:00:00:00.00
 7:               40               0     000:00:00:00.00
 8:               92               0     000:00:00:00.00
 9:              352               0     000:00:00:00.00
10:              724               0     000:00:00:00.00
11:             2680               0     000:00:00:00.00
12:            14200               0     000:00:00:00.00
13:            73712               0     000:00:00:00.00
14:           365596               0     000:00:00:00.02
15:          2279184               0     000:00:00:00.09
16:         14772512               0     000:00:00:00.44
17:         95815104               0     000:00:00:03.93


進行方針（ご確認ください）
1. Codon/Python最適化のポイント一覧化
21Py_constellations_optimized_codon.py で採用された 全最適化手法を箇条書きで整理します。

2. CUDA側の現状ロジックと差分抽出
CUDA側でまだ実装されていない、追加/置換すべきポイントをリストアップ。

3. CUDA用に書き換える際の注意点や方針
変数の型やビット操作、構造体の使い方、メモリ管理、並列化（atomicAdd等）、パフォーマンス重視のCUDA流書き方に適合させる案。

4. ソース単位のマージ・移植案提示
最も差分が大きい部分は「constellation生成・事前配置の最適化」「symmetry判定・重複排除」「盤面遷移のビット演算最適化」等。

Python+codon側の関数を、CUDAのC++/CUDA C流に具体的に落とし込みます。
CUDAカーネル・データ構造への適用例を示し、部分的にソース移植も行います。

「Codon/Python最適化テクニック一覧」を下記にまとめます。
Python+Codon最適化テクニック（抜粋）
【作業量が少ない順（実装しやすい順）】
1. 盤面状態のビットマスク化（各種マスクを事前計算、毎回再計算しない）
　→ CUDA実装でもローカル変数で完結する＆事前計算済みを渡すだけで即実装可

11. ループのビット演算高速化（while avail: bit=avail&-avail ...）
　→ CUDAでも全く同じ構文で使える。
　→for/whileループをbitmask展開に置き換えるだけ。実装コスト最小

9. next_freeを正確に伝搬（freeの使い回し防止）
　→ CUDAのスレッド・再帰変数としてしっかり管理するだけ。変数設計の問題で即着手可

10. 使い捨て一時変数を徹底排除（再計算抑制）
　→ CUDA再帰・ループ本体でローカル変数を適切に使うだけ。Pythonと同じ設計思想

8. symmetry/回転・ミラー重複排除による探索空間縮小
　→ symmetry()などの関数をマクロ化すればそのまま移植可
　→ 実装量は少ないが、最初に効果が出やすい

5. 多段分岐・特別ケースの個別最適化（中央列特別処理/コーナー処理など）
　→ Pythonと同じ分岐・if設計をCUDAカーネルでも展開するだけ
　→ コード量は増えるが、各関数/カーネル分岐で管理しやすい

6. 関数分割・テンプレート化による分岐削減
　→ CUDAはテンプレートもサポート、または関数分割で分岐パターンを個別化しやすい

2. constellation（星座）の候補事前生成とキャッシュ化
　→ 盤面候補の全列挙＆事前キャッシュ配列化はCUDA向けにも基本設計で実装済み
　→ 実装量も中くらいで済む（for/array/listの生成ロジックをCで再現）

7. atomic操作や並列化による高速カウント
　→ CUDA独自のatomicAddの使い方・warp/block単位集約の設計が必要
　→ Python側でのmultiprocessingよりはGPU側固有知識が必要

4. ジャスミン変換（回転・ミラーによる盤面正規化）
　→ CUDAでrot90/mirvert/jasmin関数等を高速化＆一時配列の扱い最適化
　→ 移植しやすいが、バグると出力が合わないので要テスト

12. state_hashの高速衝突回避設計
　→ CUDAで高速hashテーブルを設計するのは重い（Zobristハッシュや共有メモリ活用が必要）
　→ 並列hashテーブル設計は難易度高め（効くケースは超高速化）

3. Zobrist/状態ハッシュによる重複排除・枝刈り
　→ CUDAで衝突管理やハッシュ値管理・枝刈りの実装は最も移植コスト高（shared memory、atomic操作、lock管理などが必要）
　→ ただし大Nで効果絶大なので最終段階の目玉


21Py_constellations_optimized_codon.pyから要点をまとめ
A. コンステレーション（部分盤面事前配置）戦略
・N-Queens全体を一気に探索せず、盤面の一部（例：四隅/中央）にクイーンを事前配置し、その部分盤面（constellation）を使って以降のバックトラック探索を大幅削減。
・事前生成されたconstellationリストをforループで分割・並列探索（Codonでは@par/CUDAではブロックorスレッド分割）。

B. 対称性除去・重複排除の徹底
・盤面の90度・180度・270度回転／左右ミラーを全パターン生成、「最小表現」だけ探索。
・jasmin関数：盤面情報を回転・ミラーで“正規化”し、代表値だけ使う。
・状態ハッシュ＋signatureセットでconstellation重複排除（set/ハッシュテーブル）。

C. ビット演算による状態管理・遷移の最適化
・盤面の状態はすべてbitmask（int, uint32_t/uint64_t）で持ち、ビット演算で遷移。
・free & -freeで最下位の候補だけ取得→配置→消去のイテレータ化。
・mask/precalc：盤面サイズによるマスクや一時変数は関数外で1回だけ計算し、再利用。

D. next_free伝搬＆関数分割による無駄排除
・next_free（次の盤面で空いているbit群）を正確に伝搬、recalc禁止。
・各種「SQ*B」関数分割で、「コーナー配置」「中央列」「特定行」「連続配置」等に分岐した盤面探索の特殊最適化**。

E. 並列化・atomicカウント
・Codonでは@par＋atomicで分散。
・CUDAではconstellation単位でスレッド/ブロック割り当て→atomicAddで解数集約。

F. キャッシュ利用（部分盤面/状態ハッシュ/結果再利用）
・subconst_cacheなどの辞書/セットで「すでに探索した状態」を記録、同じ部分盤面は再帰を繰り返さない。
・constellation signature, state_hashなど高速衝突回避設計。

G. ループ・分岐削減/最小化
・一時変数と分岐を減らし分岐予測ミス低減/キャッシュ効率最大化。


CUDA現状の「抜け」
基本的なconstellationロジックのみ。
対称性除去やjasmin正規化、部分盤面キャッシュ・signature判定が未実装、もしくは簡易実装。
各種分岐最適化（中央列特別処理、コーナー配置の分岐、SQ***B関数分割）は未反映か単一ロジック。
状態管理はuint/int型だが、マスクやblocked計算が毎回再計算or複雑化していることが多い。
GPU並列化は各constellationごとにスレッド化・atomicAddで集約しているが、constellationリスト自体が非最適（重複や冗長パターンが多い）。

3. CUDAへの移植設計・マージ戦略
以下は移植設計の方針サマリです。
実際にはこのあと「実装例」として該当ソースを生成します（段階的に出力可能）。

(1) constellationリスト生成部の強化
Python側の gen_constellations()・jasmin正規化ロジックをCUDA側のホストコード（C++部）で事前生成。
必要に応じてCPUで事前生成→GPU転送方式 or GPU上で生成も可能。

(2) 重複排除・対称性除去
signature hashやjasmin変換をCUDA流（関数化 or precompute）で実装。
回転・ミラーを定数式／bit演算関数に最適化。

(3) 状態・ビットマスク管理
各種マスク・blocked/next_freeをconst int/constexprで保持。
配置遷移は必ず bit = avail & -avail／avail &= avail - 1で展開。

(4) 分岐最適化・SQ*B分割**
特別ケースや盤面分割ごとにCUDA関数分割（__device__関数）として整理。
スレッドごとにコンステレーションを担当→細かい分岐後は関数ポインタ／switchなどでもOK。

(5) キャッシュ＆atomic操作
subconst_cache相当は難しいが、「状態済みパターンの共有」は（必要なら）__shared__やグローバルメモリhashset。
atomicAddでcount集約。


 主な流れ・構成
constellation生成：N-Queens盤面の一部を事前にクイーン配置し、その部分配置（constellation）ごとに残りを探索する方式
・Python+codonの「gen_constellations」と同趣旨

constellationごとにGPUスレッド（またはwarp）割り当て
・各スレッドが部分盤面から探索を開始し、解数を計算
・atomicAdd等で解数を集約

対称性除去（symmetry）：90度/180度回転やミラーを部分的に実装（ただしPython最終形ほど徹底的ではない印象）
・各constellationごとに回転・ミラーを判定、ユニークなものだけ集計

ビットマスクによる状態管理
・ld, rd, colなどビット演算で盤面状態を保持
・free & -freeで次の配置候補を決定
・探索関数は基本的に再帰型 or 明示ループ型（C++/CUDA流）
・atomic操作やスレッド分散で並列集約
・状態キャッシュや部分盤面重複排除はやや限定的 or 未実装

主要関数・構造体
・struct Constellation：部分盤面の状態を持つ構造体
・generate_constellations()：事前に部分盤面を全生成し、constellations配列に格納
・symmetry判定関数：回転やミラーによる盤面の同型判定（90/180/270度、左右ミラー等）
・search_kernel（または同様の関数）：constellationごとに残りを探索
・main()：全体のフロー管理（盤面サイズ、constellation生成、デバイスメモリ確保、カーネル起動、結果集約）

 現状の「限界点」・未実装最適化
・jasmin正規化（盤面の最小表現化）が未実装or弱い
・signatureによる重複排除（state_hash, constellation signature等）が未実装
・細かい特殊ケース分岐（中央列特別処理、連続配置ケース最適化など）が未分離
・キャッシュ/メモリ効率の徹底化（部分盤面の再帰キャッシュ、探索済み状態の高速参照）が未反映
・constellation生成自体も単純全探索 or ミラー重複許容の場合がある
・next_free伝搬や再計算抑制が一部弱い場合あり

【追加・移植すべき主な最適化】
jasmin変換による盤面正規化
constellation生成時に全てをjasmin正規化し、同型盤面を一意化
signature（盤面シグネチャ）による重複排除
事前生成や探索途中でのstate_hash/signature記録による枝刈り
中央列特別処理・コーナー処理の個別分岐（細かい関数分割・最適化）
SQd0B/SQd1B/…各特殊ケースをCUDAの__device__関数として分割
next_freeの厳密伝搬と一時変数最適化
各探索関数でビット演算伝搬、一時計算・マスク等を極力抑制
部分盤面再帰・探索済みキャッシュ（subconst_cache相当）
CUDA流には制限があるが、warp内/ブロック内で共有メモリ利用も検討
探索関数の細分化・パターン分岐最適化
Pythonで大量に分割したSQ***B系関数群を必要に応じCUDA流関数に分割
state_hashによる高速判定
探索枝の高速ユニーク判定
atomic操作の最適配置
並列化部分をatomicAdd等で安全集約
メモリ転送・constellation管理の最適化
事前計算→GPU配列転送、盤面サイズで柔軟化


 CUDA版での「constellation生成＋jasmin正規化」設計
// 1. 盤面情報からij-klコードを生成（Python:to_ijkl相当）
__host__ __device__ uint32_t to_ijkl(int i, int j, int k, int l) {
    return (i << 15) | (j << 10) | (k << 5) | l;
}

// 2. jasmin正規化（回転・ミラーで最小値盤面を生成）関数
__host__ __device__ uint32_t jasmin(uint32_t ijkl, int N) {
    // Python側のロジックをC++/CUDAに変換
    // getj/geti/getk/getl, rot90, mirvert等を順次関数化
    // for _ in range(arg): ijkl = rot90(ijkl, N);
    // if getj(ijkl) < N-1-getj(ijkl): ijkl = mirvert(ijkl, N);
    // 詳細はご要望あれば全部書き下します
    // （このまま関数群のC++/CUDA翻訳できます）
}

// 3. signature用hash生成（state_hash相当）
__host__ __device__ uint64_t state_hash(uint32_t ld, uint32_t rd, uint32_t col, int row) {
    // Pythonのハッシュ式をC++に
    return (uint64_t(ld) * 0x9e3779b9ULL) ^ (uint64_t(rd) * 0x7f4a7c13ULL) ^ (uint64_t(col) * 0x6a5d39e9ULL) ^ row;
}

// 4. constellation生成ループで正規化・重複排除
std::unordered_set<uint32_t> signatures;
for (...) {
    uint32_t ijkl = to_ijkl(i, j, k, l);
    uint32_t jasmin_ijkl = jasmin(ijkl, N);
    if (signatures.count(jasmin_ijkl)) continue;
    signatures.insert(jasmin_ijkl);
    // 以降、CUDA配列への格納
}


*/
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#ifdef _WIN32
#include <windows.h>
#else
#include <sys/time.h>
#endif

#include <hip/hip_runtime.h>



#define INITIAL_CAPACITY 1000
#define presetQueens 4
#define THREAD_NUM		96
/**
  Constellation構造体の定義
*/
typedef struct{
  int id;
  int ld;
  int rd;
  int col;
  int startijkl;
  long solutions;
}Constellation;
/**
  IntHashSet構造体の定義
*/
typedef struct{
  int* data;
  int size;
  int capacity;
}IntHashSet;
/**
  ConstellationArrayList構造体の定義
*/
typedef struct{
  Constellation* data;
  int size;
  int capacity;
}ConstellationArrayList;
/**
 * 関数プロトタイプ
 */
void setPreQueens(int ld,int rd,int col,int k,int l,int row,int queens,int LD,int RD,int *counter,ConstellationArrayList* constellations,int N);
void execSolutions(ConstellationArrayList* constellations,int N);
void genConstellations(IntHashSet* ijklList,ConstellationArrayList* constellations,int N);
long calcSolutions(ConstellationArrayList* constellations,long solutions);
__host__ __device__ uint32_t jasmin(uint32_t ijkl, int N) ;
void add_constellation(int ld,int rd,int col,int startijkl,ConstellationArrayList* constellations);
__host__ __device__ void SQBkBlBjrB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQBklBjrB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQBlBjrB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQBjrB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQBlBkBjrB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N); 
__host__ __device__ void SQBlkBjrB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQBkBjrB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQBjlBkBlBjrB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQBjlBklBjrB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQBjlBlBkBjrB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQBjlBlkBjrB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQd2BkBlB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQd2BklB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQd2BlB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQd2B(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQd2BlBkB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQd2BlkB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQd1BkBlB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQd1BklB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQd1BlB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQd1B(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQd1BlBkB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQd1BlkB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQd0B(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQd0BkB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQd2BkB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
__host__ __device__ void SQd1BkB(int ld,int rd,int col,int start,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N);
/**
 * 盤面ユーティリティ群（ビットパック式盤面インデックス変換）
 *
 * Python実装のgeti/getj/getk/getl/toijklに対応。
 *
 * [i, j, k, l] 各クイーンの位置情報を5ビットずつ整数値（ijkl）にパック／アンパックするためのマクロ。
 * 15ビット～0ビットまでに [i|j|k|l] を格納する設計で、constellationのsignatureや
 * 回転・ミラー等の盤面操作を高速化する。
 *
 * 例：
 *   - geti(ijkl): 上位5ビット（15-19）からiインデックスを取り出す
 *   - toijkl(i, j, k, l): 各値を5ビット単位で連結し一意な整数値（signature）に変換
 *
 * [注意] N≦32 まで対応可能
 */
#define geti(ijkl) ( (ijkl>>15)&0x1F )
#define getj(ijkl) ( (ijkl>>10) &0x1F )
#define getk(ijkl) ( (ijkl>>5) &0x1F )
#define getl(ijkl) ( ijkl &0x1F )
#define toijkl(i,j,k,l) ( ((i)<<15)|((j)<<10)|((k)<<5)|(l) )
/**
  時計回りに90度回転
  rot90 メソッドは、90度の右回転（時計回り）を行います
  元の位置 (row,col) が、回転後の位置 (col,N-1-row) になります。
*/
#define rot90(ijkl,N) ( ((N-1-getk(ijkl))<<15) | ((N-1-getl(ijkl))<<10) | (getj(ijkl)<<5) | geti(ijkl) )
/**
  対称性のための計算と、ijklを扱うためのヘルパー関数。
  開始コンステレーションが回転90に対して対称である場合
*/
#define rot180(ijkl,N) ( ((N-1-getj(ijkl))<<15) | ((N-1-geti(ijkl))<<10) | ((N-1-getl(ijkl))<<5) | (N-1-getk(ijkl)) )
#define symmetry90(ijkl,N)( ((geti(ijkl)<<15) | (getj(ijkl)<<10) | (getk(ijkl)<<5) | getl(ijkl)) == ((N-1-getk(ijkl))<<15 | (N-1-getl(ijkl))<<10 | (getj(ijkl)<<5) | geti(ijkl)) )
/**
  symmetry: 回転・ミラー対称性ごとの重複補正 (90度:2, 180度:4, その他:8)
*/
#define symmetry(ijkl,N) ( (geti(ijkl)==N-1-getj(ijkl) && getk(ijkl)==N-1-getl(ijkl)) ? (symmetry90(ijkl,N) ? 2 : 4 ) : 8 )
/**
  左右のミラー 与えられたクイーンの配置を左右ミラーリングします。
  各クイーンの位置を取得し、列インデックスを N-1 から引いた位置に変更します（左右反転）。
  行インデックスはそのままにします。
*/
#define mirvert(ijkl,N) ( toijkl(N-1-geti(ijkl),N-1-getj(ijkl),getl(ijkl),getk(ijkl)) )
/**
 * 大小を比較して小さい最値を返却
 */
#define ffmin(a,b)(a<b ? a : b)
/**
  i,j,k,lをijklに変換し、特定のエントリーを取得する関数
  各クイーンの位置を取得し、最も左上に近い位置を見つけます
  最小の値を持つクイーンを基準に回転とミラーリングを行い、配置を最も左上に近い標準形に変換します。
  最小値を持つクイーンの位置を最下行に移動させる
  i は最初の行（上端） 90度回転2回
  j は最後の行（下端） 90度回転0回
  k は最初の列（左端） 90度回転3回
  l は最後の列（右端） 90度回転1回
  優先順位が l>k>i>j の理由は？
  l は右端の列に位置するため、その位置を基準に回転させることで、配置を最も標準形に近づけることができます。
  k は左端の列に位置しますが、l ほど標準形に寄せる影響が大きくないため、次に優先されます。
  i は上端の行に位置するため、行の位置を基準にするよりも列の位置を基準にする方が配置の標準化に効果的です。
  j は下端の行に位置するため、優先順位が最も低くなります。
*/
__host__ __device__ uint32_t jasmin(uint32_t ijkl, int N) 
{
    int arg = 0;
    int min_val = ffmin(getj(ijkl), N - 1 - getj(ijkl));
    if (ffmin(geti(ijkl), N - 1 - geti(ijkl)) < min_val) {
        arg = 2; min_val = ffmin(geti(ijkl), N - 1 - geti(ijkl));
    }
    if (ffmin(getk(ijkl), N - 1 - getk(ijkl)) < min_val) {
        arg = 3; min_val = ffmin(getk(ijkl), N - 1 - getk(ijkl));
    }
    if (ffmin(getl(ijkl), N - 1 - getl(ijkl)) < min_val) {
        arg = 1; min_val = ffmin(getl(ijkl), N - 1 - getl(ijkl));
    }
    for (int i = 0; i < arg; ++i) ijkl = rot90(ijkl, N);
    if (getj(ijkl) < N - 1 - getj(ijkl)) ijkl = mirvert(ijkl, N);
    return ijkl;
}
/**
  CUDA 初期化
  */
bool InitCUDA()
{
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    struct hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
/**
 * IntHashSet構造体のインスタンスを生成し、初期化する関数
 * @return 初期化済みのIntHashSetへのポインタ
 *         （data配列はINITIAL_CAPACITYで確保、size=0, capacity=INITIAL_CAPACITY）
 *         使用後は free_int_hashset() で解放すること
 */
IntHashSet* create_int_hashset()
{
  IntHashSet* set=(IntHashSet*)malloc(sizeof(IntHashSet));
  set->data=(int*)malloc(INITIAL_CAPACITY * sizeof(int));
  set->size=0;
  set->capacity=INITIAL_CAPACITY;
  return set;
}
/**
 * IntHashSet構造体が確保したメモリ領域を解放する関数
 * @param set 解放対象のIntHashSetポインタ
 *        （内部のdata配列と構造体本体をfreeする。多重freeに注意）
 */
void free_int_hashset(IntHashSet* set)
{
  free(set->data);
  free(set);
}
/**
 * IntHashSet内に指定した値が含まれているかを線形探索で判定する関数
 * @param set 探索対象のIntHashSetポインタ
 * @param value 判定したい整数値
 * @return 1: 含まれる / 0: 含まれない
 */
int int_hashset_contains(IntHashSet* set,int value)
{
  for(int i=0;i<set->size;i++){
    if(set->data[i]==value){ return 1; }
  }
  return 0;
}
/**
 * IntHashSetに指定した値を追加する関数
 * @param set 追加先のIntHashSetポインタ
 * @param value 追加したい整数値
 * @details
 *   既に同じ値が含まれている場合は何もしない（重複不可）。
 *   data配列が満杯の場合は容量を2倍に拡張（realloc）。
 */
void int_hashset_add(IntHashSet* set,int value)
{
  if(!int_hashset_contains(set,value)){
    if(set->size==set->capacity){
      set->capacity *= 2;
      set->data=(int*)realloc(set->data,set->capacity * sizeof(int));
    }
    set->data[set->size++]=value;
  }
}
/**
 * ConstellationArrayList構造体のインスタンスを生成し、初期化する関数
 * @return 初期化済みのConstellationArrayListへのポインタ
 *         （data配列はINITIAL_CAPACITYで確保、size=0, capacity=INITIAL_CAPACITY）
 *         使用後は free_constellation_arraylist() で解放すること
 */
ConstellationArrayList* create_constellation_arraylist()
{
  ConstellationArrayList* list=(ConstellationArrayList*)malloc(sizeof(ConstellationArrayList));
  list->data=(Constellation*)malloc(INITIAL_CAPACITY * sizeof(Constellation));
  list->size=0;
  list->capacity=INITIAL_CAPACITY;
  return list;
}
/**
 * ConstellationArrayList構造体と、その内部のdata配列を解放する関数
 * @param list 解放対象のConstellationArrayListポインタ
 * @note
 *   内部配列も本体もfreeされるので、多重解放に注意。
 */
void free_constellation_arraylist(ConstellationArrayList* list)
{
  free(list->data);
  free(list);
}
/**
 * ConstellationArrayListに要素を追加する関数
 * @param list 追加先のConstellationArrayListポインタ
 * @param value 追加するConstellation構造体
 * @details
 *   配列が満杯のときは容量を2倍に拡張（realloc）してから追加。
 */
void constellation_arraylist_add(ConstellationArrayList* list,Constellation value)
{
  if(list->size==list->capacity){
    list->capacity *= 2;
    list->data=(Constellation*)realloc(list->data,list->capacity * sizeof(Constellation));
  }
  list->data[list->size++]=value;
}
/**
 * すべてのフィールドを初期化したConstellation構造体のインスタンスを生成する関数
 * @return 初期化済みのConstellationへのポインタ（id,ld,rd,col,startijkl=0, solutions=-1）
 *         使用後はfree()でメモリ解放が必要
 */
Constellation* create_constellation()
{
  Constellation* new_constellation=(Constellation*)malloc(sizeof(Constellation));
  if(new_constellation){
    new_constellation->id=0;
    new_constellation->ld=0;
    new_constellation->rd=0;
    new_constellation->col=0;
    new_constellation->startijkl=0;
    new_constellation->solutions=-1;
  }
  return new_constellation;
}
/**
 * 引数で指定した値でConstellation構造体を生成・初期化する関数
 * @param id, ld, rd, col, startijkl, solutions 各フィールドにセットする値
 * @return フィールドがセット済みのConstellationへのポインタ
 *         使用後はfree()でメモリ解放が必要
 */
Constellation* create_constellation_with_values(int id,int ld,int rd,int col,int startijkl,long solutions)
{
  Constellation* new_constellation=(Constellation*)malloc(sizeof(Constellation));
  if(new_constellation){
    new_constellation->id=id;
    new_constellation->ld=ld;
    new_constellation->rd=rd;
    new_constellation->col=col;
    new_constellation->startijkl=startijkl;
    new_constellation->solutions=solutions;
  }
  return new_constellation;
}
/**
 * 指定したビットマスク・signatureからConstellationを生成し、リストに追加する関数
 * @param ld   クイーン配置の左斜め方向のビットマスク
 * @param rd   クイーン配置の右斜め方向のビットマスク
 * @param col  クイーン配置の縦方向のビットマスク
 * @param startijkl  盤面のsignature値
 * @param constellations  追加先のConstellationArrayListポインタ
 * @details
 *   solutionsフィールドは-1で初期化される。値はコピーされて配列に追加される。
 */
void add_constellation(int ld,int rd,int col,int startijkl,ConstellationArrayList* constellations)
{
  Constellation new_constellation={0,ld,rd,col,startijkl,-1};
  constellation_arraylist_add(constellations,new_constellation);
}
/**
 * Constellation構造体のstartijklの下位15ビット（jkl値）で昇順ソートするための比較関数
 * @param a 比較対象1（Constellation*へのvoid*）
 * @param b 比較対象2（Constellation*へのvoid*）
 * @return -1: a < b / 1: a > b / 0: 等しい
 * @details
 *   qsort等で使うことを想定。jkl値のみで比較する。
 */
int compareConstellations(const void* a, const void* b)
{
  Constellation* const1 = (Constellation*)a;
  Constellation* const2 = (Constellation*)b;
  // startijkl の最初の 15 ビットを取得
  int jkl1 = const1->startijkl & ((1 << 15) - 1);
  int jkl2 = const2->startijkl & ((1 << 15) - 1);
  // jkl に基づいてソート
  if (jkl1 < jkl2) {
      return -1;
  } else if (jkl1 > jkl2) {
      return 1;
  } else {
      return 0;
  }
}
/**
 * ConstellationArrayListのデータを、startijklの下位15ビット（jkl値）で昇順ソートする関数
 * @param constellations ソート対象のConstellationArrayListポインタ
 * @details
 *   比較関数 compareConstellations() を用いてqsortでソートされる。
 *   盤面のsignature（jkl値）でグルーピングや重複排除等を行う前処理にも使える。
 */
void sortConstellations(ConstellationArrayList* constellations) 
{
    // qsort を使ってソート
    qsort(constellations->data, constellations->size, sizeof(Constellation), compareConstellations);
}
/**
 * トラッシュ（無効・削除予定）用のダミーConstellationをリストに追加する関数
 * @param list 追加先のConstellationArrayListポインタ
 * @param ijkl トラッシュマーク対象の盤面signature値（下位ビット）
 * @details
 *   ld/rd/colを-1で埋め、startijklは(69<<20)|ijklとすることで
 *   「本来の探索対象ではない」ことを明示。探索・計数から除外したい時の管理用に利用。
 */
void addTrashConstellation(ConstellationArrayList* list, int ijkl) 
{
  // トラッシュ用のダミーコンステレーションを作成
  int ld = -1;
  int rd = -1;
  int col = -1;
  // 「69<<20」は“magic number”であり、通常のstartijklとは重複しない特殊値として扱う
  int startijkl = (69 << 20) | ijkl;
  // トラッシュコンステレーションをリストに追加
  add_constellation(ld, rd, col, startijkl, list);
}
/**
 * ConstellationArrayListをworkgroupSizeの倍数で区切るため、各グループ末尾に
 * トラッシュ（無効ダミー）コンステレーションを追加してリスト長を調整する関数
 *
 * @param constellations 入力となるConstellationArrayList（ソート済みを期待）
 * @param workgroupSize   1グループのスレッド数（CUDAのblockDimなど）
 * @return workgroupSizeの倍数にパディング済みの新ConstellationArrayList
 *
 * @details
 *   - 各startijkl（下位15ビット単位）のグループごとに、リスト長がworkgroupSizeで割り切れるまで
 *     addTrashConstellation() でダミーを追加。
 *   - 最後のグループも同様にパディング。
 *   - すでにsolutions>=0（解が既知）の要素は追加しない。
 *   - CUDAカーネルで「warp/block単位での等分散」に必須の前処理。
 */
ConstellationArrayList* fillWithTrash(ConstellationArrayList* constellations, int workgroupSize) 
{
  sortConstellations(constellations); // コンステレーションのリストをソート
  ConstellationArrayList* newConstellations = create_constellation_arraylist();// 新しいリストを作成
  int currentJkl = constellations->data[0].startijkl & ((1 << 15) - 1); // 最初のコンステレーションの currentJkl を取得
  for (int i = 0; i < constellations->size; i++) { // 各コンステレーションに対してループ
    Constellation c = constellations->data[i];
    if (c.solutions >= 0) continue;// 既にソリューションがあるものは無視
    if ((c.startijkl & ((1 << 15) - 1)) != currentJkl) { // 新しい ijkl グループの開始を確認
      while (newConstellations->size % workgroupSize != 0) { // workgroupSize の倍数になるまでトラッシュを追加
        addTrashConstellation(newConstellations, currentJkl);
      }
      currentJkl = c.startijkl & ((1 << 15) - 1);
    }
    add_constellation(c.ld, c.rd, c.col, c.startijkl, newConstellations);// コンステレーションを追加
  }
  while (newConstellations->size % workgroupSize != 0) { // 最後に残った分を埋める
    addTrashConstellation(newConstellations, currentJkl);
  }
  return newConstellations;
}
/**
 * 開始コンステレーション（部分盤面）の生成関数
 *
 * N-Queens探索の初期状態を最適化するため、3つまたは4つのクイーン（presetQueens）を
 * あらかじめ盤面に配置した全ての部分盤面（サブコンステレーション）を列挙・生成する。
 * 再帰的に呼び出され、各行ごとに可能な配置をすべて検証。
 *
 * @param ld   左対角線のビットマスク（既にクイーンがある位置は1）
 * @param rd   右対角線のビットマスク
 * @param col  縦方向（列）のビットマスク
 * @param k    事前にクイーンを必ず置く行のインデックス1
 * @param l    事前にクイーンを必ず置く行のインデックス2
 * @param row  現在の再帰探索行
 * @param queens 現在までに盤面に配置済みのクイーン数
 * @param LD/RD 探索初期状態用のマスク（使用例次第で追記）
 * @param counter 生成されたコンステレーション数を書き込むカウンタ
 * @param constellations 生成したコンステレーション（部分盤面配置）のリスト
 * @param N     盤面サイズ
 * @details
 *   - row==k/lの場合は必ずクイーンを配置し次の行へ進む
 *   - queens==presetQueensに到達したら、現時点の盤面状態をコンステレーションとして記録
 *   - その他の行では、空いている位置すべてにクイーンを順次試し、再帰的に全列挙
 *   - 生成された部分盤面は、対称性除去・探索分割等の高速化に用いる
 */
void setPreQueens(int ld,int rd,int col,int k,int l,int row,int queens,int LD,int RD,int *counter,ConstellationArrayList* constellations,int N)
{
  int mask=(1<<N)-1;//setPreQueensで使用
  if(row==k || row==l){ // k行とl行はさらに進む
    setPreQueens(ld<<1,rd>>1,col,k,l,row+1,queens,LD,RD,counter,constellations,N);
    return;
  }
  // preQueensのクイーンが揃うまでクイーンを追加する。現在のクイーンの数が presetQueens に達した場合、現在の状態を新しいコンステレーションとして追加し、カウンターを増加させる。
  if(queens==presetQueens){
    // リストに４個クイーンを置いたセットを追加する
    add_constellation(ld,rd,col,row<<20,constellations);
    (*counter)++;
    return;
  }
  // k列かl列が終わっていなければ、クイーンを置いてボードを占領し、さらに先に進む。
  else{
    // 現在の行にクイーンを配置できる位置（自由な位置）を計算
    int free=~(ld | rd | col | (LD>>(N-1-row)) | (RD<<(N-1-row))) & mask;
    int bit;
    while(free){
      bit=free & (-free);
      free -= bit;
      // クイーンをおける場所があれば、その位置にクイーンを配置し、再帰的に次の行に進む
      setPreQueens((ld | bit)<<1,(rd | bit)>>1,col | bit,k,l,row+1,queens+1,LD,RD,counter,constellations,N);
    }
  }
}
/**
 * 指定した盤面 (i, j, k, l) を90度・180度・270度回転したいずれかの盤面が
 * すでにIntHashSetに存在しているかをチェックする関数
 *
 * @param ijklList 既出盤面signature（ijkl値）の集合（HashSet）
 * @param i,j,k,l  チェック対象の盤面インデックス
 * @param N        盤面サイズ
 * @return         いずれかの回転済み盤面が登録済みなら1、なければ0
 * @details
 *   - N-Queens探索で、既存盤面の90/180/270度回転形と重複する配置を高速に排除する。
 *   - 回転後のijklをそれぞれ計算し、HashSetに含まれていれば即1を返す（重複扱い）。
 *   - 真の“unique配置”のみ探索・カウントしたい場合の前処理として必須。
 */
int checkRotations(IntHashSet* ijklList,int i,int j,int k,int l,int N)
{
  int rot90=((N-1-k)<<15)+((N-1-l)<<10)+(j<<5)+i;
  int rot180=((N-1-j)<<15)+((N-1-i)<<10)+((N-1-l)<<5)+(N-1-k);
  int rot270=(l<<15)+(k<<10)+((N-1-i)<<5)+(N-1-j);
  if(int_hashset_contains(ijklList,rot90)){ return 1; }
  if(int_hashset_contains(ijklList,rot180)){ return 1; }
  if(int_hashset_contains(ijklList,rot270)){ return 1; }
  return 0;
}
/**
 * ConstellationArrayList内の全Constellationのsolutionsフィールド値を合計する関数
 * 
 * @param constellations 合計対象のConstellationArrayListポインタ
 * @param solutions      合計値の初期値（0を渡すのが標準だが累積加算も可）
 * @return 全要素のsolutions値（0より大きいもののみ）の合計
 * 
 * @details
 *   - 各Constellationのsolutions > 0 のものだけを加算（未計算=-1はスキップ）
 *   - N-Queens探索で全グループ/分割探索の解数を集約する用途
 *   - 戻り値は累積加算値なので、通常は0で初期化して使う
 */
long calcSolutions(ConstellationArrayList* constellations,long solutions)
{
  Constellation* c;
  for(int i=0;i<constellations->size;i++){
    c=&constellations->data[i];
    if(c->solutions > 0){
      solutions += c->solutions;
    }
  }
  return solutions;
}
/**
 * CUDAカーネル：各Constellation（部分盤面）ごとにN-Queens解数を並列探索し、block単位で合計値を出力する関数
 *
 * @param constellations 入力となるConstellation配列（部分盤面群）
 * @param _total         各blockごとの解数合計を書き込む配列（block数分）
 * @param N              盤面サイズ
 * @param totalSize      探索対象Constellationの総数
 *
 * @details
 *   - 各threadは自身のidxに対応するConstellationに対し、該当ソルバ（SQ...関数群）で解数を探索
 *   - dummy data（start==69）はスキップ（トラッシュ処理）
 *   - 盤面の対称性補正（symmetry(ijkl,N)）も適用し、正確なユニーク解数を求める
 *   - block内のthreadで部分和（sum[tid]）を計算し、warp・block内で段階的に加算・集約
 *   - 最終的にblockごとの合計値を_total[bid]に格納（CPU側で全block合計すれば総解数）
 *   - スレッド間同期（__syncthreads, __syncwarp）により正確な集約処理を実装
 *   - 大規模並列GPU探索時でも「高速・正確・スケーラブル」なN-Queens全解数計算を実現
 */
__global__ void execSolutionsKernel(Constellation* constellations,unsigned int* _total,int N, int totalSize)
{
    unsigned const int tid=threadIdx.x;
    unsigned const int bid=blockIdx.x;
    unsigned const int idx = bid*blockDim.x+tid;
    // 範囲外アクセスのチェック
    __shared__ unsigned int sum[THREAD_NUM];
    if (idx >= totalSize){
       sum[tid]=0;
       return;
    }
    Constellation* constellation = &constellations[idx];
    int start = constellation->startijkl >> 20;
    //dummy dataはスキップする
    if (start == 69){
      sum[tid]=0;
      return;
    } 
    int j = getj(constellation->startijkl);
    int k = getk(constellation->startijkl);
    int l = getl(constellation->startijkl);
    int ijkl = constellation->startijkl & ((1 << 20) - 1);
    int ld = constellation->ld >> 1;
    int rd = constellation->rd >> 1;
    int col = (constellation->col >> 1) | (~((1 << (N - 2)) - 1));
    long tempcounter = 0;
    int LD = (1 << (N - 1) >> j) | (1 << (N - 1) >> l);
    ld |= LD>>(N-start);
    if(start>k){
      rd |= (1<<(N-1)>>(start-k+1));
    }
    if(j >= 2 * N-33-start){// クイーンjからのrdがない場合のみ追加する
      rd |= (1<<(N-1)>>j)<<(N-2-start);// 符号ビットを占有する！
    }
    int free=~(ld | rd | col);
    int jmark = j + 1;
    int endmark = N - 2;
    int mark1, mark2;
    /**
      どのソリングアルゴリズムを使うかを決めるための大きなケースの区別
      クイーンjがコーナーから2列以上離れている場合
    */
    if(j<(N-3)){
      jmark=j+1;
      endmark=N-2;
      /**
        クイーンjがコーナーから2列以上離れているが、jクイーンからのrdが開始時
        に正しく設定できる場合。
      */
      if(j>2 * N-34-start){
        if(k<l){
          mark1=k-1;
          mark2=l-1;
          if(start<l){// 少なくともlがまだ来ていない場合
            if(start<k){// もしkがまだ来ていないなら
              if(l != k+1){ // kとlの間に空行がある場合
                SQBkBlBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
              }else{// kとlの間に空行がない場合
                SQBklBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
              }
            }else{// もしkがすでに開始前に来ていて、lだけが残っている場合
              SQBlBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
            }
          }else{// kとlの両方が開始前にすでに来ていた場合
            SQBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
          }
        }else{// l<k
          mark1=l-1;
          mark2=k-1;
          if(start<k){// 少なくともkがまだ来ていない場合
            if(start<l){// lがまだ来ていない場合
              if(k != l+1){// lとkの間に少なくとも1つの自由行がある場合
                SQBlBkBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
              }else{// lとkの間に自由行がない場合
                SQBlkBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
              }
            }else{ // lがすでに来ていて、kだけがまだ来ていない場合
              SQBkBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
            }
          }else{// lとkの両方が開始前にすでに来ていた場合
            SQBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
          }
        }
      }else{
        /**
          クイーンjのrdをセットできる行N-1-jmarkに到達するために、
          最初にいくつかのクイーンをセットしなければならない場合。
        */
        if(k<l){
          mark1=k-1;
          mark2=l-1;

          if(l != k+1){// k行とl行の間に少なくとも1つの空行がある。
            SQBjlBkBlBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
          }else{// lがkの直後に来る場合
            SQBjlBklBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
          }
        }else{  // l<k
          mark1=l-1;
          mark2=k-1;
          if(k != l+1){// l行とk行の間には、少なくともefree行が存在する。
            SQBjlBlBkBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
          }else{// kがlの直後に来る場合
            SQBjlBlkBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
          }
        }
      }
    }else if(j==(N-3)){// クイーンjがコーナーからちょうど2列離れている場合。
     // これは、最終行が常にN-2行になることを意味する。
      endmark=N-2;
      if(k<l){
        mark1=k-1;
        mark2=l-1;
        if(start<l){// 少なくともlがまだ来ていない場合
          if(start<k){// もしkもまだ来ていないなら
            if(l != k+1){// kとlの間に空行がある場合
              SQd2BkBlB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
            }else{
              SQd2BklB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
            }
          }else{// k が開始前に設定されていた場合
            mark2=l-1;
            SQd2BlB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
          }
        }else{ // もしkとlが開始前にすでに来ていた場合
          SQd2B(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
        }
      }else{// l<k
        mark1=l-1;
        mark2=k-1;
        endmark=N-2;
        if(start<k){// 少なくともkがまだ来ていない場合
          if(start<l){// lがまだ来ていない場合
            if(k != l+1){// lとkの間に空行がある場合
              SQd2BlBkB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
            }else{// lとkの間に空行がない場合
              SQd2BlkB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
            }
          }else{ // l が開始前に来た場合
            mark2=k-1;
            SQd2BkB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
          }
        }else{ // lとkの両方が開始前にすでに来ていた場合
          SQd2B(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
        }
      }
    }else if(j==N-2){ // クイーンjがコーナーからちょうど1列離れている場合
      if(k<l){// kが最初になることはない、lはクイーンの配置の関係で
                  // 最後尾にはなれないので、常にN-2行目で終わる。
        endmark=N-2;

        if(start<l){// 少なくともlがまだ来ていない場合
          if(start<k){// もしkもまだ来ていないなら
            mark1=k-1;

            if(l != k+1){// kとlが隣り合っている場合
              mark2=l-1;
              SQd1BkBlB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
            }else{
              SQd1BklB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
            }
          }else{// lがまだ来ていないなら
            mark2=l-1;
            SQd1BlB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
          }
        }else{// すでにkとlが来ている場合
          SQd1B(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
        }
      }else{ // l<k
        if(start<k){// 少なくともkがまだ来ていない場合
          if(start<l){ // lがまだ来ていない場合
            if(k<N-2){// kが末尾にない場合
              mark1=l-1;
              endmark=N-2;

              if(k != l+1){// lとkの間に空行がある場合
                mark2=k-1;
                SQd1BlBkB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
              }else{// lとkの間に空行がない場合
                SQd1BlkB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
              }
            }else{// kが末尾の場合
              if(l != (N-3)){// lがkの直前でない場合
                mark2=l-1;
                endmark=(N-3);
                SQd1BlB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
              }else{// lがkの直前にある場合
                endmark=(N-4);
                SQd1B(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
              }
            }
          }else{ // もしkがまだ来ていないなら
            if(k != N-2){// kが末尾にない場合
              mark2=k-1;
              endmark=N-2;
              SQd1BkB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
            }else{// kが末尾の場合
              endmark=(N-3);
              SQd1B(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
            }
          }
        }else{// kとlはスタートの前
          endmark=N-2;
          SQd1B(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
        }
      }
    }else{// クイーンjがコーナーに置かれている場合
      endmark=N-2;
      if(start>k){
        SQd0B(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
      }else{
        /**
          クイーンをコーナーに置いて星座を組み立てる方法と、ジャスミンを適用
          する方法によって、Kは最後列に入ることはできない。
        */
        mark1=k-1;
        SQd0BkB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
      }
    }
    // 完成した開始コンステレーションを削除する。
    sum[tid]=tempcounter * symmetry(ijkl,N);
    __syncthreads();if(tid<64&&tid+64<THREAD_NUM){
      sum[tid]+=sum[tid+64];
    }
    __syncwarp();if(tid<32){
      sum[tid]+=sum[tid+32];
    } 
    __syncwarp();if(tid<16){
      sum[tid]+=sum[tid+16];
    } 
    __syncwarp();if(tid<8){
      sum[tid]+=sum[tid+8];
    } 
    __syncwarp();if(tid<4){
      sum[tid]+=sum[tid+4];
    } 
    __syncwarp();if(tid<2){
      sum[tid]+=sum[tid+2];
    } 
    __syncwarp();if(tid<1){
      sum[tid]+=sum[tid+1];
    } 
    __syncwarp();if(tid==0){
      _total[bid]=sum[0];
    }

}
/**
 * ConstellationArrayListの各Constellation（部分盤面）ごとに
 * N-Queens探索を分岐し、そのユニーク解数をsolutionsフィールドに記録する関数（CPU版）
 *
 * @param constellations 解探索対象のConstellationArrayListポインタ
 * @param N              盤面サイズ
 *
 * @details
 *   - 各Constellation（部分盤面）ごとにj, k, l, 各マスク値を展開し、
 *     複雑な分岐で最適な再帰ソルバー（SQ...関数群）を呼び出して解数を計算
 *   - 分岐ロジックは、部分盤面・クイーンの位置・コーナーからの距離などで高速化
 *   - 解数はtemp_counterに集約し、各Constellationのsolutionsフィールドに記録
 *   - symmetry(ijkl, N)で回転・ミラー重複解を補正
 *   - GPUバージョン(execSolutionsKernel)のCPU移植版（デバッグ・逐次確認にも活用）
 *
 * @note
 *   - N-Queens最適化アルゴリズムの核心部
 *   - temp_counterは再帰呼び出しで合計を受け渡し
 *   - 実運用時は、より多くの分岐パターンを組み合わせることで最大速度を発揮
 */
void execSolutions(ConstellationArrayList* constellations,int N)
{
  int j=0;
  int k=0;
  int l=0;
  int ijkl=0;
  int ld=0;
  int rd=0;
  int col=0;
  int startIjkl=0;
  int start=0;
  int free=0;
  int LD=0;
  int jmark=0;
  int endmark=0;
  int mark1=0;
  int mark2=0;
  long tempcounter=0;
  int smallmask=(1<<(N-2))-1;
  for(int i=0;i<constellations->size;i++){
    Constellation* constellation=&constellations->data[i];
    startIjkl=constellation->startijkl;
    start=startIjkl>>20;
    ijkl=startIjkl & ((1<<20)-1);
    j=getj(ijkl);
    k=getk(ijkl);
    l=getl(ijkl);
    /**
      重要な注意：ldとrdを1つずつ右にずらすが、これは右列は重要ではないから
      （常に女王lが占有している）。
    */
    // 最下段から上に、jとlのクイーンによるldの占有を追加する。
    // LDとrdを1つずつ右にずらすが、これは右列は重要ではないから（常に女王lが占有している）。
    LD=(1<<(N-1)>>j) | (1<<(N-1)>>l);
    ld=constellation->ld>>1;
    ld |= LD>>(N-start);
    rd=constellation->rd>>1;// クイーンjとkのrdの占有率を下段から上に加算する。
    if(start>k){
      rd |= (1<<(N-1)>>(start-k+1));
    }
    if(j >= 2 * N-33-start){// クイーンjからのrdがない場合のみ追加する
      rd |= (1<<(N-1)>>j)<<(N-2-start);// 符号ビットを占有する！
    }
    // また、colを占有し、次にフリーを計算する
    col=(constellation->col>>1) | (~smallmask);
    free=~(ld | rd | col);
    /**
      どのソリングアルゴリズムを使うかを決めるための大きなケースの区別
      クイーンjがコーナーから2列以上離れている場合
    */
    if(j<(N-3)){
      jmark=j+1;
      endmark=N-2;
      /**
        クイーンjがコーナーから2列以上離れているが、jクイーンからのrdが開始時
        に正しく設定できる場合。
      */
      if(j>2 * N-34-start){
        if(k<l){
          mark1=k-1;
          mark2=l-1;
          if(start<l){// 少なくともlがまだ来ていない場合
            if(start<k){// もしkがまだ来ていないなら
              if(l != k+1){ // kとlの間に空行がある場合
                SQBkBlBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
              }else{// kとlの間に空行がない場合
                SQBklBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
              }
            }else{// もしkがすでに開始前に来ていて、lだけが残っている場合
              SQBlBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
            }
          }else{// kとlの両方が開始前にすでに来ていた場合
            SQBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
          }
        }else{// l<k 
          mark1=l-1;
          mark2=k-1;
          if(start<k){// 少なくともkがまだ来ていない場合
            if(start<l){// lがまだ来ていない場合
              if(k != l+1){// lとkの間に少なくとも1つの自由行がある場合
                SQBlBkBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
              }else{// lとkの間に自由行がない場合
                SQBlkBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
              }
            }else{ // lがすでに来ていて、kだけがまだ来ていない場合
              SQBkBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
            }
          }else{// lとkの両方が開始前にすでに来ていた場合
            SQBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
          }
        }
      }else{
        /**
          クイーンjのrdをセットできる行N-1-jmarkに到達するために、
          最初にいくつかのクイーンをセットしなければならない場合。
        */
        if(k<l){
          mark1=k-1;
          mark2=l-1;

          if(l != k+1){// k行とl行の間に少なくとも1つの空行がある。
            SQBjlBkBlBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
          }else{// lがkの直後に来る場合
            SQBjlBklBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
          }
        }else{  // l<k
          mark1=l-1;
          mark2=k-1;
          if(k != l+1){// l行とk行の間には、少なくともefree行が存在する。
            SQBjlBlBkBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
          }else{// kがlの直後に来る場合 
            SQBjlBlkBjrB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
          }
        }
      }
    }else if(j==(N-3)){// クイーンjがコーナーからちょうど2列離れている場合。
     // これは、最終行が常にN-2行になることを意味する。
      endmark=N-2;
      if(k<l){
        mark1=k-1;
        mark2=l-1;
        if(start<l){// 少なくともlがまだ来ていない場合
          if(start<k){// もしkもまだ来ていないなら
            if(l != k+1){// kとlの間に空行がある場合
              SQd2BkBlB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
            }else{
              SQd2BklB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
            }
          }else{// k が開始前に設定されていた場合
            mark2=l-1;
            SQd2BlB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
          }
        }else{ // もしkとlが開始前にすでに来ていた場合
          SQd2B(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
        }
      }else{// l<k
        mark1=l-1;
        mark2=k-1;
        endmark=N-2;
        if(start<k){// 少なくともkがまだ来ていない場合
          if(start<l){// lがまだ来ていない場合
            if(k != l+1){// lとkの間に空行がある場合
              SQd2BlBkB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
            }else{// lとkの間に空行がない場合
              SQd2BlkB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
            }
          }else{ // l が開始前に来た場合
            mark2=k-1;
            SQd2BkB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
          }
        }else{ // lとkの両方が開始前にすでに来ていた場合
          SQd2B(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
        }
      }
    }else if(j==N-2){ // クイーンjがコーナーからちょうど1列離れている場合
      if(k<l){// kが最初になることはない、lはクイーンの配置の関係で
                  // 最後尾にはなれないので、常にN-2行目で終わる。
        endmark=N-2;

        if(start<l){// 少なくともlがまだ来ていない場合
          if(start<k){// もしkもまだ来ていないなら
            mark1=k-1;

            if(l != k+1){// kとlが隣り合っている場合
              mark2=l-1;
              SQd1BkBlB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
            }else{
              SQd1BklB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
            }
          }else{// lがまだ来ていないなら
            mark2=l-1;
            SQd1BlB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
          }
        }else{// すでにkとlが来ている場合
          SQd1B(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
        }
      }else{ // l<k
        if(start<k){// 少なくともkがまだ来ていない場合
          if(start<l){ // lがまだ来ていない場合
            if(k<N-2){// kが末尾にない場合
              mark1=l-1;
              endmark=N-2;

              if(k != l+1){// lとkの間に空行がある場合
                mark2=k-1;
                SQd1BlBkB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
              }else{// lとkの間に空行がない場合
                SQd1BlkB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
              }
            }else{// kが末尾の場合
              if(l != (N-3)){// lがkの直前でない場合
                mark2=l-1;
                endmark=(N-3);
                SQd1BlB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
              }else{// lがkの直前にある場合
                endmark=(N-4);
                SQd1B(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
              }
            }
          }else{ // もしkがまだ来ていないなら
            if(k != N-2){// kが末尾にない場合
              mark2=k-1;
              endmark=N-2;
              SQd1BkB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
            }else{// kが末尾の場合
              endmark=(N-3);
              SQd1B(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
            }
          }
        }else{// kとlはスタートの前
          endmark=N-2;
          SQd1B(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
        }
      }
    }else{// クイーンjがコーナーに置かれている場合
      endmark=N-2;
      if(start>k){
        SQd0B(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
      }else{
        /**
          クイーンをコーナーに置いて星座を組み立てる方法と、ジャスミンを適用
          する方法によって、Kは最後列に入ることはできない。
        */
        mark1=k-1;
        SQd0BkB(ld,rd,col,start,free,jmark,endmark,mark1,mark2,&tempcounter,N);
      }
    }
    // 完成した開始コンステレーションを削除する。
    constellation->solutions=tempcounter * symmetry(ijkl,N);
    tempcounter=0;
  }
}
/**
 * 開始コンステレーション（部分盤面配置パターン）の列挙・重複排除を行う関数
 *
 * @param ijklList        uniqueな部分盤面signature（ijkl値）の格納先HashSet
 * @param constellations  Constellation本体リスト（実際の盤面は後続で生成）
 * @param N               盤面サイズ
 *
 * @details
 *   - コーナー・エッジ・対角・回転対称性を考慮し、「代表解」となるuniqueな開始盤面のみ抽出する。
 *   - forループの入れ子により、N-Queens盤面の「最小単位部分盤面」を厳密な順序で列挙。
 *   - k, l, i, j 各インデックスの取り方・範囲・重複排除のための判定ロジックが最適化されている。
 *   - checkRotations()で既出盤面（回転対称）を排除、必要なものだけをijklListに追加。
 *   - このunique setをもとに、後段でConstellation構造体の生成・分割探索を展開可能。
 *
 * @note
 *   - 「部分盤面分割＋代表解のみ探索」戦略は大規模Nの高速化の要！
 *   - このループ構造・排除ロジックがN-Queensソルバの根幹。
 */
void genConstellations(IntHashSet* ijklList,ConstellationArrayList* constellations,int N)
{
  int halfN=(N+1) / 2;// N の半分を切り上げる
  int L=1<<(N-1);//Lは左端に1を立てる
  /**
    コーナーにクイーンがいない場合の開始コンステレーションを計算する
    最初のcolを通過する
    k: 最初の列（左端）に配置されるクイーンの行のインデックス。
  */
  for(int k=1;k<halfN;k++){
    /**
      l: 最後の列（右端）に配置されるクイーンの行のインデックス。
      l を k より後の行に配置する理由は、回転対称性を考慮して配置の重複を避け
      るためです。
      このアプローチにより、探索空間が効率化され、N-クイーン問題の解決が迅速
      かつ効率的に行えるようになります。
      最後のcolを通過する
    */
    for(int l=k+1;l<(N-1);l++){
      /**
        i: 最初の行（上端）に配置されるクイーンの列のインデックス。
        最初の行を通過する
        k よりも下の行に配置することで、ボード上の対称性や回転対称性を考慮し
        て、重複した解を避けるための配慮がされています。
      */
      for(int i=k+1;i<(N-1);i++){
        // i==N-1-lは、行iが列lの「対角線上」にあるかどうかをチェックしています。
        if(i==(N-1)-l){
          continue;
        }
        /**
            j: 最後の行（下端）に配置されるクイーンの列のインデックス。
            最後の行を通過する
        */
        for(int j=N-k-2;j>0;j--){
        /**
          同じ列や行にクイーンが配置されている場合は、その配置が有効でない
          ためスキップ
        */
          if(j==i || l==j){
            continue;
          }
          /**
            回転対称でスタートしない場合
            checkRotationsで回転対称性をチェックし、対称でない場合にijklList
            に配置を追加します。
          */
          if(!checkRotations(ijklList,i,j,k,l,N)){
            int_hashset_add(ijklList,toijkl(i,j,k,l));
          }
        }
      }
    }
  }
  /**
    コーナーにクイーンがある場合の開始コンステレーションを計算する
    最初のクイーンを盤面の左上隅（0,0）に固定
    j は最後の行に置かれるクイーンの列インデックスです。これは 1 から N-3 ま
    での値を取ります。
  */
  for(int j=1;j<N-2;j++){// jは最終行のクイーンのidx
    for(int l=j+1;l<(N-1);l++){// lは最終列のクイーンのidx
      int_hashset_add(ijklList,toijkl(0,j,0,l));
    }
  }
  IntHashSet* ijklListJasmin=create_int_hashset();
  int startConstellation;
  for(int i=0;i<ijklList->size;i++){
    startConstellation=ijklList->data[i];
    int_hashset_add(ijklListJasmin,jasmin(startConstellation,N));
  }
  //free_int_hashset(ijklList);
  ijklList=ijklListJasmin;
  /**
    jasmin関数を使用して、クイーンの配置を回転およびミラーリングさせて、最
    も左上に近い標準形に変換します。
    同じクイーンの配置が標準形に変換された場合、同じ整数値が返されます。
    ijkListJasmin は HashSet です。
    jasmin メソッドを使用して変換された同じ値のクイーンの配置は、HashSet に
    一度しか追加されません。
    したがって、同じ値を持つクイーンの配置が複数回追加されても、HashSet の
    サイズは増えません。
  */
  //int i,j,k,l,ld,rd,col,currentSize=0;
  int sc=0;
  int i=0;
  int j=0;
  int k=0;
  int l=0;
  int ld=0;
  int rd=0;
  int col=0;
  int LD=0;
  int RD=0;
  int counter=0;
  int currentSize=0;
  for(int s=0;s<ijklList->size;s++){
    sc=ijklList->data[s];
    i=geti(sc);
    j=getj(sc);
    k=getk(sc);
    l=getl(sc);
    /**
      プレクイーンでボードを埋め、対応する変数を生成する。
      各星座に対して ld,rd,col,start_queens_ijkl を設定する。
      碁盤の境界線上のクイーンに対応する碁盤を占有する。
      空いている最初の行、すなわち1行目から開始する。
      クイーンの左対角線上の攻撃範囲を設定する。
      L>>>(i-1) は、Lを (i-1) ビット右にシフトします。これにより、クイーンの
      位置 i に対応するビットが右に移動します。
      1<<(N-k) は、1を (N-k) ビット左にシフトします。これにより、位置 k に対
      応するビットが左に移動します。
      両者をビットOR (|) することで、クイーンの位置 i と k に対応するビットが
      1となり、これが左対角線の攻撃範囲を表します。
    */
    ld=(L>>(i-1)) | (1<<(N-k));
    /**
      クイーンの右対角線上の攻撃範囲を設定する。
      L>>>(i+1) は、Lを (i+1) ビット右にシフトします。これにより、クイーンの
      位置 i に対応するビットが右に移動します。
      1<<(l-1) は、1を (l-1) ビット左にシフトします。これにより、位置 l に対
      応するビットが左に移動します。
      両者をビットOR (|) することで、クイーンの位置 i と l に対応するビットが
      1となり、これが右対角線の攻撃範囲を表します。
    */
    rd=(L>>(i+1)) | (1<<(l-1));
    /**
      クイーンの列の攻撃範囲を設定する。
      1 は、最初の列（左端）にクイーンがいることを示します。
      L は、最上位ビットが1であるため、最初の行にクイーンがいることを示します。
      L>>>i は、Lを i ビット右にシフトし、クイーンの位置 i に対応する列を占有します
      L>>>j は、Lを j ビット右にシフトし、クイーンの位置 j に対応する列を占有します。
      これらをビットOR (|) することで、クイーンの位置 i と j に対応する列が1
      となり、これが列の攻撃範囲を表します。
    */
    col=1 | L | (L>>i) | (L>>j);
    /**
      最後の列のクイーンj、k、lの対角線を占領しボード上方に移動させる
      L>>>j は、Lを j ビット右にシフトし、クイーンの位置 j に対応する左対角線を占有します。
      L>>>l は、Lを l ビット右にシフトし、クイーンの位置 l に対応する左対角線を占有します。
      両者をビットOR (|) することで、クイーンの位置 j と l に対応する左対角線
      が1となり、これが左対角線の攻撃範囲を表します。
    */
    LD=(L>>j) | (L>>l);
    /**
      最後の列の右対角線上の攻撃範囲を設定する。
      L>>>j は、Lを j ビット右にシフトし、クイーンの位置 j に対応する右対角線を占有します。
      1<<k は、1を k ビット左にシフトし、クイーンの位置 k に対応する右対角線を占有します。
      両者をビットOR (|) することで、クイーンの位置 j と k に対応する右対角線
      が1となり、これが右対角線の攻撃範囲を表します。
    */
    RD=(L>>j) | (1<<k);
    // すべてのサブコンステレーションを数える
    counter=0;
    // すべてのサブコンステレーションを生成する
    setPreQueens(ld,rd,col,k,l,1,j==N-1 ? 3 : 4,LD,RD,&counter,constellations,N);
    currentSize=constellations->size;
     // jklとsymとstartはすべてのサブコンステレーションで同じである
    for(int a=0;a<counter;a++){
      constellations->data[currentSize-a-1].startijkl |= toijkl(i,j,k,l);
    }
  }
}
/**
 * 未使用変数警告（-Wunused-parameter等）を抑制するためのダミー関数
 *
 * @param unuse 未使用int変数（何でもOK、実際には使わない）
 * @param argv  未使用char*配列（何でもOK、実際には使わない）
 * @details
 *   - コンパイラの「未使用引数」警告抑制のため、型に応じて何らかの操作（printf等）を
 *     実装しておくのが定番。
 *   - 本番コードで未使用変数が残る場合や、必須関数のダミー実装等で用いる。
 *   - 最適化時に消される可能性もあるので、「本当に必要な値」には使わないこと。
 */
void f(int unuse,char* argv[]){
  printf("%d%s\n",unuse,argv[0]);
}
/** */
__host__ __device__ void SQd0B(int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N)
{
  if(row==endmark){
    (*tempcounter)++;
    return;
  }
  int bit;
  int nextfree;
  int next_ld;
  int next_rd;
  int next_col;
  while(free){
    free-=bit=free&(-free);;
    next_ld=((ld|bit)<<1);
    next_rd=((rd|bit)>>1);
    next_col=(col|bit);
    nextfree=~(next_ld|next_rd|next_col);
    if(nextfree){
      if(row<endmark-1){
        if(~((next_ld<<1)|(next_rd>>1)|(next_col))>0)
          SQd0B(next_ld,next_rd,next_col,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }else{
        SQd0B(next_ld,next_rd,next_col,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
  }
}
__host__ __device__ void SQd0BkB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int N3=N-3;
  int bit;
  int nextfree;
  if(row==mark1){
    while(free){
      free-=bit=free&(-free);;
      nextfree=~(((ld|bit)<<2)|((rd|bit)>>2)|(col|bit)|1<<(N3));
      if(nextfree){
        SQd0B((ld|bit)<<2,((rd|bit)>>2)|1<<(N3),col|bit,row+2,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
    return;
  }
  while(free){
    free-=bit=free&(-free);;
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQd0BkB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQd1BklB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int N4=N-4;
  int bit;
  int nextfree;
  if(row==mark1){
    while(free){
      free-=bit=free&(-free);;
      nextfree=~(((ld|bit)<<3)|((rd|bit)>>3)|(col|bit)|1|1<<(N4));
      if(nextfree){
        SQd1B(((ld|bit)<<3)|1,((rd|bit)>>3)|1<<(N4),col|bit,row+3,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
    return;
  }
  while(free){
    free-=bit=free&(-free);;
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQd1BklB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQd1B(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  if(row==endmark){
    (*tempcounter)++;
    return;
  }
  int bit;
  int nextfree;
  int next_ld;
  int next_rd;
  int next_col;
  while(free){
    free-=bit=free&(-free);;
    next_ld=((ld|bit)<<1);
    next_rd=((rd|bit)>>1);
    next_col=(col|bit);
    nextfree=~(next_ld|next_rd|next_col);
    if(nextfree){
      if(row+1<endmark){
        if(~((next_ld<<1)|(next_rd>>1)|(next_col))>0)
          SQd1B(next_ld,next_rd,next_col,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }else{
        SQd1B(next_ld,next_rd,next_col,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
  }
}
__host__ __device__ void SQd1BkBlB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int N3=N-3;
  int bit;
  int nextfree;
  if(row==mark1){
    while(free){
      free-=bit=free&(-free);;
      nextfree=~(((ld|bit)<<2)|((rd|bit)>>2)|(col|bit)|1<<(N3));
      if(nextfree){
        SQd1BlB(((ld|bit)<<2),((rd|bit)>>2)|1<<(N3),col|bit,row+2,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQd1BkBlB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQd1BlB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int bit;
  int nextfree;
  int next_ld;
  int next_rd;
  int next_col;
  if(row==mark2){
    while(free){
      free-=bit=free&(-free);
      next_ld=((ld|bit)<<2)|1;
      next_rd=((rd|bit)>>2);
      next_col=(col|bit);
      nextfree=~(next_ld|next_rd|next_col);
      if(nextfree){
        if(row+2<endmark){
          if(~((next_ld<<1)|(next_rd>>1)|(next_col))>0)
            SQd1B(next_ld,next_rd,next_col,row+2,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
        }else{
          SQd1B(next_ld,next_rd,next_col,row+2,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
        }
      }
    }
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQd1BlB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQd1BlkB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int N3=N-3;
  int bit;
  int nextfree;
  if(row==mark1){
    while(free){
      free-=bit=free&(-free);;
      nextfree=~(((ld|bit)<<3)|((rd|bit)>>3)|(col|bit)|2|1<<(N3));
      if(nextfree){
        SQd1B(((ld|bit)<<3)|2,((rd|bit)>>3)|1<<(N3),col|bit,row+3,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQd1BlkB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQd1BlBkB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int bit;
  int nextfree;
  if(row==mark1){
    while(free){
      free-=bit=free&(-free);
      nextfree=~(((ld|bit)<<2)|((rd|bit)>>2)|(col|bit)|1);
      if(nextfree){
        SQd1BkB(((ld|bit)<<2)|1,(rd|bit)>>2,col|bit,row+2,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQd1BlBkB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQd1BkB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int N3=N-3;
  int bit;
  int nextfree;
  if(row==mark2){
    while(free){
      free-=bit=free&(-free);
      nextfree=~(((ld|bit)<<2)|((rd|bit)>>2)|(col|bit)|1<<(N3));
      if(nextfree){
        SQd1B(((ld|bit)<<2),((rd|bit)>>2)|1<<(N3),col|bit,row+2,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQd1BkB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQd2BlkB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int N3=N-3;
  int bit;
  int nextfree;
  if(row==mark1){
    while(free){
      free-=bit=free&(-free);
      nextfree=~(((ld|bit)<<3)|((rd|bit)>>3)|(col|bit)|1<<(N3)|2);
      if(nextfree){
        SQd2B(((ld|bit)<<3)|2,((rd|bit)>>3)|1<<(N3),col|bit,row+3,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQd2BlkB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQd2BklB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int N4=N-4;
  int bit;
  int nextfree;
  if(row==mark1){
    while(free){
      free-=bit=free&(-free);
      nextfree=~(((ld|bit)<<3)|((rd|bit)>>3)|(col|bit)|1<<(N4)|1);
      if(nextfree){
        SQd2B(((ld|bit)<<3)|1,((rd|bit)>>3)|1<<(N4),col|bit,row+3,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQd2BklB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQd2BkB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int N3=N-3;
  int bit;
  int nextfree;
  if(row==mark2){
    while(free){
      free-=bit=free&(-free);
      nextfree=~(((ld|bit)<<2)|((rd|bit)>>2)|(col|bit)|1<<(N3));
      if(nextfree){
        SQd2B(((ld|bit)<<2),((rd|bit)>>2)|1<<(N3),col|bit,row+2,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQd2BkB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQd2BlBkB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int bit;
  int nextfree;
  if(row==mark1){
    while(free){
      free-=bit=free&(-free);
      nextfree=~(((ld|bit)<<2)|((rd|bit)>>2)|(col|bit)|1);
      if(nextfree){
        SQd2BkB(((ld|bit)<<2)|1,(rd|bit)>>2,col|bit,row+2,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQd2BlBkB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQd2BlB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int bit;
  int nextfree;
  if(row==mark2){
    while(free){
      free-=bit=free&(-free);
      nextfree=~(((ld|bit)<<2)|((rd|bit)>>2)|(col|bit)|1);
      if(nextfree){
        SQd2B(((ld|bit)<<2)|1,(rd|bit)>>2,col|bit,row+2,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQd2BlB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQd2BkBlB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int N3=N-3;
  int bit;
  int nextfree;
  if(row==mark1){
    while(free){
      free-=bit=free&(-free);
      nextfree=~(((ld|bit)<<2)|((rd|bit)>>2)|(col|bit)|(1<<(N3)));
      if(nextfree){
        SQd2BlB(((ld|bit)<<2),((rd|bit)>>2)|(1<<(N3)),col|bit,row+2,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQd2BkBlB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQd2B(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  if(row==endmark){
    if((free&(~1))>0){
      (*tempcounter)++;
    }
    return;
  }
  int bit;
  int nextfree;
  int next_ld;
  int next_rd;
  int next_col;
  while(free){
    free-=bit=free&(-free);
    next_ld=((ld|bit)<<1);
    next_rd=((rd|bit)>>1);
    next_col=(col|bit);
    nextfree=~(next_ld|next_rd|next_col);
    if(nextfree){
      if(row<endmark-1){
        if(~((next_ld<<1)|(next_rd>>1)|(next_col))>0)
          SQd2B(next_ld,next_rd,next_col,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }else{
        SQd2B(next_ld,next_rd,next_col,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
  }
}
__host__ __device__ void SQBlBjrB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int bit;
  int nextfree;
  if(row==mark2){
    while(free){
      free-=bit=free&(-free);
      nextfree=~(((ld|bit)<<2)|((rd|bit)>>2)|(col|bit)|1);
      if(nextfree){
        SQBjrB(((ld|bit)<<2)|1,(rd|bit)>>2,col|bit,row+2,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQBlBjrB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQBkBlBjrB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int N3=N-3;
  int bit;
  int nextfree;
  if(row==mark1){
    while(free){
      free-=bit=free&(-free);
      nextfree=~(((ld|bit)<<2)|((rd|bit)>>2)|(col|bit)|(1<<(N3)));
      if(nextfree){
        SQBlBjrB(((ld|bit)<<2),((rd|bit)>>2)|(1<<(N3)),col|bit,row+2,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQBkBlBjrB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQBjrB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int bit;
  int nextfree;
  if(row==jmark){
    free&=(~1);
    ld|=1;
    while(free){
      free-=bit=free&(-free);
      nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
      if(nextfree){
        SQB(((ld|bit)<<1),(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQBjrB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  if(row==endmark){
    (*tempcounter)++;
    return;
  }
  int bit;
  int nextfree;
  int next_ld;
  int next_rd;
  int next_col;
  while(free){
    free-=bit=free&(-free);
    next_ld=((ld|bit)<<1);
    next_rd=((rd|bit)>>1);
    next_col=(col|bit);
    nextfree=~(next_ld|next_rd|next_col);
    if(nextfree){
      if(row<endmark-1){
        if(~((next_ld<<1)|(next_rd>>1)|(next_col))>0){
          SQB(next_ld,next_rd,next_col,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
        }
      }else{
        SQB(next_ld,next_rd,next_col,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
  }
}
__host__ __device__ void SQBlBkBjrB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int bit;
  int nextfree;
  if(row==mark1){
    while(free){
      free-=bit=free&(-free);
      nextfree=~(((ld|bit)<<2)|((rd|bit)>>2)|(col|bit)|1);
      if(nextfree){
        SQBkBjrB(((ld|bit)<<2)|1,(rd|bit)>>2,col|bit,row+2,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQBlBkBjrB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQBkBjrB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int bit;
  int nextfree;
  int N3=N-3;
  if(row==mark2){
    while(free){
      free-=bit=free&(-free);
      nextfree=~(((ld|bit)<<2)|((rd|bit)>>2)|(col|bit)|1<<(N3));
      if(nextfree){
        SQBjrB(((ld|bit)<<2),((rd|bit)>>2)|1<<(N3),col|bit,row+2,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQBkBjrB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQBklBjrB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int N4=N-4;
  int bit;
  int nextfree;
  if(row==mark1){
    while(free){
      free-=bit=free&(-free);
      nextfree=~(((ld|bit)<<3)|((rd|bit)>>3)|(col|bit)|1<<(N4)|1);
      if(nextfree){
        SQBjrB(((ld|bit)<<3)|1,((rd|bit)>>3)|1<<(N4),col|bit,row+3,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
      }
    }
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQBklBjrB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQBlkBjrB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int N3=N-3;
  int bit;
  int nextfree;
  if(row==mark1){
    while(free){
      free-=bit=free&(-free);
      nextfree=~(((ld|bit)<<3)|((rd|bit)>>3)|(col|bit)|1<<(N3)|2);
      if(nextfree)
        SQBjrB(((ld|bit)<<3)|2,((rd|bit)>>3)|1<<(N3),col|bit,row+3,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQBlkBjrB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQBjlBkBlBjrB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int N1=N-1;
  int bit;
  int nextfree;
  if(row==N1-jmark){
    rd|=1<<(N1);
    free&=~1<<(N1);
    SQBkBlBjrB(ld,rd,col,row,free,jmark,endmark,mark1,mark2,tempcounter,N);
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQBjlBkBlBjrB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQBjlBlBkBjrB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int N1=N-1;
  int bit;
  int nextfree;
  if(row==N1-jmark){
    rd|=1<<(N1);
    free&=~1<<(N1);
    SQBlBkBjrB(ld,rd,col,row,free,jmark,endmark,mark1,mark2,tempcounter,N);
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQBjlBlBkBjrB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
__host__ __device__ void SQBjlBklBjrB(
    int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N
    )
{
  int N1=N-1;
  int bit;
  int nextfree;
  if(row==N1-jmark){
    rd|=1<<(N1);
    free&=~1<<(N1);
    SQBklBjrB(ld,rd,col,row,free,jmark,endmark,mark1,mark2,tempcounter,N);
    return;
  }
  while(free){
    free-=bit=free&(-free);
    nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    if(nextfree){
      SQBjlBklBjrB((ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
/**
  【バックトラック関数の修正】
1. freeをavailに
ループ用にint avail = free;

2. next_ld, next_rd, next_colの新設
それぞれld << 1, rd >> 1, colを入れて毎回再計算しない

3. blockedをnext_ld|next_rd|next_colで算出
クイーンが利く場所をまとめて一発計算

4. next_freeの計算式を簡潔に
next_free = board_mask & ~blocked;

5. 再帰の際はif(next_free){...}で分岐
そのまま次のバックトラック関数を呼び出す

6.「free」は「この行での“配置可能な位置”」を示す変数です。
再帰呼び出し時には「次の行での配置可能位置」＝next_freeを渡します。

7. tempcounter除去・int return化
各バックトラック関数は合計値をintでreturn
execSolutions()側で呼び出しごとにcnt等に加算

8. board_maskをexecSolutions()で事前計算・全バックトラック関数に引数で渡す
int board_mask = (1 << N) - 1;
バックトラック関数の全引数にboard_mask追加


  */
__host__ __device__ 
void SQBjlBlkBjrB(int ld,int rd,int col,int row,int free,int jmark,int endmark,int mark1,int mark2,long* tempcounter,int N)
{
  int N1=N-1;
  int board_mask=(1<<(N1))-1;
  int bit;
  //int nextfree;
  int avail=free;
  if(row==N1-jmark){
    rd|=1<<(N1);
    free&=~1<<(N1);
    int next_ld=ld<<1;
    int next_rd=rd>>1;
    int next_col=col;
    int blocked=next_ld|next_rd|next_col;
    int next_free=board_mask&~blocked;
    // SQBlkBjrB(ld,rd,col,row,free,jmark,endmark,mark1,mark2,tempcounter,N);
    SQBlkBjrB(next_ld,next_rd,next_col,row,next_free,jmark,endmark,mark1,mark2,tempcounter,N);
    return;
  }
  // while(free){
  while(avail){
    bit=avail&-avail;
    avail&=avail-1;
    int next_ld=(ld|bit)<<1;
    int next_rd=(rd|bit)>>1;
    int next_col=col|bit;
    int blocked=next_ld|next_rd|next_col;
    int next_free=board_mask&=blocked;
    //free-=bit=free&(-free);
    //nextfree=~(((ld|bit)<<1)|((rd|bit)>>1)|(col|bit));
    //if(nextfree){
    if(next_free){
      //SQBjlBlkBjrB( (ld|bit)<<1,(rd|bit)>>1,col|bit,row+1,nextfree ,jmark,endmark,mark1,mark2,tempcounter,N);
      SQBjlBlkBjrB( next_ld,next_rd,next_col,row+1,next_free ,jmark,endmark,mark1,mark2,tempcounter,N);
    }
  }
}
/**
 * メインエントリポイント
 *
 * N-Queensの解探索をCPU/GPU両モードでベンチマーク・集計する。
 * コマンドラインで -c:CPU, -g:GPU の切り替え可（デフォルトGPU）
 *
 * - 各サイズ(size)ごとに：
 *   1. unique部分盤面(ijklList)を生成（genConstellations）
 *   2. パディング＆ソート（fillWithTrash）
 *   3. CPUならexecSolutions()で逐次計算、GPUならexecSolutionsKernelで並列集計
 *   4. 合計値を出力
 *   5. リソースを後処理
 *
 * タイミング計測はgettimeofdayで行い、1行で解数・所要時間等を出力。
 * CUDAメモリ確保/解放、CPU/GPU切り替え、コマンドライン引数の柔軟処理など、現場で求められる機能が一通り揃う。
 *
 * @param argc コマンドライン引数数
 * @param argv コマンドライン引数配列
 * @return 終了コード（正常終了で0）
 */
int main(int argc,char** argv)
{
  bool cpu=false,gpu=false;
  int argstart=2;
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){gpu=true;}
    else{ gpu=true; } //デフォルトをgpuとする
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g] n steps\n",argv[0]);
    printf("  -c: CPU\n");
    printf("  -g: GPU\n");
  }
  if(cpu){ printf("\n\nCPU Constellations\n"); }
  else if(gpu){ printf("\n\nGPU Constellations\n");
   if(!InitCUDA()){return 0;}
  }
    int min=4; 
    int targetN=17;
    struct timeval t0;
    struct timeval t1;
    printf("%s\n"," N:        Total      Unique      dd:hh:mm:ss.ms");
    IntHashSet* ijklList;
    ConstellationArrayList* constellations;
    long TOTAL;
    long UNIQUE;
    int ss;
    int ms;
    int dd;
    int hh;
    int mm;
    for(int size=min;size<=targetN;++size){
      ijklList=create_int_hashset();
      constellations=create_constellation_arraylist();
      TOTAL=0;
      UNIQUE=0;
      gettimeofday(&t0,NULL);
      genConstellations(ijklList,constellations,size);
      // ソート
      ConstellationArrayList* fillconstellations = fillWithTrash(constellations, THREAD_NUM);	
      if(cpu){    
    	execSolutions(fillconstellations,size);
    	TOTAL=calcSolutions(fillconstellations,TOTAL);
      }
      if(gpu){
        int steps=24576;
	      int totalSize = fillconstellations->size;
        for (int offset = 0; offset < totalSize; offset += steps) {
      	  int currentSize = fmin(steps, totalSize - offset);
          int gridSize = (currentSize + THREAD_NUM - 1) / THREAD_NUM;  // グリッドサイズ
          unsigned int* hostTotal;
          hipHostMalloc((void**) &hostTotal,sizeof(int)*gridSize, hipHostMallocDefault);
          unsigned int* deviceTotal;
          hipMalloc((void**) &deviceTotal,sizeof(int)*gridSize);

          Constellation* deviceMemory;
          hipMalloc((void**)&deviceMemory, currentSize * sizeof(Constellation));
          // デバイスにコピー
          hipMemcpy(deviceMemory, &fillconstellations->data[offset], currentSize * sizeof(Constellation), hipMemcpyHostToDevice);
          // カーネルを実行
          execSolutionsKernel<<<gridSize, THREAD_NUM>>>(deviceMemory,deviceTotal, size, currentSize);
          // カーネル実行後にデバイスメモリからホストにコピー
          hipMemcpy(hostTotal, deviceTotal, sizeof(int)*gridSize,hipMemcpyDeviceToHost);
          // 取得したsolutionsをホスト側で集計
          // 取得したsolutionsをホスト側で集計
          for (int i = 0; i < gridSize; i++) {
            TOTAL += hostTotal[i];
          }
          //cudaFreeを追加
          hipFree(deviceMemory);
          hipFree(deviceTotal);
          hipHostFree(hostTotal);
        }
     }
     gettimeofday(&t1,NULL);
     if(t1.tv_usec<t0.tv_usec){
       dd=(t1.tv_sec-t0.tv_sec-1)/86400;
       ss=(t1.tv_sec-t0.tv_sec-1)%86400;
       ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
     }else{
       dd=(t1.tv_sec-t0.tv_sec)/86400;
       ss=(t1.tv_sec-t0.tv_sec)%86400;
       ms=(t1.tv_usec-t0.tv_usec+500)/10000;
     }
     hh=ss/3600;
     mm=(ss-hh*3600)/60;
     ss%=60;
     printf("%2d:%17ld%16ld%8.3d:%02d:%02d:%02d.%02d\n",size,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
     // 後処理
     free_int_hashset(ijklList);
     free_constellation_arraylist(constellations);
  } 
  return 0;
}
