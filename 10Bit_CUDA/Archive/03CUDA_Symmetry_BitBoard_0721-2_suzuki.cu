/**
 *
 * bash版対称解除法のC言語版のGPU/CUDA移植版
 *
 詳しい説明はこちらをどうぞ
 https://suzukiiichiro.github.io/search/?keyword=Ｎクイーン問題

非再帰でのコンパイルと実行
$ nvcc -O3 -arch=sm_61 03CUDA_Symmetry_BitBoard.cu && ./a.out -c

再帰でのコンパイルと実行
$ nvcc -O3 -arch=sm_61 03CUDA_Symmetry_BitBoard.cu && ./a.out -r

GPU で並列処理せずに実行
$ nvcc -O3 -arch=sm_61 03CUDA_Symmetry_BitBoard.cu && ./a.out -n

GPU で並列処理で実行（ビットボード）
$ nvcc -O3 -arch=sm_61 03CUDA_Symmetry_BitBoard.cu && ./a.out -n



 *
*/
#include <iostream>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define MAX 27
#define THREAD_NUM 96
using std::cout; using std::endl;
using std::vector; using std::string;
// システムによって以下のマクロが必要であればコメントを外してください。
//#define UINT64_C(c) c ## ULL
//
// グローバル変数
unsigned long TOTAL=0;
unsigned long UNIQUE=0;
//GPU で使うローカル構造体
typedef struct local
{
  unsigned int BOUND1,BOUND2;
  unsigned int TOPBIT,ENDBIT,SIDEMASK,LASTMASK;
  unsigned long board[MAX];
  unsigned long COUNT2,COUNT4,COUNT8,TOTAL,UNIQUE;
  unsigned long TYPE;
}local;
// CPU 再帰/非再帰共通 対称解除法
void symmetryOps(unsigned int size,struct local* l)
{
  /**
  ２．クイーンが右上角以外にある場合、
  (1) 90度回転させてオリジナルと同型になる場合、さらに90度回転(オリジナルか
  ら180度回転)させても、さらに90度回転(オリジナルから270度回転)させてもオリ
  ジナルと同型になる。
  こちらに該当するユニーク解が属するグループの要素数は、左右反転させたパター
  ンを加えて２個しかありません。
  */
  if(l->board[l->BOUND2]==1){
    unsigned int ptn;
    unsigned int own;
    for(ptn=2,own=1;own<size;++own,ptn<<=1){
      unsigned int bit;
      unsigned int you;
      for(bit=1,you=size-1;(l->board[you]!=ptn)&&l->board[own]>=bit;--you){
        bit<<=1;
      }
      if(l->board[own]>bit){
        return ;
      }
      if(l->board[own]<bit){
        break;
      }
    }//end for
    // ９０度回転して同型なら１８０度回転しても２７０度回転しても同型である
    if(own>size-1){
      l->COUNT2++;
      return ;
    }//end if
  }//end if
  /**
  ２．クイーンが右上角以外にある場合、
    (2) 90度回転させてオリジナルと異なる場合は、270度回転させても必ずオリジナル
    とは異なる。ただし、180度回転させた場合はオリジナルと同型になることも有り得
    る。こちらに該当するユニーク解が属するグループの要素数は、180度回転させて同
    型になる場合は４個(左右反転×縦横回転)
   */
  //１８０度回転
  if(l->board[size-1]==l->ENDBIT){
    unsigned int you;
    unsigned int own;
    for(you=size-1-1,own=1;own<=size-1;++own,--you){
      unsigned int bit;
      unsigned int ptn;
      for(bit=1,ptn=l->TOPBIT;(ptn!=l->board[you])&&(l->board[own]>=bit);ptn>>=1){
        bit<<=1;
      }
      if(l->board[own]>bit){
        return ;
      }
      if(l->board[own]<bit){
        break;
      }
    }//end for
    //９０度回転が同型でなくても１８０度回転が同型であることもある
    if(own>size-1){
      l->COUNT4++;
      return ;
    }
  }//end if
  /**
  ２．クイーンが右上角以外にある場合、
    (3)180度回転させてもオリジナルと異なる場合は、８個(左右反転×縦横回転×上下反転)
  */
  //２７０度回転
  if(l->board[l->BOUND1]==l->TOPBIT){
    unsigned int ptn;
    unsigned int own;
    unsigned int you;
    unsigned int bit;
    for(ptn=l->TOPBIT>>1,own=1;own<=size-1;++own,ptn>>=1){
      for(bit=1,you=0;(l->board[you]!=ptn)&&(l->board[own]>=bit);++you){
        bit<<=1;
      }
      if(l->board[own]>bit){
        return ;
      }
      if(l->board[own]<bit){
        break;
      }
    }//end for
  }//end if
  l->COUNT8++;
}
/**
  CPU -c
  */
// 非再帰 角にQがないときのバックトラック
void symmetry_backTrack_NR(unsigned int size,unsigned int row,unsigned int _left,unsigned int _down,unsigned int _right,struct local *l)
{
  unsigned int mask=(1<<size)-1;
  unsigned int down[size];
  unsigned int left[size];
  unsigned int right[size];
  unsigned int bitmap[size];
  left[row]=_left;
  down[row]=_down;
  right[row]=_right;
  bitmap[row]=mask&~(left[row]|down[row]|right[row]);
  while(row>0){
    if(bitmap[row]>0){
      if(row<l->BOUND1){ //上部サイド枝刈り
        bitmap[row]|=l->SIDEMASK;
        bitmap[row]^=l->SIDEMASK;
      }else if(row==l->BOUND2){ //下部サイド枝刈り
        if((down[row]&l->SIDEMASK)==0){
          row--; 
        }
        if((down[row]&l->SIDEMASK)!=l->SIDEMASK){
          bitmap[row]&=l->SIDEMASK;
        }
      }
      unsigned int save_bitmap=bitmap[row];
      unsigned int bit=-bitmap[row]&bitmap[row];
      bitmap[row]^=bit;
      l->board[row]=bit; //Qを配置
      if((bit&mask)!=0){
        if(row==(size-1)){
          if( (save_bitmap&l->LASTMASK)==0){
            symmetryOps(size,l);  //対称解除法
          }
          row--;
        }else{
          unsigned int n=row++;
          left[row]=(left[n]|bit)<<1;
          down[row]=(down[n]|bit);
          right[row]=(right[n]|bit)>>1;
          bitmap[row]=mask&~(left[row]|down[row]|right[row]);
        }
      }else{
        row--;
      }
    }else{
      row--;
    }
  }//end while
}
// 非再帰 角にQがあるときのバックトラック
void symmetry_backTrack_corner_NR(unsigned int size,unsigned int row,unsigned int _left,unsigned int _down,unsigned int _right,struct local *l)
{
  unsigned int mask=(1<<size)-1;
  unsigned int bit=0;
  unsigned int down[size];
  unsigned int left[size];
  unsigned int right[size];
  unsigned int bitmap[size];
  left[row]=_left;
  down[row]=_down;
  right[row]=_right;
  bitmap[row]=mask&~(left[row]|down[row]|right[row]);
  while(row>=2){
    if(row<l->BOUND1){
      // bitmap[row]=bitmap[row]|2;
      // bitmap[row]=bitmap[row]^2;
      bitmap[row]&=~2;
    }
    if(bitmap[row]>0){
      bit=-bitmap[row]&bitmap[row];
      bitmap[row]^=bit;
      if(row==(size-1)){
        l->COUNT8++;
        row--;
      }else{
        unsigned int n=row++;
        left[row]=(left[n]|bit)<<1;
        down[row]=(down[n]|bit);
        right[row]=(right[n]|bit)>>1;
        l->board[row]=bit; //Qを配置
        //クイーンが配置可能な位置を表す
        bitmap[row]=mask&~(left[row]|down[row]|right[row]);
      }
    }else{
      row--;
    }
  }//end while
}
// 非再帰 対称解除法
void symmetry_NR(unsigned int size,struct local* l)
{
  l->TOTAL=l->UNIQUE=l->COUNT2=l->COUNT4=l->COUNT8=0;
  unsigned int bit=0;
  l->TOPBIT=1<<(size-1);
  l->ENDBIT=l->SIDEMASK=l->LASTMASK=0;
  l->BOUND1=2;
  l->BOUND2=0;
  l->board[0]=1;
  while(l->BOUND1>1&&l->BOUND1<size-1){
    if(l->BOUND1<size-1){
      bit=1<<l->BOUND1;
      l->board[1]=bit;   //２行目にQを配置
      //角にQがあるときのバックトラック
      symmetry_backTrack_corner_NR(size,2,(2|bit)<<1,1|bit,(2|bit)>>1,l);
    }
    l->BOUND1++;
  }
  l->TOPBIT=1<<(size-1);
  l->ENDBIT=l->TOPBIT>>1;
  l->SIDEMASK=l->TOPBIT|1;
  l->LASTMASK=l->TOPBIT|1;
  l->BOUND1=1;
  l->BOUND2=size-2;
  while(l->BOUND1>0 && l->BOUND2<size-1 && l->BOUND1<l->BOUND2){
    if(l->BOUND1<l->BOUND2){
      bit=1<<l->BOUND1;
      l->board[0]=bit;   //Qを配置
      //角にQがないときのバックトラック
      symmetry_backTrack_NR(size,1,bit<<1,bit,bit>>1,l);
    }
    l->BOUND1++;
    l->BOUND2--;
    l->ENDBIT=l->ENDBIT>>1;
    l->LASTMASK=l->LASTMASK<<1|l->LASTMASK|l->LASTMASK>>1;
  }//ene while
  UNIQUE=l->COUNT2+l->COUNT4+l->COUNT8;
  TOTAL=l->COUNT2*2+l->COUNT4*4+l->COUNT8*8;
}
/**
  CPU -r
  */
// 再帰 角にQがないときのバックトラック
void symmetry_backTrack(unsigned int size,unsigned int row,unsigned int left,unsigned int down,unsigned int right,struct local* l)
{
  unsigned int mask=(1<<size)-1;
  unsigned int bitmap=mask&~(left|down|right);
  if(row==(size-1)){
    if(bitmap){
      if( (bitmap&l->LASTMASK)==0){
        l->board[row]=bitmap;  //Qを配置
        symmetryOps(size,l);    //対称解除
      }
    }
  }else{
    if(row<l->BOUND1){
      bitmap=bitmap|l->SIDEMASK;
      bitmap=bitmap^l->SIDEMASK;
    }else{
      if(row==l->BOUND2){
        if((down&l->SIDEMASK)==0){
          return;
        }
        if( (down&l->SIDEMASK)!=l->SIDEMASK){
          bitmap=bitmap&l->SIDEMASK;
        }
      }
    }
    while(bitmap){
      unsigned int bit=-bitmap&bitmap;
      bitmap=bitmap^bit;
      l->board[row]=bit;
      symmetry_backTrack(size,row+1,(left|bit)<<1,down|bit,(right|bit)>>1,l);
    }
  }
}
// 再帰 角にQがあるときのバックトラック
void symmetry_backTrack_corner(unsigned int size,unsigned int row,unsigned int left,unsigned int down,unsigned int right,struct local* l)
{
  unsigned int mask=(1<<size)-1;
  unsigned int bitmap=mask&~(left|down|right);
  unsigned int bit=0;
  if(row==(size-1)){
    if(bitmap){
      l->board[row]=bitmap;
      l->COUNT8++;
    }
  }else{
    if(row<l->BOUND1){   //枝刈り
      bitmap=bitmap|2;
      bitmap=bitmap^2;
    }
    while(bitmap){
      bit=-bitmap&bitmap;
      bitmap=bitmap^bit;
      l->board[row]=bit;   //Qを配置
      symmetry_backTrack_corner(size,row+1,(left|bit)<<1,down|bit,(right|bit)>>1,l);
    }
  }
}
// 再帰 対称解除法
void symmetry_R(unsigned int size,struct local* l)
{
  l->TOTAL=l->UNIQUE=l->COUNT2=l->COUNT4=l->COUNT8=0;
  unsigned int bit=0;
  l->TOPBIT=1<<(size-1);
  l->ENDBIT=l->LASTMASK=l->SIDEMASK=0;
  l->BOUND1=2;
  l->BOUND2=0;
  l->board[0]=1;
  while(l->BOUND1>1 && l->BOUND1<size-1){
    if(l->BOUND1<size-1){
      bit=1<<l->BOUND1;
      l->board[1]=bit;   //２行目にQを配置
      //角にQがあるときのバックトラック
      symmetry_backTrack_corner(size,2,(2|bit)<<1,1|bit,(2|bit)>>1,l);
    }
    l->BOUND1++;
  }//end while
  l->TOPBIT=1<<(size-1);
  l->ENDBIT=l->TOPBIT>>1;
  l->SIDEMASK=l->TOPBIT|1;
  l->LASTMASK=l->TOPBIT|1;
  l->BOUND1=1;
  l->BOUND2=size-2;
  while(l->BOUND1>0 && l->BOUND2<size-1 && l->BOUND1<l->BOUND2){
    if(l->BOUND1<l->BOUND2){
      bit=1<<l->BOUND1;
      l->board[0]=bit;   //Qを配置
      //角にQがないときのバックトラック
      symmetry_backTrack(size,1,bit<<1,bit,bit>>1,l);
    }
    l->BOUND1++;
    l->BOUND2--;
    l->ENDBIT=l->ENDBIT>>1;
    l->LASTMASK=l->LASTMASK<<1|l->LASTMASK|l->LASTMASK>>1;
  }//ene while
  UNIQUE=l->COUNT2+l->COUNT4+l->COUNT8;
  TOTAL=l->COUNT2*2+l->COUNT4*4+l->COUNT8*8;
}
/**
  GPU -g
  */
__device__
struct dlocal
{
  unsigned int BOUND1,BOUND2;
  unsigned int TOPBIT,ENDBIT,SIDEMASK,LASTMASK;
  unsigned long board[MAX];
  unsigned long COUNT2,COUNT4,COUNT8,TOTAL,UNIQUE;
  unsigned long TYPE;
}dlocal;
__device__ struct dlocal gdl[9999];
// GPU 対称解除法
__host__ __device__
long GPU_symmetryOps(unsigned int size,struct dlocal* l)
{
  /**
  ２．クイーンが右上角以外にある場合、
  (1) 90度回転させてオリジナルと同型になる場合、さらに90度回転(オリジナルか
  ら180度回転)させても、さらに90度回転(オリジナルから270度回転)させてもオリ
  ジナルと同型になる。
  こちらに該当するユニーク解が属するグループの要素数は、左右反転させたパター
  ンを加えて２個しかありません。
  */
  if(l->board[l->BOUND2]==1){
    unsigned int ptn;
    unsigned int own;
    for(ptn=2,own=1;own<size;++own,ptn<<=1){
      unsigned int bit;
      unsigned int you;
      for(bit=1,you=size-1;(l->board[you]!=ptn)&& l->board[own]>=bit;--you){
        bit<<=1;
      }
      if(l->board[own]>bit){
        return 0;
      }
      if(l->board[own]<bit){
        break;
      }
    }//end for
    // ９０度回転して同型なら１８０度回転しても２７０度回転しても同型である
    if(own>size-1){
      l->COUNT2++;
      return 2;
    }//end if
  }//end if
  /**
  ２．クイーンが右上角以外にある場合、
    (2) 90度回転させてオリジナルと異なる場合は、270度回転させても必ずオリジナル
    とは異なる。ただし、180度回転させた場合はオリジナルと同型になることも有り得
    る。こちらに該当するユニーク解が属するグループの要素数は、180度回転させて同
    型になる場合は４個(左右反転×縦横回転)
   */
  //１８０度回転
  if(l->board[size-1]==l->ENDBIT){
    unsigned int you;
    unsigned int own;
    for(you=size-1-1,own=1;own<=size-1;++own,--you){
      unsigned int bit;
      unsigned int ptn;
      for(bit=1,ptn=l->TOPBIT;(ptn!=l->board[you])&&(l->board[own]>=bit);ptn>>=1){
        bit<<=1;
      }
      if(l->board[own]>bit){
        return 0;
      }
      if(l->board[own]<bit){
        break;
      }
    }//end for
    //９０度回転が同型でなくても１８０度回転が同型であることもある
    if(own>size-1){
      l->COUNT4++;
      return 4;
    }
  }//end if
  /**
  ２．クイーンが右上角以外にある場合、
    (3)180度回転させてもオリジナルと異なる場合は、８個(左右反転×縦横回転×上下反転)
  */
  //２７０度回転
  if(l->board[l->BOUND1]==l->TOPBIT){
    unsigned int ptn;
    unsigned int own;
    unsigned int you;
    unsigned int bit;
    for(ptn=l->TOPBIT>>1,own=1;own<=size-1;++own,ptn>>=1){
      for(bit=1,you=0;(l->board[you]!=ptn)&&(l->board[own]>=bit);++you){
        bit<<=1;
      }
      if(l->board[own]>bit){
        return 0;
      }
      if(l->board[own]<bit){
        break;
      }
    }//end for
  }//end if
  l->COUNT8++;
  return 8;
}
// GPU 角にQがないときのバックトラック
__host__ __device__
long GPU_symmetry_backTrack(unsigned int size,unsigned int row,unsigned int left,unsigned int down,unsigned int right,struct dlocal* l)
{
  unsigned long counter=0;
  unsigned int mask=(1<<size)-1;
  unsigned int bitmap=mask&~(left|down|right);
  if(row==(size-1)){
    if(bitmap){
      if( (bitmap& l->LASTMASK)==0){
        l->board[row]=bitmap;  //Qを配置
        counter+=GPU_symmetryOps(size,l);    //対称解除
      }
    }
  }else{
    if(row<l->BOUND1){
      bitmap=bitmap|l->SIDEMASK;
      bitmap=bitmap^l->SIDEMASK;
    }else{
      if(row==l->BOUND2){
        if((down&l->SIDEMASK)==0){
          return 0;
        }
        if( (down&l->SIDEMASK)!=l->SIDEMASK){
          bitmap=bitmap&l->SIDEMASK;
        }
      }
    }
    while(bitmap){
      unsigned int bit=-bitmap&bitmap;
      bitmap=bitmap^bit;
      l->board[row]=bit;
      counter+=GPU_symmetry_backTrack(size,row+1,(left|bit)<<1,down|bit,(right|bit)>>1,l);
    }
  }
  return counter;
}
// GPU 角にQがあるときのバックトラック
__host__ __device__
long GPU_symmetry_backTrack_corner(unsigned int size,unsigned int row,unsigned int left,unsigned int down,unsigned int right,struct dlocal* l)
{
  unsigned long counter=0;
  unsigned int mask=(1<<size)-1;
  unsigned int bitmap=mask&~(left|down|right);
  unsigned int bit=0;
  if(row==(size-1)){
    if(bitmap){
      l->board[row]=bitmap;
      l->COUNT8++;
      counter+=8;
    }
  }else{
    if(row<l->BOUND1){   //枝刈り
      bitmap=bitmap|2;
      bitmap=bitmap^2;
    }
    while(bitmap){
      bit=-bitmap&bitmap;
      bitmap=bitmap^bit;
      l->board[row]=bit;   //Qを配置
      counter+=GPU_symmetry_backTrack_corner(size,row+1,(left|bit)<<1,down|bit,(right|bit)>>1,l);
    }
  }
  return counter;
}
// GPU 対称解除法 -g の実行時のみ呼び出されます
__host__ __device__
void GPU_symmetry_R(unsigned int size,struct local* hostLocal)
{
  // GPU内部で使うための dlocal構造体
  struct dlocal l;
  l.TOTAL=l.UNIQUE=l.COUNT2=l.COUNT4=l.COUNT8=0;
  unsigned int bit=0;
  l.TOPBIT=1<<(size-1);
  l.ENDBIT=l.LASTMASK=l.SIDEMASK=0;
  l.BOUND1=2;
  l.BOUND2=0;
  l.board[0]=1;
  while(l.BOUND1>1 && l.BOUND1<size-1){
    if(l.BOUND1<size-1){
      bit=1<<l.BOUND1;
      l.board[1]=bit;   //２行目にQを配置
      //角にQがあるときのバックトラック
      GPU_symmetry_backTrack_corner(size,2,(2|bit)<<1,1|bit,(2|bit)>>1,&l);
    }
    l.BOUND1++;
  }//end while
  l.TOPBIT=1<<(size-1);
  l.ENDBIT=l.TOPBIT>>1;
  l.SIDEMASK=l.TOPBIT|1;
  l.LASTMASK=l.TOPBIT|1;
  l.BOUND1=1;
  l.BOUND2=size-2;
  while(l.BOUND1>0 && l.BOUND2<size-1 && l.BOUND1<l.BOUND2){
    if(l.BOUND1<l.BOUND2){
      bit=1<<l.BOUND1;
      l.board[0]=bit;   //Qを配置
      //角にQがないときのバックトラック
      GPU_symmetry_backTrack(size,1,bit<<1,bit,bit>>1,&l);
    }
    l.BOUND1++;
    l.BOUND2--;
    l.ENDBIT=l.ENDBIT>>1;
    l.LASTMASK=l.LASTMASK<<1|l.LASTMASK|l.LASTMASK>>1;
  }//ene while
  // 集計値は hostLocalへ代入
  hostLocal->UNIQUE=l.COUNT2+l.COUNT4+l.COUNT8;
  hostLocal->TOTAL=l.COUNT2*2+l.COUNT4*4+l.COUNT8*8;
}
/**
  CUDA13
  */
// GPU -n 対称解除法
__device__ 
int BitBoard_symmetryOps(int si,unsigned int *d_aBoard,int BOUND1,int BOUND2,int TOPBIT,int ENDBIT)
{
  int own,ptn,you,bit;
  //90度回転
  if(d_aBoard[BOUND2]==1){ own=1; ptn=2;
    while(own<=si-1){ bit=1; you=si-1;
      while((d_aBoard[you]!=ptn)&&(d_aBoard[own]>=bit)){ bit<<=1; you--; }
      if(d_aBoard[own]>bit){ return 0; } else if(d_aBoard[own]<bit){ break; }
      own++; ptn<<=1;
    }
    /** 90度回転して同型なら180度/270度回転も同型である */
    if(own>si-1){ return 2; }
  }
  //180度回転
  if(d_aBoard[si-1]==ENDBIT){ own=1; you=si-1-1;
    while(own<=si-1){ bit=1; ptn=TOPBIT;
      while((d_aBoard[you]!=ptn)&&(d_aBoard[own]>=bit)){ bit<<=1; ptn>>=1; }
      if(d_aBoard[own]>bit){ return 0; } else if(d_aBoard[own]<bit){ break; }
      own++; you--;
    }
    /** 90度回転が同型でなくても180度回転が同型である事もある */
    if(own>si-1){ return 4; }
  }
  //270度回転
  if(d_aBoard[BOUND1]==TOPBIT){ own=1; ptn=TOPBIT>>1;
    while(own<=si-1){ bit=1; you=0;
      while((d_aBoard[you]!=ptn)&&(d_aBoard[own]>=bit)){ bit<<=1; you++; }
      if(d_aBoard[own]>bit){ return 0; } else if(d_aBoard[own]<bit){ break; }
      own++; ptn>>=1;
    }
  }
  return 8; 
}
// GPU -n Ｑが角にある場合のバックトラック内の再帰処理をカーネルで行う
__global__
void BitBoard_cuda_kernel_b1(int size,int mark,unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,unsigned int* d_results,unsigned int* d_uniq,long totalCond,int h_row,int B1)
{
  const unsigned int mask=(1<<size)-1;
  unsigned long total=0;
  unsigned int unique=0;
  int row=0;
  unsigned int bit;
  //
  //スレッド
  //
  //ブロック内のスレッドID
  unsigned const int tid=threadIdx.x;
  //グリッド内のブロックID
  unsigned const int bid=blockIdx.x;
  //全体通してのID
  unsigned const int idx=bid*blockDim.x+tid;
  //
  //シェアードメモリ
  //
  //sharedメモリを使う ブロック内スレッドで共有
  //10固定なのは現在のmask設定で
  //GPUで実行するのは最大10だから
  //THREAD_NUMはブロックあたりのスレッド数
  __shared__ unsigned int down[THREAD_NUM][10];
  down[tid][row]=totalDown[idx];
  __shared__ unsigned int left[THREAD_NUM][10];
  left[tid][row]=totalLeft[idx];
  __shared__ unsigned int right[THREAD_NUM][10];
  right[tid][row]=totalRight[idx];
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  //down,left,rightからbitmapを出す
  bitmap[tid][row]
    =mask&~(
        down[tid][row]
        |left[tid][row]
        |right[tid][row]);
  __shared__ unsigned int sum[THREAD_NUM];
  /***11 backTrack1ではaBoard不要 *********************/
  //unsigned int c_aBoard[MAX];
  __shared__ unsigned int usum[THREAD_NUM];
  //余分なスレッドは動かさない 
  //GPUはsteps数起動するがtotalCond以上は空回しする
  if(idx<totalCond){
    //totalDown,totalLeft,totalRightの情報を
    //down,left,rightに詰め直す 
    //CPU で詰め込んだ t_はsteps個あるが
    //ブロック内ではブロックあたりのスレッド数に限定
    //されるので idxでよい
    //
    /***11 backTrack1ではaBoard不要*********************/
    //for(int i=0;i<h_row;i++){
    //  c_aBoard[i]=t_aBoard[idx*h_row+i]; //２次元配列だが1次元的に利用  
    //}
    unsigned int bitmap_tid_row;
    unsigned int down_tid_row;
    unsigned int left_tid_row;
    unsigned int right_tid_row;
    while(row>=0){
      bitmap_tid_row=bitmap[tid][row];
      down_tid_row=down[tid][row];
      left_tid_row=left[tid][row];
      right_tid_row=right[tid][row];
      if(bitmap_tid_row==0){
        row--;
      }else{
        /**11 枝刈り**********/
        if(row+h_row<B1) {
          bitmap_tid_row=bitmap[tid][row]&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
        }  
        //クイーンを置く
        //置く場所があるかどうか
        bitmap[tid][row]
          /***11 backTrack1ではaBoard不要のためコメント*********************/
          //^=c_aBoard[row+h_row]
          //=bit
          ^=bit
          =(-bitmap_tid_row&bitmap_tid_row);       
        if((bit&mask)!=0){
          //最終行?最終行から１個前の行まで
          //無事到達したら 加算する
          if(row+1==mark){
            /**11 backTradk1ではsymmetryOps不要のためコメント*********************/
            //int s=symmetryOps(size,c_aBoard); 
            //if(s!=0){
            //print(size); //print()でTOTALを++しない
            //ホストに戻す配列にTOTALを入れる
            //スレッドが１つの場合は配列は１個
            unique++; 
            total+=8;   //対称解除で得られた解数を加算
            //}
            row--;
          }else{
            int rowP=row+1;
            down[tid][rowP]=down_tid_row|bit;
            left[tid][rowP]=(left_tid_row|bit)<<1;
            right[tid][rowP]=(right_tid_row|bit)>>1;
            bitmap[tid][rowP]
              =mask&~(
                  down[tid][rowP]
                  |left[tid][rowP]
                  |right[tid][rowP]);
            row++;
          }
        }else{
          //置く場所がなければ１個上に
          row--;
        }
      }
    }
    //最後sum[tid]に加算する
    sum[tid]=total;
    usum[tid]=unique;
  }else{
    //totalCond未満は空回しするのでtotalは加算しない
    sum[tid]=0;
    usum[tid]=0;
  } 
  //__syncthreads()でブロック内のスレッド間の同期
  //全てのスレッドが__syncthreads()に辿り着くのを待つ
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){
    sum[tid]+=sum[tid+64];
    usum[tid]+=usum[tid+64];
  }
  __syncwarp();if(tid<32){
    sum[tid]+=sum[tid+32];
    usum[tid]+=usum[tid+32];
  } 
  __syncwarp();if(tid<16){
    sum[tid]+=sum[tid+16];
    usum[tid]+=usum[tid+16];
  } 
  __syncwarp();if(tid<8){
    sum[tid]+=sum[tid+8];
    usum[tid]+=usum[tid+8];
  } 
  __syncwarp();if(tid<4){
    sum[tid]+=sum[tid+4];
    usum[tid]+=usum[tid+4];
  } 
  __syncwarp();if(tid<2){
    sum[tid]+=sum[tid+2];
    usum[tid]+=usum[tid+2];
  } 
  __syncwarp();if(tid<1){
    sum[tid]+=sum[tid+1];
    usum[tid]+=usum[tid+1];
  } 
  __syncwarp();if(tid==0){
    d_results[bid]=sum[0];
    d_uniq[bid]=usum[0];
  }
}
// GPU -n Ｑが角にない場合のバックトラック内の再帰処理をカーネルで行う
__global__
void BitBoard_cuda_kernel_b2(int size,int mark,unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,unsigned int* d_results,unsigned int* d_uniq,long totalCond,unsigned int* t_aBoard,int h_row,int B1,int B2,int SM,int LM,int TB,int EB)
{
  const unsigned int mask=(1<<size)-1;
  unsigned long total=0;
  unsigned int unique=0;
  int row=0;
  unsigned int bit;
  //
  //スレッド
  //
  //ブロック内のスレッドID
  unsigned const int tid=threadIdx.x;
  //グリッド内のブロックID
  unsigned const int bid=blockIdx.x;
  //全体通してのID
  unsigned const int idx=bid*blockDim.x+tid;
  //
  //シェアードメモリ
  //
  //sharedメモリを使う ブロック内スレッドで共有
  //10固定なのは現在のmask設定で
  //GPUで実行するのは最大10だから
  //THREAD_NUMはブロックあたりのスレッド数
  __shared__ unsigned int down[THREAD_NUM][10];
  down[tid][row]=totalDown[idx];
  __shared__ unsigned int left[THREAD_NUM][10];
  left[tid][row]=totalLeft[idx];
  __shared__ unsigned int right[THREAD_NUM][10];
  right[tid][row]=totalRight[idx];
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  //down,left,rightからbitmapを出す
  bitmap[tid][row]
    =mask&~(
        down[tid][row]
        |left[tid][row]
        |right[tid][row]);
  __shared__ unsigned int sum[THREAD_NUM];
  unsigned int c_aBoard[MAX];
  __shared__ unsigned int usum[THREAD_NUM];
  //余分なスレッドは動かさない 
  //GPUはsteps数起動するがtotalCond以上は空回しする
  if(idx<totalCond){
    //totalDown,totalLeft,totalRightの情報を
    //down,left,rightに詰め直す 
    //CPU で詰め込んだ t_はsteps個あるが
    //ブロック内ではブロックあたりのスレッド数に限定
    //されるので idxでよい
    //
    for(int i=0;i<h_row;i++){
      c_aBoard[i]=t_aBoard[idx*h_row+i]; //２次元配列だが1次元的に利用  
    }
    unsigned int bitmap_tid_row;
    unsigned int down_tid_row;
    unsigned int left_tid_row;
    unsigned int right_tid_row;
    while(row>=0){
      bitmap_tid_row=bitmap[tid][row];
      down_tid_row=down[tid][row];
      left_tid_row=left[tid][row];
      right_tid_row=right[tid][row];
      //
      //bitmap[tid][row]=00000000 クイーンを
      //どこにも置けないので1行上に戻る
      if(bitmap_tid_row==0){
        row--;
      }else{
        /**11 枝刈り追加**********/
        //【枝刈り】上部サイド枝刈り
        if(row+h_row<B1){             	
          //printf("BOUND1_row:%d:h_row:%d:row+hrow:%d:bit:%d\n",row,h_row,row+h_row,bitmap[tid][row]);
          bitmap_tid_row=bitmap[tid][row]&=~SM;
          //【枝刈り】下部サイド枝刈り
        }else if(row+h_row==B2) {     	
          //printf("BOUND2_row:%d:h_row:%d:row+hrow:%d:bit:%d\n",row,h_row,row+h_row,bitmap[tid][row]);
          if((down_tid_row&SM)==0){ 
            row--; 
            continue;
            //printf("BOUND2_row\n");
          }
          if((down_tid_row&SM)!=SM){ 
            bitmap_tid_row=bitmap[tid][row]&=SM; 
            //printf("BOUND2_SIDEMASK\n");            
          }
        }
        int save_bitmap=bitmap[tid][row];
        //クイーンを置く
        //置く場所があるかどうか
        bitmap[tid][row]
          ^=c_aBoard[row+h_row]
          =bit
          =(-bitmap_tid_row&bitmap_tid_row);       
        if((bit&mask)!=0){
          //最終行?最終行から１個前の行まで
          //無事到達したら 加算する
          if(row+1==mark){
            /***11 LASTMASK枝刈り*********************/ 
            if((save_bitmap&LM)==0){ 
              /***12 symmetryOps 省力化のためBOUND1,BOUND2,TOPBIT,ENDBITを渡す*****/ 
              int s=BitBoard_symmetryOps(size,c_aBoard,B1,B2,TB,EB); 
              if(s!=0){
                //print(size); //print()でTOTALを++しない
                //ホストに戻す配列にTOTALを入れる
                //スレッドが１つの場合は配列は１個
                unique++; 
                total+=s;   //対称解除で得られた解数を加算
              }
              row--;
            }
          }else{
            int rowP=row+1;
            down[tid][rowP]=down_tid_row|bit;
            left[tid][rowP]=(left_tid_row|bit)<<1;
            right[tid][rowP]=(right_tid_row|bit)>>1;
            bitmap[tid][rowP]
              =mask&~(
                  down[tid][rowP]
                  |left[tid][rowP]
                  |right[tid][rowP]);
            row++;
          }
        }else{
          //置く場所がなければ１個上に
          row--;
        }
      }
    }
    //最後sum[tid]に加算する
    sum[tid]=total;
    usum[tid]=unique;
  }else{
    //totalCond未満は空回しするのでtotalは加算しない
    sum[tid]=0;
    usum[tid]=0;
  } 
  //__syncthreads()でブロック内のスレッド間の同期
  //全てのスレッドが__syncthreads()に辿り着くのを待つ
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){
    sum[tid]+=sum[tid+64];
    usum[tid]+=usum[tid+64];
  }
  __syncwarp();if(tid<32){
    sum[tid]+=sum[tid+32];
    usum[tid]+=usum[tid+32];
  } 
  __syncwarp();if(tid<16){
    sum[tid]+=sum[tid+16];
    usum[tid]+=usum[tid+16];
  } 
  __syncwarp();if(tid<8){
    sum[tid]+=sum[tid+8];
    usum[tid]+=usum[tid+8];
  } 
  __syncwarp();if(tid<4){
    sum[tid]+=sum[tid+4];
    usum[tid]+=usum[tid+4];
  } 
  __syncwarp();if(tid<2){
    sum[tid]+=sum[tid+2];
    usum[tid]+=usum[tid+2];
  } 
  __syncwarp();if(tid<1){
    sum[tid]+=sum[tid+1];
    usum[tid]+=usum[tid+1];
  } 
  __syncwarp();if(tid==0){
    d_results[bid]=sum[0];
    d_uniq[bid]=usum[0];
  }
}
// GPU -n Ｑが角にない
long BitBoard_backTrack2G(int size,int mask,int row,int n_left,int n_down,int n_right,int steps,int BOUND1,int BOUND2,int SIDEMASK,int LASTMASK,int TOPBIT,int ENDBIT,unsigned int* aBoard)
{
  //何行目からGPUで行くか。ここの設定は変更可能、設定値を多くするほどGPUで並行して動く
  /***11 size<8の時はmarkが2*********************/
  unsigned int mark=size>12?size-10:3;
  //unsigned int mark=size>11?size-9:3;
  if(size<8){ mark=2; }
  const unsigned int h_mark=row;
  long total=0;
  long totalCond=0;
  bool matched=false;
  //host
  unsigned int down[32];  down[row]=n_down;
  unsigned int right[32]; right[row]=n_right;
  unsigned int left[32];  left[row]=n_left;
  //bitmapを配列で持つことにより
  //stackを使わないで1行前に戻れる
  unsigned int bitmap[32];
  bitmap[row]=mask&~(left[row]|down[row]|right[row]);
  unsigned int bit;
  unsigned int* totalDown;
  hipHostMalloc((void**) &totalDown,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* totalLeft;
  hipHostMalloc((void**) &totalLeft,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* totalRight;
  hipHostMalloc((void**) &totalRight,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* h_results;
  hipHostMalloc((void**) &h_results,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* h_uniq;
  hipHostMalloc((void**) &h_uniq,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* t_aBoard;
  hipHostMalloc((void**) &t_aBoard,sizeof(int)*steps*mark, hipHostMallocDefault);
  //device
  unsigned int* downCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  unsigned int* leftCuda;
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  unsigned int* rightCuda;
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  unsigned int* resultsCuda;
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);
  unsigned int* d_uniq;
  hipMalloc((void**) &d_uniq,sizeof(int)*steps/THREAD_NUM);
  unsigned int* d_aBoard;
  hipMalloc((void**) &d_aBoard,sizeof(int)*steps*mark);
  //12行目までは3行目までCPU->row==mark以下で 3行目までの
  //down,left,right情報を totalDown,totalLeft,totalRight
  //に格納
  //する->3行目以降をGPUマルチスレッドで実行し結果を取得
  //13行目以降はCPUで実行する行数が１個ずつ増えて行く
  //例えばn15だとrow=5までCPUで実行し、
  //それ以降はGPU(現在の設定だとGPUでは最大10行実行する
  //ようになっている)
  int rowP=0;
  while(row>=h_mark) {
    //bitmap[row]=00000000 クイーンを
    //どこにも置けないので1行上に戻る
    //06GPU こっちのほうが優秀
    if(bitmap[row]==0){ row--; }
    else{//おける場所があれば進む
      /***11 枝刈り追加*********************/
      //【枝刈り】上部サイド枝刈り
      if(row<BOUND1){             	
        bitmap[row]&=~SIDEMASK;
        //【枝刈り】下部サイド枝刈り
      }else if(row==BOUND2) {     	
        if((down[row]&SIDEMASK)==0){ row--; }
        if((down[row]&SIDEMASK)!=SIDEMASK){ bitmap[row]&=SIDEMASK; }
      }
      //06SGPU
      bitmap[row]^=aBoard[row]=bit=(-bitmap[row]&bitmap[row]);
      if((bit&mask)!=0){//置く場所があれば先に進む
        rowP=row+1;
        down[rowP]=down[row]|bit;
        left[rowP]=(left[row]|bit)<<1;
        right[rowP]=(right[row]|bit)>>1;
        bitmap[rowP]=mask&~(down[rowP]|left[rowP]|right[rowP]);
        row++;
        if(row==mark){
          //3行目(mark)にクイーンを１個ずつ置いていって、
          //down,left,right情報を格納、
          //その次の行へは進まない。その行で可能な場所にクイー
          //ン置き終わったらGPU並列実行
          //totalCond がthreadIdになる 各スレッドに down,left,right情報を渡す
          //row=2(13行目以降は増えていく。例えばn15だとrow=5)の情報を
          //totalDown,totalLeft,totalRightに格納する
          totalDown[totalCond]=down[row];
          totalLeft[totalCond]=left[row];
          totalRight[totalCond]=right[row];
          for(int i=0;i<mark;i++){
            t_aBoard[totalCond*mark+i]=aBoard[i];
          }
          //スレッド数をインクリメントする
          totalCond++;
          //最大GPU数に達してしまったら一旦ここでGPUを実行する。stepsはGPUの同
          //時並行稼働数を制御
          //nの数が少ないうちはtotalCondがstepsを超えることはないがnの数が増え
          //て行くと超えるようになる。
          //ここではtotalCond==stepsの場合だけこの中へ         
          if(totalCond==steps){
            //matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUか
            //ら出たらmatched=trueになってる
            if(matched){
              hipMemcpy(h_results,resultsCuda,
                  sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
              hipMemcpy(h_uniq,d_uniq,
                  sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
              for(int col=0;col<steps/THREAD_NUM;col++){
                total+=h_results[col];
                UNIQUE+=h_uniq[col];
              }
              matched=false;
            }
            hipMemcpy(downCuda,totalDown,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(leftCuda,totalLeft,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(rightCuda,totalRight,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(d_aBoard,t_aBoard,
                sizeof(int)*totalCond*mark,hipMemcpyHostToDevice);
            /***12 TOPBIT,ENDBIT追加*********************/
            //cuda_kernel_b2<<<steps/THREAD_NUM,THREAD_NUM
            //  >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,d_aBoard,row,BOUND1,BOUND2,SIDEMASK,LASTMASK);
            BitBoard_cuda_kernel_b2<<<steps/THREAD_NUM,THREAD_NUM
              >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,d_aBoard,row,BOUND1,BOUND2,SIDEMASK,LASTMASK,TOPBIT,ENDBIT);
            //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われ
            //るのはtotalCondの数だけでそれ以外は空回しになる
            //GPU内でカウントしているので、GPUから出たらmatched=trueになってる
            matched=true;
            //totalCond==stepsルートでGPUを実行したらスレッドをまた0から開始す
            //る(これによりなんどもsteps数分だけGPUを起動できる)
            totalCond=0;           
          }
          //totalDown,totalLeft,totalRightに情報を格納したら1行上に上がる
          //これを繰り返すことにより row=2で可能な場所全てにクイーンを置いて
          //totalDown,totalLeft,totalRightに情報を格納する
          row--;
        }
      }else{
        //置く場所がなければ上に上がる。row==mark行に達するまではCPU側で普通に
        //nqueenをやる
        row--;
      }
    }
  }
  //matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUから出たら
  //matched=trueになってる
  if(matched){
    hipMemcpy(h_results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    hipMemcpy(h_uniq,d_uniq,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){
      total+=h_results[col];
      UNIQUE+=h_uniq[col];
    }
    matched=false;
  }
  hipMemcpy(downCuda,totalDown,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(d_aBoard,t_aBoard,
      sizeof(int)*totalCond*mark,hipMemcpyHostToDevice);
  //size-mark は何行GPUを実行するか totalCondはスレッド数
  //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われるのは
  //totalCondの数だけでそれ以外は空回しになる
  /***12 TOPBIT,ENDBIT追加*********************/
  //cuda_kernel_b2<<<steps/THREAD_NUM,THREAD_NUM
  //  >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,d_aBoard,mark,BOUND1,BOUND2,SIDEMASK,LASTMASK);
  BitBoard_cuda_kernel_b2<<<steps/THREAD_NUM,THREAD_NUM
    >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,d_aBoard,mark,BOUND1,BOUND2,SIDEMASK,LASTMASK,TOPBIT,ENDBIT);
  hipMemcpy(h_results,resultsCuda,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  hipMemcpy(h_uniq,d_uniq,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  for(int col=0;col<steps/THREAD_NUM;col++){
    total+=h_results[col];
    UNIQUE+=h_uniq[col];
  }
  //
  hipFree(downCuda);
  hipFree(leftCuda);
  hipFree(rightCuda);
  hipFree(resultsCuda);
  hipFree(d_uniq);
  hipFree(d_aBoard);
  hipHostFree(totalDown);
  hipHostFree(totalLeft);
  hipHostFree(totalRight);
  hipHostFree(h_results);
  hipHostFree(h_uniq);
  hipHostFree(t_aBoard);
  return total;
}
// GPU -n Ｑが角にある
long BitBoard_backTrack1G(int size,int mask,int row,int n_left,int n_down,int n_right,int steps,int BOUND1)
{
  //何行目からGPUで行くか。ここの設定は変更可能、設定値を多くするほどGPUで並行して動く
  /***08 クイーンを２行目まで固定で置くためmarkが3以上必要*********************/
  const unsigned int mark=size>12?size-10:3;
  const unsigned int h_mark=row;
  long total=0;
  long totalCond=0;
  bool matched=false;
  //host
  unsigned int down[32];  down[row]=n_down;
  unsigned int right[32]; right[row]=n_right;
  unsigned int left[32];  left[row]=n_left;
  //bitmapを配列で持つことにより
  //stackを使わないで1行前に戻れる
  unsigned int bitmap[32];
  bitmap[row]=mask&~(left[row]|down[row]|right[row]);
  unsigned int bit;
  unsigned int* totalDown;
  hipHostMalloc((void**) &totalDown,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* totalLeft;
  hipHostMalloc((void**) &totalLeft,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* totalRight;
  hipHostMalloc((void**) &totalRight,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* h_results;
  hipHostMalloc((void**) &h_results,sizeof(int)*steps, hipHostMallocDefault);
  unsigned int* h_uniq;
  hipHostMalloc((void**) &h_uniq,sizeof(int)*steps, hipHostMallocDefault);
  /***11 backTrack1ではaBoard不要のためコメント*********************/
  //unsigned int* t_aBoard;
  //cudaMallocHost((void**) &t_aBoard,sizeof(int)*steps*mark);
  //device
  unsigned int* downCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  unsigned int* leftCuda;
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  unsigned int* rightCuda;
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  unsigned int* resultsCuda;
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);
  unsigned int* d_uniq;
  hipMalloc((void**) &d_uniq,sizeof(int)*steps/THREAD_NUM);
  /***11 backTrack1ではaBoard不要のためコメント*********************/
  //unsigned int* d_aBoard;
  //cudaMalloc((void**) &d_aBoard,sizeof(int)*steps*mark);
  //12行目までは3行目までCPU->row==mark以下で 3行目までの
  //down,left,right情報を totalDown,totalLeft,totalRight
  //に格納
  //する->3行目以降をGPUマルチスレッドで実行し結果を取得
  //13行目以降はCPUで実行する行数が１個ずつ増えて行く
  //例えばn15だとrow=5までCPUで実行し、
  //それ以降はGPU(現在の設定だとGPUでは最大10行実行する
  //ようになっている)
  //while(row>=0) {
  int rowP=0;
  while(row>=h_mark) {
    //bitmap[row]=00000000 クイーンを
    //どこにも置けないので1行上に戻る
    //06GPU こっちのほうが優秀
    if(bitmap[row]==0){ row--; }
    else{//おける場所があれば進む
      /***11 枝刈り*********************/
      if(row<BOUND1) {
        bitmap[row]&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
      }
      //06SGPU
      /***11 aBoard不要*********************/
      //bitmap[row]^=aBoard[row]=bit=(-bitmap[row]&bitmap[row]);
      bitmap[row]^=bit=(-bitmap[row]&bitmap[row]);
      if((bit&mask)!=0){//置く場所があれば先に進む
        rowP=row+1;
        down[rowP]=down[row]|bit;
        left[rowP]=(left[row]|bit)<<1;
        right[rowP]=(right[row]|bit)>>1;
        bitmap[rowP]=mask&~(down[rowP]|left[rowP]|right[rowP]);
        row++;
        if(row==mark){
          //3行目(mark)にクイーンを１個ずつ置いていって、
          //down,left,right情報を格納、
          //その次の行へは進まない。その行で可能な場所にクイー
          //ン置き終わったらGPU並列実行
          //totalCond がthreadIdになる 各スレッドに down,left,right情報を渡す
          //row=2(13行目以降は増えていく。例えばn15だとrow=5)の情報を
          //totalDown,totalLeft,totalRightに格納する         
          totalDown[totalCond]=down[row];
          totalLeft[totalCond]=left[row];
          totalRight[totalCond]=right[row];
          /***11 aBoardコメント*********************/
          //for(int i=0;i<mark;i++){
          //  t_aBoard[totalCond*mark+i]=aBoard[i];
          //}
          //スレッド数をインクリメントする
          totalCond++;
          //最大GPU数に達してしまったら一旦ここでGPUを実行する。stepsはGPUの同
          //時並行稼働数を制御
          //nの数が少ないうちはtotalCondがstepsを超えることはないがnの数が増え
          //て行くと超えるようになる。
          //ここではtotalCond==stepsの場合だけこの中へ         
          if(totalCond==steps){
            //matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUか
            //ら出たらmatched=trueになってる
            if(matched){
              hipMemcpy(h_results,resultsCuda,
                  sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
              hipMemcpy(h_uniq,d_uniq,
                  sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
              for(int col=0;col<steps/THREAD_NUM;col++){
                total+=h_results[col];
                UNIQUE+=h_uniq[col];
              }
              matched=false;
            }
            hipMemcpy(downCuda,totalDown,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(leftCuda,totalLeft,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            hipMemcpy(rightCuda,totalRight,
                sizeof(int)*totalCond,hipMemcpyHostToDevice);
            /***11 aBoard不要のためコメント*********************/
            //cudaMemcpy(d_aBoard,t_aBoard,
            //    sizeof(int)*totalCond*mark,cudaMemcpyHostToDevice);
            /***11 BOUND1追加*********************/
            //cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
            //  >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,d_aBoard,row);
            BitBoard_cuda_kernel_b1<<<steps/THREAD_NUM,THREAD_NUM
              >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,row,BOUND1);

            //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われ
            //るのはtotalCondの数だけでそれ以外は空回しになる
            //GPU内でカウントしているので、GPUから出たらmatched=trueになってる
            matched=true;
            //totalCond==stepsルートでGPUを実行したらスレッドをまた0から開始す
            //る(これによりなんどもsteps数分だけGPUを起動できる)
            totalCond=0;           
          }
          //totalDown,totalLeft,totalRightに情報を格納したら1行上に上がる
          //これを繰り返すことにより row=2で可能な場所全てにクイーンを置いて
          //totalDown,totalLeft,totalRightに情報を格納する
          row--;
        }
      }else{
        //置く場所がなければ上に上がる。row==mark行に達するまではCPU側で普通に
        //nqueenをやる
        row--;
      }
    }
  }
  //matched=trueの時にCOUNT追加 //GPU内でカウントしているので、GPUから出たら
  //matched=trueになってる
  if(matched){
    hipMemcpy(h_results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    hipMemcpy(h_uniq,d_uniq,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){
      total+=h_results[col];
      UNIQUE+=h_uniq[col];
    }
    matched=false;
  }
  hipMemcpy(downCuda,totalDown,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  /***11 aBoard不要のためコメント*********************/
  //cudaMemcpy(d_aBoard,t_aBoard,
  //    sizeof(int)*totalCond*mark,cudaMemcpyHostToDevice);
  //size-mark は何行GPUを実行するか totalCondはスレッド数
  //steps数の数だけマルチスレッドで起動するのだが、実際に計算が行われるのは
  //totalCondの数だけでそれ以外は空回しになる
  /***11 BOUND1追加*********************/
  //cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM
  //  >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,d_aBoard,mark);
  BitBoard_cuda_kernel_b1<<<steps/THREAD_NUM,THREAD_NUM
    >>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,d_uniq,totalCond,mark,BOUND1);
  hipMemcpy(h_results,resultsCuda,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  hipMemcpy(h_uniq,d_uniq,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  for(int col=0;col<steps/THREAD_NUM;col++){
    total+=h_results[col];
    UNIQUE+=h_uniq[col];
  }
  //
  hipFree(downCuda);
  hipFree(leftCuda);
  hipFree(rightCuda);
  hipFree(resultsCuda);
  hipFree(d_uniq);
  /***11 aBoardコメント**/
  //cudaFree(d_aBoard);
  hipHostFree(totalDown);
  hipHostFree(totalLeft);
  hipHostFree(totalRight);
  hipHostFree(h_results);
  hipHostFree(h_uniq);
  /***11 aBoardコメント**/
  //cudaFreeHost(t_aBoard);
  return total;
}
// GPU -n ビットボードの実行 角にＱがある・ないの分岐を行う
void symmetry_build_bitBoard(unsigned int size,int steps)
{
  if(size<=0||size>32){return;}
  /**
    パラメータは渡す変数はregisterとする
    int型は unsigned とする
    total: グローバル変数TOTALへのアクセスを極小化する
    sizeE:size-1といった計算を変数に格納しフラット化する 
    */
  unsigned long total=0;
  unsigned int sizeE=size-1;
  unsigned int aBoard[MAX];
  int bit=0;
  int mask=((1<<size)-1);
  int col=0;//1行め右端 0
  aBoard[0]=bit=(1<<col);
  int left=bit<<1,down=bit,right=bit>>1;
  /**
    2行目は右から3列目から左端から2列目まで
  */
  for(int BOUND1=2;BOUND1<sizeE;BOUND1++){
    aBoard[1]=bit=(1<<BOUND1);
    total+=BitBoard_backTrack1G(size,mask,2,
        (left|bit)<<1,(down|bit),(right|bit)>>1,
        steps,BOUND1);
  }
  int LASTMASK,SIDEMASK;
  int TOPBIT=1<<(sizeE);
  SIDEMASK=LASTMASK=(TOPBIT|1);
  int ENDBIT=(TOPBIT>>1);
  /**
    1行目右から2列目から
    偶数個は1/2 n=8 なら 1,2,3 奇数個は1/2+1 n=9 なら 1,2,3,4
  */
  for(int BOUND1=1,BOUND2=sizeE-1;BOUND1<BOUND2;BOUND1++,BOUND2--){
    aBoard[0]=bit=(1<<BOUND1);
    total+=BitBoard_backTrack2G(size,mask,1,
        bit<<1,bit,bit>>1,
        steps,BOUND1,BOUND2,SIDEMASK,LASTMASK,TOPBIT,ENDBIT,aBoard);
    LASTMASK|=LASTMASK>>1|LASTMASK<<1;
    ENDBIT>>=1;
  }
  /**
    グローバル変数へのアクセスを極小化する
    */
  TOTAL=total;
}
// CUDA 初期化
bool InitCUDA()
{
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  unsigned int i;
  for(i=0;i<count;++i){
    struct hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
//メイン
int main(int argc,char** argv)
{
  bool cpu=false,cpur=false,gpu=false,gpuBitBoard=false;
  unsigned int argstart=2;
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='r'||argv[1][1]=='R'){cpur=true;}
    else if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){gpu=true;}
    else if(argv[1][1]=='n'||argv[1][1]=='N'){gpuBitBoard=true;}
    else{ gpuBitBoard=true; } //デフォルトをgpuとする
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g|-r|-s] n steps\n",argv[0]);
    printf("  -r: CPU 再帰\n");
    printf("  -c: CPU 非再帰\n");
    printf("  -g: GPU 再帰\n");
    printf("  -n: GPU ビットボード\n");
  }
  if(cpur){ printf("\n\n対称解除法 再帰 \n"); }
  else if(cpu){ printf("\n\n対称解除法 非再帰 \n"); }
  else if(gpu){ printf("\n\n対称解除法 GPU\n"); }
  else if(gpuBitBoard){ printf("\n\n対称解除法 GPUビットボード \n"); }
  if(cpu||cpur)
  {
    unsigned int min=4; 
    unsigned int targetN=17;
    struct timeval t0;
    struct timeval t1;
    printf("%s\n"," N:        Total      Unique      dd:hh:mm:ss.ms");
    for(unsigned int size=min;size<=targetN;size++){
      local l;
      gettimeofday(&t0,NULL);//計測開始
      if(cpur){ //再帰
        symmetry_R(size,&l);
      }
      if(cpu){ //非再帰
        symmetry_NR(size,&l);
      }
      //
      gettimeofday(&t1,NULL);//計測終了
      unsigned int ss;
      unsigned int ms;
      unsigned int dd;
      if(t1.tv_usec<t0.tv_usec) {
        dd=(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      }else {
        dd=(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }//end if
      unsigned int hh=ss/3600;
      unsigned int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%13ld%12ld%8.2d:%02d:%02d:%02d.%02d\n",size,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    } //end for
  }//end if
  if(gpu||gpuBitBoard)
  {
    int steps=24576;
    if(!InitCUDA()){return 0;}
    unsigned int min=4;
    unsigned int targetN=21;
    struct timeval t0;
    struct timeval t1;
    printf("%s\n"," N:        Total      Unique      dd:hh:mm:ss.ms");
    for(unsigned int size=min;size<=targetN;size++){
      gettimeofday(&t0,NULL);
      if(gpu){
        TOTAL=UNIQUE=0;
        local l[MAX];
        GPU_symmetry_R(size,&l[0]);
        TOTAL=l->TOTAL;
        UNIQUE=l->UNIQUE;
      }else if(gpuBitBoard){
        TOTAL=UNIQUE=0;
        symmetry_build_bitBoard(size,steps);
      }
      gettimeofday(&t1,NULL);
      unsigned int ss;
      unsigned int ms;
      unsigned int dd;
      if (t1.tv_usec<t0.tv_usec) {
        dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      } else {
        dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }//end if
      unsigned int hh=ss/3600;
      unsigned int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%13ld%12ld%8.2d:%02d:%02d:%02d.%02d\n",size,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    }
  }
  return 0;
}
