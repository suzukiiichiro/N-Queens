/**
 CUDAで学ぶアルゴリズムとデータ構造
 ステップバイステップでＮ−クイーン問題を最適化
 一般社団法人  共同通信社  情報技術局  鈴木  維一郎(suzuki.iichiro@kyodonews.jp)

 コンパイルと実行
 $ nvcc -O3 CUDA**_N-Queen.cu && ./a.out (-c|-r|-g)
                    -c:cpu 
                    -r cpu再帰 
                    -g GPU 
*/

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define THREAD_NUM		96
#define MAX 27
//変数宣言
long Total=0 ;      //GPU
long Unique=0;			//GPU
int COUNT=0;     		//カウント用
int aBoard[MAX]; 		//版の配列
int down[2*MAX-1]; 	//down:flagA 縦 配置フラグ　
//関数宣言CUDA
__global__ void nqueen_cuda(int *d_aBoard,int *d_results,int *d_count, int COUNT,int row,int size);
void solve_nqueen_cuda(int si,int steps);
bool InitCUDA();
//関数宣言CPU
void print(int size);
void NQueen(int row,int size);
void NQueenR(int row,int size);
//
__global__
void nqueen_cuda(int *d_aBoard,int *d_down,int *d_results,int *d_count,int COUNT,int row,int size){
    bool matched;
  while(row>=0){
    matched=false;
    for(int col=d_aBoard[row]+1;col<size;col++){
      if(d_down[col]==0){      //downは効き筋ではない
        if(d_aBoard[row]!=-1){ //Qは配置済み
          d_down[d_aBoard[row]]=0;//downの効き筋を外す
        }
        d_aBoard[row]=col;     //Qを配置
        d_down[col]=1;         //downは効き筋である
        matched=true;
        break;
      }
    }
    if(matched){
      row++;
      if(row==size){
        //cudaの中で　printせず配列に格納して　hostに返却する
        //ex 0,1,1,3 だったら　3110
        int sum=0;
        for(int j=0;j<size;j++){
          sum+=d_aBoard[j]*pow(10,j);   
        }
        d_results[COUNT++]=sum;
        row--;
      }
    }else{                   //置けるところがない
      if(d_aBoard[row]!=-1){
        int col=d_aBoard[row]; /** colの代用 */
        d_down[col]=0;         //downの効き筋を解除
        d_aBoard[row]=-1;      //空き地に戻す
      }
      row--;
    }
  }
	d_count[0]=COUNT;//カウントを代入
}
//
void solve_nqueen_cuda(int si,int steps){
    //メモリ登録
    int *h_aBoard;
    int *h_down;
    int *h_results;
    int *h_count;
    hipHostMalloc((void**)&h_aBoard,sizeof(int)*MAX, hipHostMallocDefault);
    hipHostMalloc((void**)&h_down,sizeof(int)*2*MAX-1, hipHostMallocDefault);
    hipHostMalloc((void**)&h_results,sizeof(int)*steps, hipHostMallocDefault);
    hipHostMalloc((void**)&h_count,sizeof(int), hipHostMallocDefault);
    int *d_aBoard;
    int *d_down;
    int *d_results;
    int *d_count;
    hipMalloc((void**)&d_aBoard,sizeof(int)*MAX);
    hipMalloc((void**)&d_down,sizeof(int)*2*MAX-1);
    hipMalloc((void**)&d_results,sizeof(int)*steps);
    hipMalloc((void**)&d_count,sizeof(int));
    //初期化
    for(int i=0;i<si;i++){
        h_aBoard[i]=-1;
    }
    //カウンターを初期化
    h_count[0]=0;
    //host to device
    hipMemcpy(d_aBoard,h_aBoard,
      sizeof(int)*MAX,hipMemcpyHostToDevice);
    hipMemcpy(d_down,h_down,
      sizeof(int)*2*MAX-1,hipMemcpyHostToDevice);
    hipMemcpy(d_results,h_results,
      sizeof(int)*steps,hipMemcpyHostToDevice);
    hipMemcpy(d_count,h_count,
      sizeof(int),hipMemcpyHostToDevice);
    //実行
    nqueen_cuda<<<1,1>>>(d_aBoard,d_down,d_results,d_count,0,0,si);
    //device to host
    hipMemcpy(h_results,d_results,
      sizeof(int)*steps,hipMemcpyDeviceToHost);
    hipMemcpy(h_count,d_count,
      sizeof(int),hipMemcpyDeviceToHost);
    //出力
    for(int i=0;i<h_count[0];i++){
      printf("%d:%08d\n",i+1,h_results[i]);  
    }
    //開放
    hipHostFree(h_aBoard);
    hipHostFree(h_down);
    hipHostFree(h_results);
    hipHostFree(h_count);
    hipFree(d_aBoard);
    hipFree(d_down);
    hipFree(d_results);
    hipFree(d_count);
}
//
/** CUDA 初期化 **/
bool InitCUDA(){
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
//出力用のメソッド
void print(int size){
  printf("%d: ",++COUNT);
  for(int j=0;j<size;j++){
    printf("%d ",aBoard[j]);
  }
  printf("\n");
}
//CPU 非再帰 ロジックメソッド
void NQueen(int row,int size){
  bool matched;
  while(row>=0){
    matched=false;
    for(int col=aBoard[row]+1;col<size;col++){
      if(down[col]==0){      //downは効き筋ではない
        if(aBoard[row]!=-1){ //Qは配置済み
          down[aBoard[row]]=0;//downの効き筋を外す
        }
        aBoard[row]=col;     //Qを配置
        down[col]=1;         //downは効き筋である
        matched=true;
        break;
      }
    }
    if(matched){
      row++;
      if(row==size){
        print(size);
        row--;
      }
    }else{                   //置けるところがない
      if(aBoard[row]!=-1){
        int col=aBoard[row]; /** colの代用 */
        down[col]=0;         //downの効き筋を解除
        aBoard[row]=-1;      //空き地に戻す
      }
      row--;
    }
  }
}
//CPUR 再帰 ロジックメソッド
void NQueenR(int row,int size){
  if(row==size){
    print(size);
  }else{
    for(int col=aBoard[row]+1;col<size;col++){
      aBoard[row]=col;  //Qを配置
      if(down[col]==0){
        down[col]=1;
        NQueenR(row+1,size);
        down[col]=0;
      }
      aBoard[row]=-1;   //空き地に戻す
    }
  }
}
//メインメソッド
int main(int argc,char** argv) {
  int size=5;
  bool cpu=false,cpur=false,gpu=false;
  int argstart=1,steps=24576;
  /** パラメータの処理 */
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='r'||argv[1][1]=='R'){cpur=true;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){gpu=true;}
    else{
      cpur=true;
		}
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g|-r|-s]\n",argv[0]);
    printf("  -c: CPU only\n");
    printf("  -r: CPUR only\n");
    printf("  -g: GPU only\n");
    printf("Default CPUR to 8 queen\n");
  }
  /** 出力と実行 */
  //aBoard配列を-1で初期化
  for(int i=0;i<size;i++){ aBoard[i]=-1; }
  if(cpu){ 
    printf("\n\n２．CPU 非再帰 配置フラグ（制約テスト高速化）\n");
    NQueen(0,size); 
  }
  if(cpur){ 
    printf("\n\n２．CPU 再帰 配置フラグ（制約テスト高速化）\n");
    NQueenR(0,size); 
  }
  if(gpu){
    printf("\n\n２．GPU 非再帰 配置フラグ（制約テスト高速化）\n");
    if(!InitCUDA()){return 0;}
    solve_nqueen_cuda(size,steps);
  }
  return 0;
}
