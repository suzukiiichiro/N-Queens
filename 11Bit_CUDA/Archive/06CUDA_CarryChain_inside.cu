/**
 *
 * bash版キャリーチェーンのC言語版のGPU/CUDA移植版
 * CUDAの実行をfor文の一番内側、クイーンを上下左右2行2列置いたあと
 
 詳しい説明はこちらをどうぞ
 https://suzukiiichiro.github.io/search/?keyword=Ｎクイーン問題
 *
・carryChain GPU inside backTrack部分でGPUを起動 stepsに達するまで貯めた

NQueens_suzuki$ nvcc -O3 -arch=sm_61  -Xcompiler -mcmodel=medium  06CUDA_CarryChain_inside.cu && ./a.out -n
 N:            Total          Unique      dd:hh:mm:ss.ms
 4:            2           0      00:00:00:00.13
 5:           10           0      00:00:00:00.00
 6:            4           0      00:00:00:00.00
 7:           40           0      00:00:00:00.00
 8:           92           0      00:00:00:00.00
 9:          352           0      00:00:00:00.00
10:          724           0      00:00:00:00.00
11:         2680           0      00:00:00:00.00
12:        14200           0      00:00:00:00.01
13:        73712           0      00:00:00:00.04
14:       365596           0      00:00:00:00.12
15:      2279184           0      00:00:00:00.43
16:     14772512           0      00:00:00:02.10
17:     95815104           0      00:00:00:13.29
18:    666090624           0      00:00:01:36.21
19:   4968057848           0      00:00:12:16.30
*/

#include <iostream>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define THREAD_NUM		96
#define MAX 27
#define steps 24576
/**
  * システムによって以下のマクロが必要であればコメントを外してください。
  */
//#define UINT64_C(c) c ## ULL

typedef unsigned int uint;
typedef unsigned long ulong;

ulong TOTAL=0; 
ulong UNIQUE=0;
ulong totalCond=0;
typedef struct
{
  uint size;
  uint pres_a[930]; 
  uint pres_b[930];
}Global; Global g;
typedef struct
{
  ulong row;
  ulong down;
  ulong left;
  ulong right;
  long long x[MAX];
}Board ;
typedef struct
{
  Board B;
  Board nB;
  Board eB;
  Board sB;
  Board wB;
  uint n;
  uint e;
  uint s;
  uint w;
  ulong dimx;
  ulong dimy;
  ulong COUNTER[3];      
  uint COUNT2;
  uint COUNT4;
  uint COUNT8;
  uint type;
}Local;


ulong* totalDown=new ulong[steps];
ulong* totalLeft=new ulong[steps];
ulong* totalRight=new ulong[steps];
ulong* totalRow=new ulong[steps];
uint* totalType=new uint[steps];
ulong* results =new ulong[steps];
bool matched=false;
ulong* rowCuda;
ulong* downCuda;
ulong* leftCuda;
ulong* rightCuda;
ulong* resultsCuda;
uint* typeCuda;

/**
  *
  */
__global__ void solve(uint size,int current,uint* totalType,ulong* totalRow,ulong* totalDown,ulong* totalLeft,ulong* totalRight,ulong* results,uint totalCond)
{
  const uint tid=threadIdx.x;
  const uint bid=blockIdx.x;
  const uint idx=bid*blockDim.x+tid;
  ulong  row_a[MAX];
  ulong  down_a[MAX];
  ulong  left_a[MAX];
  ulong  right_a[MAX];
  ulong  bitmap_a[MAX];
  __shared__ int  sum[THREAD_NUM];
  ulong row=row_a[current]=totalRow[idx];
  ulong left=left_a[current]=totalLeft[idx];
  ulong down=down_a[current]=totalDown[idx];
  ulong right=right_a[current]=totalRight[idx];
  ulong bitmap=bitmap_a[current]=~(left_a[current]|down_a[current]|right_a[current]);
  uint total=0;
  ulong bit;
  uint ttype=totalType[idx];
  if(idx<totalCond){
  while(current>-1){
    if((bitmap!=0||row&1)&&current<size){
      if(!(down+1)){

        total+=ttype;
        current--;
        row=row_a[current];
        left=left_a[current];
        right=right_a[current];
        down=down_a[current];
        bitmap=bitmap_a[current];
        continue;
      }else if(row&1){
        while( row&1 ){
          row>>=1;
          left<<=1;
          right>>=1;
        }
        bitmap=~(left|down|right);  //再帰に必要な変数は必ず定義する必要があります。
        continue;
      }else{
        bit=-bitmap&bitmap;
        bitmap=bitmap^bit;
        if(current<size){
          row_a[current]=row;
          left_a[current]=left;
          down_a[current]=down;
          right_a[current]=right;
          bitmap_a[current]=bitmap;
          current++;
        }
        row>>=1;      //１行下に移動する
        left=(left|bit)<<1;
        down=down|bit;
        right=(right|bit)>>1;
        bitmap=~(left|down|right);  //再帰に必要な変数は必ず定義する必要があります。
      }
    }else{
      current--;
      row=row_a[current];
      left=left_a[current];
      right=right_a[current];
      down=down_a[current];
      bitmap=bitmap_a[current];
    }
  }
  sum[tid]=total;
  }else{
    sum[tid]=0;
  }
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){sum[tid]+=sum[tid+64];} 
  __syncthreads();if(tid<32){sum[tid]+=sum[tid+32];} 
  __syncthreads();if(tid<16){sum[tid]+=sum[tid+16];} 
  __syncthreads();if(tid<8){sum[tid]+=sum[tid+8];} 
  __syncthreads();if(tid<4){sum[tid]+=sum[tid+4];} 
  __syncthreads();if(tid<2){sum[tid]+=sum[tid+2];} 
  __syncthreads();if(tid<1){sum[tid]+=sum[tid+1];} 
  __syncthreads();if(tid==0){results[bid]=sum[0];}
}
/**
  *
  */
void append(void* args)
{
  Local *l=(Local *)args;
  totalRow[totalCond]=l->B.row>>2;
  totalDown[totalCond]=((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1;
  totalLeft[totalCond]=l->B.left>>4;
  totalRight[totalCond]=(l->B.right>>4)<<(g.size-5);
  totalType[totalCond]=l->type;
  totalCond++;
  if(totalCond==steps){
    if(matched){
      hipMemcpy(results,resultsCuda,sizeof(long)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
      for(uint col=0;col<steps/THREAD_NUM;col++){TOTAL+=results[col];}
        matched=false;
      }
      hipMemcpy(rowCuda,totalRow,sizeof(ulong)*totalCond,hipMemcpyHostToDevice);
      hipMemcpy(downCuda,totalDown,sizeof(ulong)*totalCond,hipMemcpyHostToDevice);
      hipMemcpy(leftCuda,totalLeft,sizeof(ulong)*totalCond,hipMemcpyHostToDevice);
      hipMemcpy(rightCuda,totalRight,sizeof(ulong)*totalCond,hipMemcpyHostToDevice);
      hipMemcpy(typeCuda,totalType,sizeof(uint)*totalCond,hipMemcpyHostToDevice);
      solve<<<steps/THREAD_NUM,THREAD_NUM>>>(g.size,0,typeCuda,rowCuda,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
      hipMemcpy(results,resultsCuda,
      sizeof(uint)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
      matched=true;
      totalCond=0;
  }
}
/**
  * 非再帰 対称解除法
  */
void carryChain_symmetry(void* args)
{
  Local *l=(Local *)args;
  // 対称解除法
  const uint ww=(g.size-2)*(g.size-1)-1-l->w;
  const uint w2=(g.size-2)*(g.size-1)-1;
  // # 対角線上の反転が小さいかどうか確認する
  if((l->s==ww)&&(l->n<(w2-l->e))){ return ; }
  // # 垂直方向の中心に対する反転が小さいかを確認
  if((l->e==ww)&&(l->n>(w2-l->n))){ return; }
  // # 斜め下方向への反転が小さいかをチェックする
  if((l->n==ww)&&(l->e>(w2-l->s))){ return; }
  // 枝刈り １行目が角の場合回転対称チェックせずCOUNT8にする
  if(l->B.x[0]==0){
    l->type=8;
    append(l);
    //l->COUNTER[l->COUNT8]+=solve(g.size,0,l->B.row>>2,
    //l->B.left>>4,((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1,(l->B.right>>4)<<(g.size-5));
    return ;
  }
  // n,e,s==w の場合は最小値を確認する。右回転で同じ場合は、
  // w=n=e=sでなければ値が小さいのでskip  w=n=e=sであれば90度回転で同じ可能性
  if(l->s==l->w){ if((l->n!=l->w)||(l->e!=l->w)){ return; }
    l->type=2;
    append(l);
    //l->COUNTER[l->COUNT2]+=solve(g.size,0,l->B.row>>2,
    //l->B.left>>4,((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1,(l->B.right>>4)<<(g.size-5));
    return;
  }
  // e==wは180度回転して同じ 180度回転して同じ時n>=sの時はsmaller?
  if((l->e==l->w)&&(l->n>=l->s)){ if(l->n>l->s){ return; }
    l->type=4;
    append(l);
    //l->COUNTER[l->COUNT4]+=solve(g.size,0,l->B.row>>2,
    //l->B.left>>4,((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1,(l->B.right>>4)<<(g.size-5));
    return;
  }
  l->type=8;
  append(l);
  //l->COUNTER[l->COUNT8]+=solve(g.size,0,l->B.row>>2,
  //l->B.left>>4,((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1,(l->B.right>>4)<<(g.size-5));
  return;
}
/**
  * クイーンの効きをチェック
  */
bool placement(void* args)
{
  Local *l=(Local *)args;
  if(l->B.x[l->dimx]==l->dimy){ return true;  }  
  if (l->B.x[0]==0){
    if (l->B.x[1]!=(ulong)-1){
      if((l->B.x[1]>=l->dimx)&&(l->dimy==1)){ return false; }
    }
  }else{
    if( (l->B.x[0]!=(ulong)-1) ){
      if(( (l->dimx<l->B.x[0]||l->dimx>=g.size-l->B.x[0])
        && (l->dimy==0 || l->dimy==g.size-1)
      )){ return 0; } 
      if ((  (l->dimx==g.size-1)&&((l->dimy<=l->B.x[0])||
          l->dimy>=g.size-l->B.x[0]))){
        return 0;
      } 
    }
  }
  l->B.x[l->dimx]=l->dimy;                    //xは行 yは列
  ulong row=UINT64_C(1)<<l->dimx;
  ulong down=UINT64_C(1)<<l->dimy;
  ulong left=UINT64_C(1)<<(g.size-1-l->dimx+l->dimy); //右上から左下
  ulong right=UINT64_C(1)<<(l->dimx+l->dimy);       // 左上から右下
  if((l->B.row&row)||(l->B.down&down)||(l->B.left&left)||(l->B.right&right)){ return false; }     
  l->B.row|=row; l->B.down|=down; l->B.left|=left; l->B.right|=right;
  return true;
}
/**
  * 
  */
void thread_run(void* args)
{
  Local *l=(Local *)args;
  // memcpy(&l->B,&l->wB,sizeof(Board));       // B=wB;
  l->B=l->wB;
  l->dimx=0; l->dimy=g.pres_a[l->w];
  //if(!placement(l)){ continue; }
  if(!placement(l)){ return; }
  l->dimx=1; l->dimy=g.pres_b[l->w];
  // if(!placement(l)){ continue; }
  if(!placement(l)){ return; }
  //２ 左２行に置く
  // memcpy(&l->nB,&l->B,sizeof(Board));       // nB=B;
  l->nB=l->B;
  for(l->n=l->w;l->n<(g.size-2)*(g.size-1)-l->w;++l->n){
    // memcpy(&l->B,&l->nB,sizeof(Board));     // B=nB;
    l->B=l->nB;
    l->dimx=g.pres_a[l->n]; l->dimy=g.size-1;
    if(!placement(l)){ continue; }
    l->dimx=g.pres_b[l->n]; l->dimy=g.size-2;
    if(!placement(l)){ continue; }
    // ３ 下２行に置く
    // memcpy(&l->eB,&l->B,sizeof(Board));     // eB=B;
    l->eB=l->B;
    for(l->e=l->w;l->e<(g.size-2)*(g.size-1)-l->w;++l->e){
      // memcpy(&l->B,&l->eB,sizeof(Board));   // B=eB;
      l->B=l->eB;
      l->dimx=g.size-1; l->dimy=g.size-1-g.pres_a[l->e];
      if(!placement(l)){ continue; }
      l->dimx=g.size-2; l->dimy=g.size-1-g.pres_b[l->e];
      if(!placement(l)){ continue; }
      // ４ 右２列に置く
      // memcpy(&l->sB,&l->B,sizeof(Board));   // sB=B;
      l->sB=l->B;
      for(l->s=l->w;l->s<(g.size-2)*(g.size-1)-l->w;++l->s){
        // memcpy(&l->B,&l->sB,sizeof(Board)); // B=sB;
        l->B=l->sB;
        l->dimx=g.size-1-g.pres_a[l->s]; l->dimy=0;
        if(!placement(l)){ continue; }
        l->dimx=g.size-1-g.pres_b[l->s]; l->dimy=1;
        if(!placement(l)){ continue; }
        // 対称解除法
        carryChain_symmetry(l);
      } //w
    } //e
  } //n
}
/**
  * 非再帰  チェーンのビルド
  */
void buildChain()
{
  Local l[(g.size/2)*(g.size-3)];
  hipMalloc((void**) &rowCuda,sizeof(ulong)*steps);
  hipMalloc((void**) &downCuda,sizeof(ulong)*steps);
  hipMalloc((void**) &leftCuda,sizeof(ulong)*steps);
  hipMalloc((void**) &rightCuda,sizeof(ulong)*steps);
  hipMalloc((void**) &typeCuda,sizeof(uint)*steps);
  hipMalloc((void**) &resultsCuda,sizeof(ulong)*steps/THREAD_NUM);
  // カウンターの初期化
  l->COUNT2=0; l->COUNT4=1; l->COUNT8=2;
  l->COUNTER[l->COUNT2]=l->COUNTER[l->COUNT4]=l->COUNTER[l->COUNT8]=0;
  // Board の初期化 nB,eB,sB,wB;
  l->B.row=l->B.down=l->B.left=l->B.right=0;
  // Board x[]の初期化
  for(uint i=0;i<g.size;++i){ l->B.x[i]=-1; }
  //１ 上２行に置く
  // memcpy(&l->wB,&l->B,sizeof(Board));         // wB=B;
  l->wB=l->B;
  for(l->w=0;l->w<=(unsigned)(g.size/2)*(g.size-3);++l->w){
    thread_run(&l);
  }
  if(matched){
    hipMemcpy(results,resultsCuda,sizeof(long)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(uint col=0;col<steps/THREAD_NUM;col++){TOTAL+=results[col];}
    matched=false;
  }
  hipMemcpy(rowCuda,totalRow,sizeof(ulong)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(downCuda,totalDown,sizeof(ulong)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,sizeof(ulong)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,sizeof(ulong)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(typeCuda,totalType,sizeof(uint)*totalCond,hipMemcpyHostToDevice);
  solve<<<steps/THREAD_NUM,THREAD_NUM>>>(g.size,0,typeCuda,rowCuda,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
  hipMemcpy(results,resultsCuda,sizeof(long)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  for(uint col=0;col<steps/THREAD_NUM;col++){TOTAL+=results[col];}	
}
/**
  * チェーンのリストを作成
  */
void listChain()
{
  uint idx=0;
  for(uint a=0;a<(unsigned)g.size;++a){
    for(uint b=0;b<(unsigned)g.size;++b){
      if(((a>=b)&&(a-b)<=1)||((b>a)&&(b-a)<=1)){ continue; }
      g.pres_a[idx]=a;
      g.pres_b[idx]=b;
      ++idx;
    }
  }
}
/**
  * キャリーチェーン
  */
void carryChain()
{
  listChain();  //チェーンのリストを作成
  buildChain(); // チェーンのビルド
}
/**
  * CUDA 初期化
  */
bool InitCUDA()
{
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    struct hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
/**
  * メイン
  */
int main(int argc,char** argv)
{
  if(!InitCUDA()){return 0;}
  /* int steps=24576; */
  int min=4;
  int targetN=19;
  struct timeval t0;
  struct timeval t1;
  printf("%s\n"," N:            Total          Unique      dd:hh:mm:ss.ms");
  for(int size=min;size<=targetN;size++){
    gettimeofday(&t0,NULL);   // 計測開始
    totalCond=0;
    TOTAL=UNIQUE=0;
    g.size=size;
    carryChain();
    gettimeofday(&t1,NULL);   // 計測終了
    int ss;int ms;int dd;
    if (t1.tv_usec<t0.tv_usec) {
      dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
      ss=(t1.tv_sec-t0.tv_sec-1)%86400;
      ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
    } else {
      dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
      ss=(t1.tv_sec-t0.tv_sec)%86400;
      ms=(t1.tv_usec-t0.tv_usec+500)/10000;
    }//end if
    int hh=ss/3600;
    int mm=(ss-hh*3600)/60;
    ss%=60;
    printf("%2d:%17ld%16ld%8.3d:%02d:%02d:%02d.%02d\n",size,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
  }
  hipFree(rowCuda);
  hipFree(downCuda);
  hipFree(leftCuda);
  hipFree(rightCuda);
  hipFree(typeCuda);
  hipFree(resultsCuda);
  delete[] totalRow;
  delete[] totalDown;
  delete[] totalLeft;
  delete[] totalRight;
  delete[] totalType;
  delete[] results;
  return 0;
}
