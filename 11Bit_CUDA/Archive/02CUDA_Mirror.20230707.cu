/**

  }
  にあわせて % 対となる { にジャンプする
  応用例
  } にカーソルを合わせて Shift + % 関数内が範囲指定される。
  そこで、zf でたたまれる

  zfでたたまれた関数は za で展開される

  zM -- 折りたたみ（すべて） (Close all folds)
  zR -- 展開（すべて） (Open all folds)
  zd	折畳削除
  zD	折畳を全て削除
  zE	ページ全体の折畳みを全て削除


 *
 * bash版ミラーのC言語版のGPU/CUDA移植版
 *
 詳しい説明はこちらをどうぞ
 https://suzukiiichiro.github.io/search/?keyword=Ｎクイーン問題
 *
*/
#include <iostream>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define THREAD_NUM		96
#define MAX 27
// システムによって以下のマクロが必要であればコメントを外してください。
//#define UINT64_C(c) c ## ULL
//
// グローバル変数
unsigned long TOTAL=0; 
unsigned long UNIQUE=0;
//ミラー処理部分 非再帰版
void mirror_solve_NR(unsigned int size,unsigned int row,unsigned int _left,unsigned int _down, unsigned int _right)
{
  unsigned int mask=(1<<size)-1;
  unsigned int bit=0;
  unsigned int down[MAX];   //ポストフラグ/ビットマップ/ミラー
  unsigned int left[MAX];   //ポストフラグ/ビットマップ/ミラー
  unsigned int right[MAX];  //ポストフラグ/ビットマップ/ミラー
  unsigned int bitmap[MAX]; //ミラー
  left[row]=_left;
  down[row]=_down;
  right[row]=_right;
  bitmap[row]=mask&~(left[row]|down[row]|right[row]);
  while(row>0){
    if(bitmap[row]>0){
      bit=-bitmap[row]&bitmap[row];
      bitmap[row]=bitmap[row]^bit;
      if(row==(size-1)){
        TOTAL++;
        row--;
      }else{
        unsigned int n=row++;
        left[row]=(left[n]|bit)<<1;
        down[row]=(down[n]|bit);
        right[row]=(right[n]|bit)>>1;
        //クイーンが配置可能な位置を表す
        bitmap[row]=mask&~(left[row]|down[row]|right[row]);
      }
    }else{
      row--;
    }
  }
}
// ミラー 非再帰版
void mirror_NR(unsigned int size)
{
  unsigned int bit=0;
  unsigned int limit=size%2 ? size/2-1 : size/2;
  for(unsigned int i=0;i<size/2;++i){ //奇数でも偶数でも通過
    bit=1<<i;
    mirror_solve_NR(size,1,bit<<1,bit,bit>>1);
  }
  if(size%2){                 //奇数で通過
    bit=1<<(size-1)/2;
    unsigned int left=bit<<1;
    unsigned int down=bit;
    unsigned int right=bit>>1;
    for(unsigned int i=0;i<limit;++i){
      bit=1<<i;
      mirror_solve_NR(size,2,(left|bit)<<1,down|bit,(right|bit)>>1);
    }
  }
  TOTAL=TOTAL<<1;    //倍にする
}
//ミラーロジック 再帰版
void mirror_solve_R(unsigned int size,unsigned int row,unsigned int left,unsigned int down,unsigned int right)
{
  unsigned int mask=(1<<size)-1;
  unsigned int bit=0;
  if(row==size){
    TOTAL++;
  }else{
    for(unsigned int bitmap=mask&~(left|down|right);bitmap;bitmap=bitmap&~bit){
      bit=-bitmap&bitmap;
      mirror_solve_R(size,row+1,(left|bit)<<1,down|bit,(right|bit)>>1);
    }
  }
}
// ミラー 再帰版
void mirror_R(unsigned int size)
{
  unsigned int bit=0;
  unsigned int limit=size%2 ? size/2-1 : size/2;
  for(unsigned int i=0;i<size/2;++i){
    bit=1<<i;
    mirror_solve_R(size,1,bit<<1,bit,bit>>1);
  }
  if(size%2){               //奇数で通過
    bit=1<<(size-1)/2;
    unsigned int left=bit<<1;
    unsigned int down=bit;
    unsigned int right=bit>>1;
    for(unsigned int i=0;i<limit;++i){
      bit=1<<i;
      mirror_solve_R(size,2,(left|bit)<<1,down|bit,(right|bit)>>1);
    }
  }
  TOTAL=TOTAL<<1;    //倍にする
}
// クイーンの効きを判定して解を返す
__host__ __device__ 
long mirror_solve_nodeLayer(int size,long left,long down,long right)
{
  long mask=(1<<size)-1;
  long counter = 0;
  if (down==mask) { // downがすべて専有され解が見つかる
    return 1;
  }
  long bit=0;
  for(long bitmap=mask&~(left|down|right);bitmap;bitmap^=bit){
    bit=-bitmap&bitmap;
    counter += mirror_solve_nodeLayer(size,(left|bit)>>1,(down|bit),(right|bit)<< 1); 
  }
  return counter;
}
// i 番目のメンバを i 番目の部分木の解で埋める
__global__ 
void dim_nodeLayer(int size,long* nodes, long* solutions, int numElements)
{
  int i=blockDim.x * blockIdx.x + threadIdx.x;
  if(i<numElements){
    solutions[i]=mirror_solve_nodeLayer(size,nodes[3 * i],nodes[3 * i + 1],nodes[3 * i + 2]);
  }
}
// 0以外のbitをカウント
int countBits_nodeLayer(long n)
{
  int counter = 0;
  while (n){
    n &= (n - 1); // 右端のゼロ以外の数字を削除
    counter++;
  }
  return counter;
}
// ノードをk番目のレイヤーのノードで埋める
long kLayer_nodeLayer(int size,std::vector<long>& nodes, int k, long left, long down, long right)
{
  long counter=0;
  long mask=(1<<size)-1;
  // すべてのdownが埋まったら、解決策を見つけたことになる。
  if (countBits_nodeLayer(down) == k) {
    nodes.push_back(left);
    nodes.push_back(down);
    nodes.push_back(right);
    return 1;
  }
  long bit=0;
  for(long bitmap=mask&~(left|down|right);bitmap;bitmap^=bit){
    bit=-bitmap&bitmap;
    // 解を加えて対角線をずらす
    counter+=kLayer_nodeLayer(size,nodes,k,(left|bit)>>1,(down|bit),(right|bit)<<1); 
  }
  return counter;
}
// k 番目のレイヤのすべてのノードを含むベクトルを返す。
std::vector<long> kLayer_nodeLayer(int size,int k)
{
  std::vector<long> nodes{};
  kLayer_nodeLayer(size,nodes, k, 0, 0, 0);
  return nodes;
}
// 【GPU ミラー】ノードレイヤーの作成
void mirror_build_nodeLayer(int size)
{
  //int size=16;
  // ツリーの3番目のレイヤーにあるノード
  //（それぞれ連続する3つの数字でエンコードされる）のベクトル。
  // レイヤー2以降はノードの数が均等なので、対称性を利用できる。
  // レイヤ4には十分なノードがある（N16の場合、9844）。
  std::vector<long> nodes = kLayer_nodeLayer(size,4); 

  // デバイスにはクラスがないので、
  // 最初の要素を指定してからデバイスにコピーする。
  size_t nodeSize = nodes.size() * sizeof(long);
  long* hostNodes = (long*)malloc(nodeSize);
  hostNodes = &nodes[0];
  long* deviceNodes = NULL;
  hipMalloc((void**)&deviceNodes, nodeSize);
  hipMemcpy(deviceNodes, hostNodes, nodeSize, hipMemcpyHostToDevice);

  // デバイス出力の割り当て
  long* deviceSolutions = NULL;
  // 必要なのはノードの半分だけで、各ノードは3つの整数で符号化される。
  int numSolutions = nodes.size() / 6; 
  size_t solutionSize = numSolutions * sizeof(long);
  hipMalloc((void**)&deviceSolutions, solutionSize);

  // CUDAカーネルを起動する。
  int threadsPerBlock = 256;
  int blocksPerGrid = (numSolutions + threadsPerBlock - 1) / threadsPerBlock;
  dim_nodeLayer <<<blocksPerGrid, threadsPerBlock >>> (size,deviceNodes, deviceSolutions, numSolutions);

  // 結果をホストにコピー
  long* hostSolutions = (long*)malloc(solutionSize);
  hipMemcpy(hostSolutions, deviceSolutions, solutionSize, hipMemcpyDeviceToHost);

  // 部分解を加算し、結果を表示する。
  long solutions = 0;
  for (long i = 0; i < numSolutions; i++) {
      solutions += 2*hostSolutions[i]; // Symmetry
  }

  // 出力
  //std::cout << "We have " << solutions << " solutions on a " << size << " by " << size << " board." << std::endl;
  TOTAL=solutions;
  //return 0;
}
// CUDA 初期化
bool InitCUDA()
{
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    struct hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
//メイン
int main(int argc,char** argv)
{
  bool cpu=false,cpur=false,gpu=false,gpuNodeLayer=false;
  int argstart=2;
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='r'||argv[1][1]=='R'){cpur=true;}
    else if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){gpu=true;}
    else if(argv[1][1]=='n'||argv[1][1]=='N'){gpuNodeLayer=true;}
    else{ gpuNodeLayer=true; } //デフォルトをgpuとする
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g|-r|-s] n steps\n",argv[0]);
    printf("  -r: CPU 再帰\n");
    printf("  -c: CPU 非再帰\n");
    printf("  -g: GPU 再帰\n");
    printf("  -n: GPU ノードレイヤー\n");
  }
  if(cpur){ printf("\n\nミラー 再帰 \n"); }
  else if(cpu){ printf("\n\nミラー 非再帰 \n"); }
  else if(gpu){ printf("\n\nミラー GPU\n"); }
  else if(gpuNodeLayer){ printf("\n\nミラー GPUノードレイヤー \n"); }
  if(cpu||cpur){
    int min=4; 
    int targetN=17;
    struct timeval t0;
    struct timeval t1;
    printf("%s\n"," N:           Total           Unique          dd:hh:mm:ss.ms");
    for(int size=min;size<=targetN;size++){
      TOTAL=UNIQUE=0;
      gettimeofday(&t0, NULL);//計測開始
      if(cpur){ //再帰
        mirror_R(size);
      }
      if(cpu){ //非再帰
        mirror_NR(size);
      }
      //
      gettimeofday(&t1, NULL);//計測終了
      int ss;int ms;int dd;
      if(t1.tv_usec<t0.tv_usec) {
        dd=(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      }else {
        dd=(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }//end if
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%16ld%17ld%12.2d:%02d:%02d:%02d.%02d\n",
          size,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    } //end for
  }//end if
  if(gpu||gpuNodeLayer){
    if(!InitCUDA()){return 0;}
    /* int steps=24576; */
    int min=4;
    int targetN=21;
    struct timeval t0;
    struct timeval t1;
    printf("%s\n"," N:        Total      Unique      dd:hh:mm:ss.ms");
    for(int size=min;size<=targetN;size++){
      gettimeofday(&t0,NULL);   // 計測開始
      if(gpu){
        TOTAL=UNIQUE=0;
        // GPUは起動するがノードレイヤーは行わない
        TOTAL=mirror_solve_nodeLayer(size,0,0,0); //ミラー
      }else if(gpuNodeLayer){
        TOTAL=UNIQUE=0;
        // GPUを起動し、ノードレイヤーも行う
        mirror_build_nodeLayer(size); // ミラー
      }
      gettimeofday(&t1,NULL);   // 計測終了
      int ss;int ms;int dd;
      if (t1.tv_usec<t0.tv_usec) {
        dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      } else {
        dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }//end if
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%13ld%16ld%4.2d:%02d:%02d:%02d.%02d\n",
          size,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    }//end for
  }//end if
  return 0;
}
