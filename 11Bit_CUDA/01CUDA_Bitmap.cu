/**
 *
 * bash版ビットマップのC言語版のGPU/CUDA移植版
 *
 詳しい説明はこちらをどうぞ
 https://suzukiiichiro.github.io/search/?keyword=Ｎクイーン問題
 *
アーキテクチャの指定（なくても問題なし、あれば高速）
-arch=sm_13 or -arch=sm_61

CPUの再帰での実行
$ nvcc -O3 -arch=sm_61 01CUDA_Bitmap.cu && ./a.out -r

CPUの非再帰での実行
$ nvcc -O3 -arch=sm_61 01CUDA_Bitmap.cu && ./a.out -c

GPUのシングルスレッド
$ nvcc -O3 -arch=sm_61 01CUDA_Bitmap.cu && ./a.out -g

GPUのマルチスレッド
ビットマップ GPUノードレイヤー
$ nvcc -O3 -arch=sm_61 01CUDA_Bitmap.cu && ./a.out -n
 N:        Total      Unique      dd:hh:mm:ss.ms
 4:            2               0  00:00:00:00.15
 5:           10               0  00:00:00:00.00
 6:            4               0  00:00:00:00.00
 7:           40               0  00:00:00:00.00
 8:           92               0  00:00:00:00.00
 9:          352               0  00:00:00:00.00
10:          724               0  00:00:00:00.00
11:         2680               0  00:00:00:00.00
12:        14200               0  00:00:00:00.00
13:        73712               0  00:00:00:00.00
14:       365596               0  00:00:00:00.04
15:      2279184               0  00:00:00:00.21
16:     14772512               0  00:00:00:02.05
17:     95815104               0  00:00:00:19.56
18:    666090624               0  00:00:03:15.21

コメント追加
・kLayer_nodeLayer 
GPUで並列実行するためのleft,right,downを作成する
kLayer_nodeLayer(size,4)
第2引数の4は4行目までnqueenを実行し、それまでのleft,down,rightをnodes配列に格納する

nodesはベクター配列で構造体でもなんでも格納できる
push_backで追加。
nodes配列は3個で１セットleft,dwon,rightの情報を同じ配列に格納する
[0]left[1]down[2]right

・bitmap_build_nodeLayer
  int numSolutions = nodes.size() / 6; 
  3個で1セットなので/3 さらにnodesの2分の1だけ実行すればミラーになるので/6

  
  solutions += 2*hostSolutions[i]; // Symmetry
  GPUごとのTOTALを集計している。ミラー分最後に2倍する

・dim_nodeLayer 
GPU並列処理
bitmap_solve_nodeLayerを再帰呼び出しし、counter(最終行までクイーンを置けると+1)をsolutionsに格納する
solutionsは配列でGPUのステップ数分ある

・bitmap_solve_ndoeLayer
down==maskが最終行までクイーンを置けた状態
ビットだとクイーンを置けない場所に1が立つ
downだとクイーンを置いた場所に1が立つ

maskは、size分1が立っているもの
n8だと11111111

downはクイーンが配置されるたびに配置された列に1が立って行くので最終行までクイーンを置くと全列に1が立った状態になりmaskと同じ内容になる



*/
#include <iostream>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define THREAD_NUM		96
#define MAX 27
// システムによって以下のマクロが必要であればコメントを外してください。
//#define UINT64_C(c) c ## ULL
//
// グローバル変数
unsigned long TOTAL=0; 
unsigned long UNIQUE=0;
// ビットマップ 非再帰版
void bitmap_NR(unsigned int size,int row)
{
  unsigned int mask=(1<<size)-1;
  unsigned int bitmap[size];
  unsigned int bit=0;
  unsigned int left[size];
  unsigned int down[size];
  unsigned int right[size];
  left[0]=0;
  down[0]=0;
  right[0]=0;
  bitmap[row]=mask;
  while(row>-1){
    if(bitmap[row]>0){
      bit=-bitmap[row]&bitmap[row];//一番右のビットを取り出す
      bitmap[row]=bitmap[row]^bit;//配置可能なパターンが一つずつ取り出される
      if(row==(size-1)){
        TOTAL++;
        row--;
      }else{
        unsigned int n=row++;
        left[row]=(left[n]|bit)<<1;
        down[row]=down[n]|bit;
        right[row]=(right[n]|bit)>>1;
        //クイーンが配置可能な位置を表す
        bitmap[row]=mask&~(left[row]|down[row]|right[row]);
      }
    }else{
      row--;
    }
  }//end while
}
// ビットマップ 再帰版
void bitmap_R(unsigned int size,unsigned int row,unsigned int left,unsigned int down, unsigned int right)
{
  unsigned int mask=(1<<size)-1;
  unsigned int bit=0;
  if(row==size){
    TOTAL++;
  }else{
    // クイーンが配置可能な位置を表す
    for(unsigned int bitmap=mask&~(left|down|right);bitmap;bitmap=bitmap&~bit){
      bit=bitmap&-bitmap;
      bitmap_R(size,row+1,(left|bit)<<1,down|bit,(right|bit)>>1);
    }
  }
}
// クイーンの効きを判定して解を返す
__host__ __device__ 
long bitmap_solve_nodeLayer(int size,long left,long down,long right)
{
  long mask=(1<<size)-1;
  long counter = 0;
  if (down==mask) { // downがすべて専有され解が見つかる
    return 1;
  }
  long bit=0;
  for(long bitmap=mask&~(left|down|right);bitmap;bitmap^=bit){
    bit=-bitmap&bitmap;
    counter += bitmap_solve_nodeLayer(size,(left|bit)>>1,(down|bit),(right|bit)<< 1); 
  }
  return counter;
}
// i 番目のメンバを i 番目の部分木の解で埋める
__global__ 
void dim_nodeLayer(int size,long* nodes, long* solutions, int numElements)
{
  int i=blockDim.x * blockIdx.x + threadIdx.x;
  if(i<numElements){
    solutions[i]=bitmap_solve_nodeLayer(size,nodes[3 * i],nodes[3 * i + 1],nodes[3 * i + 2]);
  }
}
// 0以外のbitをカウント
int countBits_nodeLayer(long n)
{
  int counter = 0;
  while (n){
    n &= (n - 1); // 右端のゼロ以外の数字を削除
    counter++;
  }
  return counter;
}
// ノードをk番目のレイヤーのノードで埋める
long kLayer_nodeLayer(int size,std::vector<long>& nodes, int k, long left, long down, long right)
{
  long counter=0;
  long mask=(1<<size)-1;
  // すべてのdownが埋まったら、解決策を見つけたことになる。
  if (countBits_nodeLayer(down) == k) {
    nodes.push_back(left);
    nodes.push_back(down);
    nodes.push_back(right);
    return 1;
  }
  long bit=0;
  for(long bitmap=mask&~(left|down|right);bitmap;bitmap^=bit){
    bit=-bitmap&bitmap;
    // 解を加えて対角線をずらす
    counter+=kLayer_nodeLayer(size,nodes,k,(left|bit)>>1,(down|bit),(right|bit)<<1); 
  }
  return counter;
}
// k 番目のレイヤのすべてのノードを含むベクトルを返す。
std::vector<long> kLayer_nodeLayer(int size,int k)
{
  std::vector<long> nodes{};
  kLayer_nodeLayer(size,nodes, k, 0, 0, 0);
  return nodes;
}
// 【GPU ビットマップ】ノードレイヤーの作成
void bitmap_build_nodeLayer(int size)
{
  //int size=16;
  // ツリーの3番目のレイヤーにあるノード
  //（それぞれ連続する3つの数字でエンコードされる）のベクトル。
  // レイヤー2以降はノードの数が均等なので、対称性を利用できる。
  // レイヤ4には十分なノードがある（N16の場合、9844）。
  std::vector<long> nodes = kLayer_nodeLayer(size,4); 

  // デバイスにはクラスがないので、
  // 最初の要素を指定してからデバイスにコピーする。
  size_t nodeSize = nodes.size() * sizeof(long);
  long* hostNodes = (long*)malloc(nodeSize);
  hostNodes = &nodes[0];
  long* deviceNodes = NULL;
  hipMalloc((void**)&deviceNodes, nodeSize);
  hipMemcpy(deviceNodes, hostNodes, nodeSize, hipMemcpyHostToDevice);

  // デバイス出力の割り当て
  long* deviceSolutions = NULL;
  // 必要なのはノードの半分だけで、各ノードは3つの整数で符号化される。
  int numSolutions = nodes.size() / 6; 
  size_t solutionSize = numSolutions * sizeof(long);
  hipMalloc((void**)&deviceSolutions, solutionSize);

  // CUDAカーネルを起動する。
  int threadsPerBlock = 256;
  int blocksPerGrid = (numSolutions + threadsPerBlock - 1) / threadsPerBlock;
  dim_nodeLayer <<<blocksPerGrid, threadsPerBlock >>> (size,deviceNodes, deviceSolutions, numSolutions);

  // 結果をホストにコピー
  long* hostSolutions = (long*)malloc(solutionSize);
  hipMemcpy(hostSolutions, deviceSolutions, solutionSize, hipMemcpyDeviceToHost);

  // 部分解を加算し、結果を表示する。
  long solutions = 0;
  for (long i = 0; i < numSolutions; i++) {
      solutions += 2*hostSolutions[i]; // Symmetry
  }

  // 出力
  //std::cout << "We have " << solutions << " solutions on a " << size << " by " << size << " board." << std::endl;
  TOTAL=solutions;
  //return 0;
}
// CUDA 初期化
bool InitCUDA()
{
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    struct hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
//メイン
int main(int argc,char** argv)
{
  bool cpu=false,cpur=false,gpu=false,gpuNodeLayer=false;
  int argstart=2;
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='r'||argv[1][1]=='R'){cpur=true;}
    else if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){gpu=true;}
    else if(argv[1][1]=='n'||argv[1][1]=='N'){gpuNodeLayer=true;}
    else{ gpuNodeLayer=true; } //デフォルトをgpuとする
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g|-r|-s] n steps\n",argv[0]);
    printf("  -r: CPU 再帰\n");
    printf("  -c: CPU 非再帰\n");
    printf("  -g: GPU 再帰\n");
    printf("  -n: GPU ノードレイヤー\n");
  }
  if(cpur){ printf("\n\nビットマップ 再帰 \n"); }
  else if(cpu){ printf("\n\nビットマップ 非再帰 \n"); }
  else if(gpu){ printf("\n\nビットマップ GPU\n"); }
  else if(gpuNodeLayer){ printf("\n\nビットマップ GPUノードレイヤー \n"); }
  if(cpu||cpur){
    int min=4; 
    int targetN=17;
    struct timeval t0;
    struct timeval t1;
    printf("%s\n"," N:           Total           Unique          dd:hh:mm:ss.ms");
    for(int size=min;size<=targetN;size++){
      TOTAL=UNIQUE=0;
      gettimeofday(&t0, NULL);//計測開始
      if(cpur){ //再帰
        bitmap_R(size,0,0,0,0);
      }
      if(cpu){ //非再帰
        bitmap_NR(size,0);
      }
      //
      gettimeofday(&t1, NULL);//計測終了
      int ss;int ms;int dd;
      if(t1.tv_usec<t0.tv_usec) {
        dd=(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      }else {
        dd=(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }//end if
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%16ld%17ld%12.2d:%02d:%02d:%02d.%02d\n",
          size,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    } //end for
  }//end if
  if(gpu||gpuNodeLayer){
    if(!InitCUDA()){return 0;}
    /* int steps=24576; */
    int min=4;
    int targetN=21;
    struct timeval t0;
    struct timeval t1;
    printf("%s\n"," N:        Total      Unique      dd:hh:mm:ss.ms");
    for(int size=min;size<=targetN;size++){
      gettimeofday(&t0,NULL);   // 計測開始
      if(gpu){
        TOTAL=UNIQUE=0;
        TOTAL=bitmap_solve_nodeLayer(size,0,0,0); //ビットマップ
      }else if(gpuNodeLayer){
        TOTAL=UNIQUE=0;
        bitmap_build_nodeLayer(size); // ビットマップ
      }
      gettimeofday(&t1,NULL);   // 計測終了
      int ss;int ms;int dd;
      if (t1.tv_usec<t0.tv_usec) {
        dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      } else {
        dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }//end if
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%13ld%16ld%4.2d:%02d:%02d:%02d.%02d\n",
          size,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    }//end for
  }//end if
  return 0;
}
