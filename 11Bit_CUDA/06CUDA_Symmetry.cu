/**
 *
 * bash版対称解除法のC言語版のGPU/CUDA移植版
 *
 詳しい説明はこちらをどうぞ
 https://suzukiiichiro.github.io/search/?keyword=Ｎクイーン問題
 *
*/
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define THREAD_NUM		96
#define MAX 27
// システムによって以下のマクロが必要であればコメントを外してください。
//#define UINT64_C(c) c ## ULL
//
// グローバル変数
unsigned long TOTAL=0; 
unsigned long UNIQUE=0;
int board[MAX];  //ボード配列
unsigned int down[MAX];   //ポストフラグ/ビットマップ/ミラー
unsigned int left[MAX];   //ポストフラグ/ビットマップ/ミラー
unsigned int right[MAX];  //ポストフラグ/ビットマップ/ミラー
unsigned int bitmap[MAX]; //ミラー
unsigned long COUNT2=0;   //ミラー/対称解除法
unsigned long COUNT4=0;   //対称解除法
unsigned long COUNT8=0;   //対称解除法
unsigned int BOUND1=0;    //対称解除法
unsigned int BOUND2=0;    //対称解除法
unsigned int SIDEMASK=0;  //対称解除法
unsigned int LASTMASK=0;  //対称解除法
unsigned int TOPBIT=0;    //対称解除法
unsigned int ENDBIT=0;    //対称解除法
// 対称解除法
void symmetryOps(unsigned int size)
{
  /**
  ２．クイーンが右上角以外にある場合、
  (1) 90度回転させてオリジナルと同型になる場合、さらに90度回転(オリジナルか
  ら180度回転)させても、さらに90度回転(オリジナルから270度回転)させてもオリ
  ジナルと同型になる。
  こちらに該当するユニーク解が属するグループの要素数は、左右反転させたパター
  ンを加えて２個しかありません。
  */
  if(board[BOUND2]==1){
    unsigned int ptn;
    unsigned int own;
    for(ptn=2,own=1;own<size;++own,ptn<<=1){
      unsigned int bit;
      unsigned int you;
      for(bit=1,you=size-1;(board[you]!=ptn)&&board[own]>=bit;--you){
        bit<<=1;
      }
      if(board[own]>bit){
        return ;
      }
      if(board[own]<bit){
        break;
      }
    }//end for
    // ９０度回転して同型なら１８０度回転しても２７０度回転しても同型である
    if(own>size-1){
      COUNT2++;
      return ;
    }//end if
  }//end if
  /**
  ２．クイーンが右上角以外にある場合、
    (2) 90度回転させてオリジナルと異なる場合は、270度回転させても必ずオリジナル
    とは異なる。ただし、180度回転させた場合はオリジナルと同型になることも有り得
    る。こちらに該当するユニーク解が属するグループの要素数は、180度回転させて同
    型になる場合は４個(左右反転×縦横回転)
   */
  //１８０度回転
  if(board[size-1]==ENDBIT){
    unsigned int you;
    unsigned int own;
    for(you=size-1-1,own=1;own<=size-1;++own,--you){
      unsigned int bit;
      unsigned int ptn;
      for(bit=1,ptn=TOPBIT;(ptn!=board[you])&&(board[own]>=bit);ptn>>=1){
        bit<<=1;
      }
      if(board[own]>bit){
        return ;
      }
      if(board[own]<bit){
        break;
      }
    }//end for
    //９０度回転が同型でなくても１８０度回転が同型であることもある
    if(own>size-1){
      COUNT4++;
      return ;
    }
  }//end if
  /**
  ２．クイーンが右上角以外にある場合、
    (3)180度回転させてもオリジナルと異なる場合は、８個(左右反転×縦横回転×上下反転)
  */
  //２７０度回転
  if(board[BOUND1]==TOPBIT){
    unsigned int ptn;
    unsigned int own;
    unsigned int you;
    unsigned int bit;
    for(ptn=TOPBIT>>1,own=1;own<=size-1;++own,ptn>>=1){
      for(bit=1,you=0;(board[you]!=ptn)&&(board[own]>=bit);++you){
        bit<<=1;
      }
      if(board[own]>bit){
        return ;
      }
      if(board[own]<bit){
        break;
      }
    }//end for
  }//end if
  COUNT8++;
}
// 非再帰 角にQがないときのバックトラック
void symmetry_backTrack_NR(unsigned int size,unsigned int row,unsigned int _left,unsigned int _down,unsigned int _right)
{
  unsigned int mask=(1<<size)-1;
  left[row]=_left;
  down[row]=_down;
  right[row]=_right;
  bitmap[row]=mask&~(left[row]|down[row]|right[row]);
  while(row>0){
    if(bitmap[row]>0){
      if(row<BOUND1){ //上部サイド枝刈り
        bitmap[row]|=SIDEMASK;
        bitmap[row]^=SIDEMASK;
      }else if(row==BOUND2){ //下部サイド枝刈り
        if((down[row]&SIDEMASK)==0){
          row--; 
        }
        if((down[row]&SIDEMASK)!=SIDEMASK){
          bitmap[row]&=SIDEMASK;
        }
      }
      unsigned int save_bitmap=bitmap[row];
      unsigned int bit=-bitmap[row]&bitmap[row];
      bitmap[row]^=bit;
      board[row]=bit; //Qを配置
      if((bit&mask)!=0){
        if(row==(size-1)){
          if( (save_bitmap&LASTMASK)==0){
            symmetryOps(size);  //対称解除法
          }
          row--;
        }else{
          unsigned int n=row++;
          left[row]=(left[n]|bit)<<1;
          down[row]=(down[n]|bit);
          right[row]=(right[n]|bit)>>1;
          bitmap[row]=mask&~(left[row]|down[row]|right[row]);
        }
      }else{
        row--;
      }
    }else{
      row--;
    }
  }//end while
}
// 非再帰 角にQがあるときのバックトラック
void symmetry_backTrack_corner_NR(unsigned int size,unsigned int row,unsigned int _left,unsigned int _down, unsigned int _right)
{
  unsigned int mask=(1<<size)-1;
  unsigned int bit=0;
  left[row]=_left;
  down[row]=_down;
  right[row]=_right;
  bitmap[row]=mask&~(left[row]|down[row]|right[row]);
  while(row>=2){
    if(row<BOUND1){
      // bitmap[row]=bitmap[row]|2;
      // bitmap[row]=bitmap[row]^2;
      bitmap[row]&=~2;
    }
    if(bitmap[row]>0){
      bit=-bitmap[row]&bitmap[row];
      bitmap[row]^=bit;
      if(row==(size-1)){
        COUNT8++;
        row--;
      }else{
        unsigned int n=row++;
        left[row]=(left[n]|bit)<<1;
        down[row]=(down[n]|bit);
        right[row]=(right[n]|bit)>>1;
        board[row]=bit; //Qを配置
        //クイーンが配置可能な位置を表す
        bitmap[row]=mask&~(left[row]|down[row]|right[row]);
      }
    }else{
      row--;
    }
  }//end while
}
// 対称解除法 非再帰版
void symmetry_NR(unsigned int size)
{
  TOTAL=UNIQUE=COUNT2=COUNT4=COUNT8=0;
  unsigned int bit=0;
  TOPBIT=1<<(size-1);
  ENDBIT=SIDEMASK=LASTMASK=0;
  BOUND1=2;
  BOUND2=0;
  board[0]=1;
  while(BOUND1>1&&BOUND1<size-1){
    if(BOUND1<size-1){
      bit=1<<BOUND1;
      board[1]=bit;   //２行目にQを配置
      //角にQがあるときのバックトラック
      symmetry_backTrack_corner_NR(size,2,(2|bit)<<1,1|bit,(2|bit)>>1);
    }
    BOUND1++;
  }
  TOPBIT=1<<(size-1);
  ENDBIT=TOPBIT>>1;
  SIDEMASK=TOPBIT|1;
  LASTMASK=TOPBIT|1;
  BOUND1=1;
  BOUND2=size-2;
  while(BOUND1>0 && BOUND2<size-1 && BOUND1<BOUND2){
    if(BOUND1<BOUND2){
      bit=1<<BOUND1;
      board[0]=bit;   //Qを配置
      //角にQがないときのバックトラック
      symmetry_backTrack_NR(size,1,bit<<1,bit,bit>>1);
    }
    BOUND1++;
    BOUND2--;
    ENDBIT=ENDBIT>>1;
    LASTMASK=LASTMASK<<1|LASTMASK|LASTMASK>>1;
  }//ene while
  UNIQUE=COUNT2+COUNT4+COUNT8;
  TOTAL=COUNT2*2+COUNT4*4+COUNT8*8;
}
// 再帰 角にQがないときのバックトラック
void symmetry_backTrack(unsigned int size,unsigned int row,unsigned int left,unsigned int down,unsigned int right)
{
  unsigned int mask=(1<<size)-1;
  unsigned int bitmap=mask&~(left|down|right);
  if(row==(size-1)){
    if(bitmap){
      if( (bitmap&LASTMASK)==0){
        board[row]=bitmap;  //Qを配置
        symmetryOps(size);    //対称解除
      }
    }
  }else{
    if(row<BOUND1){
      bitmap=bitmap|SIDEMASK;
      bitmap=bitmap^SIDEMASK;
    }else{
      if(row==BOUND2){
        if((down&SIDEMASK)==0){
          return;
        }
        if( (down&SIDEMASK)!=SIDEMASK){
          bitmap=bitmap&SIDEMASK;
        }
      }
    }
    while(bitmap){
      unsigned int bit=-bitmap&bitmap;
      bitmap=bitmap^bit;
      board[row]=bit;
      symmetry_backTrack(size,row+1,(left|bit)<<1,down|bit,(right|bit)>>1);
    }//end while
  }//end if
}
// 再帰 角にQがあるときのバックトラック
void symmetry_backTrack_corner(unsigned int size,unsigned int row,unsigned int left,unsigned int down,unsigned int right)
{
  unsigned int mask=(1<<size)-1;
  unsigned int bitmap=mask&~(left|down|right);
  unsigned int bit=0;
  if(row==(size-1)){
    if(bitmap){
      board[row]=bitmap;
      COUNT8++;
    }
  }else{
    if(row<BOUND1){   //枝刈り
      bitmap=bitmap|2;
      bitmap=bitmap^2;
    }
    while(bitmap){
      bit=-bitmap&bitmap;
      bitmap=bitmap^bit;
      board[row]=bit;   //Qを配置
      symmetry_backTrack_corner(size,row+1,(left|bit)<<1,down|bit,(right|bit)>>1);
    }
  }
}
// 対称解除法 再帰版
void symmetry_R(unsigned int size)
{
  TOTAL=UNIQUE=COUNT2=COUNT4=COUNT8=0;
  unsigned int bit=0;
  TOPBIT=1<<(size-1);
  ENDBIT=LASTMASK=SIDEMASK=0;
  BOUND1=2;
  BOUND2=0;
  board[0]=1;
  while(BOUND1>1 && BOUND1<size-1){
    if(BOUND1<size-1){
      bit=1<<BOUND1;
      board[1]=bit;   //２行目にQを配置
      //角にQがあるときのバックトラック
      symmetry_backTrack_corner(size,2,(2|bit)<<1,1|bit,(2|bit)>>1);
    }
    BOUND1++;
  }//end while
  TOPBIT=1<<(size-1);
  ENDBIT=TOPBIT>>1;
  SIDEMASK=TOPBIT|1;
  LASTMASK=TOPBIT|1;
  BOUND1=1;
  BOUND2=size-2;
  while(BOUND1>0 && BOUND2<size-1 && BOUND1<BOUND2){
    if(BOUND1<BOUND2){
      bit=1<<BOUND1;
      board[0]=bit;   //Qを配置
      //角にQがないときのバックトラック
      symmetry_backTrack(size,1,bit<<1,bit,bit>>1);
    }
    BOUND1++;
    BOUND2--;
    ENDBIT=ENDBIT>>1;
    LASTMASK=LASTMASK<<1|LASTMASK|LASTMASK>>1;
  }//ene while
  UNIQUE=COUNT2+COUNT4+COUNT8;
  TOTAL=COUNT2*2+COUNT4*4+COUNT8*8;
}
// CUDA 初期化
bool InitCUDA()
{
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    struct hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
//メイン
int main(int argc,char** argv)
{
  bool cpu=false,cpur=false,gpu=false,sgpu=false;
  int argstart=1;
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='r'||argv[1][1]=='R'){cpur=true;}
    else if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){gpu=true;}
    else{ gpu=true; } //デフォルトをgpuとする
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g|-r|-s] n steps\n",argv[0]);
    printf("  -r: CPUR only\n");
    printf("  -c: CPU only\n");
    printf("  -g: GPU only\n");
  }
  if(cpu){ printf("\n\n対称解除法 非再帰 \n"); }
  else if(cpur){ printf("\n\n対称解除法 再帰 \n"); }
  else if(gpu){ printf("\n\n対称解除法 GPGPU/CUDA\n"); }
  if(cpu||cpur){
    int min=4; 
    int targetN=17;
    struct timeval t0;
    struct timeval t1;
    printf("%s\n"," N:           Total           Unique          dd:hh:mm:ss.ms");
    for(int size=min;size<=targetN;size++){
      TOTAL=UNIQUE=0;
      gettimeofday(&t0, NULL);//計測開始
      if(cpur){ //再帰
        // bluteForce_R(size,0);//ブルートフォース
        // backTracking_R(size,0); //バックトラック
        // postFlag_R(size,0);     //配置フラグ
        // bitmap_R(size,0,0,0,0); //ビットマップ
        // mirror_R(size);         //ミラー
        symmetry_R(size);       //対称解除法
      }
      if(cpu){ //非再帰
        //bluteForce_NR(size,0);//ブルートフォース
        // backTracking_NR(size,0);//バックトラック
        // postFlag_NR(size,0);     //配置フラグ
        // bitmap_NR(size,0);  //ビットマップ
        // mirror_NR(size);         //ミラー
        symmetry_NR(size);       //対称解除法
      }
      //
      gettimeofday(&t1, NULL);//計測終了
      int ss;int ms;int dd;
      if(t1.tv_usec<t0.tv_usec) {
        dd=(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      }else {
        dd=(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }//end if
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%16ld%17ld%12.2d:%02d:%02d:%02d.%02d\n",
          size,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    } //end for
  }//end if
  if(gpu||sgpu){
    if(!InitCUDA()){return 0;}
    /* int steps=24576; */
    int min=4;
    int targetN=21;
    struct timeval t0;
    struct timeval t1;
    printf("%s\n"," N:        Total      Unique      dd:hh:mm:ss.ms");
    for(int i=min;i<=targetN;i++){
      gettimeofday(&t0,NULL);   // 計測開始
      if(gpu){
        TOTAL=UNIQUE=0;
        //
      }
      gettimeofday(&t1,NULL);   // 計測終了
      int ss;int ms;int dd;
      if (t1.tv_usec<t0.tv_usec) {
        dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      } else {
        dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }//end if
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%13ld%16ld%4.2d:%02d:%02d:%02d.%02d\n",
          i,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    }//end for
  }//end if
  return 0;
}
