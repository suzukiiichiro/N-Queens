/**
 *
 * bash版キャリーチェーンのC言語版のGPU/CUDA移植版
 *
 詳しい説明はこちらをどうぞ
 https://suzukiiichiro.github.io/search/?keyword=Ｎクイーン問題
 *
アーキテクチャの指定（なくても問題なし、あれば高速）
-arch=sm_13 or -arch=sm_61

CPUの再帰での実行
$ nvcc -O3 -arch=sm_61 05CUDA_CarryChain.cu && ./a.out -r

CPUの非再帰での実行
$ nvcc -O3 -arch=sm_61 05CUDA_CarryChain.cu && ./a.out -c

GPUのシングルスレッド
$ nvcc -O3 -arch=sm_61 05CUDA_CarryChain.cu && ./a.out -g

GPUのマルチスレッド
$ nvcc -O3 -arch=sm_61 05CUDA_CarryChain.cu && ./a.out -n
*/
#include <iostream>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define THREAD_NUM		96
#define MAX 27
// システムによって以下のマクロが必要であればコメントを外してください。
//#define UINT64_C(c) c ## ULL
//
// グローバル変数
unsigned long TOTAL=0; 
unsigned long UNIQUE=0;
// キャリーチェーン 非再帰版
// 構造体
typedef struct
{
  unsigned int size;
  unsigned int pres_a[930]; 
  unsigned int pres_b[930];
  // uint64_t COUNTER[3];      
  // //カウンター配列
  // unsigned int COUNT2;
  // unsigned int COUNT4;
  // unsigned int COUNT8;
}Global; Global g;
// 構造体
typedef struct Board
{
  uint64_t row;
  uint64_t down;
  uint64_t left;
  uint64_t right;
  long long x[MAX];
}Board ;
typedef struct Local
{
  unsigned int size;
  struct Board B;
  struct Board nB;
  struct Board eB;
  struct Board sB;
  struct Board wB;
  unsigned n;
  unsigned e;
  unsigned s;
  unsigned w;
  uint64_t dimx;
  uint64_t dimy;
  uint64_t COUNTER[3];      
  //カウンター配列
  unsigned int COUNT2;
  unsigned int COUNT4;
  unsigned int COUNT8;
  unsigned int STEPS;
}Local;
/**
  CPU/CPUR 再帰・非再帰共通
  */
// チェーンのリストを作成
void listChain()
{
  unsigned int idx=0;
  for(unsigned int a=0;a<(unsigned)g.size;++a){
    for(unsigned int b=0;b<(unsigned)g.size;++b){
      if(((a>=b)&&(a-b)<=1)||((b>a)&&(b-a)<=1)){ continue; }
      g.pres_a[idx]=a;
      g.pres_b[idx]=b;
      ++idx;
    }
  }
}
/**
  CPU 非再帰
*/
// クイーンの効きをチェック
bool placement(void* args)
{
  Local *l=(Local *)args;
  if(l->B.x[l->dimx]==l->dimy){ return true;  }  
  if (l->B.x[0]==0){
    if (l->B.x[1]!=(uint64_t)-1){
      if((l->B.x[1]>=l->dimx)&&(l->dimy==1)){ return false; }
    }
  }else{
    if( (l->B.x[0]!=(uint64_t)-1) ){
      if(( (l->dimx<l->B.x[0]||l->dimx>=g.size-l->B.x[0])
        && (l->dimy==0 || l->dimy==g.size-1)
      )){ return 0; } 
      if ((  (l->dimx==g.size-1)&&((l->dimy<=l->B.x[0])||
          l->dimy>=g.size-l->B.x[0]))){
        return 0;
      } 
    }
  }
  l->B.x[l->dimx]=l->dimy;                    //xは行 yは列
  uint64_t row=UINT64_C(1)<<l->dimx;
  uint64_t down=UINT64_C(1)<<l->dimy;
  uint64_t left=UINT64_C(1)<<(g.size-1-l->dimx+l->dimy); //右上から左下
  uint64_t right=UINT64_C(1)<<(l->dimx+l->dimy);       // 左上から右下
  if((l->B.row&row)||(l->B.down&down)||(l->B.left&left)||(l->B.right&right)){ return false; }     
  l->B.row|=row; l->B.down|=down; l->B.left|=left; l->B.right|=right;
  return true;
}
//非再帰
uint64_t solve(int size,int current,uint64_t row,uint64_t left,uint64_t down,uint64_t right)
{
  uint64_t row_a[MAX];
  uint64_t right_a[MAX];
  uint64_t left_a[MAX];
  uint64_t down_a[MAX];
  uint64_t bitmap_a[MAX];
  for (int i=0;i<size;i++){
    row_a[i]=0;
    left_a[i]=0;
    down_a[i]=0;
    right_a[i]=0;
    bitmap_a[i]=0;
  }
  row_a[current]=row;
  left_a[current]=left;
  down_a[current]=down;
  right_a[current]=right;
  uint64_t bitmap=bitmap_a[current]=~(left_a[current]|down_a[current]|right_a[current]);
  uint64_t total=0;
  uint64_t bit;

  while(current>-1){
    if((bitmap!=0||row&1)&&current<size){
      if(!(down+1)){
        total++;
        current--;
        row=row_a[current];
        left=left_a[current];
        right=right_a[current];
        down=down_a[current];
        bitmap=bitmap_a[current];
        continue;
      }else if(row&1){
        while( row&1 ){
          row>>=1;
          left<<=1;
          right>>=1;
        }
        bitmap=~(left|down|right);  //再帰に必要な変数は必ず定義する必要があります。
        continue;
      }else{
        bit=-bitmap&bitmap;
        bitmap=bitmap^bit;
        if(current<size){
          row_a[current]=row;
          left_a[current]=left;
          down_a[current]=down;
          right_a[current]=right;
          bitmap_a[current]=bitmap;
          current++;
        }
        row>>=1;      //１行下に移動する
        left=(left|bit)<<1;
        down=down|bit;
        right=(right|bit)>>1;
        bitmap=~(left|down|right);  //再帰に必要な変数は必ず定義する必要があります。
      }
    }else{
      current--;
      row=row_a[current];
      left=left_a[current];
      right=right_a[current];
      down=down_a[current];
      bitmap=bitmap_a[current];
    }
  }
  return total;
}
//非再帰 対称解除法
void carryChain_symmetry(void* args)
{
  Local *l=(Local *)args;
  // 対称解除法
  unsigned const int ww=(g.size-2)*(g.size-1)-1-l->w;
  unsigned const int w2=(g.size-2)*(g.size-1)-1;
  // # 対角線上の反転が小さいかどうか確認する
  if((l->s==ww)&&(l->n<(w2-l->e))){ return ; }
  // # 垂直方向の中心に対する反転が小さいかを確認
  if((l->e==ww)&&(l->n>(w2-l->n))){ return; }
  // # 斜め下方向への反転が小さいかをチェックする
  if((l->n==ww)&&(l->e>(w2-l->s))){ return; }
  // 枝刈り １行目が角の場合回転対称チェックせずCOUNT8にする
  if(l->B.x[0]==0){
    l->COUNTER[l->COUNT8]+=solve(g.size,0,l->B.row>>2,
    l->B.left>>4,((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1,(l->B.right>>4)<<(g.size-5));
    return ;
  }
  // n,e,s==w の場合は最小値を確認する。右回転で同じ場合は、
  // w=n=e=sでなければ値が小さいのでskip  w=n=e=sであれば90度回転で同じ可能性
  if(l->s==l->w){ if((l->n!=l->w)||(l->e!=l->w)){ return; }
    l->COUNTER[l->COUNT2]+=solve(g.size,0,l->B.row>>2,
    l->B.left>>4,((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1,(l->B.right>>4)<<(g.size-5));
    return;
  }
  // e==wは180度回転して同じ 180度回転して同じ時n>=sの時はsmaller?
  if((l->e==l->w)&&(l->n>=l->s)){ if(l->n>l->s){ return; }
    l->COUNTER[l->COUNT4]+=solve(g.size,0,l->B.row>>2,
    l->B.left>>4,((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1,(l->B.right>>4)<<(g.size-5));
    return;
  }
  l->COUNTER[l->COUNT8]+=solve(g.size,0,l->B.row>>2,
  l->B.left>>4,((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1,(l->B.right>>4)<<(g.size-5));
  return;
}
//非再帰  pthread run()
void thread_run(void* args)
{
  Local *l=(Local *)args;

  // memcpy(&l->B,&l->wB,sizeof(Board));       // B=wB;
  l->B=l->wB;
  l->dimx=0; l->dimy=g.pres_a[l->w];
  //if(!placement(l)){ continue; }
  if(!placement(l)){ return; }
  l->dimx=1; l->dimy=g.pres_b[l->w];
  // if(!placement(l)){ continue; }
  if(!placement(l)){ return; }
  //２ 左２行に置く
  // memcpy(&l->nB,&l->B,sizeof(Board));       // nB=B;
  l->nB=l->B;
  for(l->n=l->w;l->n<(g.size-2)*(g.size-1)-l->w;++l->n){
    // memcpy(&l->B,&l->nB,sizeof(Board));     // B=nB;
    l->B=l->nB;
    l->dimx=g.pres_a[l->n]; l->dimy=g.size-1;
    if(!placement(l)){ continue; }
    l->dimx=g.pres_b[l->n]; l->dimy=g.size-2;
    if(!placement(l)){ continue; }
    // ３ 下２行に置く
    // memcpy(&l->eB,&l->B,sizeof(Board));     // eB=B;
    l->eB=l->B;
    for(l->e=l->w;l->e<(g.size-2)*(g.size-1)-l->w;++l->e){
      // memcpy(&l->B,&l->eB,sizeof(Board));   // B=eB;
      l->B=l->eB;
      l->dimx=g.size-1; l->dimy=g.size-1-g.pres_a[l->e];
      if(!placement(l)){ continue; }
      l->dimx=g.size-2; l->dimy=g.size-1-g.pres_b[l->e];
      if(!placement(l)){ continue; }
      // ４ 右２列に置く
      // memcpy(&l->sB,&l->B,sizeof(Board));   // sB=B;
      l->sB=l->B;
      for(l->s=l->w;l->s<(g.size-2)*(g.size-1)-l->w;++l->s){
        // memcpy(&l->B,&l->sB,sizeof(Board)); // B=sB;
        l->B=l->sB;
        l->dimx=g.size-1-g.pres_a[l->s]; l->dimy=0;
        if(!placement(l)){ continue; }
        l->dimx=g.size-1-g.pres_b[l->s]; l->dimy=1;
        if(!placement(l)){ continue; }
        // 対称解除法
        carryChain_symmetry(l);
      } //w
    } //e
  } //n
}
//非再帰  チェーンのビルド
void buildChain()
{
  Local l[(g.size/2)*(g.size-3)];

  // カウンターの初期化
  l->COUNT2=0; l->COUNT4=1; l->COUNT8=2;
  l->COUNTER[l->COUNT2]=l->COUNTER[l->COUNT4]=l->COUNTER[l->COUNT8]=0;
  // Board の初期化 nB,eB,sB,wB;
  l->B.row=l->B.down=l->B.left=l->B.right=0;
  // Board x[]の初期化
  for(unsigned int i=0;i<g.size;++i){ l->B.x[i]=-1; }
  //１ 上２行に置く
  // memcpy(&l->wB,&l->B,sizeof(Board));         // wB=B;
  l->wB=l->B;
  for(l->w=0;l->w<=(unsigned)(g.size/2)*(g.size-3);++l->w){
    thread_run(&l);
  } //w
  /**
   * 集計
   */
  UNIQUE= l->COUNTER[l->COUNT2]+
          l->COUNTER[l->COUNT4]+
          l->COUNTER[l->COUNT8];
  TOTAL=  l->COUNTER[l->COUNT2]*2+
          l->COUNTER[l->COUNT4]*4+
          l->COUNTER[l->COUNT8]*8;
}
//非再帰  キャリーチェーン
void carryChain()
{
  listChain();  //チェーンのリストを作成
  buildChain(); // チェーンのビルド
  // calcChain(&l);  // 集計
}
/**
  CPUR 再帰
  */
//再帰 ボード外側２列を除く内側のクイーン配置処理
uint64_t solveR(uint64_t row,uint64_t left,uint64_t down,uint64_t right)
{
  if(down+1==0){ return  1; }
  while((row&1)!=0) { 
    row>>=1;
    left<<=1;
    right>>=1;
  }
  row>>=1;
  uint64_t total=0;
  for(uint64_t carryChain=~(left|down|right);carryChain!=0;){
    uint64_t const bit=carryChain&-carryChain;
    total+=solveR(row,(left|bit)<<1,down|bit,(right|bit)>>1);
    carryChain^=bit;
  }
  return total;
} 
//再帰 対称解除法
void carryChain_symmetryR(void* args)
{
  Local *l=(Local *)args;
  // 対称解除法
  unsigned const int ww=(g.size-2)*(g.size-1)-1-l->w;
  unsigned const int w2=(g.size-2)*(g.size-1)-1;
  // # 対角線上の反転が小さいかどうか確認する
  if((l->s==ww)&&(l->n<(w2-l->e))){ return ; }
  // # 垂直方向の中心に対する反転が小さいかを確認
  if((l->e==ww)&&(l->n>(w2-l->n))){ return; }
  // # 斜め下方向への反転が小さいかをチェックする
  if((l->n==ww)&&(l->e>(w2-l->s))){ return; }
  // 枝刈り １行目が角の場合回転対称チェックせずCOUNT8にする
  if(l->B.x[0]==0){
    l->COUNTER[l->COUNT8]+=solveR(l->B.row>>2,
    l->B.left>>4,((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1,(l->B.right>>4)<<(g.size-5));
    return ;
  }
  // n,e,s==w の場合は最小値を確認する。右回転で同じ場合は、
  // w=n=e=sでなければ値が小さいのでskip  w=n=e=sであれば90度回転で同じ可能性
  if(l->s==l->w){ if((l->n!=l->w)||(l->e!=l->w)){ return; }
    l->COUNTER[l->COUNT2]+=solveR(l->B.row>>2,
    l->B.left>>4,((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1,(l->B.right>>4)<<(g.size-5));
    return;
  }
  // e==wは180度回転して同じ 180度回転して同じ時n>=sの時はsmaller?
  if((l->e==l->w)&&(l->n>=l->s)){ if(l->n>l->s){ return; }
    l->COUNTER[l->COUNT4]+=solveR(l->B.row>>2,
    l->B.left>>4,((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1,(l->B.right>>4)<<(g.size-5));
    return;
  }
  l->COUNTER[l->COUNT8]+=solveR(l->B.row>>2,
  l->B.left>>4,((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1,(l->B.right>>4)<<(g.size-5));
  return;
}
//再帰  pthread run()
void thread_runR(void* args)
{
  Local *l=(Local *)args;

  // memcpy(&l->B,&l->wB,sizeof(Board));       // B=wB;
  l->B=l->wB;
  l->dimx=0; l->dimy=g.pres_a[l->w];
  //if(!placement(l)){ continue; }
  if(!placement(l)){ return; }
  l->dimx=1; l->dimy=g.pres_b[l->w];
  // if(!placement(l)){ continue; }
  if(!placement(l)){ return; }
  //２ 左２行に置く
  // memcpy(&l->nB,&l->B,sizeof(Board));       // nB=B;
  l->nB=l->B;
  for(l->n=l->w;l->n<(g.size-2)*(g.size-1)-l->w;++l->n){
    // memcpy(&l->B,&l->nB,sizeof(Board));     // B=nB;
    l->B=l->nB;
    l->dimx=g.pres_a[l->n]; l->dimy=g.size-1;
    if(!placement(l)){ continue; }
    l->dimx=g.pres_b[l->n]; l->dimy=g.size-2;
    if(!placement(l)){ continue; }
    // ３ 下２行に置く
    // memcpy(&l->eB,&l->B,sizeof(Board));     // eB=B;
    l->eB=l->B;
    for(l->e=l->w;l->e<(g.size-2)*(g.size-1)-l->w;++l->e){
      // memcpy(&l->B,&l->eB,sizeof(Board));   // B=eB;
      l->B=l->eB;
      l->dimx=g.size-1; l->dimy=g.size-1-g.pres_a[l->e];
      if(!placement(l)){ continue; }
      l->dimx=g.size-2; l->dimy=g.size-1-g.pres_b[l->e];
      if(!placement(l)){ continue; }
      // ４ 右２列に置く
      // memcpy(&l->sB,&l->B,sizeof(Board));   // sB=B;
      l->sB=l->B;
      for(l->s=l->w;l->s<(g.size-2)*(g.size-1)-l->w;++l->s){
        // memcpy(&l->B,&l->sB,sizeof(Board)); // B=sB;
        l->B=l->sB;
        l->dimx=g.size-1-g.pres_a[l->s]; l->dimy=0;
        if(!placement(l)){ continue; }
        l->dimx=g.size-1-g.pres_b[l->s]; l->dimy=1;
        if(!placement(l)){ continue; }
        // 対称解除法
        carryChain_symmetryR(l);
      } //w
    } //e
  } //n
}
//再帰  チェーンのビルド
void buildChainR()
{
  Local l[(g.size/2)*(g.size-3)];

  // カウンターの初期化
  l->COUNT2=0; l->COUNT4=1; l->COUNT8=2;
  l->COUNTER[l->COUNT2]=l->COUNTER[l->COUNT4]=l->COUNTER[l->COUNT8]=0;
  // Board の初期化 nB,eB,sB,wB;
  l->B.row=l->B.down=l->B.left=l->B.right=0;
  // Board x[]の初期化
  for(unsigned int i=0;i<g.size;++i){ l->B.x[i]=-1; }
  //１ 上２行に置く
  // memcpy(&l->wB,&l->B,sizeof(Board));         // wB=B;
  l->wB=l->B;
  for(l->w=0;l->w<=(unsigned)(g.size/2)*(g.size-3);++l->w){
    thread_runR(&l);
  } //w
  /**
   * 集計
   */
  UNIQUE= l->COUNTER[l->COUNT2]+
          l->COUNTER[l->COUNT4]+
          l->COUNTER[l->COUNT8];
  TOTAL=  l->COUNTER[l->COUNT2]*2+
          l->COUNTER[l->COUNT4]*4+
          l->COUNTER[l->COUNT8]*8;
}
//再帰  キャリーチェーン
void carryChainR()
{
  listChain();  //チェーンのリストを作成
  buildChainR(); // チェーンのビルド
  // calcChain(&l);  // 集計
}
/**
  GPU 
 */
// GPU クイーンの効きをチェック
bool GPU_placement(void* args)
{
  Local *l=(Local *)args;
  if(l->B.x[l->dimx]==l->dimy){ return true;  }  
  if (l->B.x[0]==0){
    if (l->B.x[1]!=(uint64_t)-1){
      if((l->B.x[1]>=l->dimx)&&(l->dimy==1)){ return false; }
    }
  }else{
    if( (l->B.x[0]!=(uint64_t)-1) ){
      if(( (l->dimx<l->B.x[0]||l->dimx>=g.size-l->B.x[0])
        && (l->dimy==0 || l->dimy==g.size-1)
      )){ return 0; } 
      if ((  (l->dimx==g.size-1)&&((l->dimy<=l->B.x[0])||
          l->dimy>=g.size-l->B.x[0]))){
        return 0;
      } 
    }
  }
  l->B.x[l->dimx]=l->dimy;                    //xは行 yは列
  uint64_t row=UINT64_C(1)<<l->dimx;
  uint64_t down=UINT64_C(1)<<l->dimy;
  uint64_t left=UINT64_C(1)<<(g.size-1-l->dimx+l->dimy); //右上から左下
  uint64_t right=UINT64_C(1)<<(l->dimx+l->dimy);       // 左上から右下
  if((l->B.row&row)||(l->B.down&down)||(l->B.left&left)||(l->B.right&right)){ return false; }     
  l->B.row|=row; l->B.down|=down; l->B.left|=left; l->B.right|=right;
  return true;
}
//GPU 再帰 ボード外側２列を除く内側のクイーン配置処理
uint64_t GPU_solveR(uint64_t row,uint64_t left,uint64_t down,uint64_t right)
{
  if(down+1==0){ return  1; }
  while((row&1)!=0) { 
    row>>=1;
    left<<=1;
    right>>=1;
  }
  row>>=1;
  uint64_t total=0;
  for(uint64_t carryChain=~(left|down|right);carryChain!=0;){
    uint64_t const bit=carryChain&-carryChain;
    total+=solveR(row,(left|bit)<<1,down|bit,(right|bit)>>1);
    carryChain^=bit;
  }
  return total;
} 
//GPU 再帰 対称解除法
void GPU_carryChain_symmetryR(void* args)
{
  Local *l=(Local *)args;
  // 対称解除法
  unsigned const int ww=(g.size-2)*(g.size-1)-1-l->w;
  unsigned const int w2=(g.size-2)*(g.size-1)-1;
  // # 対角線上の反転が小さいかどうか確認する
  if((l->s==ww)&&(l->n<(w2-l->e))){ return ; }
  // # 垂直方向の中心に対する反転が小さいかを確認
  if((l->e==ww)&&(l->n>(w2-l->n))){ return; }
  // # 斜め下方向への反転が小さいかをチェックする
  if((l->n==ww)&&(l->e>(w2-l->s))){ return; }
  // 枝刈り １行目が角の場合回転対称チェックせずCOUNT8にする
  if(l->B.x[0]==0){
    l->COUNTER[l->COUNT8]+=solveR(l->B.row>>2,
    l->B.left>>4,((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1,(l->B.right>>4)<<(g.size-5));
    return ;
  }
  // n,e,s==w の場合は最小値を確認する。右回転で同じ場合は、
  // w=n=e=sでなければ値が小さいのでskip  w=n=e=sであれば90度回転で同じ可能性
  if(l->s==l->w){ if((l->n!=l->w)||(l->e!=l->w)){ return; }
    l->COUNTER[l->COUNT2]+=solveR(l->B.row>>2,
    l->B.left>>4,((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1,(l->B.right>>4)<<(g.size-5));
    return;
  }
  // e==wは180度回転して同じ 180度回転して同じ時n>=sの時はsmaller?
  if((l->e==l->w)&&(l->n>=l->s)){ if(l->n>l->s){ return; }
    l->COUNTER[l->COUNT4]+=solveR(l->B.row>>2,
    l->B.left>>4,((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1,(l->B.right>>4)<<(g.size-5));
    return;
  }
  l->COUNTER[l->COUNT8]+=solveR(l->B.row>>2,
  l->B.left>>4,((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1,(l->B.right>>4)<<(g.size-5));
  return;
}
//GPU 再帰  pthread run()
void GPU_thread_runR(void* args)
{
  Local *l=(Local *)args;

  // memcpy(&l->B,&l->wB,sizeof(Board));       // B=wB;
  l->B=l->wB;
  l->dimx=0; l->dimy=g.pres_a[l->w];
  //if(!GPU_placement(l)){ continue; }
  if(!GPU_placement(l)){ return; }
  l->dimx=1; l->dimy=g.pres_b[l->w];
  // if(!GPU_placement(l)){ continue; }
  if(!GPU_placement(l)){ return; }
  //２ 左２行に置く
  // memcpy(&l->nB,&l->B,sizeof(Board));       // nB=B;
  l->nB=l->B;
  for(l->n=l->w;l->n<(g.size-2)*(g.size-1)-l->w;++l->n){
    // memcpy(&l->B,&l->nB,sizeof(Board));     // B=nB;
    l->B=l->nB;
    l->dimx=g.pres_a[l->n]; l->dimy=g.size-1;
    if(!GPU_placement(l)){ continue; }
    l->dimx=g.pres_b[l->n]; l->dimy=g.size-2;
    if(!GPU_placement(l)){ continue; }
    // ３ 下２行に置く
    // memcpy(&l->eB,&l->B,sizeof(Board));     // eB=B;
    l->eB=l->B;
    for(l->e=l->w;l->e<(g.size-2)*(g.size-1)-l->w;++l->e){
      // memcpy(&l->B,&l->eB,sizeof(Board));   // B=eB;
      l->B=l->eB;
      l->dimx=g.size-1; l->dimy=g.size-1-g.pres_a[l->e];
      if(!GPU_placement(l)){ continue; }
      l->dimx=g.size-2; l->dimy=g.size-1-g.pres_b[l->e];
      if(!GPU_placement(l)){ continue; }
      // ４ 右２列に置く
      // memcpy(&l->sB,&l->B,sizeof(Board));   // sB=B;
      l->sB=l->B;
      for(l->s=l->w;l->s<(g.size-2)*(g.size-1)-l->w;++l->s){
        // memcpy(&l->B,&l->sB,sizeof(Board)); // B=sB;
        l->B=l->sB;
        l->dimx=g.size-1-g.pres_a[l->s]; l->dimy=0;
        if(!GPU_placement(l)){ continue; }
        l->dimx=g.size-1-g.pres_b[l->s]; l->dimy=1;
        if(!GPU_placement(l)){ continue; }
        // 対称解除法
        carryChain_symmetryR(l);
      } //w
    } //e
  } //n
}
//GPU 再帰  チェーンのビルド
void GPU_buildChainR(const unsigned int size,unsigned int STEPS)
{
  Local l[(g.size/2)*(g.size-3)];
  l->STEPS=STEPS;
  l->size=size;
  Local lDevice[(g.size/2)*(g.size-3)];
  hipHostMalloc((void**) &l,   sizeof(struct Local)*l->STEPS, hipHostMallocDefault);
  hipMalloc((void**) &lDevice, sizeof(struct Local)*l->STEPS);

  // カウンターの初期化
  l->COUNT2=0; l->COUNT4=1; l->COUNT8=2;
  l->COUNTER[l->COUNT2]=l->COUNTER[l->COUNT4]=l->COUNTER[l->COUNT8]=0;
  // Board の初期化 nB,eB,sB,wB;
  l->B.row=l->B.down=l->B.left=l->B.right=0;
  // Board x[]の初期化
  for(unsigned int i=0;i<g.size;++i){ l->B.x[i]=-1; }
  //１ 上２行に置く
  // memcpy(&l->wB,&l->B,sizeof(Board));         // wB=B;
  l->wB=l->B;
  unsigned int limit=(unsigned)(g.size/2)*(g.size-3);
  hipMemcpy(lDevice,l,
      sizeof(struct Local)*limit,hipMemcpyHostToDevice);
  for(l->w=0;l->w<=(unsigned)(g.size/2)*(g.size-3);++l->w){
    thread_runR(&l);
    //GPU_thread_runR<<<l->STEPS/THREAD_NUM,THREAD_NUM>>>(&l);
  } //w
  hipMemcpy(l,lDevice,
      sizeof(struct Local)*limit,hipMemcpyDeviceToHost);
  /**
   * 集計
   */
  UNIQUE= l->COUNTER[l->COUNT2]+
          l->COUNTER[l->COUNT4]+
          l->COUNTER[l->COUNT8];
  TOTAL=  l->COUNTER[l->COUNT2]*2+
          l->COUNTER[l->COUNT4]*4+
          l->COUNTER[l->COUNT8]*8;
}
//GPU 再帰  キャリーチェーン
void GPU_carryChainR(const unsigned int size,unsigned int STEPS)
{
  listChain();  //チェーンのリストを作成
  GPU_buildChainR(size,STEPS); // チェーンのビルド
  // calcChain(&l);  // 集計
}
// CUDA 初期化
bool InitCUDA()
{
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    struct hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
//メイン
int main(int argc,char** argv)
{
  bool cpu=false,cpur=false,gpu=false,gpuNodeLayer=false;
  int argstart=2;
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='r'||argv[1][1]=='R'){cpur=true;}
    else if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){gpu=true;}
    else if(argv[1][1]=='n'||argv[1][1]=='N'){gpuNodeLayer=true;}
    else{ gpuNodeLayer=true; } //デフォルトをgpuとする
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g|-r|-s] n steps\n",argv[0]);
    printf("  -r: CPU 再帰\n");
    printf("  -c: CPU 非再帰\n");
    printf("  -g: GPU 再帰\n");
    printf("  -n: GPU キャリーチェーン\n");
  }
  if(cpur){ printf("\n\nCPU キャリーチェーン 再帰 \n"); }
  else if(cpu){ printf("\n\nCPU キャリーチェーン 非再帰 \n"); }
  else if(gpu){ printf("\n\nGPU キャリーチェーン シングルスレッド\n"); }
  else if(gpuNodeLayer){ printf("\n\nGPU キャリーチェーン マルチスレッド\n"); }
  if(cpu||cpur)
  {
    int min=4; 
    int targetN=17;
    struct timeval t0;
    struct timeval t1;
    printf("%s\n"," N:        Total      Unique      dd:hh:mm:ss.ms");
    for(int size=min;size<=targetN;size++){
      TOTAL=UNIQUE=0;
      gettimeofday(&t0, NULL);//計測開始
      if(cpur){ //再帰
        g.size=size;
        carryChainR();
      }
      if(cpu){ //非再帰
        g.size=size;
        carryChain();
      }
      //
      gettimeofday(&t1, NULL);//計測終了
      int ss;int ms;int dd;
      if(t1.tv_usec<t0.tv_usec) {
        dd=(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      }else {
        dd=(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }//end if
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%13ld%12ld%8.2d:%02d:%02d:%02d.%02d\n",size,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    } //end for
  }//end if
  if(gpu||gpuNodeLayer)
  {
    if(!InitCUDA()){return 0;}
    int STEPS=24576;
    int min=4;
    int targetN=21;
    struct timeval t0;
    struct timeval t1;
    printf("%s\n"," N:        Total      Unique      dd:hh:mm:ss.ms");
    for(int size=min;size<=targetN;size++){
      gettimeofday(&t0,NULL);   // 計測開始
      if(gpu){
        TOTAL=UNIQUE=0;
        g.size=size;
        GPU_carryChainR(size,STEPS); //キャリーチェーン
      }else if(gpuNodeLayer){
        TOTAL=UNIQUE=0;
        g.size=size;
        GPU_carryChainR(size,STEPS); // キャリーチェーン
      }
      gettimeofday(&t1,NULL);   // 計測終了
      int ss;int ms;int dd;
      if (t1.tv_usec<t0.tv_usec) {
        dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      } else {
        dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }//end if
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%13ld%12ld%8.2d:%02d:%02d:%02d.%02d\n",size,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    }//end for
  }//end if
  return 0;
}
