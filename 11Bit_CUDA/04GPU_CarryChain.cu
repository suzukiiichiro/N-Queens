/**
 *
 * bash版キャリーチェーンのC言語版のGPU/CUDA移植版
 *
 詳しい説明はこちらをどうぞ
 https://suzukiiichiro.github.io/search/?keyword=Ｎクイーン問題
 *
*/
#include <iostream>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define THREAD_NUM		96
#define MAX 27
// システムによって以下のマクロが必要であればコメントを外してください。
//#define UINT64_C(c) c ## ULL
//
// グローバル変数
unsigned long TOTAL=0; 
unsigned long UNIQUE=0;
// キャリーチェーン 非再帰版
// 構造体
typedef struct{
  unsigned int size;
  unsigned int pres_a[930]; 
  unsigned int pres_b[930];
  // uint64_t COUNTER[3];      
  // //カウンター配列
  // unsigned int COUNT2;
  // unsigned int COUNT4;
  // unsigned int COUNT8;
}Global; Global g;
// 構造体
typedef struct{
  uint64_t row;
  uint64_t down;
  uint64_t left;
  uint64_t right;
  long long x[MAX];
}Board ;
typedef struct{
  Board B;
  Board nB;
  Board eB;
  Board sB;
  Board wB;
  unsigned n;
  unsigned e;
  unsigned s;
  unsigned w;
  uint64_t dimx;
  uint64_t dimy;
  uint64_t COUNTER[3];      
  //カウンター配列
  unsigned int COUNT2;
  unsigned int COUNT4;
  unsigned int COUNT8;
}Local;
// ボード外側２列を除く内側のクイーン配置処理
uint64_t solve(uint64_t row,uint64_t left,uint64_t down,uint64_t right)
{
  if(down+1==0){ return  1; }
  while((row&1)!=0) { 
    row>>=1;
    left<<=1;
    right>>=1;
  }
  row>>=1;
  uint64_t total=0;
  for(uint64_t carryChain=~(left|down|right);carryChain!=0;){
    uint64_t const bit=carryChain&-carryChain;
    total+=solve(row,(left|bit)<<1,down|bit,(right|bit)>>1);
    carryChain^=bit;
  }
  return total;
} 
// クイーンの効きをチェック
bool placement(void* args)
{
  Local *l=(Local *)args;
  if(l->B.x[l->dimx]==l->dimy){ return true;  }  
  if (l->B.x[0]==0){
    if (l->B.x[1]!=(uint64_t)-1){
      if((l->B.x[1]>=l->dimx)&&(l->dimy==1)){ return false; }
    }
  }else{
    if( (l->B.x[0]!=(uint64_t)-1) ){
      if(( (l->dimx<l->B.x[0]||l->dimx>=g.size-l->B.x[0])
        && (l->dimy==0 || l->dimy==g.size-1)
      )){ return 0; } 
      if ((  (l->dimx==g.size-1)&&((l->dimy<=l->B.x[0])||
          l->dimy>=g.size-l->B.x[0]))){
        return 0;
      } 
    }
  }
  l->B.x[l->dimx]=l->dimy;                    //xは行 yは列
  uint64_t row=UINT64_C(1)<<l->dimx;
  uint64_t down=UINT64_C(1)<<l->dimy;
  uint64_t left=UINT64_C(1)<<(g.size-1-l->dimx+l->dimy); //右上から左下
  uint64_t right=UINT64_C(1)<<(l->dimx+l->dimy);       // 左上から右下
  if((l->B.row&row)||(l->B.down&down)||(l->B.left&left)||(l->B.right&right)){ return false; }     
  l->B.row|=row; l->B.down|=down; l->B.left|=left; l->B.right|=right;
  return true;
}
//対称解除法
void carryChain_symmetry(void* args)
{
  Local *l=(Local *)args;
  // 対称解除法 
  unsigned const int ww=(g.size-2)*(g.size-1)-1-l->w;
  unsigned const int w2=(g.size-2)*(g.size-1)-1;
  // # 対角線上の反転が小さいかどうか確認する
  if((l->s==ww)&&(l->n<(w2-l->e))){ return ; }
  // # 垂直方向の中心に対する反転が小さいかを確認
  if((l->e==ww)&&(l->n>(w2-l->n))){ return; }
  // # 斜め下方向への反転が小さいかをチェックする
  if((l->n==ww)&&(l->e>(w2-l->s))){ return; }
  // 枝刈り １行目が角の場合回転対称チェックせずCOUNT8にする
  if(l->B.x[0]==0){ 
    l->COUNTER[l->COUNT8]+=solve(l->B.row>>2,
    l->B.left>>4,((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1,(l->B.right>>4)<<(g.size-5));
    return ;
  }
  // n,e,s==w の場合は最小値を確認する。右回転で同じ場合は、
  // w=n=e=sでなければ値が小さいのでskip  w=n=e=sであれば90度回転で同じ可能性
  if(l->s==l->w){ if((l->n!=l->w)||(l->e!=l->w)){ return; } 
    l->COUNTER[l->COUNT2]+=solve(l->B.row>>2,
    l->B.left>>4,((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1,(l->B.right>>4)<<(g.size-5));
    return;
  }
  // e==wは180度回転して同じ 180度回転して同じ時n>=sの時はsmaller?
  if((l->e==l->w)&&(l->n>=l->s)){ if(l->n>l->s){ return; } 
    l->COUNTER[l->COUNT4]+=solve(l->B.row>>2,
    l->B.left>>4,((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1,(l->B.right>>4)<<(g.size-5));
    return;
  }
  l->COUNTER[l->COUNT8]+=solve(l->B.row>>2,
  l->B.left>>4,((((l->B.down>>2)|(~0<<(g.size-4)))+1)<<(g.size-5))-1,(l->B.right>>4)<<(g.size-5));
  return;
}
// pthread run()
void thread_run(void* args)
{
  Local *l=(Local *)args;

  // memcpy(&l->B,&l->wB,sizeof(Board));       // B=wB;
  l->B=l->wB;
  l->dimx=0; l->dimy=g.pres_a[l->w]; 
  //if(!placement(l)){ continue; } 
  if(!placement(l)){ return; } 
  l->dimx=1; l->dimy=g.pres_b[l->w]; 
  // if(!placement(l)){ continue; } 
  if(!placement(l)){ return; } 
  //２ 左２行に置く
  // memcpy(&l->nB,&l->B,sizeof(Board));       // nB=B;
  l->nB=l->B;
  for(l->n=l->w;l->n<(g.size-2)*(g.size-1)-l->w;++l->n){
    // memcpy(&l->B,&l->nB,sizeof(Board));     // B=nB;
    l->B=l->nB;
    l->dimx=g.pres_a[l->n]; l->dimy=g.size-1; 
    if(!placement(l)){ continue; } 
    l->dimx=g.pres_b[l->n]; l->dimy=g.size-2; 
    if(!placement(l)){ continue; } 
    // ３ 下２行に置く
    // memcpy(&l->eB,&l->B,sizeof(Board));     // eB=B;
    l->eB=l->B;
    for(l->e=l->w;l->e<(g.size-2)*(g.size-1)-l->w;++l->e){
      // memcpy(&l->B,&l->eB,sizeof(Board));   // B=eB;
      l->B=l->eB;
      l->dimx=g.size-1; l->dimy=g.size-1-g.pres_a[l->e]; 
      if(!placement(l)){ continue; } 
      l->dimx=g.size-2; l->dimy=g.size-1-g.pres_b[l->e]; 
      if(!placement(l)){ continue; } 
      // ４ 右２列に置く
      // memcpy(&l->sB,&l->B,sizeof(Board));   // sB=B;
      l->sB=l->B;
      for(l->s=l->w;l->s<(g.size-2)*(g.size-1)-l->w;++l->s){
        // memcpy(&l->B,&l->sB,sizeof(Board)); // B=sB;
        l->B=l->sB;
        l->dimx=g.size-1-g.pres_a[l->s]; l->dimy=0; 
        if(!placement(l)){ continue; } 
        l->dimx=g.size-1-g.pres_b[l->s]; l->dimy=1; 
        if(!placement(l)){ continue; } 
        // 対称解除法
        carryChain_symmetry(l);
      } //w
    } //e
  } //n
}
// チェーンのビルド
void buildChain()
{
  Local l[(g.size/2)*(g.size-3)];

  // カウンターの初期化
  l->COUNT2=0; l->COUNT4=1; l->COUNT8=2;
  l->COUNTER[l->COUNT2]=l->COUNTER[l->COUNT4]=l->COUNTER[l->COUNT8]=0;
  // Board の初期化 nB,eB,sB,wB;
  l->B.row=l->B.down=l->B.left=l->B.right=0;
  // Board x[]の初期化
  for(unsigned int i=0;i<g.size;++i){ l->B.x[i]=-1; }
  //１ 上２行に置く
  // memcpy(&l->wB,&l->B,sizeof(Board));         // wB=B;
  l->wB=l->B;
  for(l->w=0;l->w<=(unsigned)(g.size/2)*(g.size-3);++l->w){
    thread_run(&l);
  } //w
  /**
   * 集計
   */
  UNIQUE= l->COUNTER[l->COUNT2]+
          l->COUNTER[l->COUNT4]+
          l->COUNTER[l->COUNT8];
  TOTAL=  l->COUNTER[l->COUNT2]*2+
          l->COUNTER[l->COUNT4]*4+
          l->COUNTER[l->COUNT8]*8;
}
// チェーンのリストを作成
void listChain()
{
  unsigned int idx=0;
  for(unsigned int a=0;a<(unsigned)g.size;++a){
    for(unsigned int b=0;b<(unsigned)g.size;++b){
      if(((a>=b)&&(a-b)<=1)||((b>a)&&(b-a)<=1)){ continue; }
      g.pres_a[idx]=a;
      g.pres_b[idx]=b;
      ++idx;
    }
  }
}
// キャリーチェーン
void carryChain()
{
  listChain();  //チェーンのリストを作成
  buildChain(); // チェーンのビルド
  // calcChain(&l);  // 集計
}
// クイーンの効きを判定して解を返す
__host__ __device__ 
long carryChain_solve_nodeLayer(int size,long left,long down,long right)
{
  long mask=(1<<size)-1;
  long counter = 0;
  if (down==mask) { // downがすべて専有され解が見つかる
    return 1;
  }
  long bit=0;
  for(long carryChain=mask&~(left|down|right);carryChain;carryChain^=bit){
    bit=-carryChain&carryChain;
    counter += carryChain_solve_nodeLayer(size,(left|bit)>>1,(down|bit),(right|bit)<< 1); 
  }
  return counter;
}
// i 番目のメンバを i 番目の部分木の解で埋める
__global__ 
void dim_nodeLayer(int size,long* nodes, long* solutions, int numElements)
{
  int i=blockDim.x * blockIdx.x + threadIdx.x;
  if(i<numElements){
    solutions[i]=carryChain_solve_nodeLayer(size,nodes[3 * i],nodes[3 * i + 1],nodes[3 * i + 2]);
  }
}
// 0以外のbitをカウント
int countBits_nodeLayer(long n)
{
  int counter = 0;
  while (n){
    n &= (n - 1); // 右端のゼロ以外の数字を削除
    counter++;
  }
  return counter;
}
// ノードをk番目のレイヤーのノードで埋める
long kLayer_nodeLayer(int size,std::vector<long>& nodes, int k, long left, long down, long right)
{
  long counter=0;
  long mask=(1<<size)-1;
  // すべてのdownが埋まったら、解決策を見つけたことになる。
  if (countBits_nodeLayer(down) == k) {
    nodes.push_back(left);
    nodes.push_back(down);
    nodes.push_back(right);
    return 1;
  }
  long bit=0;
  for(long carryChain=mask&~(left|down|right);carryChain;carryChain^=bit){
    bit=-carryChain&carryChain;
    // 解を加えて対角線をずらす
    counter+=kLayer_nodeLayer(size,nodes,k,(left|bit)>>1,(down|bit),(right|bit)<<1); 
  }
  return counter;
}
// k 番目のレイヤのすべてのノードを含むベクトルを返す。
std::vector<long> kLayer_nodeLayer(int size,int k)
{
  std::vector<long> nodes{};
  kLayer_nodeLayer(size,nodes, k, 0, 0, 0);
  return nodes;
}
// 【GPU キャリーチェーン】ノードレイヤーの作成
void carryChain_build_nodeLayer(int size)
{
  //int size=16;
  // ツリーの3番目のレイヤーにあるノード
  //（それぞれ連続する3つの数字でエンコードされる）のベクトル。
  // レイヤー2以降はノードの数が均等なので、対称性を利用できる。
  // レイヤ4には十分なノードがある（N16の場合、9844）。
  std::vector<long> nodes = kLayer_nodeLayer(size,4); 

  // デバイスにはクラスがないので、
  // 最初の要素を指定してからデバイスにコピーする。
  size_t nodeSize = nodes.size() * sizeof(long);
  long* hostNodes = (long*)malloc(nodeSize);
  hostNodes = &nodes[0];
  long* deviceNodes = NULL;
  hipMalloc((void**)&deviceNodes, nodeSize);
  hipMemcpy(deviceNodes, hostNodes, nodeSize, hipMemcpyHostToDevice);

  // デバイス出力の割り当て
  long* deviceSolutions = NULL;
  int numSolutions = nodes.size() / 6; // We only need half of the nodes, and each node is encoded by 3 integers.
  size_t solutionSize = numSolutions * sizeof(long);
  hipMalloc((void**)&deviceSolutions, solutionSize);

  // CUDAカーネルを起動する。
  int threadsPerBlock = 256;
  int blocksPerGrid = (numSolutions + threadsPerBlock - 1) / threadsPerBlock;
  dim_nodeLayer <<<blocksPerGrid, threadsPerBlock >>> (size,deviceNodes, deviceSolutions, numSolutions);

  // 結果をホストにコピー
  long* hostSolutions = (long*)malloc(solutionSize);
  hipMemcpy(hostSolutions, deviceSolutions, solutionSize, hipMemcpyDeviceToHost);

  // 部分解を加算し、結果を表示する。
  long solutions = 0;
  for (long i = 0; i < numSolutions; i++) {
      solutions += 2*hostSolutions[i]; // Symmetry
  }

  // 出力
  //std::cout << "We have " << solutions << " solutions on a " << size << " by " << size << " board." << std::endl;
  TOTAL=solutions;
  //return 0;
}
// CUDA 初期化
bool InitCUDA()
{
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    struct hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
//メイン
int main(int argc,char** argv)
{
  bool cpu=false,cpur=false,gpu=false,gpuNodeLayer=false;
  int argstart=2;
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='r'||argv[1][1]=='R'){cpur=true;}
    else if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){gpu=true;}
    else if(argv[1][1]=='n'||argv[1][1]=='N'){gpuNodeLayer=true;}
    else{ gpuNodeLayer=true; } //デフォルトをgpuとする
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g|-r|-s] n steps\n",argv[0]);
    printf("  -r: CPU 再帰\n");
    printf("  -c: CPU 非再帰\n");
    printf("  -g: GPU 再帰\n");
    printf("  -n: GPU ノードレイヤー\n");
  }
  if(cpur){ printf("\n\nキャリーチェーン 再帰 \n"); }
  else if(cpu){ printf("\n\nキャリーチェーン 非再帰 \n"); }
  else if(gpu){ printf("\n\nキャリーチェーン GPU\n"); }
  else if(gpuNodeLayer){ printf("\n\nキャリーチェーン GPUノードレイヤー \n"); }
  if(cpu||cpur){
    int min=4; 
    int targetN=17;
    struct timeval t0;
    struct timeval t1;
    printf("%s\n"," N:           Total           Unique          dd:hh:mm:ss.ms");
    for(int size=min;size<=targetN;size++){
      TOTAL=UNIQUE=0;
      gettimeofday(&t0, NULL);//計測開始
      if(cpur){ //再帰
        g.size=size;
        carryChain();
      }
      if(cpu){ //非再帰
        g.size=size;
        carryChain();
      }
      //
      gettimeofday(&t1, NULL);//計測終了
      int ss;int ms;int dd;
      if(t1.tv_usec<t0.tv_usec) {
        dd=(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      }else {
        dd=(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }//end if
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%16ld%17ld%12.2d:%02d:%02d:%02d.%02d\n",
          size,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    } //end for
  }//end if
  if(gpu||gpuNodeLayer){
    if(!InitCUDA()){return 0;}
    /* int steps=24576; */
    int min=4;
    int targetN=21;
    struct timeval t0;
    struct timeval t1;
    printf("%s\n"," N:        Total      Unique      dd:hh:mm:ss.ms");
    for(int size=min;size<=targetN;size++){
      gettimeofday(&t0,NULL);   // 計測開始
      if(gpu){
        TOTAL=UNIQUE=0;
        g.size=size;
        TOTAL=carryChain_solve_nodeLayer(size,0,0,0); //キャリーチェーン
      }else if(gpuNodeLayer){
        TOTAL=UNIQUE=0;
        g.size=size;
        carryChain_build_nodeLayer(size); // キャリーチェーン
      }
      gettimeofday(&t1,NULL);   // 計測終了
      int ss;int ms;int dd;
      if (t1.tv_usec<t0.tv_usec) {
        dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      } else {
        dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }//end if
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%13ld%16ld%4.2d:%02d:%02d:%02d.%02d\n",
          size,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    }//end for
  }//end if
  return 0;
}
